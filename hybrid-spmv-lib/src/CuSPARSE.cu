///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Nvidia CuSPARSE wrapper for benchmarking
///

#include <cstdio>
#include "cuSPARSE.hpp"

cuSPARSE_wrap *cuSPARSE_desc() {
  ddebug(" -> cuSPARSE_desc()\n");
  cuSPARSE_wrap *tmp = (cuSPARSE_wrap *)malloc(sizeof(cuSPARSE_wrap));
  tmp->target_mem = SPMV_MEMTYPE_HOST;
  int double_device = -1, unified_device = -1, nDevices = 0;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&tmp->properties, i);
    if (tmp->properties.major >= 2) double_device = i;
    if (tmp->properties.major >= 3) unified_device = i;
    if ((unified_device + 1) && (double_device + 1)) break;
  }
  massert(nDevices, "cuSPARSE_desc: No CUDA device found");
  if (unified_device + 1)
    tmp->target_mem = SPMV_MEMTYPE_UNIFIED;
  else if (double_device < 0)
    massert(0, "cuSPARSE_desc: Device does not support double values");
  else
    tmp->target_mem = SPMV_MEMTYPE_DEVICE;
  /// TODO: Add multiple gpu support?
  hipSetDevice(unified_device);
  // printf("Using Device: %d\n", unified_device);
  hipStreamCreate(&tmp->stream);
  hipsparseCreate(&tmp->handle);
  hipsparseSetStream(tmp->handle, tmp->stream);
  hipsparseCreateMatDescr(&tmp->descA);
  hipsparseSetMatType(tmp->descA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(tmp->descA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseCreateMatDescr(&tmp->descB);
  hipsparseSetMatType(tmp->descB, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(tmp->descB, HIPSPARSE_INDEX_BASE_ZERO);
  tmp->dir = HIPSPARSE_DIRECTION_COLUMN;
  hipDeviceSynchronize();
  ddebug("  <- cuSPARSE_desc()\n");
  return tmp;
}

void SpmvOperator::cuSPARSE_check_compatibility() {
  ddebug(" -> SpmvOperator::cuSPARSE_check_compatibility()\n");
  if (0)
    massert(0,
            "SpmvOperator::cuSPARSE_check_compatibility -> SpmvOperator not "
            "compatible");
  ddebug(" <- SpmvOperator::cuSPARSE_check_compatibility()\n");
}

void SpmvOperator::cuSPARSE_init() {
  ddebug(" -> SpmvOperator::cuSPARSE_init()\n");
  cuSPARSE_check_compatibility();
  free_lib_struct();
  lib_struct = cuSPARSE_desc();
  lib = SPMV_LIBRARY_CUSPARSE;
  mem_convert(((cuSPARSE_wrap *)lib_struct)->target_mem);
  ddebug(" <- SpmvOperator::cuSPARSE_init()\n");
}

void cuSPARSE_free(cuSPARSE_wrap *tmp) {
  ddebug(" -> cuSPARSE_free(wrapper)\n");
  hipStreamDestroy(tmp->stream);
  hipsparseDestroy(tmp->handle);
  hipsparseDestroyMatDescr(tmp->descA);
  hipsparseDestroyMatDescr(tmp->descB);
  hipDeviceSynchronize();
  free(tmp);
  ddebug(" <- cuSPARSE_free(wrapper)\n");
}

void SpmvOperator::format_convert_uni_coo2csr() {
  ddebug(" -> SpmvOperator::format_convert_uni_coo2csr()\n");
  SpmvCooData *coo_data = (SpmvCooData *)format_data;
  SpmvCsrData *csr_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));
  csr_data->colInd = coo_data->colInd;
  csr_data->values = coo_data->values;
  hipMallocManaged(&csr_data->rowPtr, (m + 1) * sizeof(int));

  timer = csecond();
  hipsparseXcoo2csr(((cuSPARSE_wrap *)lib_struct)->handle, coo_data->rowInd, nz,
                   m, csr_data->rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  timer = csecond() - timer;

  gpu_free(coo_data->rowInd);
  cudaCheckErrors();
  format_data = csr_data;
  format = SPMV_FORMAT_CSR;
  ddebug(" <- SpmvOperator::format_convert_uni_coo2csr()\n");
}

void SpmvOperator::format_convert_uni_csr2hyb() {
  ddebug(" -> SpmvOperator::format_convert_uni_csr2hyb()\n");
  SpmvCsrData *csr_data = (SpmvCsrData *)format_data;
  SpmvHybData *hyb_data = (SpmvHybData *)malloc(sizeof(SpmvHybData));
  timer = csecond();
  hipsparseCreateHybMat(&hyb_data->hybMatrix);
  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipsparseScsr2hyb(((cuSPARSE_wrap *)lib_struct)->handle, m, n,
                       ((cuSPARSE_wrap *)lib_struct)->descA,
                       (float *)csr_data->values, csr_data->rowPtr,
                       csr_data->colInd, hyb_data->hybMatrix, 0,
                       HIPSPARSE_HYB_PARTITION_AUTO); // == HYB. HIPSPARSE_HYB_PARTITION_MAX); // == ELL. 
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipsparseDcsr2hyb(((cuSPARSE_wrap *)lib_struct)->handle, m, n,
                       ((cuSPARSE_wrap *)lib_struct)->descA,
                       (double *)csr_data->values, csr_data->rowPtr,
                       csr_data->colInd, hyb_data->hybMatrix, 0,
                       HIPSPARSE_HYB_PARTITION_AUTO); // == HYB. HIPSPARSE_HYB_PARTITION_MAX); // == ELL.
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_csr2hyb() -> Unsupported SpMV "
              "value datatype");
  }
  hipDeviceSynchronize();
  timer = csecond() - timer;
  spmv_free();
  cudaCheckErrors();
  format_data = hyb_data;
  format = SPMV_FORMAT_HYB;
  ddebug(" <- SpmvOperator::format_convert_uni_csr2hyb()\n");
}

void SpmvOperator::format_convert_uni_csr2bsr() {
  ddebug(" -> SpmvOperator::format_convert_uni_csr2bsr()\n");
  SpmvCsrData *csr_data = (SpmvCsrData *)format_data;
  SpmvBsrData *bsr_data = (SpmvBsrData *)malloc(sizeof(SpmvBsrData));
  const int nb = (n + bsr_blockDim - 1) / bsr_blockDim;
  const int mb = (m + bsr_blockDim - 1) / bsr_blockDim;
  bsr_data->blockDim = bsr_blockDim;
  timer = csecond();
  hipMallocManaged(&bsr_data->rowPtr, (mb + 1) * sizeof(int));

  hipsparseXcsr2bsrNnz(
      ((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->dir,
      m, n, ((cuSPARSE_wrap *)lib_struct)->descA, csr_data->rowPtr,
      csr_data->colInd, bsr_data->blockDim,
      ((cuSPARSE_wrap *)lib_struct)->descB, bsr_data->rowPtr, &bsr_data->nnzb);
  hipDeviceSynchronize();
  hipMallocManaged(&bsr_data->colInd, bsr_data->nnzb * sizeof(int));

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipMallocManaged(&bsr_data->values,
                        (bsr_data->blockDim * bsr_data->blockDim) *
                            bsr_data->nnzb * sizeof(float));
      hipsparseScsr2bsr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, m, n,
          ((cuSPARSE_wrap *)lib_struct)->descA, (float *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd, bsr_data->blockDim,
          ((cuSPARSE_wrap *)lib_struct)->descB, (float *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipMallocManaged(&bsr_data->values,
                        (bsr_data->blockDim * bsr_data->blockDim) *
                            bsr_data->nnzb * sizeof(double));
      hipsparseDcsr2bsr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, m, n,
          ((cuSPARSE_wrap *)lib_struct)->descA, (double *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd, bsr_data->blockDim,
          ((cuSPARSE_wrap *)lib_struct)->descB, (double *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd);
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_csr2bsr -> Unsupported SpMV "
              "value datatype");
  }
  hipDeviceSynchronize();
  cudaCheckErrors();
  timer = csecond() - timer;
  spmv_free();
  cudaCheckErrors();
  format_data = bsr_data;
  format = SPMV_FORMAT_BSR;

  /// BSR vector padding TODO: This is kind of a cheat, maybe n, m should change
  /// too and vec_alloc_uni used for this
  void *xp, *yp;
  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipMallocManaged(&xp, nb * bsr_data->blockDim * sizeof(float));
      hipMallocManaged(&yp, mb * bsr_data->blockDim * sizeof(float));
      vec_copy<float>((float *)xp, (float *)x, m, mb * bsr_data->blockDim - m);
      for (int i = 0; i < m; i++) ((float *)yp)[i] = 0;
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipMallocManaged(&xp, nb * bsr_data->blockDim * sizeof(double));
      hipMallocManaged(&yp, mb * bsr_data->blockDim * sizeof(double));
      vec_copy<double>((double *)xp, (double *)x, m,
                       mb * bsr_data->blockDim - m);
      for (int i = 0; i < m; i++) ((double *)yp)[i] = 0;
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_csr2bsr -> Unsupported SpMV "
              "value datatype");
  }
  massert(xp && yp,
          "SpmvOperator::format_convert_uni_csr2bsr -> Padded Vector Unified "
          "Alloc failed");
  gpu_free(x);
  gpu_free(y);
  x = xp;
  y = yp;
  ddebug(" <- SpmvOperator::format_convert_uni_csr2bsr()\n");
}

void SpmvOperator::format_convert_uni_csr2coo() {
  ddebug(" -> SpmvOperator::format_convert_uni_csr2coo()\n");
  SpmvCsrData *csr_data = (SpmvCsrData *)format_data;
  SpmvCooData *coo_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
  coo_data->colInd = csr_data->colInd;
  coo_data->values = csr_data->values;
  hipMallocManaged(&coo_data->rowInd, nz * sizeof(int));

  timer = csecond();
  hipsparseXcsr2coo(((cuSPARSE_wrap *)lib_struct)->handle, csr_data->rowPtr, nz,
                   m, coo_data->rowInd, HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  timer = csecond() - timer;

  gpu_free(csr_data->rowPtr);
  cudaCheckErrors();
  format_data = coo_data;
  format = SPMV_FORMAT_COO;
  ddebug(" <- SpmvOperator::format_convert_uni_csr2coo()\n");
}

void SpmvOperator::format_convert_uni_bsr2csr() {
  ddebug(" -> SpmvOperator::format_convert_uni_bsr2csr()\n");
massert(false, "SpmvOperator::format_convert_uni_bsr2csr -> Implementation is not correct");
  SpmvBsrData *bsr_data = (SpmvBsrData *)format_data;
  SpmvCsrData *csr_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));
  const int nb = (n + bsr_data->blockDim - 1) / bsr_data->blockDim;
  const int mb = (m + bsr_data->blockDim - 1) / bsr_data->blockDim;

  timer = csecond();
  hipMallocManaged(&csr_data->rowPtr, (m + 1) * sizeof(int));
  hipMallocManaged(&csr_data->colInd, nz * sizeof(int));

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipMallocManaged(&csr_data->values,
                        nz * sizeof(float));
      hipsparseSbsr2csr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, 
          ((cuSPARSE_wrap *)lib_struct)->descA, (float *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd, bsr_data->blockDim, 
          ((cuSPARSE_wrap *)lib_struct)->descB, (float *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipMallocManaged(&csr_data->values,
                        nz * sizeof(double));
      hipsparseDbsr2csr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, 
          ((cuSPARSE_wrap *)lib_struct)->descA, (double *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd, bsr_data->blockDim, 
          ((cuSPARSE_wrap *)lib_struct)->descB, (double *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd);
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_bsr2csr -> Unsupported SpMV "
              "value datatype");
  }
  hipDeviceSynchronize();
  timer = csecond() - timer;
  spmv_free();
  cudaCheckErrors();
  format_data = csr_data;
  format = SPMV_FORMAT_CSR;


  ddebug(" <- SpmvOperator::format_convert_uni_bsr2csr()\n");
}

void SpmvOperator::format_convert_uni_bsr2bsr() {
  ddebug(" -> SpmvOperator::format_convert_uni_bsr2bsr()\n");
massert(false, "SpmvOperator::format_convert_uni_bsr2bsr -> Implementation is not correct");
  SpmvBsrData *bsr_data = (SpmvBsrData *)format_data;
  SpmvBsrData *bsr_data1 = (SpmvBsrData *)malloc(sizeof(SpmvBsrData));
  const int nb = (n + bsr_data->blockDim - 1) / bsr_data->blockDim;
  const int mb = (m + bsr_data->blockDim - 1) / bsr_data->blockDim;

  timer = csecond();
  int bufferSize, *nnzTotalDevHostPtr;
  void *pBuffer;

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
     hipsparseSgebsr2gebsr_bufferSize(((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, bsr_data->nnzb,
    ((cuSPARSE_wrap *)lib_struct)->descA, (float *)bsr_data->values , bsr_data->rowPtr, bsr_data->colInd,
    bsr_data->blockDim, bsr_data->blockDim,
    bsr_blockDim, bsr_blockDim,
    &bufferSize);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipsparseDgebsr2gebsr_bufferSize(((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, bsr_data->nnzb,
    ((cuSPARSE_wrap *)lib_struct)->descA, (double *)bsr_data->values , bsr_data->rowPtr, bsr_data->colInd,
    bsr_data->blockDim, bsr_data->blockDim,
    bsr_blockDim, bsr_blockDim,
    &bufferSize);
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_csr2bsr -> Unsupported SpMV "
              "value datatype");
  }
hipDeviceSynchronize();
hipMallocManaged(&pBuffer, bufferSize);

  bsr_data1->blockDim = bsr_blockDim;
  const int nb1 = (nb* bsr_data->blockDim + bsr_data1->blockDim - 1) / bsr_data1->blockDim;
  const int mb1 = (mb* bsr_data->blockDim + bsr_data1->blockDim - 1) / bsr_data1->blockDim;
  hipMallocManaged(&bsr_data1->rowPtr, (mb + 1) * sizeof(int));


hipsparseXgebsr2gebsrNnz( ((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->dir,
      mb, nb, bsr_data->nnzb, ((cuSPARSE_wrap *)lib_struct)->descA, bsr_data->rowPtr,
      bsr_data->colInd, bsr_data->blockDim, bsr_data->blockDim,
((cuSPARSE_wrap *)lib_struct)->descB, bsr_data1->rowPtr, bsr_data1->blockDim, bsr_data1->blockDim, nnzTotalDevHostPtr, pBuffer);
hipDeviceSynchronize();

if (NULL != nnzTotalDevHostPtr) bsr_data1->nnzb = *nnzTotalDevHostPtr;
else bsr_data1->nnzb =  bsr_data1->rowPtr[mb1] - bsr_data1->rowPtr[0];

  hipMallocManaged(&bsr_data1->colInd, bsr_data1->nnzb * sizeof(int));

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipMallocManaged(&bsr_data1->values,
                        (bsr_data1->blockDim * bsr_data1->blockDim) *
                            bsr_data1->nnzb * sizeof(float));
      hipsparseSgebsr2gebsr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, bsr_data->nnzb,
          ((cuSPARSE_wrap *)lib_struct)->descA, (float *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd, bsr_data->blockDim, bsr_data->blockDim,
          ((cuSPARSE_wrap *)lib_struct)->descB, (float *)bsr_data1->values,
          bsr_data1->rowPtr, bsr_data1->colInd, bsr_data1->blockDim, bsr_data1->blockDim, pBuffer);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipMallocManaged(&bsr_data1->values,
                        (bsr_data1->blockDim * bsr_data1->blockDim) *
                            bsr_data1->nnzb * sizeof(double));
      hipsparseDgebsr2gebsr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, bsr_data->nnzb,
          ((cuSPARSE_wrap *)lib_struct)->descA, (double *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd, bsr_data->blockDim, bsr_data->blockDim,
          ((cuSPARSE_wrap *)lib_struct)->descB, (double *)bsr_data1->values,
          bsr_data1->rowPtr, bsr_data1->colInd, bsr_data1->blockDim, bsr_data1->blockDim, pBuffer);
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_bsr2bsr -> Unsupported SpMV "
              "value datatype");
  }
  hipDeviceSynchronize();
  timer = csecond() - timer;
  spmv_free();
  cudaCheckErrors();
  format_data = bsr_data1;
  format = SPMV_FORMAT_BSR;

  /// BSR vector padding TODO: This is kind of a cheat, maybe n, m should change
  /// too and vec_alloc_uni used for this
  void *xp, *yp;
  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipMallocManaged(&xp, nb1 * bsr_data1->blockDim * sizeof(float));
      hipMallocManaged(&yp, mb1 * bsr_data1->blockDim * sizeof(float));
      vec_copy<float>((float *)xp, (float *)x, nb* bsr_data->blockDim, nb1 * bsr_data1->blockDim - nb* bsr_data->blockDim);
      for (int i = 0; i < mb* bsr_data->blockDim; i++) ((float *)yp)[i] = 0;
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipMallocManaged(&xp, nb1 * bsr_data1->blockDim * sizeof(double));
      hipMallocManaged(&yp, mb1 * bsr_data1->blockDim * sizeof(double));
      vec_copy<double>((double *)xp, (double *)x, nb* bsr_data->blockDim,
                       nb1 * bsr_data1->blockDim - nb* bsr_data->blockDim);
      for (int i = 0; i < mb* bsr_data->blockDim; i++) ((double *)yp)[i] = 0;
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_bsr2bsr -> Unsupported SpMV "
              "value datatype");
  }
  massert(xp && yp,
          "SpmvOperator::format_convert_uni_bsr2bsr -> Padded Vector Unified "
          "Alloc failed");
  gpu_free(x);
  gpu_free(y);
  x = xp;
  y = yp;
  ddebug(" <- SpmvOperator::format_convert_uni_bsr2bsr()\n");
}

void SpmvOperator::format_convert_uni_hyb2csr() {
  ddebug(" -> SpmvOperator::format_convert_uni_hyb2csr()\n");
  SpmvHybData *hyb_data = (SpmvHybData *)format_data;
  SpmvCsrData *csr_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));

  hipMallocManaged(&csr_data->rowPtr, (m + 1) * sizeof(int));
  hipMallocManaged(&csr_data->colInd, nz * sizeof(int));

timer = csecond();
switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      hipMallocManaged(&csr_data->values, nz * sizeof(float));
hipsparseShyb2csr(((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->descA, 
                 hyb_data->hybMatrix, (float *) csr_data->values, (int *) csr_data->rowPtr, (int *) csr_data->colInd);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      hipMallocManaged(&csr_data->values,  nz * sizeof(double));
hipsparseDhyb2csr(((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->descA, 
                 hyb_data->hybMatrix, (double *) csr_data->values, (int *) csr_data->rowPtr, (int *) csr_data->colInd);
    } break;
    default:
      massert(false,
              "SpmvOperator::format_convert_uni_hyb2csr -> Unsupported SpMV "
              "value datatype");
  }
  hipDeviceSynchronize();
  timer = csecond() - timer;
spmv_free();
  cudaCheckErrors();
  format_data = csr_data;
  format = SPMV_FORMAT_CSR;
  ddebug(" <- SpmvOperator::format_convert_uni_hyb2csr()\n");
}

void SpmvOperator::cuSPARSE_csr() {
  ddebug(" -> SpmvOperator::cuSPARSE_csr()\n");
  massert(format == SPMV_FORMAT_CSR,
          "SpmvOperator::cuSPARSE_csr -> Wrong input format");

  SpmvCsrData *data = (SpmvCsrData *)format_data;
  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      const float alf = 1.0;
      const float beta = 0;
      hipsparseScsrmv(((cuSPARSE_wrap *)lib_struct)->handle,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nz, &alf,
                     ((cuSPARSE_wrap *)lib_struct)->descA,
                     (float *)data->values, data->rowPtr, data->colInd,
                     (float *)x, &beta, (float *)y);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      const double alf = 1.0;
      const double beta = 0;
      hipsparseDcsrmv(((cuSPARSE_wrap *)lib_struct)->handle,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nz, &alf,
                     ((cuSPARSE_wrap *)lib_struct)->descA,
                     (double *)data->values, data->rowPtr, data->colInd,
                     (double *)x, &beta, (double *)y);
    } break;
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(false,
              "SpmvOperator::cuSPARSE_csr -> Unsupported SpMV value datatype");
  }
  ddebug(" <- SpmvOperator::cuSPARSE_csr()\n");
}

void SpmvOperator::cuSPARSE_hyb() {
  ddebug(" -> SpmvOperator::cuSPARSE_hyb()\n");
  massert(format == SPMV_FORMAT_HYB,
          "SpmvOperator::cuSPARSE_hyb -> Wrong input format");

  SpmvHybData *data = (SpmvHybData *)format_data;

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      const float alf = 1.0;
      const float beta = 0;
      hipsparseShybmv(((cuSPARSE_wrap *)lib_struct)->handle,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE, &alf,
                     ((cuSPARSE_wrap *)lib_struct)->descA, data->hybMatrix,
                     (float *)x, &beta, (float *)y);
      break;
    }
    case (SPMV_VALUETYPE_DOUBLE): {
      const double alf = 1.0;
      const double beta = 0;
      hipsparseDhybmv(((cuSPARSE_wrap *)lib_struct)->handle,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE, &alf,
                     ((cuSPARSE_wrap *)lib_struct)->descA, data->hybMatrix,
                     (double *)x, &beta, (double *)y);
      break;
    }
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(false,
              "SpmvOperator::cuSPARSE_hyb -> Unsupported SpMV value datatype");
  }
  ddebug(" <- SpmvOperator::cuSPARSE_hyb()\n");
}

void SpmvOperator::cuSPARSE_bsr() {
  ddebug(" -> SpmvOperator::cuSPARSE_bsr()\n");
  massert(format == SPMV_FORMAT_BSR,
          "SpmvOperator::cuSPARSE_bsr -> Wrong input format");

  SpmvBsrData *data = (SpmvBsrData *)format_data;
  const int nb = (n + data->blockDim - 1) / data->blockDim;
  const int mb = (m + data->blockDim - 1) / data->blockDim;

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT): {
      const float alf = 1.0;
      const float beta = 0;
      hipsparseSbsrmv(((cuSPARSE_wrap *)lib_struct)->handle,
                     ((cuSPARSE_wrap *)lib_struct)->dir,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, data->nnzb, &alf,
                     ((cuSPARSE_wrap *)lib_struct)->descA,
                     (float *)data->values, data->rowPtr, data->colInd,
                     data->blockDim, (float *)x, &beta, (float *)y);
    } break;
    case (SPMV_VALUETYPE_DOUBLE): {
      const double alf = 1.0;
      const double beta = 0;
      hipsparseDbsrmv(((cuSPARSE_wrap *)lib_struct)->handle,
                     ((cuSPARSE_wrap *)lib_struct)->dir,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, nb, data->nnzb, &alf,
                     ((cuSPARSE_wrap *)lib_struct)->descA,
                     (double *)data->values, data->rowPtr, data->colInd,
                     data->blockDim, (double *)x, &beta, (double *)y);
    } break;
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(false,
              "SpmvOperator::cuSPARSE_bsr -> Unsupported SpMV value datatype");
  }
  ddebug(" <- SpmvOperator::cuSPARSE_bsr()\n");
}
