///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Input functions for .mtx
///

#include <stdio.h>
#include <stdlib.h>
#include "gpu_utils.hpp"
#include "mmio.h"
#include "spmv_utils.hpp"

void SpmvOperator::mtx_read_uni(){
    ddebug(" -> SpmvOperator::mtx_read_uni()\n");
    int nnzA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;
    
	// read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;
    // load matrix
    if ((f = fopen(mtx_name, "r")) == NULL)
        exit(1);

    if (mm_read_banner(f, &matcode) != 0)
    {
        cout << "Could not process Matrix Market banner." << endl;
        exit(2);
    }

    if ( mm_is_complex( matcode ) )
    {
        cout <<"Sorry, data type 'COMPLEX' is not supported. " << endl;
        exit(3);
    }

    if ( mm_is_pattern( matcode ) )  { isPattern = 1; /*cout << "type = Pattern" << endl;*/ }
    if ( mm_is_real ( matcode) )     { isReal = 1; /*cout << "type = real" << endl;*/ }
    if ( mm_is_integer ( matcode ) ) { isInteger = 1; /*cout << "type = integer" << endl;*/ }

    /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
        exit(4);

    if ( mm_is_symmetric( matcode ) || mm_is_hermitian( matcode ) )
    {
        isSymmetric = 1;
        //cout << "symmetric = true" << endl;
    }
    else
    {
        //cout << "symmetric = false" << endl;
    }

    int *csrRowPtrA_counter = (int *)malloc((m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    int *csrRowIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    int *csrColIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    VALUE_TYPE *csrValA_tmp    = (VALUE_TYPE *)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (int i = 0; i < nnzA_mtx_report; i++)
    {
        int idxi, idxj;
        VALUE_TYPE fval;
        int ival;

        if (isReal)
            fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
        else if (isInteger)
        {
            fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
            fval = ival;
        }
        else if (isPattern)
        {
            fscanf(f, "%d %d\n", &idxi, &idxj);
            fval = 1.0;
        }

        // adjust from 1-based to 0-based
        idxi--;
        idxj--;

        csrRowPtrA_counter[idxi]++;
        csrRowIdxA_tmp[i] = idxi;
        csrColIdxA_tmp[i] = idxj;
        csrValA_tmp[i] = fval;
    }

    if (f != stdin)
        fclose(f);

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
        }
    }

    // exclusive scan for csrRowPtrA_counter
    int old_val, new_val;

    old_val = csrRowPtrA_counter[0];
    csrRowPtrA_counter[0] = 0;
    for (int i = 1; i <= m; i++)
    {
        new_val = csrRowPtrA_counter[i];
        csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i-1];
        old_val = new_val;
    }

	hipMallocManaged(&csrRowPtrA, (m+1) * sizeof(int));
	hipMallocManaged(&csrColIdxA, nnzA * sizeof(int));
	hipMallocManaged(&csrValA, nnzA * sizeof(VALUE_TYPE));
	hipDeviceSynchronize();
	cudaCheckErrors();
  
    nnzA = csrRowPtrA_counter[m];
    memcpy(csrRowPtrA, csrRowPtrA_counter, (m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

                offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
                csrColIdxA[offset] = csrRowIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
            }
            else
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
            }
        }
    }
    else
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
            csrColIdxA[offset] = csrColIdxA_tmp[i];
            csrValA[offset] = csrValA_tmp[i];
            csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
        }
    }

    // free tmp space
    free(csrColIdxA_tmp);
    free(csrValA_tmp);
    free(csrRowIdxA_tmp);
    free(csrRowPtrA_counter);
    
    SpmvCsrData* csr_output = (SpmvCsrData *) malloc(sizeof(SpmvCsrData));
	nz = nnzA;
	mem_bytes += (nz) * sizeof(VALUE_TYPE) + (2 * nz) * sizeof(int);
	gpu_mem_bytes += (nz) * sizeof(VALUE_TYPE) + (2 * nz) * sizeof(int);
  	csr_output->rowPtr = csrRowPtrA;
  	csr_output->colInd = csrColIdxA;
  	csr_output->values = csrValA;
  	
  	format_data = csr_output;
  	ddebug(" <- SpmvOperator::mtx_read_uni()\n");
}

/*void SpmvOperator::mtx_read_host() {
  ddebug(" -> SpmvOperator::mtx_read_host()\n");
  massert(value_type == SPMV_VALUE_TYPE_DOUBLE,
          "SpmvOperator::mtx_read_host -> only double value_type supported");
  int ret_code, nz1, *I, *J, ctr;
  double *val;
  MM_typecode matcode;
  FILE *f;
  int i;

  if ((f = fopen(mtx_name, "r")) == NULL)
    massert(0, "SpmvOperator::mtx_read_host -> Failed to open mtx file");


  if (mm_read_banner(f, &matcode) != 0)
    massert(0,
            "SpmvOperator::mtx_read_host -> Could not process Matrix Market "
            "banner");

  //  This is how one can screen matrix types if their application
  //  only supports a subset of the Matrix Market data types. 

  massert(mm_is_valid(matcode),
          "SpmvOperator::mtx_read_host -> mm_is_valid(matcode) returned false");
  massert(!mm_is_complex(matcode),
          "SpmvOperator::mtx_read_host -> Complex Matrices not supported");
  massert(mm_is_sparse(matcode),
          "SpmvOperator::mtx_read_host -> Dense Matrices not supported");


  if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nz1)) != 0)
    massert(
        0,
        "SpmvOperator::mtx_read_host -> Error in finding size of mtx matrix");

  //massert(n == m, "SpmvOperator::mtx_read_host -> Only square Matrices supported in this version");

  I = (int *)malloc(nz1 * sizeof(int));
  J = (int *)malloc(nz1 * sizeof(int));
  val = (double *)malloc(nz1 * sizeof(double));
  nz = nz1;


  for (i = 0; i < nz1; i++) {
    if (mm_is_pattern(matcode)) {
      fscanf(f, "%d %d\n", &(I[i]), &(J[i]));
      val[i] = 1.0;
    } else
      fscanf(f, "%d %d %lf\n", &(I[i]), &(J[i]), &(val[i]));
    if (mm_is_symmetric(matcode) && (I[i] != J[i])) nz++;
  }
  SpmvCooData *data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
  data->rowInd = (int *)malloc(nz * sizeof(int));
  data->colInd = (int *)malloc(nz * sizeof(int));
  data->values = malloc(nz * sizeof(double));
  mem_bytes += (nz) * sizeof(double) + (2 * nz) * sizeof(int);
  double *values = (double *)data->values;

  ctr = nz1;
  for (i = 0; i < nz1; i++) {
    data->rowInd[i] = I[i];
    data->colInd[i] = J[i];
    values[i] = val[i];
    data->rowInd[i]--;
    data->colInd[i]--;
    if (mm_is_symmetric(matcode) && (data->rowInd[i] != data->colInd[i])) {
      data->rowInd[ctr] = data->colInd[i];
      data->colInd[ctr] = data->rowInd[i];
      values[ctr] = values[i];
      ctr++;
    }
  }

  if (f != stdin) fclose(f);

  int *tmp_X = (int *)malloc(nz * sizeof(int));
  int *tmp_Y = (int *)malloc(nz * sizeof(int));
  double *tmp_Z = (double *)malloc(nz * sizeof(double));

  if (!isArraySorted(data->rowInd, nz)) mergeSort(data->rowInd, data->colInd, values, nz - 1, tmp_X, tmp_Y, tmp_Z);
  ctr = 0;
  for (i = 1; i < nz; i++)
    if (data->rowInd[i] > data->rowInd[i - 1]) {
      if (!isArraySorted(&(data->colInd[ctr]), i - ctr)) mergeSort(&(data->colInd[ctr]), &(data->rowInd[ctr]), &(values[ctr]),
                i - 1 - ctr, tmp_X, tmp_Y, tmp_Z);
      ctr = i;
    }
  if (!isArraySorted(&(data->colInd[ctr]), i - ctr)) mergeSort(&(data->colInd[ctr]), &(data->rowInd[ctr]), &(values[ctr]),
            i - 1 - ctr, tmp_X, tmp_Y, tmp_Z);
  free(I);
  free(J);
  free(val);
  // vec_print<int>(data->rowInd, nz, "rowInd");
  // vec_print<int>(data->colInd, nz, "colInd");
  // vec_print<double>((double*)data->values, nz, "values");
  massert(data->rowInd && data->colInd && data->values,
          "SpmvOperator::mtx_read_host -> Format Struct Alloc failed");
  format_data = data;
  ddebug(" <- SpmvOperator::mtx_read_host()\n");
}

void SpmvOperator::mtx_read_uni() {
  ddebug(" -> SpmvOperator::mtx_read_uni()\n");
  massert(value_type == SPMV_VALUE_TYPE_DOUBLE,
          "SpmvOperator::mtx_read_uni -> only double value_type supported");
  int ret_code, nz1, *I, *J, ctr;
  double *val;
  MM_typecode matcode;
  FILE *f;
  int i;

  if ((f = fopen(mtx_name, "r")) == NULL)
    massert(0, "SpmvOperator::mtx_read_uni -> Failed to open mtx file");

  if (mm_read_banner(f, &matcode) != 0)
    massert(
        0,
        "SpmvOperator::mtx_read_uni -> Could not process Matrix Market banner");


  massert(mm_is_valid(matcode),
          "SpmvOperator::mtx_read_host -> mm_is_valid(matcode) returned false");
  massert(!mm_is_complex(matcode),
          "SpmvOperator::mtx_read_host -> Complex Matrices not supported");
  massert(mm_is_sparse(matcode),
          "SpmvOperator::mtx_read_host -> Dense Matrices not supported");


  if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nz1)) != 0)
    massert(
        0, "SpmvOperator::mtx_read_uni -> Error in finding size of mtx matrix");

  massert(n == m,
          "SpmvOperator::mtx_read_uni -> Only square Matrices supported in "
          "this version");

  I = (int *)malloc(nz1 * sizeof(int));
  J = (int *)malloc(nz1 * sizeof(int));
  val = (double *)malloc(nz1 * sizeof(double));
  nz = nz1;


  for (i = 0; i < nz1; i++) {
    if (mm_is_pattern(matcode)) {
      fscanf(f, "%d %d\n", &(I[i]), &(J[i]));
      val[i] = 1.0;
    } else
      fscanf(f, "%d %d %lf\n", &(I[i]), &(J[i]), &(val[i]));
    if (mm_is_symmetric(matcode) && (I[i] != J[i])) nz++;
  }
  SpmvCooData *data = (SpmvCooData *)malloc(sizeof(SpmvCooData));

  hipMallocManaged(&data->rowInd, nz * sizeof(int));
  hipMallocManaged(&data->colInd, nz * sizeof(int));
  hipMallocManaged(&data->values, nz * sizeof(double));
  hipDeviceSynchronize();
  cudaCheckErrors();
  mem_bytes += (nz) * sizeof(double) + (2 * nz) * sizeof(int);
  gpu_mem_bytes += (nz) * sizeof(double) + (2 * nz) * sizeof(int);
  double *values = (double *)data->values;

  ctr = nz1;
  for (i = 0; i < nz1; i++) {
    data->rowInd[i] = I[i];
    data->colInd[i] = J[i];
    values[i] = val[i];
    data->rowInd[i]--; 
    data->colInd[i]--;
    if (mm_is_symmetric(matcode) && (data->rowInd[i] != data->colInd[i])) {
      data->rowInd[ctr] = data->colInd[i];
      data->colInd[ctr] = data->rowInd[i];
      values[ctr] = values[i];
      ctr++;
    }
  }

  if (f != stdin) fclose(f);

  int *tmp_X = (int *)malloc(nz * sizeof(int));
  int *tmp_Y = (int *)malloc(nz * sizeof(int));
  double *tmp_Z = (double *)malloc(nz * sizeof(double));

  if (!isArraySorted(data->rowInd, nz)) mergeSort(data->rowInd, data->colInd, values, nz - 1, tmp_X, tmp_Y, tmp_Z);
  ctr = 0;
  for (i = 1; i < nz; i++)
    if (data->rowInd[i] > data->rowInd[i - 1]) {
      if (!isArraySorted(&(data->colInd[ctr]), i - ctr)) mergeSort(&(data->colInd[ctr]), &(data->rowInd[ctr]), &(values[ctr]),
                i - 1 - ctr, tmp_X, tmp_Y, tmp_Z);
      ctr = i;
    }

  free(I);
  free(J);
  free(val);
  massert(data->rowInd && data->colInd && data->values,
          "SpmvOperator::mtx_read_host -> Format Struct Alloc failed");
  format_data = data;
  ddebug(" <- SpmvOperator::mtx_read_uni()\n");
}
*/

void SpmvOperator::mtx_read_device() {
  ddebug(" -> SpmvOperator::mtx_read_device()\n");
  massert(0, "SpmvOperator::mtx_read_device -> Not implemented");
  ddebug(" <- SpmvOperator::mtx_read_device()\n");
}

void SpmvOperator::mtx_generate_device() {
  ddebug(" -> SpmvOperator::mtx_generate_device()\n");
  massert(0, "SpmvOperator::mtx_generate_device -> Not implemented");
  ddebug(" <- SpmvOperator::mtx_generate_device()\n");
}

void SpmvOperator::mtx_generate_uni() {
  ddebug(" -> SpmvOperator::mtx_generate_uni()\n");
  massert(0, "SpmvOperator::mtx_generate_uni -> Not implemented");
  ddebug(" <- SpmvOperator::mtx_generate_uni()\n");
}
