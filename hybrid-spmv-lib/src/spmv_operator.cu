///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some helpfull functions for SpMV
///

#include <unistd.h>
#include <cassert>
#include <cstdio>
#include "avx512CSR5.hpp"
#include "cuCSR5.hpp"
#include "cuSPARSE.hpp"
#include "mkl.hpp"

#include "gpu_utils.hpp"

/// SpmvOperator functions
SpmvOperator::SpmvOperator(int argc, char *argv[], int start_of_matrix_generation_args, int verbose) {
  ddebug(" -> SpmvOperator::SpmvOperator()\n");
  /// Default intialization values for an operator
  mem_bytes = 0;
  mem_bytes += sizeof(SpmvOperator);
  n = m = nz = bytes = flops = bsr_blockDim = 0;
  x = y = NULL;
  mem_alloc = SPMV_MEMTYPE_HOST;
  if (sizeof(VALUE_TYPE) == sizeof(double)) value_type = SPMV_VALUETYPE_DOUBLE;
  else if (sizeof(VALUE_TYPE) == sizeof(float)) value_type = SPMV_VALUETYPE_FLOAT;
  else massert(false, "SpmvOperator::SpmvOperator() -> Unsupported VALUE_TYPE\n");
  format_data = NULL;
  format = SPMV_FORMAT_CSR;
  lib_struct = NULL;
  lib = SPMV_NONE;
  mtx_name = "Synthetic";
  mtx_generate(argc, argv, start_of_matrix_generation_args, verbose);
  bytes = 2 * sizeof(int) * n + 1 * sizeof(int) * nz + 2 * sizeof(double) * nz +
          2 * sizeof(double) * n;
  flops = 2 * nz;
  ddebug(" <- SpmvOperator::SpmvOperator()\n");
}

SpmvOperator::SpmvOperator(char *matrix_name) {
  ddebug(" -> SpmvOperator::SpmvOperator(matrix_name)\n");
  /// Default intialization values for an operator
  mem_bytes = 0;
  mem_bytes += sizeof(SpmvOperator);
  mtx_name = matrix_name;
  n = m = nz = bytes = flops = bsr_blockDim = 0;
  x = y = NULL;
  mem_alloc = SPMV_MEMTYPE_HOST;
  if (sizeof(VALUE_TYPE) == sizeof(double)) value_type = SPMV_VALUETYPE_DOUBLE;
  else if (sizeof(VALUE_TYPE) == sizeof(float)) value_type = SPMV_VALUETYPE_FLOAT;
  else massert(false, "SpmvOperator::SpmvOperator() -> Unsupported VALUE_TYPE\n");
  format_data = NULL;
  format = SPMV_FORMAT_CSR;
  lib_struct = NULL;
  lib = SPMV_NONE;
  mtx_read();
  bytes = 2 * sizeof(int) * n + 1 * sizeof(int) * nz + 2 * sizeof(double) * nz +
          2 * sizeof(double) * n;
  flops = 2 * nz;
  ddebug(" <- SpmvOperator::SpmvOperator(matrix_name)\n");
}

SpmvOperator::SpmvOperator(SpmvOperator &op) {
  ddebug(" -> SpmvOperator::SpmvOperator(copy)\n");
  lib = op.lib;
  // TODO: This switch could be replaced with an actual function
  switch (lib) {
    case (SPMV_LIBRARY_CUSPARSE):
      lib_struct = cuSPARSE_desc();
      debug(
          "SpmvOperator::SpmvOperator(copy) -> Generated new cuSPARSE_desc\n");
      break;
    case (SPMV_NONE):
      debug(
          "SpmvOperator::SpmvOperator(copy) -> warning... copying SPMV_NONE "
          "operator\n");
      break;
    case (SPMV_LIBRARY_OPENMP):
      lib_struct = NULL;
      break;
    default: {
      massert(false,
              "SpmvOperator::SpmvOperator(copy) -> Unreachable lib default "
              "reached\n");
      break;
    }
  }
  bsr_blockDim = op.bsr_blockDim;
  mem_alloc = op.mem_alloc;
  mtx_name = op.mtx_name;
  format = op.format;
  mem_alloc = op.mem_alloc;
  value_type = op.value_type;
  m = op.m;
  n = op.n;
  nz = op.nz;
  density =  op.density;
	//bytes = matrix->mem_footprint;
  avg_nz_row = op.avg_nz_row;
  std_nz_row = op.std_nz_row;
  avg_bandwidth = op.avg_bandwidth;
  std_bandwidth = op.std_bandwidth;
  avg_scattering = op.avg_scattering;
  std_scattering = op.std_scattering;
  strcpy(distribution, op.distribution);
  strcpy(placement, op.placement);
  diagonal_factor = op.diagonal_factor;
  seed = op.seed;
  flops = op.flops;
  bytes = op.bytes;
  format_data = op.spmv_data_get_copy();
  debug("SpmvOperator::SpmvOperator(copy) -> Copied format struct\n");

  // TODO: This switch could be replaced with an actual function
  switch (op.mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
    case (SPMV_MEMTYPE_UNIFIED):
      vec_alloc(op.x);
      debug("SpmvOperator::SpmvOperator(copy) -> Copied x vector\n");
      break;
    case (SPMV_MEMTYPE_DEVICE):
      massert(0,
              "SpmvOperator::SpmvOperator(copy) -> Copy from device vector not "
              "supported\n");
      break;
    default:
      massert(0,
              "SpmvOperator::SpmvOperator(copy) -> Unreachable mem_alloc "
              "default reached\n");
      break;
  }
  ddebug(" <- SpmvOperator::SpmvOperator(copy)\n");
}

SpmvOperator::SpmvOperator(SpmvOperator &op, int start, int end, int mode) {
  ddebug(" -> SpmvOperator::SpmvOperator(copy_op,start,end)\n");
  massert(start >= 0 && start < op.nz,
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> start is not "
          "within accepted limits");
  massert(end > 0 && end <= op.nz,
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> end is not within "
          "accepted limits");
  massert(end > start,
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> end is not greater "
          "than start");
  op.format_convert(SPMV_FORMAT_COO);
  lib = op.lib;
  // TODO: This switch could be replaced with an actual function
  switch (lib) {
    case (SPMV_LIBRARY_CUSPARSE):
      lib_struct = cuSPARSE_desc();
      debug(
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> Generated new "
          "cuSPARSE_desc\n");
      break;
    case (SPMV_NONE):
      debug(
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> warning... copying "
          "SPMV_NONE "
          "operator\n");
      break;
    case (SPMV_LIBRARY_OPENMP):
      lib_struct = NULL;
      break;
    default: {
      massert(false,
              "SpmvOperator::SpmvOperator(copy_op,start,end) -> Unreachable "
              "lib default "
              "reached\n");
      break;
    }
  }
  bsr_blockDim = op.bsr_blockDim;
  mem_alloc = op.mem_alloc;
  mtx_name = op.mtx_name;
  format = op.format;
  mem_alloc = op.mem_alloc;
  value_type = op.value_type;
  // TODO: All the splitting mechanism will be defined here
  m = n = op.m;
  nz = end - start;
  strcpy(distribution, op.distribution);
  strcpy(placement, op.placement);
  diagonal_factor = op.diagonal_factor;
  seed = op.seed;
  density = 0; 
  avg_nz_row = 0;
  std_nz_row = 0;
  avg_bandwidth = 0;
  std_bandwidth = 0;
  avg_scattering = 0;
  std_scattering = 0;
  format_data = op.spmv_data_get_subcopy(&start, &nz, mode);
  /// FIXME:EXP
  /*
  if (mode == 0) n = ((SpmvCooData *)format_data)->rowInd[nz - 1] + 1;
  else  if (mode == 1) n = m - ((SpmvCooData *)op.format_data)->rowInd[op.nz -
  nz];

    bytes = 2 * sizeof(int) * n + 1 * sizeof(int) * nz + 2 * sizeof(double) * nz
  +
            2 * sizeof(double) * n;
    /// FLOPS
    flops = 2 * nz;
  */

  debug(
      "SpmvOperator::SpmvOperator(copy_op,start,end) -> Copied format "
      "struct\n");

  // TODO: This switch could be replaced with an actual function
  switch (op.mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
    case (SPMV_MEMTYPE_UNIFIED):
      vec_alloc(op.x);
      debug(
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> Copied x vector\n");
      break;
    case (SPMV_MEMTYPE_DEVICE):
      massert(0,
              "SpmvOperator::SpmvOperator(copy_op,start,end) -> Copy from "
              "device vector not "
              "supported\n");
      break;
    default:
      massert(0,
              "SpmvOperator::SpmvOperator(copy_op,start,end) -> Unreachable "
              "mem_alloc "
              "default reached\n");
      break;
  }
  if (mode == 0)
    n = ((SpmvCooData *)format_data)->rowInd[nz - 1] + 1;
  else if (mode == 1)
    n = m - ((SpmvCooData *)op.format_data)->rowInd[op.nz - nz];

  bytes = 2 * sizeof(int) * n + 1 * sizeof(int) * nz + 2 * sizeof(double) * nz +
          2 * sizeof(double) * n;
  /// FLOPS
  flops = 2 * nz;

  ddebug(" <- SpmvOperator::SpmvOperator(copy_op,start,end)\n");
}

SpmvOperator::~SpmvOperator() {
  ddebug(" -> SpmvOperator::~SpmvOperator()\n");
  vec_free(x, n * sizeof(double), mem_alloc);
  vec_free(y, m * sizeof(double), mem_alloc);
  spmv_free();
  free(lib_struct);
  debug("SpmvOperator::~SpmvOperator -> Operator successfully destroyed\n");
  ddebug(" <- SpmvOperator::~SpmvOperator()\n");
}

void SpmvOperator::spmv_free_host() {
  ddebug(" -> SpmvOperator::spmv_free_host()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      free(data->rowInd);
      free(data->colInd);
      free(data->values);
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      free(data->rowPtr);
      free(data->colInd);
      free(data->values);
    } break;
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      free(data->rowPtr);
      free(data->colInd);
      free(data->values);
    } break;
    case (SPMV_FORMAT_HYB): {
      massert(false,
              "SpmvOperator::Spmv_free_host -> Unsupported format = "
              "SPMV_FORMAT_HYB");
    } break;
    default:
      massert(false, "SpmvOperator::Spmv_free_host -> format default reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_free_host()\n");
}

void SpmvOperator::spmv_free_numa() {
  ddebug(" -> SpmvOperator::spmv_free_numa()\n");
	massert(false, "SpmvOperator::spmv_free_numa -> No numa please");
/*
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      vec_free(data->rowInd, nz * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->colInd, nz * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->values, nz * sizeof(double), SPMV_MEMTYPE_NUMA);
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      vec_free(data->rowPtr, (n + 1) * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->colInd, nz * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->values, nz * sizeof(double), SPMV_MEMTYPE_NUMA);
    } break;
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      const int nb = (n + data->blockDim - 1) / data->blockDim;
      const int mb = (m + data->blockDim - 1) / data->blockDim;
      vec_free(data->rowPtr, (nb + 1) * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->colInd, data->nnzb * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->values,
               (data->blockDim * data->blockDim) * data->nnzb * sizeof(double),
               SPMV_MEMTYPE_NUMA);
    } break;
    case (SPMV_FORMAT_HYB): {
      massert(false,
              "SpmvOperator::spmv_free_numa -> Unsupported format = "
              "SPMV_FORMAT_HYB");
    } break;
    default:
      massert(false, "SpmvOperator::spmv_free_numa -> format default reached");
      break;
  }
*/
  ddebug(" <- SpmvOperator::spmv_free_numa()\n");
}

void SpmvOperator::spmv_free_device() {
  ddebug(" -> SpmvOperator::spmv_free_device()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      gpu_free(data->rowInd);
      gpu_free(data->colInd);
      gpu_free(data->values);
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      gpu_free(data->rowPtr);
      gpu_free(data->colInd);
      gpu_free(data->values);
    } break;
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      gpu_free(data->rowPtr);
      gpu_free(data->colInd);
      gpu_free(data->values);
    } break;
    case (SPMV_FORMAT_HYB): {
      SpmvHybData *data = (SpmvHybData *)format_data;
      hipsparseDestroyHybMat(data->hybMatrix);
    } break;
    case (SPMV_FORMAT_CSR5): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      gpu_free(data->rowPtr);
      gpu_free(data->colInd);
      gpu_free(data->values);
      switch (lib) {
        case (SPMV_LIBRARY_CSR5_AVX512):
	  massert(false, "SpmvOperator::Spmv_free_device -> No aCSR5_free()");
          //aCSR5_free((aCSR5_wrap *)lib_struct);
          break;
        case (SPMV_LIBRARY_CSR5_CUDA):
	  massert(false, "SpmvOperator::Spmv_free_device -> No cuCSR5_free()");
          //cuCSR5_free((cuCSR5_wrap *)lib_struct);
          break;
        default:
          massert(false,
                  "SpmvOperator::Spmv_free_device -> CSR5 lib default reached "
                  "- something is wrong");
          break;
      }

    } break;
    default:
      massert(false,
              "SpmvOperator::Spmv_free_device -> format default reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_free_device()\n");
}

void SpmvOperator::spmv_free() {
  ddebug(" -> SpmvOperator::spmv_free()\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST): {
      spmv_free_host();
    } break;
    case (SPMV_MEMTYPE_NUMA): {
      spmv_free_numa();
    } break;
    case (SPMV_MEMTYPE_DEVICE):
    case (SPMV_MEMTYPE_UNIFIED): {
      spmv_free_device();
    } break;
    default:
      massert(false, "SpmvOperator::Spmv_free -> mem_alloc default reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_free()\n");
}

// FIXME: Deprecated function for spliting, must change completely
SpmvOperator **split_nz(SpmvOperator *op, int div) {
  massert(0, "split_nz -> Not implemented\n");
}
/*
SpmvOperator **split_nz(SpmvOperator *op, int div) {
        massert(op->format == SPMV_FORMAT_COO, "split_nz -> Only coo format
supported\n");
  SpmvOperator **split_nz =
      (SpmvOperator **)malloc(div * sizeof(SpmvOperator *));
  const int nze = op->nz / div;
  printf("Initializing spliting with div= %d -> nze= %d\n", div, nze);
  for (int i = 0; i < div; i++) {
    split_nz[i] = new SpmvOperator(*op);
    split_nz[i]->nz = nze;
    split_nz[i]->bytes = split_nz[i]->bytes / div;
    split_nz[i]->flops = 2 * split_nz[i]->nz;
    if (i == div - 1) {
      split_nz[i]->nz = nze + op->nz % div;
      split_nz[i]->flops = 2 * split_nz[i]->nz;
    }
    SpmvCooData *tmp_data = (SpmvCooData *)split_nz[i]->format_data;
    SpmvCooData *cp_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
    hipMallocManaged(&cp_data->rowInd, split_nz[i]->nz * sizeof(int));
    hipMallocManaged(&cp_data->colInd, split_nz[i]->nz * sizeof(int));
    hipMallocManaged(&cp_data->values, split_nz[i]->nz * sizeof(double));

    vec_copy_int(cp_data->rowInd, &(((int *)tmp_data->rowInd)[nze * i]),
                 split_nz[i]->nz, 0);
    vec_copy_int(cp_data->colInd, &(((int *)tmp_data->colInd)[nze * i]),
                 split_nz[i]->nz, 0);
    /// TODO: Update this for other value_types
    vec_copy(cp_data->values, &(((double *)tmp_data->values)[nze * i]),
             split_nz[i]->nz, 0, split_nz[i]->value_type);
    //SpmvFree(split_nz[i]->format_struct, split_nz[i]->mem_alloc);
    split_nz[i]->format_data = cp_data;
  }
  return split_nz;
}
*/

void SpmvOperator::vec_alloc(void *x) {
  ddebug(" -> SpmvOperator::vec_alloc(x)\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      vec_alloc_host(x);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      vec_alloc_device(x);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      vec_alloc_uni(x);
      break;
    case (SPMV_MEMTYPE_NUMA):
      vec_alloc_numa(x);
      break;
    default:
      massert(0,
              "SpmvOperator::vec_alloc -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  ddebug(" <- SpmvOperator::vec_alloc(x)\n");
}

void SpmvOperator::vec_alloc_numa(void *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_numa(x_in)\n");
        massert(false, "SpmvOperator::vec_alloc_numa -> No numa please");
/*
  void *x_tmp, *y_tmp;
  mkl_wrap *wrapper = (mkl_wrap *)lib_struct;
  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT):
      /// Allocate vectors x,y
      x_tmp = (float *)numalloc(m * sizeof(float), 0, wrapper);
      y_tmp = (float *)numalloc(n * sizeof(float), 0, wrapper);
      memset(y_tmp, 0, n * sizeof(float));
      /// Initialize vector x to x_in (y memset to 0)
      vec_copy<float>((float *)x_tmp, (float *)x_in, m, 0);
      break;
    case (SPMV_VALUETYPE_DOUBLE):
      /// Allocate vectors x,y
      x_tmp = (double *)numalloc(m * sizeof(double), 0, wrapper);
      y_tmp = (double *)numalloc(n * sizeof(double), 0, wrapper);
      memset(y_tmp, 0, n * sizeof(double));
      /// Initialize vector x to x_in (y memset to 0)
      vec_copy<double>((double *)x_tmp, (double *)x_in, m, 0);
      break;
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(
          false,
          "SpmvOperator::vec_alloc_numa -> Unsupported SpMV value datatype");
  }

  massert(x_tmp && y_tmp,
          "SpmvOperator::vec_alloc_numa -> Vector Alloc failed");

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_numa -> previous x and y allocated, "
        "deleting...\n");
    vec_free(x, m * sizeof(double), mem_alloc);
    vec_free(y, n * sizeof(double), mem_alloc);
  } else {
    switch (value_type) {
      case (SPMV_VALUETYPE_FLOAT):
        mem_bytes += (m + n) * sizeof(float);
        break;
      case (SPMV_VALUETYPE_DOUBLE):
        mem_bytes += (m + n) * sizeof(double);
        break;
      case (SPMV_VALUETYPE_INT):
      case (SPMV_VALUETYPE_BINARY):
      default:
        massert(
            false,
            "SpmvOperator::vec_alloc_numa -> Unsupported SpMV value datatype");
    }
  }
  x = x_tmp;
  y = y_tmp;
*/
  ddebug(" <- SpmvOperator::vec_alloc_numa(x_in)\n");
}

void SpmvOperator::vec_alloc_host(void *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_host(x_in)\n");
  void *x_tmp, *y_tmp;

  /*
    switch (value_type) {
      case (SPMV_VALUETYPE_FLOAT):
        /// Allocate vectors x,y
        x_tmp = (float *)malloc(m * sizeof(float));
        y_tmp = (float *)calloc(n, sizeof(float));
        /// Initialize vector x to x_in (y was calloc'ed)
        vec_copy<float>((float *)x_tmp, (float *)x_in, m, 0);
        break;
      case (SPMV_VALUETYPE_DOUBLE):
        /// Allocate vectors x,y
        x_tmp = (double *)malloc(m * sizeof(double));
        y_tmp = (double *)calloc(n, sizeof(double));
        /// Initialize vector x to x_in (y was calloc'ed)
        vec_copy<double>((double *)x_tmp, (double *)x_in, m, 0);
        break;
      case (SPMV_VALUETYPE_INT):
      case (SPMV_VALUETYPE_BINARY):
      default:
        massert(
            false,
            "SpmvOperator::vec_alloc_host -> Unsupported SpMV value datatype");
    }
  */
  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT):
      /// Allocate vectors x,y
      hipHostAlloc(&x_tmp, n * sizeof(float), hipHostMallocDefault);
      hipHostAlloc(&y_tmp, m * sizeof(float), hipHostMallocDefault);
      /// Initialize vector x to x_in (y was calloc'ed)
      vec_copy<float>((float *)x_tmp, (float *)x_in, n, 0);
      break;
    case (SPMV_VALUETYPE_DOUBLE):
      /// Allocate vectors x,y
      hipHostAlloc(&x_tmp, n * sizeof(double), hipHostMallocDefault);
      hipHostAlloc(&y_tmp, m * sizeof(double), hipHostMallocDefault);
      /// Initialize vector x to x_in (y was calloc'ed)
      vec_copy<double>((double *)x_tmp, (double *)x_in, n, 0);
      break;
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(
          false,
          "SpmvOperator::vec_alloc_host -> Unsupported SpMV value datatype");
  }
  massert(x_tmp && y_tmp,
          "SpmvOperator::vec_alloc_host -> Vector Alloc failed");

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_host -> previous x and y allocated, "
        "deleting...\n");
    vec_free(x, n * sizeof(double), mem_alloc);
    vec_free(y, m * sizeof(double), mem_alloc);
  } else {
    switch (value_type) {
      case (SPMV_VALUETYPE_FLOAT):
        mem_bytes += (m + n) * sizeof(float);
        break;
      case (SPMV_VALUETYPE_DOUBLE):
        mem_bytes += (m + n) * sizeof(double);
        break;
      case (SPMV_VALUETYPE_INT):
      case (SPMV_VALUETYPE_BINARY):
      default:
        massert(
            false,
            "SpmvOperator::vec_alloc_host -> Unsupported SpMV value datatype");
    }
  }
  x = x_tmp;
  y = y_tmp;
  ddebug(" <- SpmvOperator::vec_alloc_host(x_in)\n");
}

void SpmvOperator::vec_alloc_uni(void *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_uni(x)\n");
  void *x_tmp, *y_tmp;

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT):
      /// Allocate vectors x,y
      hipMallocManaged(&x_tmp, n * sizeof(float));
      hipMallocManaged(&y_tmp, m * sizeof(float));
      /// Initialize vectors (x,y) to (x_in,0)
      vec_copy<float>((float *)x_tmp, (float *)x_in, n, 0);
      for (int i = 0; i < m; i++) ((float *)y_tmp)[i] = 0;
      break;
    case (SPMV_VALUETYPE_DOUBLE):
      /// Allocate vectors x,y
      hipMallocManaged(&x_tmp, n * sizeof(double));
      hipMallocManaged(&y_tmp, m * sizeof(double));
      /// Initialize vectors (x,y) to (x_in,0)
      vec_copy<double>((double *)x_tmp, (double *)x_in, n, 0);
      for (int i = 0; i < m; i++) ((double *)y_tmp)[i] = 0;
      break;
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(false,
              "SpmvOperator::vec_alloc_uni -> Unsupported SpMV value datatype");
  }

  massert(x_tmp && y_tmp, "SpmvOperator::vec_alloc_uni -> Vector Alloc failed");

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_uni -> previous x and y allocated, "
        "deleting...\n");
    vec_free(x, n * sizeof(double), mem_alloc);
    vec_free(y, m * sizeof(double), mem_alloc);

  } else {
    switch (value_type) {
      case (SPMV_VALUETYPE_FLOAT):
        mem_bytes += (m + n) * sizeof(float);
        gpu_mem_bytes += (m + n) * sizeof(float);
        break;
      case (SPMV_VALUETYPE_DOUBLE):
        mem_bytes += (m + n) * sizeof(double);
        gpu_mem_bytes += (m + n) * sizeof(double);
        break;
      case (SPMV_VALUETYPE_INT):
      case (SPMV_VALUETYPE_BINARY):
      default:
        massert(
            false,
            "SpmvOperator::vec_alloc_uni -> Unsupported SpMV value datatype");
    }
  }
  x = x_tmp;
  y = y_tmp;
  ddebug(" <- SpmvOperator::vec_alloc_uni(x)\n");
}

void SpmvOperator::vec_alloc_device(void *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_device(x)\n");
  void *x_tmp, *y_tmp;

  switch (value_type) {
    case (SPMV_VALUETYPE_FLOAT):
      /// Allocate vectors x,y
      x_tmp = (float *)gpu_alloc(n * sizeof(float));
      y_tmp = (float *)gpu_alloc(m * sizeof(float));
      /// Initialize vectors (x,y) to (x_in,0)
      copy_to_gpu(x_in, x_tmp, n * sizeof(float));
      hipMemset(y_tmp, 0, m * sizeof(float));
      break;
    case (SPMV_VALUETYPE_DOUBLE):
      /// Allocate vectors x,y
      x_tmp = (double *)gpu_alloc(n * sizeof(double));
      y_tmp = (double *)gpu_alloc(m * sizeof(double));
      /// Initialize vectors (x,y) to (x_in,0)
      copy_to_gpu(x_in, x_tmp, n * sizeof(double));
      hipMemset(y_tmp, 0, m * sizeof(double));
      break;
    case (SPMV_VALUETYPE_INT):
    case (SPMV_VALUETYPE_BINARY):
    default:
      massert(
          false,
          "SpmvOperator::vec_alloc_device -> Unsupported SpMV value datatype");
  }
  cudaCheckErrors();

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_device -> previous x and y allocated, "
        "deleting...");
    vec_free(x, n * sizeof(double), mem_alloc);
    vec_free(y, m * sizeof(double), mem_alloc);
  } else {
    switch (value_type) {
      case (SPMV_VALUETYPE_FLOAT):
        gpu_mem_bytes += (m + n) * sizeof(float);
        break;
      case (SPMV_VALUETYPE_DOUBLE):
        gpu_mem_bytes += (m + n) * sizeof(double);
        break;
      case (SPMV_VALUETYPE_INT):
      case (SPMV_VALUETYPE_BINARY):
      default:
        massert(false,
                "SpmvOperator::vec_alloc_device -> Unsupported SpMV value "
                "datatype");
    }
  }
  x = x_tmp;
  y = y_tmp;
  ddebug(" <- SpmvOperator::vec_alloc_device(x)\n");
}

void *SpmvOperator::spmv_data_get_copy() {
  ddebug(" -> SpmvOperator::spmv_data_get_copy()\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      return spmv_data_copy_host();
    case (SPMV_MEMTYPE_DEVICE):
      return spmv_data_copy_device();
    case (SPMV_MEMTYPE_UNIFIED):
      return spmv_data_copy_uni();
    default:
      massert(0,
              "SpmvOperator::spmv_data_copy -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  return NULL;
}

void *SpmvOperator::spmv_data_copy_uni() {
  ddebug(" -> SpmvOperator::spmv_data_get_copy_uni()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      SpmvCooData *cp_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
      if (data->rowInd && data->colInd && data->values) {
        hipMallocManaged(&cp_data->rowInd, nz * sizeof(int));
        hipMallocManaged(&cp_data->colInd, nz * sizeof(int));

        vec_copy<int>(cp_data->rowInd, data->rowInd, nz, 0);
        vec_copy<int>(cp_data->colInd, data->colInd, nz, 0);

        switch (value_type) {
          case (SPMV_VALUETYPE_FLOAT):
            hipMallocManaged(&cp_data->values, nz * sizeof(float));
            vec_copy<float>((float *)cp_data->values, (float *)data->values, nz,
                            0);
            break;
          case (SPMV_VALUETYPE_DOUBLE):
            hipMallocManaged(&cp_data->values, nz * sizeof(double));
            vec_copy<double>((double *)cp_data->values, (double *)data->values,
                             nz, 0);
            break;
          case (SPMV_VALUETYPE_INT):
          case (SPMV_VALUETYPE_BINARY):
          default:
            massert(false,
                    "SpmvOperator::spmv_data_copy_uni -> Unsupported SpMV "
                    "value datatype");
        }

      } else
        debug(
            "SpmvOperator::spmv_data_copy_uni -> warning... empty Spmv struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      SpmvCsrData *cp_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));
      if (data->rowPtr && data->colInd && data->values) {
        hipMallocManaged(&cp_data->rowPtr, (m + 1) * sizeof(int));
        hipMallocManaged(&cp_data->colInd, nz * sizeof(int));

        vec_copy<int>(cp_data->rowPtr, data->rowPtr, m + 1, 0);
        vec_copy<int>(cp_data->colInd, data->colInd, nz, 0);

        switch (value_type) {
          case (SPMV_VALUETYPE_FLOAT):
            hipMallocManaged(&cp_data->values, nz * sizeof(float));
            vec_copy<float>((float *)cp_data->values, (float *)data->values, nz,
                            0);
            break;
          case (SPMV_VALUETYPE_DOUBLE):
            hipMallocManaged(&cp_data->values, nz * sizeof(double));
            vec_copy<double>((double *)cp_data->values, (double *)data->values,
                             nz, 0);
            break;
          case (SPMV_VALUETYPE_INT):
          case (SPMV_VALUETYPE_BINARY):
          default:
            massert(false,
                    "SpmvOperator::spmv_data_copy_uni -> Unsupported SpMV "
                    "value datatype");
        }

      } else
        debug(
            "SpmvOperator::spmv_data_copy_uni -> warning... empty Spmv struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      SpmvBsrData *cp_data = (SpmvBsrData *)malloc(sizeof(SpmvBsrData));
      const int nb = (n + data->blockDim - 1) / data->blockDim;
      const int mb = (m + data->blockDim - 1) / data->blockDim;
      if (data->rowPtr && data->colInd && data->values) {
        hipMallocManaged(&cp_data->rowPtr, (nb + 1) * sizeof(int));
        hipMallocManaged(&cp_data->colInd, data->nnzb * sizeof(int));

        vec_copy<int>(cp_data->rowPtr, data->rowPtr, nb + 1, 0);
        vec_copy<int>(cp_data->colInd, data->colInd, data->nnzb, 0);

        switch (value_type) {
          case (SPMV_VALUETYPE_FLOAT):
            hipMallocManaged(
                &cp_data->values,
                (data->blockDim * data->blockDim) * data->nnzb * sizeof(float));
            vec_copy<float>((float *)cp_data->values, (float *)data->values,
                            (data->blockDim * data->blockDim) * data->nnzb, 0);
            break;
          case (SPMV_VALUETYPE_DOUBLE):
            hipMallocManaged(&cp_data->values,
                              (data->blockDim * data->blockDim) * data->nnzb *
                                  sizeof(double));
            vec_copy<double>((double *)cp_data->values, (double *)data->values,
                             (data->blockDim * data->blockDim) * data->nnzb, 0);
            break;
          case (SPMV_VALUETYPE_INT):
          case (SPMV_VALUETYPE_BINARY):
          default:
            massert(false,
                    "SpmvOperator::spmv_data_copy_uni -> Unsupported SpMV "
                    "value datatype");
        }

        cp_data->nnzb = data->nnzb;
        cp_data->blockDim = data->blockDim;
      } else
        debug(
            "SpmvOperator::spmv_data_copy_uni -> warning... empty Spmv struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_DIA):
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_ELL):
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> SPMV_FORMAT_ELL not "
              "implemented");
    case (SPMV_FORMAT_HYB):
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> SPMV_FORMAT_HYB not "
              "implemented");
    default:
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> Unreacheable format default "
              "reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
  return NULL;
}

void *SpmvOperator::spmv_data_copy_device() {
  ddebug(" -> SpmvOperator::spmv_data_get_copy_device()\n");
  massert(0, "SpmvOperator::spmv_data_copy_device -> Not implemented");
  ddebug(" <- SpmvOperator::spmv_data_get_copy_device()\n");
  return NULL;
}

void *SpmvOperator::spmv_data_get_subcopy(int *start, int *nzc, int mode) {
  ddebug(" -> SpmvOperator::spmv_data_get_subcopy()\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      return spmv_data_subcopy_host(start, nzc, mode);
    case (SPMV_MEMTYPE_DEVICE):
      return spmv_data_subcopy_device(start, nzc, mode);
    case (SPMV_MEMTYPE_UNIFIED):
      return spmv_data_subcopy_uni(start, nzc, mode);
    default:
      massert(0,
              "SpmvOperator::spmv_data_subcopy -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  return NULL;
}

void *SpmvOperator::spmv_data_subcopy_uni(int *start, int *nzc, int mode) {
  ddebug(" -> SpmvOperator::spmv_data_subcopy_uni()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      SpmvCooData *cp_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
      if (data->rowInd && data->colInd && data->values) {
        if (mode == 0)
          while (data->rowInd[*nzc - 1] == data->rowInd[*nzc]) (*nzc)++;
        else if (mode == 1)
          while (data->rowInd[*start - 1] == data->rowInd[*start]) {
            (*start)++;
            (*nzc)--;
          }
        else
          massert(false,
                  "SpmvOperator::spmv_data_subcopy_uni -> unsupported subcopy "
                  "mode");
        hipMallocManaged(&cp_data->rowInd, *nzc * sizeof(int));
        hipMallocManaged(&cp_data->colInd, *nzc * sizeof(int));

        vec_copy<int>(cp_data->rowInd, &(data->rowInd[*start]), *nzc, 0);
        vec_copy<int>(cp_data->colInd, &(data->colInd[*start]), *nzc, 0);

        if (mode == 1) {
          for (int i = 1; i < *nzc; i++)
            cp_data->rowInd[i] = cp_data->rowInd[i] - cp_data->rowInd[0];
          cp_data->rowInd[0] = 0;
        }

        switch (value_type) {
          case (SPMV_VALUETYPE_FLOAT):
            hipMallocManaged(&cp_data->values, *nzc * sizeof(float));
            vec_copy<float>((float *)cp_data->values,
                            &((float *)data->values)[*start], *nzc, 0);
            break;
          case (SPMV_VALUETYPE_DOUBLE):
            hipMallocManaged(&cp_data->values, *nzc * sizeof(double));
            vec_copy<double>((double *)cp_data->values,
                             &((double *)data->values)[*start], *nzc, 0);
            break;
          case (SPMV_VALUETYPE_INT):
          case (SPMV_VALUETYPE_BINARY):
          default:
            massert(false,
                    "SpmvOperator::spmv_data_subcopy_uni -> Unsupported SpMV "
                    "value datatype");
        }

      } else
        debug(
            "SpmvOperator::spmv_data_subcopy_uni -> warning... empty Spmv "
            "struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_subcopy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_CSR):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_BSR):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_DIA):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_ELL):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_ELL not "
              "implemented");
    case (SPMV_FORMAT_HYB):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_HYB not "
              "implemented");
    default:
      massert(
          0,
          "SpmvOperator::spmv_data_subcopy_uni -> Unreacheable format default "
          "reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_data_subcopy_uni()\n");
  return NULL;
}

void *SpmvOperator::spmv_data_subcopy_device(int *start, int *nzc, int mode) {
  ddebug(" -> SpmvOperator::spmv_data_subcopy_device()\n");
  massert(0, "SpmvOperator::spmv_data_subcopy_device -> Not implemented");
  ddebug(" <- SpmvOperator::spmv_data_subcopy_device()\n");
  return NULL;
}

void SpmvOperator::mem_convert(SpmvMemType target_mem) {
  ddebug(" -> SpmvOperator::mem_convert(target_mem)\n");
  switch (target_mem) {
    case (SPMV_MEMTYPE_HOST):
      mem_convert_host();
      break;
    case (SPMV_MEMTYPE_DEVICE):
      mem_convert_device();
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      mem_convert_uni();
      break;
    case (SPMV_MEMTYPE_NUMA):
      mem_convert_numa();
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  ddebug(" <- SpmvOperator::mem_convert(target_mem)\n");
}

void SpmvOperator::mem_convert_uni() {
  ddebug(" -> SpmvOperator::mem_convert_uni()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      newptr = spmv_data_copy_uni();
      spmv_free_host();
      format_data = newptr;
      vec_alloc_uni(x);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      debug(
          "SpmvOperator::mem_convert_uni -> Data already in unified memory\n");
      break;
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_uni -> warning... "
          "SpmvOperator::spmv_data_copy_uni from device is not properly "
          "tested\n");
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_uni -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_UNIFIED;
  ddebug(" <- SpmvOperator::mem_convert_uni()\n");
}

void SpmvOperator::mem_convert_host() {
  ddebug(" -> SpmvOperator::mem_convert_host()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      debug("SpmvOperator::mem_convert_host -> Data already in host memory\n");
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      newptr = spmv_data_copy_host();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_host(x);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_host -> warning... "
          "SpmvOperator::spmv_data_copy_uni from device is not properly "
          "tested\n");
      /// Convert to unified from device
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      /// Convert to host from unified
      newptr = spmv_data_copy_host();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_host(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_host -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_HOST;
  ddebug(" <- SpmvOperator::mem_convert_host()\n");
}

void SpmvOperator::mem_convert_numa() {
  ddebug(" -> SpmvOperator::mem_convert_numa()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_NUMA):
      debug("SpmvOperator::mem_convert_numa -> Data already in numa memory\n");
      break;
    case (SPMV_MEMTYPE_HOST):
      newptr = spmv_data_copy_numa();
      spmv_free_host();
      format_data = newptr;
      vec_alloc_numa(x);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      newptr = spmv_data_copy_numa();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_numa(x);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_numa -> warning... "
          "SpmvOperator::spmv_data_copy_uni from device is not properly "
          "tested\n");
      /// Convert to unified from device
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      /// Convert to host from unified
      newptr = spmv_data_copy_numa();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_numa(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_numa -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_NUMA;
  ddebug(" <- SpmvOperator::mem_convert_numa()\n");
}

void SpmvOperator::mem_convert_device() {
  ddebug(" -> SpmvOperator::mem_convert_device()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_device -> Data already in device memory");
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      newptr = spmv_data_copy_device();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_device(x);
    case (SPMV_MEMTYPE_HOST):
      /// Convert to unified from host
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      /// Convert to device from unified
      debug(
          "SpmvOperator::mem_convert_device -> warning... "
          "SpmvOperator::spmv_data_copy_device from unified is not properly "
          "tested");
      newptr = spmv_data_copy_device();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_device(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_device -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_DEVICE;
  ddebug(" <- SpmvOperator::mem_convert_device()\n");
}

void SpmvOperator::format_convert(SpmvFormat target_format) {
  ddebug(" -> SpmvOperator::format_convert(target_format)\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      format_convert_host(target_format);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      format_convert_device(target_format);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      format_convert_uni(target_format);
      break;
    default:
      massert(0,
              "SpmvOperator::format_convert_type -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  ddebug(" <- SpmvOperator::format_convert(target_format)\n");
}

void SpmvOperator::format_convert_uni(SpmvFormat target_format) {
  ddebug(" -> SpmvOperator::format_convert_uni(target_format)\n");
  switch (target_format) {
    case (SPMV_FORMAT_COO): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          debug(
              "SpmvOperator::format_convert_uni -> Struct already in the "
              "correct format\n");
          break;
        case (SPMV_FORMAT_CSR):
          format_convert_uni_csr2coo();
          break;
        case (SPMV_FORMAT_BSR):
          format_convert_uni_bsr2csr();
          format_convert_uni_csr2coo();
          break;
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          format_convert_uni_hyb2csr();
          format_convert_uni_csr2coo();
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    case (SPMV_FORMAT_CSR): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          format_convert_uni_coo2csr();
          break;
        case (SPMV_FORMAT_CSR):
          debug(
              "SpmvOperator::format_convert_uni -> Struct already in the "
              "correct format");
          break;
        case (SPMV_FORMAT_BSR):
          format_convert_uni_bsr2csr();
          break;
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          format_convert_uni_hyb2csr();
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    case (SPMV_FORMAT_BSR): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          format_convert_uni_coo2csr();
          format_convert_uni_csr2bsr();
          break;
        case (SPMV_FORMAT_CSR):
          format_convert_uni_csr2bsr();
          break;
        case (SPMV_FORMAT_BSR):
          if (bsr_blockDim != ((SpmvBsrData *)format_data)->blockDim)
            format_convert_uni_bsr2bsr();
          else
            debug(
                "SpmvOperator::format_convert_uni -> Already in the correct "
                "bsr format");
          break;
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          format_convert_uni_hyb2csr();
          format_convert_uni_csr2bsr();
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    case (SPMV_FORMAT_DIA):
      massert(0,
              "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA target not "
              "supported");
      break;
    case (SPMV_FORMAT_ELL):
      massert(0,
              "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL target not "
              "supported");
      break;
    case (SPMV_FORMAT_HYB): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          format_convert_uni_coo2csr();
          format_convert_uni_csr2hyb();
          break;
        case (SPMV_FORMAT_CSR):
          format_convert_uni_csr2hyb();
          break;
        case (SPMV_FORMAT_BSR):
          format_convert_uni_bsr2csr();
          format_convert_uni_csr2hyb();
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          debug(
              "SpmvOperator::format_convert_uni -> Struct already in the "
              "correct format");
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    default:
      massert(0,
              "SpmvOperator::format_convert_uni -> Unreacheable target format "
              "default reached");
      break;
  }
  ddebug(" <- SpmvOperator::format_convert_uni(target_format)\n");
}

void SpmvOperator::format_convert_device(SpmvFormat target_format) {
  ddebug(" -> SpmvOperator::format_convert_device(target_format)\n");
  massert(0, "SpmvOperator::format_convert_device -> Not Implemented");
  ddebug(" <- SpmvOperator::format_convert_device(target_format)\n");
}

void *SpmvOperator::y_get_copy() {
  ddebug(" -> SpmvOperator::y_get_copy()\n");
  void *out;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
    case (SPMV_MEMTYPE_NUMA):
    case (SPMV_MEMTYPE_UNIFIED):
      switch (value_type) {
        case (SPMV_VALUETYPE_FLOAT):
          out = malloc(m * sizeof(float));
          vec_copy<float>((float *)out, (float *)y, m, 0);
          break;
        case (SPMV_VALUETYPE_DOUBLE):
          out = malloc(m * sizeof(double));
          vec_copy<double>((double *)out, (double *)y, m, 0);
          break;
        case (SPMV_VALUETYPE_INT):
        case (SPMV_VALUETYPE_BINARY):
        default:
          massert(
              false,
              "SpmvOperator::y_get_copy -> Unsupported SpMV value datatype");
      }
      break;
    case (SPMV_MEMTYPE_DEVICE):
      switch (value_type) {
        case (SPMV_VALUETYPE_FLOAT):
          out = (float *)malloc(m * sizeof(float));
          copy_from_gpu(out, y, m * sizeof(float));
          break;
        case (SPMV_VALUETYPE_DOUBLE):
          out = (double *)malloc(m * sizeof(double));
          copy_from_gpu(out, y, m * sizeof(double));
          break;
        case (SPMV_VALUETYPE_INT):
        case (SPMV_VALUETYPE_BINARY):
        default:
          massert(
              false,
              "SpmvOperator::y_get_copy -> Unsupported SpMV value datatype");
      }

      cudaCheckErrors();
      break;
    default:
      massert(0, "SpmvOperator::y_get_copy op->mem_alloc type unsupported");
  }
  ddebug(" <- SpmvOperator::y_get_copy()\n");
  return out;
}

void SpmvOperator::free_lib_struct() {
  ddebug(" -> SpmvOperator::free_lib_struct()\n");
  switch (lib) {
    case (SPMV_NONE):
      debug(
          "SpmvOperator::free_lib_struct -> Tried to free SPMV_NONE "
          "lib_struct");
      break;
    case (SPMV_LIBRARY_CUSPARSE):
      cuSPARSE_free((cuSPARSE_wrap *)lib_struct);
      break;
    default:
      massert(
          0,
          "SpmvOperator::free_lib_struct -> lib unreachable default reached");
  }
  ddebug(" <- SpmvOperator::free_lib_struct()\n");
}

int SpmvOperator::count_transactions() {
  ddebug(" -> SpmvOperator::count_transactions()\n");
  massert(format_data != NULL, "count_transactions -> No format struct");
  int ctr = 0, *exists = (int *)calloc(n, sizeof(int));
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      for (int i = 0; i < nz; i++) exists[data->colInd[i]] = 1;
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      for (int i = 0; i < nz; i++) exists[data->colInd[i]] = 1;
    } break;
    default:
      free(exists);
      massert(false, "count_transactions -> Unsupported SpMV format");
      break;
  }
  for (int i = 0; i < n; i++)
    if (exists[i]) ctr++;
  //free(exists);
  ddebug(" <- SpmvOperator::count_transactions()\n");
  return ctr;
}

/*
void SpmvOperator::op_transmute(SpmvMemType target_mem) {
  if (format_data) {
    if (target_mem == mem_alloc)
      debug("SpmvOperator::op_transmute -> Spmv struct already in the
correct mem_alloc\n");
    else {
      switch (target_mem) {
        case (SPMV_MEMTYPE_HOST):
          massert(0, "SpmvOperator::op_transmute -> Transmute to host memory
not supported");
          break;
        case (SPMV_MEMTYPE_DEVICE):
          device_convert(op->format_struct->format);
          break;
        case (SPMV_MEMTYPE_UNIFIED):
          unified_convert(op->format_struct->format);
          break;
        case (SPMV_MEMTYPE_CUSP):
          cuSP_convert(op->format_struct->format);
          break;

        default:
          massert(0, "op_transmute: op->mem_alloc unreachable default
reached");
      }
    }
  } else {
    op->format_struct->data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
    op->mem_alloc = target_mem;  // FIXME: This could be wrong
    op->util->timer = csecond();
    mtx_read(op);
    op->util->timer = csecond() - op->util->timer;
    printf("op_transmute: .mtx file input done in %lf s\n",
op->util->timer);
  }
  if (op->vectors->x && op->vectors->y && op->vectors->y_check) {
    debug("op_transmute: vectors already initialized\n");
    if (target_mem == op->mem_alloc)
      debug("op_transmute: vectors already in the correct mem_alloc\n");
    else {
      switch (target_mem) {
        case (SPMV_MEMTYPE_HOST):
          massert(0, "op_transmute: Transmute to host memory not
supported");
          break;
        case (SPMV_MEMTYPE_DEVICE):
          /// FIXME: this is just for testing, not generic and wrong
          double *x, *gpu_x, *gpu_y;
          x = (double *)malloc(op->size->m * sizeof(double));
          vec_init_rand((double *)x, op->size->m, 0);
          hipMalloc((void **)&gpu_x, op->size->m * sizeof(double));
          hipMalloc((void **)&gpu_y, op->size->n * sizeof(double));
          hipMemcpy(gpu_x, x, op->size->m * sizeof(double),
                     hipMemcpyHostToDevice);
          hipMemset(gpu_y, 0, op->size->n * sizeof(double));
          free(x);
          op->vectors->x = gpu_x;
          op->vectors->y = gpu_y;
          break;
        case (SPMV_MEMTYPE_UNIFIED):
          massert(0, "op_transmute: Transmute to unified memory not
supported");
          break;
        case (SPMV_MEMTYPE_CUSP): {
          /// FIXME: this is just for testing, not generic and wrong
          double *x;
          x = (double *)malloc(op->size->m * sizeof(double));
          vec_init_rand((double *)x, op->size->m, 0);
          vec_alloc_cuSP(op, (void **)&x);
          free(x);
        } break;
        default:
          massert(0, "op_transmute: op->mem_alloc unreachable default
reached");
      }
    }
  } else {
    /// Initialize random x if not present
    double *x;
    x = (double *)malloc(op->size->m * sizeof(double));
    vec_init_rand((double *)x, op->size->m, 0);
    op->vectors->y_check = (double *)calloc(op->size->n, sizeof(double));
    vec_alloc(op, (void **)&x);
  }
  op->mem_alloc = target_mem;
}
*/
