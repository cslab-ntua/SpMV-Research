///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///

#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

#include "nvmlPower.hpp"

int main(int argc, char **argv) {
  /// Check Input
  massert(argc == 4 || argc ==5,
          "Incorrect arguments.\nUsage:\t./Executable logfilename spmv_implementation [blockdim] Matrix_name.mtx ./Executable logfilename spmv_implementation [blockdim] Mpakos_7_parameters_with_spaces");
          
  // Set/Check for device
  int device_id = 1;
  hipSetDevice(device_id);
  hipGetDevice(&device_id);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;
  
  char *name = argv[3], *outfile = argv[1];
  int impFlag = 0, blockdim = 4; /// Default case; run ALL versions (might break for 
  void *y_out, *y_out1;
  double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;
  /// Check which version to run!! Important: Names must match strings here
  char* tempstr = argv[2];
  if (strstr(tempstr, "OMP_csr")) impFlag = 1;
  else if (strstr(tempstr, "cuSPARSE_csr")) impFlag = 2;
  else if (strstr(tempstr, "cuSPARSE_hyb")) impFlag = 3;
  else if (strstr(tempstr, "cuSPARSE_bsr")){
    impFlag = 4;
    if(argc == 5){
    	blockdim = atoi(argv[3]);
    	name = argv[4]; 
    }
  }
  if (!impFlag) fprintf(stderr,"WARNING: Running all implementations might crash due to stuff\n"); 
  FILE *fp = fopen(name, "r");
  massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");
  
  /// Mix C & C++ file inputs, because...?
  ofstream foutp;
  foutp.open(outfile, ios::out | ios::app ); 
  massert(foutp.is_open() , "Invalid output File");
  // print_devices();
  
  exc_timer = csecond();
  SpmvOperator op(name);
  exc_timer = csecond() - exc_timer;
 
  fprintf(stdout,
      "File=%s ( n=%d m=%d trans=%d n_z=%d Sparsity=%lf ) Input time=%lf s\n",
      op.mtx_name, op.n, op.m, op.count_transactions(), op.nz,
      1.0 * op.nz / op.n / op.m, exc_timer);
  VALUE_TYPE *x = (VALUE_TYPE *)malloc(op.n * sizeof(VALUE_TYPE));
  VALUE_TYPE *out = (VALUE_TYPE *)calloc(op.n, sizeof(VALUE_TYPE));
  vec_init_rand<VALUE_TYPE>(x, op.n, 0);
  op.vec_alloc((VALUE_TYPE*)x);

  //cuSPARSE_op.print_op();
  //SpmvOperator openmp_op(cuSPARSE_op);
  SpmvOperator cuSPARSE_op(op);
  cuSPARSE_op.cuSPARSE_init();

  SpmvOperator bsr_op(op);
  bsr_op.cuSPARSE_init();
  
#ifdef TEST
  SpmvOperator openmp_op(cuSPARSE_op);
  openmp_op.mem_convert(SPMV_MEMTYPE_HOST);
  openmp_op.openmp_init();
  
  fprintf(stdout,"Serial-CSR: ");
  op.timer = csecond();
  SpmvCsrData *data = (SpmvCsrData *)op.format_data;
  spmv_csr<VALUE_TYPE>(data->rowPtr, data->colInd, (VALUE_TYPE *) data->values, (VALUE_TYPE *)op.x,
           (VALUE_TYPE *) op.y, op.m);
  op.timer = csecond() - op.timer;
  report_results(op.timer * NR_ITER, op.flops, op.bytes);
  fprintf(stdout,"\n");

  fprintf(stdout,"\nRunning tests.. \n");

  if (!impFlag || impFlag == 1){
  	fprintf(stdout,"Testing openmp_csr...\t");
  	openmp_op.openmp_csr();
  	check_result<VALUE_TYPE>((VALUE_TYPE *)openmp_op.y, (VALUE_TYPE *) op.y, openmp_op.m);
  }

  /*
  SpmvOperator csr5_op(cuSPARSE_op);
      csr5_op.format_convert(SPMV_FORMAT_CSR);
      csr5_op.cuCSR5_init();
  printf("Testing cuCSR5_csr...");
  csr5_op.cuCSR5_csr();
  hipDeviceSynchronize();
  check_result<VALUE_TYPE>((VALUE_TYPE *)csr5_op.y, (VALUE_TYPE *)op.y, openmp_op.n);


      SpmvOperator acsr5_op(cuSPARSE_op);
      acsr5_op.format_convert(SPMV_FORMAT_CSR);
      acsr5_op.mem_convert(SPMV_MEMTYPE_HOST);
      acsr5_op.avx512CSR5_init();
  printf("Testing avx512CSR5_csr...");
  acsr5_op.avx512CSR5_csr();
  check_result<VALUE_TYPE>((VALUE_TYPE *)acsr5_op.y, (VALUE_TYPE *)op.y, openmp_op.n);
  */
  if (!impFlag || impFlag == 2){
  	fprintf(stdout,"Testing cuSPARSE_csr...\t");
  	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);
  	cuSPARSE_op.cuSPARSE_csr();
  	hipDeviceSynchronize();
  	check_result<VALUE_TYPE>((VALUE_TYPE *)cuSPARSE_op.y, (VALUE_TYPE *)op.y, cuSPARSE_op.m);
  }

  if (!impFlag || impFlag == 3){
  	fprintf(stdout,"Testing cuSPARSE_hyb...\t");
  	cuSPARSE_op.format_convert(SPMV_FORMAT_HYB);
  	cuSPARSE_op.cuSPARSE_hyb();
  	hipDeviceSynchronize();
  	check_result<VALUE_TYPE>((VALUE_TYPE *)cuSPARSE_op.y, (VALUE_TYPE *)op.y, cuSPARSE_op.m);
  }

  if (!impFlag || impFlag == 4){
  	fprintf(stdout,"Testing cuSPARSE_bsr-%d...\t", blockdim);
  	/// Convert to cuSPARSE bsr
  	bsr_op.bsr_blockDim = blockdim;
  	bsr_op.format_convert(SPMV_FORMAT_BSR);
  	bsr_op.cuSPARSE_bsr();
  	hipDeviceSynchronize();
  	check_result<VALUE_TYPE>((VALUE_TYPE *)bsr_op.y, (VALUE_TYPE *)op.y, bsr_op.m);
  }

#endif

  if (!impFlag || impFlag == 1){
  	// Warmup
  	for (int i = 0; i < 100; i++) openmp_op.openmp_csr();

  	// Run OpenMP csr
  	int threadNum = get_num_threads(); 
  	fprintf(stdout,"Timing openmp_csr-%d...\n", threadNum);
  	openmp_op.timer = csecond();
  	for (int i = 0; i < NR_ITER; i++) openmp_op.openmp_csr();
  	openmp_op.timer = csecond() - openmp_op.timer;
  	gflops_s = 2*openmp_op.nz*1e-9/openmp_op.timer*NR_ITER;
  	std::cout << name << "," << "OMP_csr-" << threadNum << "," << openmp_op.m << "," << openmp_op.n << "," << openmp_op.nz << "," << openmp_op.timer << "," << gflops_s << "\n";
  	foutp << name << "," << "OMP_csr-" << threadNum << "," << openmp_op.m << "," << openmp_op.n << "," << openmp_op.nz << "," << openmp_op.timer << "," << gflops_s << "\n";

  	cpu_timer = openmp_op.timer;
  }

  if (!impFlag || impFlag == 2){
  	/// Run cuSPARSE csr
  	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);

  	// Warmup
  	for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_csr();
  	hipDeviceSynchronize();

  	// Run cuSPARSE csr
  	fprintf(stdout,"Timing cuSPARSE_csr...\n");
  	nvmlAPIRun();
  	cuSPARSE_op.timer = csecond();
  	for (int i = 0; i < NR_ITER; i++) {
    		cuSPARSE_op.cuSPARSE_csr();
    		hipDeviceSynchronize();
  	}
	cudaCheckErrors();
  	cuSPARSE_op.timer = (csecond() - cuSPARSE_op.timer)/NR_ITER;
  	nvmlAPIEnd();
  	gflops_s = 2*cuSPARSE_op.nz*1e-9/cuSPARSE_op.timer;
  	std::cout << name << "," << "cuSPARSE_csr" << "," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.nz << "," << cuSPARSE_op.timer << "," << gflops_s << "\n";
  	foutp << name << "," << "cuSPARSE_csr" << "," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.nz << "," << cuSPARSE_op.timer << "," << gflops_s << "\n";
  }

  if (!impFlag || impFlag == 3){
  	/// Convert to cuSPARSE hyb
  	cuSPARSE_op.format_convert(SPMV_FORMAT_HYB);

  	// Warmup
  	for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_hyb();
  	hipDeviceSynchronize();

  	// Run cuSPARSE hyb
  	fprintf(stdout,"Timing cuSPARSE_hyb...\n");
  	nvmlAPIRun();
  	cuSPARSE_op.timer = csecond();
  	for (int i = 0; i < NR_ITER; i++) {
  	  cuSPARSE_op.cuSPARSE_hyb();
  	  hipDeviceSynchronize();
  	}
	cudaCheckErrors();
  	cuSPARSE_op.timer = (csecond() - cuSPARSE_op.timer)/NR_ITER;
  	nvmlAPIEnd();
  	gflops_s = 2*cuSPARSE_op.nz*1e-9/cuSPARSE_op.timer;
  	std::cout << name << "," << "cuSPARSE_hyb" << "," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.nz << "," << cuSPARSE_op.timer << "," << gflops_s << "\n";
  	foutp << name << "," << "cuSPARSE_hyb" << "," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.nz << "," << cuSPARSE_op.timer << "," << gflops_s << "\n";
  }

  if (!impFlag || impFlag == 4){
  	/// Convert to cuSPARSE bsr
  	bsr_op.bsr_blockDim = blockdim;
  	bsr_op.format_convert(SPMV_FORMAT_BSR);

  	// Warmup
  	for (int i = 0; i < 100; i++) bsr_op.cuSPARSE_bsr();
  	hipDeviceSynchronize();

  	// Run cuSPARSE bsr
  	fprintf(stdout,"Timing cuSPARSE_bsr-%d...\n", blockdim);
  	nvmlAPIRun();
  	bsr_op.timer = csecond();
  	for (int i = 0; i < NR_ITER; i++) {
    		bsr_op.cuSPARSE_bsr();
    		hipDeviceSynchronize();
  	}
	cudaCheckErrors();
  	bsr_op.timer = (csecond() - bsr_op.timer)/NR_ITER;
  	nvmlAPIEnd();
  	gflops_s = 2*bsr_op.nz*1e-9/bsr_op.timer;
  	std::cout << name << "," << "cuSPARSE_bsr-" << blockdim << "," << bsr_op.m << "," << bsr_op.n << "," << bsr_op.nz << "," << bsr_op.timer << "," << gflops_s << "\n";
  	foutp << name << "," << "cuSPARSE_bsr-" << blockdim << "," << bsr_op.m << "," << bsr_op.n << "," << bsr_op.nz << "," << bsr_op.timer << "," << gflops_s << "\n";
  } 

  foutp.close();

}
