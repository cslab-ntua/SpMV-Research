#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///

#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

#include "nvmlPower.hpp"

int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 3,
	  "Incorrect arguments.\nUsage:\t./Executable logfilename Matrix_name.mtx");
	  
	// Set/Check for device
	int device_id = 1;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *name = argv[2], *outfile = argv[1];
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, diagonal_factor = %lf, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.diagonal_factor, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, 
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering );
		
	VALUE_TYPE *x = (VALUE_TYPE *)malloc(op.n * sizeof(VALUE_TYPE));
	VALUE_TYPE *out = (VALUE_TYPE *)malloc(op.m * sizeof(VALUE_TYPE));
	vec_init_rand<VALUE_TYPE>(x, op.n, 0);
	op.vec_alloc((VALUE_TYPE*)x);

	SpmvOperator cuSPARSE_op(op);
	cuSPARSE_op.cuSPARSE_init();
  
#ifdef TEST
  
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
	spmv_csr<VALUE_TYPE>(data->rowPtr, data->colInd, (VALUE_TYPE *) data->values, (VALUE_TYPE *)op.x,
		   (VALUE_TYPE *) op.y, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");


	fprintf(stdout,"Testing cuSPARSE_csr...\t");
	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);
	cuSPARSE_op.cuSPARSE_csr();
	hipDeviceSynchronize();
	check_result<VALUE_TYPE>((VALUE_TYPE *)cuSPARSE_op.y, (VALUE_TYPE *)op.y, cuSPARSE_op.m);


#endif

	/// Run cuSPARSE csr
	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);

	// Warmup
	for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_csr();
	hipDeviceSynchronize();

	// Run cuSPARSE csr
	fprintf(stdout,"Timing cuSPARSE_csr...\n");
	nvmlAPIRun();
	cuSPARSE_op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			cuSPARSE_op.cuSPARSE_csr();
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	cuSPARSE_op.timer = (csecond() - cuSPARSE_op.timer)/NR_ITER;
	if (cuSPARSE_op.timer*NR_ITER < 1.0){
		unsigned int extra_itter = ((unsigned int) 1.0/cuSPARSE_op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			cuSPARSE_op.cuSPARSE_csr();
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	nvmlAPIEnd();
	gflops_s = cuSPARSE_op.flops*1e-9/cuSPARSE_op.timer;
	fprintf(stdout, "cuSPARSE_csr: t = %lf ms (%lf Gflops/s )\n", cuSPARSE_op.timer, gflops_s);
	foutp << cuSPARSE_op.mtx_name << "," << cuSPARSE_op.distribution << "," << cuSPARSE_op.placement << "," << cuSPARSE_op.diagonal_factor << "," << cuSPARSE_op.seed <<
	"," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.density << 
	"," << cuSPARSE_op.nz << "," << cuSPARSE_op.avg_nz_row << "," << cuSPARSE_op.std_nz_row <<
	"," << cuSPARSE_op.avg_bandwidth << "," << cuSPARSE_op.std_bandwidth <<
	"," << cuSPARSE_op.avg_scattering << "," << cuSPARSE_op.std_scattering <<
	"," << "cuSPARSE_csr" <<  "," << cuSPARSE_op.timer << "," << gflops_s << "\n";

	foutp.close();

}
