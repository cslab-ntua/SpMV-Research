///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///

#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

#include "nvmlPower.hpp"

int main(int argc, char **argv) {
  /// Check Input
  massert(argc >= 3 && argc <=5,
          "Incorrect arguments.\nUsage: ./Executable Matrix_name.mtx logfilename [spmv_implementation [[blockdim]]]");
  char *name = argv[1], *outfile = argv[2];
  int impFlag = 0, blockdim = 4; /// Default case; run ALL versions (might break for 
  void *y_out, *y_out1;
  double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;
  /// Check which version to run!! Important: Names must match strings here
  if (argc >= 4){
	char* tempstr = argv[3];
	if (strstr(tempstr, "OMP_csr")) impFlag = 1;
	else if (strstr(tempstr, "cuSPARSE_csr")) impFlag = 2;
	else if (strstr(tempstr, "cuSPARSE_hyb")) impFlag = 3;
	else if (strstr(tempstr, "cuSPARSE_bsr")){
		impFlag = 4;
		if(argc == 5) blockdim = atoi(argv[4]);
	}
  }
  if (!impFlag) fprintf(stderr,"WARNING: Running all implementations might crash due to stuff\n"); 
  FILE *fp = fopen(name, "r");
  massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");
  
  /// Mix C & C++ file inputs, because...?
  ofstream foutp;
  foutp.open(outfile, ios::out | ios::app ); 
  massert(foutp.is_open() , "Invalid output File");
  // print_devices();

  exc_timer = csecond();
  SpmvOperator op(name);
  exc_timer = csecond() - exc_timer;
  
  fprintf(stderr,
      "File=%s ( n=%d m=%d trans=%d n_z=%d Sparsity=%lf ) Input time=%lf s\n",
      op.mtx_name, op.n, op.m, op.count_transactions(), op.nz,
      1.0 * op.nz / op.n / op.m, exc_timer);
  double *x = (double *)malloc(op.m * sizeof(double));
  double *out = (double *)malloc(op.n * sizeof(double));
  vec_init_rand(x, op.m, 0);
  op.vec_alloc((double*)x);

  SpmvOperator cuSPARSE_op(op);
  cuSPARSE_op.cuSPARSE_init();

  SpmvOperator bsr_op(op);
  bsr_op.cuSPARSE_init();

  //cuSPARSE_op.print_op();
  SpmvOperator openmp_op(cuSPARSE_op);
  openmp_op.format_convert(SPMV_FORMAT_CSR);
  openmp_op.mem_convert(SPMV_MEMTYPE_HOST);
  openmp_op.openmp_init();

#ifdef TEST
  // compare_op(op, cuSPARSE_op);
  // cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);
  // compare_op(cuSPARSE_op, openmp_op);

  fprintf(stderr,"Serial-CSR: ");
  op.timer = csecond();
  SpmvCsrData *data = (SpmvCsrData *)op.format_data;
  spmv_csr(data->rowPtr, data->colInd, (double *)data->values, (double *)op.x,
           (double *)op.y, op.n);
  op.timer = csecond() - op.timer;
  report_results(op.timer * NR_ITER, op.flops, op.bytes);
  fprintf(stderr,"\n");

  fprintf(stderr,"\nRunning tests.. \n");

  if (!impFlag || impFlag == 1){
  	fprintf(stderr,"Testing openmp_csr...\t");
  	openmp_op.openmp_csr();
  	check_result<double>((double *)openmp_op.y, (double *)op.y, openmp_op.n);
  }

  /*
  SpmvOperator csr5_op(cuSPARSE_op);
      csr5_op.format_convert(SPMV_FORMAT_CSR);
      csr5_op.cuCSR5_init();
  printf("Testing cuCSR5_csr...");
  csr5_op.cuCSR5_csr();
  hipDeviceSynchronize();
  check_result<double>((double *)csr5_op.y, (double *)op.y, openmp_op.n);


      SpmvOperator acsr5_op(cuSPARSE_op);
      acsr5_op.format_convert(SPMV_FORMAT_CSR);
      acsr5_op.mem_convert(SPMV_MEMTYPE_HOST);
      acsr5_op.avx512CSR5_init();
  printf("Testing avx512CSR5_csr...");
  acsr5_op.avx512CSR5_csr();
  check_result<double>((double *)acsr5_op.y, (double *)op.y, openmp_op.n);
  */
  if (!impFlag || impFlag == 2){
  	fprintf(stderr,"Testing cuSPARSE_csr...\t");
  	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);
  	cuSPARSE_op.cuSPARSE_csr();
  	hipDeviceSynchronize();
  	check_result<double>((double *)cuSPARSE_op.y, (double *)op.y, cuSPARSE_op.n);
  }

  if (!impFlag || impFlag == 3){
  	fprintf(stderr,"Testing cuSPARSE_hyb...\t");
  	cuSPARSE_op.format_convert(SPMV_FORMAT_HYB);
  	cuSPARSE_op.cuSPARSE_hyb();
  	hipDeviceSynchronize();
  	check_result<double>((double *)cuSPARSE_op.y, (double *)op.y, cuSPARSE_op.n);
  }

  if (!impFlag || impFlag == 4){
  	fprintf(stderr,"Testing cuSPARSE_bsr-%d...\t", blockdim);
  	/// Convert to cuSPARSE bsr
  	bsr_op.bsr_blockDim = blockdim;
  	bsr_op.format_convert(SPMV_FORMAT_BSR);
  	bsr_op.cuSPARSE_bsr();
  	hipDeviceSynchronize();
  	check_result<double>((double *)bsr_op.y, (double *)op.y, bsr_op.n);
  }

#endif

  if (!impFlag || impFlag == 1){
  	// Warmup
  	for (int i = 0; i < 100; i++) openmp_op.openmp_csr();

  	// Run OpenMP csr
  	int threadNum = get_num_threads(); 
  	fprintf(stderr,"Timing openmp_csr-%d...\n", threadNum);
  	openmp_op.timer = csecond();
  	for (int i = 0; i < NR_ITER; i++) openmp_op.openmp_csr();
  	openmp_op.timer = csecond() - openmp_op.timer;
  	gflops_s = 2*openmp_op.nz/openmp_op.timer*NR_ITER*1e-9;
  	foutp << name << "," << "OMP_csr-" << threadNum << "," << openmp_op.m << "," << openmp_op.n << "," << openmp_op.nz << "," << openmp_op.timer << "," << gflops_s << "\n";

  	cpu_timer = openmp_op.timer;
  }

  if (!impFlag || impFlag == 2){
  	/// Run cuSPARSE csr
  	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);

  	// Warmup
  	for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_csr();
  	hipDeviceSynchronize();

  	// Run cuSPARSE csr
  	fprintf(stderr,"Timing cuSPARSE_csr...\n");
  	cuSPARSE_op.timer = csecond();
	nvmlAPIRun();
  	for (int i = 0; i < NR_ITER; i++) {
    		cuSPARSE_op.cuSPARSE_csr();
    		hipDeviceSynchronize();
  	}
	cudaCheckErrors();
	nvmlAPIEnd();
  	cuSPARSE_op.timer = csecond() - cuSPARSE_op.timer;
  	gflops_s = 2*cuSPARSE_op.nz/cuSPARSE_op.timer*NR_ITER*1e-9;
  	foutp << name << "," << "cuSPARSE_csr" << "," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.nz << "," << cuSPARSE_op.timer << "," << gflops_s << "\n";
  }

  if (!impFlag || impFlag == 3){
  	/// Convert to cuSPARSE hyb
  	cuSPARSE_op.format_convert(SPMV_FORMAT_HYB);

  	// Warmup
  	for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_hyb();
  	hipDeviceSynchronize();

  	// Run cuSPARSE hyb
  	fprintf(stderr,"Timing cuSPARSE_hyb...\n");
  	cuSPARSE_op.timer = csecond();
	nvmlAPIRun();
  	for (int i = 0; i < NR_ITER; i++) {
  	  cuSPARSE_op.cuSPARSE_hyb();
  	  hipDeviceSynchronize();
  	}
	cudaCheckErrors();
	nvmlAPIEnd();
  	cuSPARSE_op.timer = csecond() - cuSPARSE_op.timer;
  	gflops_s = 2*cuSPARSE_op.nz/cuSPARSE_op.timer*NR_ITER*1e-9;
  	foutp << name << "," << "cuSPARSE_hyb" << "," << cuSPARSE_op.m << "," << cuSPARSE_op.n << "," << cuSPARSE_op.nz << "," << cuSPARSE_op.timer << "," << gflops_s << "\n";
  }

  if (!impFlag || impFlag == 4){
  	/// Convert to cuSPARSE bsr
  	bsr_op.bsr_blockDim = blockdim;
  	bsr_op.format_convert(SPMV_FORMAT_BSR);

  	// Warmup
  	for (int i = 0; i < 100; i++) bsr_op.cuSPARSE_bsr();
  	hipDeviceSynchronize();

  	// Run cuSPARSE bsr
  	fprintf(stderr,"Timing cuSPARSE_bsr-%d...\n", blockdim);
  	bsr_op.timer = csecond();
	nvmlAPIRun();
  	for (int i = 0; i < NR_ITER; i++) {
    		bsr_op.cuSPARSE_bsr();
    		hipDeviceSynchronize();
  	}
	cudaCheckErrors();
	nvmlAPIEnd();
  	bsr_op.timer = csecond() - bsr_op.timer;
  	gflops_s = 2*bsr_op.nz/bsr_op.timer*NR_ITER*1e-9;
  	foutp << name << "," << "cuSPARSE_bsr-" << blockdim << "," << bsr_op.m << "," << bsr_op.n << "," << bsr_op.nz << "," << bsr_op.timer << "," << gflops_s << "\n";
  } 

  foutp.close();

}
