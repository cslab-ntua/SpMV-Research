#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///

#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

#include "nvmlPower.hpp"

int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 4,
	"Incorrect arguments.\nUsage:\t./Executable logfilename blockdim Matrix_name.mtx");

	// Set/Check for device
	int device_id = 1;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;
	cout << "I seem to fail for some inputs, consider NOT using me" << endl;

	char *name = argv[3], *outfile = argv[1];
	int blockdim = 4;
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	blockdim = atoi(argv[2]);

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, diagonal_factor = %lf, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.diagonal_factor, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, 
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering );

	VALUE_TYPE *x = (VALUE_TYPE *)malloc(op.n * sizeof(VALUE_TYPE));
	VALUE_TYPE *out = (VALUE_TYPE *)malloc(op.m * sizeof(VALUE_TYPE));
	vec_init_rand<VALUE_TYPE>(x, op.n, 0);
	op.vec_alloc((VALUE_TYPE*)x);


	SpmvOperator bsr_op(op);
	bsr_op.cuSPARSE_init();
  
#ifdef TEST
  
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
	spmv_csr<VALUE_TYPE>(data->rowPtr, data->colInd, (VALUE_TYPE *) data->values, (VALUE_TYPE *)op.x,
		   (VALUE_TYPE *) op.y, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");

	fprintf(stdout,"Testing cuSPARSE_bsr-%d...\t", blockdim);
	/// Convert to cuSPARSE bsr
	bsr_op.bsr_blockDim = blockdim;
	bsr_op.format_convert(SPMV_FORMAT_BSR);
	bsr_op.cuSPARSE_bsr();
	hipDeviceSynchronize();
	check_result<VALUE_TYPE>((VALUE_TYPE *)bsr_op.y, (VALUE_TYPE *)op.y, bsr_op.m);
#else 
	/// Convert to cuSPARSE bsr
	bsr_op.bsr_blockDim = blockdim;
	bsr_op.format_convert(SPMV_FORMAT_BSR);
#endif


	// Warmup
	for (int i = 0; i < 100; i++) bsr_op.cuSPARSE_bsr();
	hipDeviceSynchronize();

	// Run cuSPARSE bsr
	fprintf(stdout,"Timing cuSPARSE_bsr-%d...\n", blockdim);
	nvmlAPIRun();
	bsr_op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			bsr_op.cuSPARSE_bsr();
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	bsr_op.timer = (csecond() - bsr_op.timer)/NR_ITER;
	if (bsr_op.timer*NR_ITER < 1.0){
		unsigned int extra_itter = ((unsigned int) 1.0/bsr_op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			bsr_op.cuSPARSE_bsr();
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	nvmlAPIEnd();
	gflops_s = 2*bsr_op.nz*1e-9/bsr_op.timer;
	gflops_s = bsr_op.flops*1e-9/bsr_op.timer;
	fprintf(stdout, "cuSPARSE_bsr-%d: t = %lf ms (%lf Gflops/s )\n", blockdim, bsr_op.timer, gflops_s);
	foutp << bsr_op.mtx_name << "," << bsr_op.distribution << "," << bsr_op.placement << "," << bsr_op.diagonal_factor << "," << bsr_op.seed <<
	"," << bsr_op.m << "," << bsr_op.n << "," << bsr_op.density << 
	"," << bsr_op.nz << "," << bsr_op.avg_nz_row << "," << bsr_op.std_nz_row <<
	"," << bsr_op.avg_bandwidth << "," << bsr_op.std_bandwidth <<
	"," << bsr_op.avg_scattering << "," << bsr_op.std_scattering <<
	"," << "cuSPARSE_bsr-" << blockdim <<  "," << bsr_op.timer << "," << gflops_s << "\n";

	foutp.close();

}
