#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///

#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

#include "nvem.hpp"

int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 4 || argc == 3,
	  "Incorrect arguments.\nUsage:\t./Executable logfilename Matrix_name.mtx [blockdim]");
	  
	// Set/Check for device
	int device_id = 0, blockdim = 4; 
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *name = argv[2], *outfile = argv[1];
	if (argc == 4) blockdim = atoi(argv[3]);
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf, mem_footprint = %lf MB, mem_range=%s\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bw=%lf, std_bw = %lf, avg_bw_scaled = %lf, std_bw_scaled = %lf\n\t\
	  avg_sc=%lf, std_sc=%lf, avg_sc_scaled = %lf, std_sc_scaled = %lf\
	  \n\t, skew =%lf, avg_num_neighbours =%lf, cross_row_similarity =%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, op.mem_footprint, op.mem_range,
	  op.nz, op.avg_nnz_per_row,  op.std_nnz_per_row, 
	  op.avg_bw,  op.std_bw, op.avg_bw_scaled, op.std_bw_scaled,
	  op.avg_sc,  op.std_sc, op.avg_sc_scaled, op.std_sc_scaled, 
	  op.skew, op.avg_num_neighbours, op.cross_row_similarity);
		
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_AX *out = (VALUE_TYPE_AX *)malloc(op.m * sizeof(VALUE_TYPE_AX));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc((VALUE_TYPE_AX*)x);

	SpmvOperator bsr_op(op);
	bsr_op.cuSPARSE_init();
  	/// Convert to cuSPARSE bsr
	bsr_op.bsr_blockDim = blockdim;
	bsr_op.format_convert(SPMV_FORMAT_BSR);
#ifdef TEST
  
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
	spmv_csr<VALUE_TYPE_AX>(data->rowPtr, data->colInd, (VALUE_TYPE_AX *) data->values, (VALUE_TYPE_AX *)op.x,
		   (VALUE_TYPE_AX *) op.y, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");


	fprintf(stdout,"Testing cuSPARSE_bsr...\t");

	bsr_op.cuSPARSE_bsr();
	hipDeviceSynchronize();
	hipMemcpy(out, bsr_op.y, op.m * sizeof(VALUE_TYPE_AX), hipMemcpyDeviceToHost);
	check_result<VALUE_TYPE_AX>((VALUE_TYPE_AX *)out, (VALUE_TYPE_AX *)op.y, bsr_op.m);


#endif
	
	// Warmup
	for (int i = 0; i < 100; i++) bsr_op.cuSPARSE_bsr();
	hipDeviceSynchronize();

	// Run cuSPARSE csr
	fprintf(stdout,"Timing cuSPARSE_bsr...\n");
	short CUDA_VALUE_TYPE_AX;
	if (std::is_same<VALUE_TYPE_AX, float>::value)  CUDA_VALUE_TYPE_AX = 0;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = 1;
	char powa_filename[256];
	sprintf(powa_filename, "cuSPARSE_bsrmv_9-2_mtx_cudatype-%d_format-CSR.log", CUDA_VALUE_TYPE_AX);
	NvemStartMeasure(device_id, powa_filename, 0); // Set to 1 for NVEM log messages. 
	op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			bsr_op.cuSPARSE_bsr();
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	op.timer = (csecond() - op.timer)/NR_ITER;
	unsigned int extra_itter = 0; 
	if (op.timer*NR_ITER < 1.0){
		extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			bsr_op.cuSPARSE_bsr();
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	NvemStats_p nvem_data = NvemStopMeasure(device_id, "Energy measure cuSPARSE_bsrmv_9-2_mtx");
	gflops_s = op.flops*1e-9/op.timer;
	double W_avg = nvem_data->W_avg, J_estimated = nvem_data->J_estimated/(NR_ITER+extra_itter); 
	fprintf(stdout, "cuSPARSE_bsr9-2: t = %lf ms (%lf Gflops/s ). Average Watts = %lf, Estimated Joules = %lf\n", op.timer*1000, gflops_s, W_avg, J_estimated);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.nz << "," << op.density << 
	"," << op.mem_footprint << "," << op.mem_range << "," << op.avg_nnz_per_row << "," << op.std_nnz_per_row <<
	"," << op.avg_bw << "," << op.std_bw <<
	"," << op.avg_bw_scaled << "," << op.std_bw_scaled <<
	"," << op.avg_sc << "," << op.std_sc <<
	"," << op.avg_sc_scaled << "," << op.std_sc_scaled <<
	"," << op.skew << "," << op.avg_num_neighbours << "," << op.cross_row_similarity <<
	"," << "cuSPARSE_bsr9-2_" << blockdim <<  "," << op.timer << "," << gflops_s << "," << W_avg <<  "," << J_estimated << endl;

	foutp.close();

}
