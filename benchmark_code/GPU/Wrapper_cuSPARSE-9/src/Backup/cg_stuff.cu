#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
///

#include <cstdio>
#include "cuSP.hpp"
#include "cuSPARSE.hpp"
/// TODO: If openmp include is put before cuSP it doesn't compile :P
#include <numeric>
#include "OpenMP.hpp"

double dot(double *v1, double *v2, int size) {
  register double sum = 0;
  for (int i = 0; i < size; i++) sum += v1[i] * v2[i];
  return sum;
}

double dot_r2(double *v1, int size) {
  register double sum = 0;
  for (int i = 0; i < size; i++) sum += v1[i] * v1[i];
  return sum;
}

bool breakdown(double inner, double *v, double *w, int size) {
  ddebug(" -> breakdown(inner,v,w,size)\n");

  /// This was copied from miniFE
  // This is code that was copied from Aztec, and originally written
  // by my hero, Ray Tuminaro.
  //
  // Assuming that inner = <v,w> (inner product of v and w),
  // v and w are considered orthogonal if
  //  |inner| < 100 * ||v||_2 * ||w||_2 * epsilon

  bool tmp = 0;
  double vnorm = sqrt(dot(v, v, size));
  double wnorm = sqrt(dot(w, w, size));
  tmp = abs(inner) <=
        100 * vnorm * wnorm * std::numeric_limits<double>::epsilon();
  if (tmp) debug("breakdown(inner,v,w,size) -> Returns breakdown\n");
  ddebug(" <- breakdown(inner,v,w,size)\n");
  return tmp;
}

void waxpby(double *dest, double a, double *x, double b, double *y, int size) {
  for (int i = 0; i < size; i++) dest[i] = a * x[i] + b * y[i];
}

void daxpby(double a, double *x, double b, double *y, int size) {
  for (int i = 0; i < size; i++) y[i] = a * x[i] + b * y[i];
}

void spmv(SpmvOperator &op) {
  SpmvCsrData *data = (SpmvCsrData *)op.format_data;
  spmv_csr(data->rowPtr, data->colInd, (double *)data->values, (double *)op.x,
           (double *)op.y, op.n);
}

double *conjugate_gradient_generic(SpmvOperator &op, double *b, double *x,
                                   double tolerance) {
  double *r, *p, *AxP, alpha, beta, rt, rt_old, *timers, r_norm, tmp, break_tol,
      p_ap_dot;
  vec_print<double>(b, 10, "b");
  vec_print<double>(x, 10, "x0");

  timers = (double *)calloc(5, sizeof(double));
  hipMallocManaged(&r, op.n * sizeof(double));
  AxP = (double *)op.y;
  p = (double *)op.x;
  timers[1] = csecond() - timers[1];
  waxpby(p, 1.0, x, 0.0, x, op.m);
  vec_print<double>(p, 10, "p");
  tmp = csecond();
  timers[1] = tmp - timers[1];
  timers[0] = tmp - timers[0];
  spmv(op);
  vec_print<double>(AxP, 10, "AxP");
  tmp = csecond();
  timers[0] = tmp - timers[0];
  timers[1] = tmp - timers[1];
  waxpby(r, 1.0, b, -1.0, AxP, op.n);
  vec_print<double>(r, 10, "r");
  tmp = csecond();
  timers[1] = tmp - timers[1];
  timers[2] = tmp - timers[2];
  rt = dot_r2(r, op.n);
  r_norm = sqrt(rt);
  timers[2] = csecond() - timers[2];
  break_tol = std::numeric_limits<double>::epsilon();

#ifdef DEBUG
  std::cout << "Initial Residual = " << r_norm << std::endl;
  std::cout << "break_tol = " << break_tol << std::endl;
  std::cout << "Starting CG Solve Phase..." << std::endl;
#endif

  for (int k = 1; k <= NR_ITER && r_norm > tolerance; ++k) {
    if (k == 1) {
      timers[1] = csecond() - timers[1];
      waxpby(p, 1.0, r, 0.0, r, op.n);
      vec_print<double>(p, 10, "p");
      timers[1] = csecond() - timers[1];
    } else {
      timers[2] = csecond() - timers[2];
      rt_old = rt;
      rt = dot_r2(r, op.n);
      beta = rt / rt_old;
      tmp = csecond();
      timers[2] = tmp - timers[2];
      timers[1] = tmp - timers[1];
      daxpby(1.0, r, beta, p, op.m);
      vec_print<double>(r, 10, "r");
      timers[1] = csecond() - timers[1];
    }

    r_norm = sqrt(rt);

    // if ((k%50==0 || k==NR_ITER)) {
    std::cout << "Iteration = " << k << "   Residual = " << r_norm
              << "   p_ap_dot = " << p_ap_dot << std::endl;
    //}

    p_ap_dot = 0;
    timers[0] = csecond() - timers[0];
    spmv(op);
    vec_print<double>(AxP, 10, "AxP");
    tmp = csecond();
    timers[0] = tmp - timers[0];
    timers[2] = tmp - timers[2];
    p_ap_dot = dot(AxP, p, op.m);
    timers[2] = csecond() - timers[2];

    if (p_ap_dot < break_tol) {
      debug(
          "conjugate_gradient_generic(op, b, tolerance) -> testing "
          "breakdown\n");
      if (p_ap_dot < 0 || breakdown(p_ap_dot, AxP, p, op.m)) {
        std::cout << "conjugate_gradient_generic(op, b, tolerance) -> ERROR, "
                     "numerical breakdown!"
                  << std::endl;

        // update the timers before jumping out.
        timers[4] = k;
        return timers;
      } else
        break_tol = 0.1 * p_ap_dot;
    }
    alpha = rt / p_ap_dot;
    printf("alpha=%lf\n", alpha);
    timers[1] = csecond() - timers[1];
    daxpby(alpha, p, 1.0, x, op.m);
    vec_print<double>(x, 10, "x");
    daxpby(-alpha, AxP, 1.0, r, op.m);
    vec_print<double>(r, 10, "r");
    timers[1] = csecond() - timers[1];
  }
  timers[4] = NR_ITER;
  return timers;
}

double *conjugate_gradient_naive(SpmvOperator &op, double *b) {
  double *rk, *pk, alpha, beta, dotr_r, dotr_rn, *timers, tmp;
  SpmvCsrData *data = (SpmvCsrData *)op.format_data;

  timers = (double *)malloc(5 * sizeof(double));
  rk = (double *)malloc(op.n * sizeof(double));
  hipMallocManaged(&pk, op.n * sizeof(double));

  timers[0] = timers[1] = timers[2] = timers[3] = timers[4] = 0;
  timers[0] = csecond() - timers[0];
  spmv_csr(data->rowPtr, data->colInd, (double *)data->values, (double *)op.x,
           (double *)op.y, op.n);
  timers[0] = csecond() - timers[0];
  for (int i = 0; i < op.n; i++) pk[i] = rk[i] = b[i] - ((double *)op.y)[i];

  for (int k = 0; k < NR_ITER; k++) {
    timers[0] = csecond() - timers[0];
    spmv_csr(data->rowPtr, data->colInd, (double *)data->values, pk,
             (double *)op.y, op.n);
    tmp = csecond();
    timers[0] = tmp - timers[0];
    timers[1] = tmp - timers[1];
    dotr_r = dot(rk, rk, op.n);
    alpha = dotr_r / dot(pk, (double *)op.y, op.n);
    tmp = csecond();
    timers[1] = tmp - timers[1];
    timers[2] = tmp - timers[2];
    for (int i = 0; i < op.n; i++) pk[i] = pk[i] * alpha;
    for (int i = 0; i < op.n; i++)
      ((double *)op.y)[i] = -((double *)op.y)[i] * alpha;
    tmp = csecond();
    timers[2] = tmp - timers[2];
    timers[3] = tmp - timers[3];
    vec_add<double>((double *)op.x, pk, op.n);
    vec_add<double>((double *)rk, (double *)op.y, op.n);
    tmp = csecond();
    timers[3] = tmp - timers[3];
    timers[1] = tmp - timers[1];
    dotr_rn = dot(rk, rk, op.n);
    if (sqrt(dotr_rn) < 1e-10) {
      timers[1] = tmp - timers[1];
      timers[4] = k + 1;
      return timers;
    }
    beta = dotr_rn / dotr_r;
    tmp = csecond();
    timers[1] = tmp - timers[1];
    timers[2] = tmp - timers[2];
    for (int i = 0; i < op.n; i++) pk[i] = pk[i] * beta / alpha;
    tmp = csecond();
    timers[2] = tmp - timers[2];
    timers[3] = tmp - timers[3];
    vec_add<double>((double *)pk, rk, op.n);
    timers[3] = tmp - timers[3];
  }
  timers[4] = NR_ITER;
  return timers;
}

double *conjugate_gradient_openmp(SpmvOperator &op, double *b) {
  double *rk, *pk, alpha, beta, dotr_r, dotr_rn, *timers, tmp;
  void *tmp_ptr;
  SpmvCsrData *data = (SpmvCsrData *)op.format_data;

  timers = (double *)malloc(5 * sizeof(double));
  rk = (double *)malloc(op.n * sizeof(double));
  hipMallocManaged(&pk, op.n * sizeof(double));

  timers[0] = timers[1] = timers[2] = timers[3] = timers[4] = 0;
  timers[0] = csecond() - timers[0];
  op.openmp_csr();
  timers[0] = csecond() - timers[0];
  for (int i = 0; i < op.n; i++) pk[i] = rk[i] = b[i] - ((double *)op.y)[i];

  for (int k = 0; k < NR_ITER; k++) {
    timers[0] = csecond() - timers[0];
    tmp_ptr = op.x;
    op.x = pk;
    op.openmp_csr();
    op.x = tmp_ptr;
    tmp = csecond();
    timers[0] = tmp - timers[0];
    timers[1] = tmp - timers[1];
    dotr_r = dot(rk, rk, op.n);
    alpha = dotr_r / dot(pk, (double *)op.y, op.n);
    tmp = csecond();
    timers[1] = tmp - timers[1];
    timers[2] = tmp - timers[2];
    for (int i = 0; i < op.n; i++) pk[i] = pk[i] * alpha;
    for (int i = 0; i < op.n; i++)
      ((double *)op.y)[i] = -((double *)op.y)[i] * alpha;
    tmp = csecond();
    timers[2] = tmp - timers[2];
    timers[3] = tmp - timers[3];
    vec_add<double>((double *)op.x, pk, op.n);
    vec_add<double>((double *)rk, (double *)op.y, op.n);
    tmp = csecond();
    timers[3] = tmp - timers[3];
    timers[1] = tmp - timers[1];
    dotr_rn = dot(rk, rk, op.n);
    if (sqrt(dotr_rn) < 1e-10) {
      timers[1] = tmp - timers[1];
      timers[4] = k;
      return timers;
    }
    beta = dotr_rn / dotr_r;
    tmp = csecond();
    timers[1] = tmp - timers[1];
    timers[2] = tmp - timers[2];
    for (int i = 0; i < op.n; i++) pk[i] = pk[i] * beta / alpha;
    tmp = csecond();
    timers[2] = tmp - timers[2];
    timers[3] = tmp - timers[3];
    vec_add<double>((double *)pk, rk, op.n);
    timers[3] = tmp - timers[3];
  }
  timers[4] = NR_ITER;
  return timers;
}

double *conjugate_gradient_cuSPARSE(SpmvOperator &op, double *b) {
  double *rk, *pk, alpha, beta, dotr_r, dotr_rn, *timers, tmp;
  void *tmp_ptr;
  SpmvCsrData *data = (SpmvCsrData *)op.format_data;

  timers = (double *)malloc(5 * sizeof(double));
  rk = (double *)malloc(op.n * sizeof(double));
  hipMallocManaged(&pk, op.n * sizeof(double));

  timers[0] = timers[1] = timers[2] = timers[3] = timers[4] = 0;
  timers[0] = csecond() - timers[0];
  op.cuSPARSE_hyb();
  hipDeviceSynchronize();
  timers[0] = csecond() - timers[0];
  for (int i = 0; i < op.n; i++) pk[i] = rk[i] = b[i] - ((double *)op.y)[i];

  for (int k = 0; k < NR_ITER; k++) {
    timers[0] = csecond() - timers[0];
    tmp_ptr = op.x;
    op.x = pk;
    op.cuSPARSE_hyb();
    hipDeviceSynchronize();
    op.x = tmp_ptr;
    tmp = csecond();
    timers[0] = tmp - timers[0];
    timers[1] = tmp - timers[1];
    dotr_r = dot(rk, rk, op.n);
    alpha = dotr_r / dot(pk, (double *)op.y, op.n);
    tmp = csecond();
    timers[1] = tmp - timers[1];
    timers[2] = tmp - timers[2];
    for (int i = 0; i < op.n; i++) pk[i] = pk[i] * alpha;
    for (int i = 0; i < op.n; i++)
      ((double *)op.y)[i] = -((double *)op.y)[i] * alpha;
    tmp = csecond();
    timers[2] = tmp - timers[2];
    timers[3] = tmp - timers[3];
    vec_add<double>((double *)op.x, pk, op.n);
    vec_add<double>((double *)rk, (double *)op.y, op.n);
    tmp = csecond();
    timers[3] = tmp - timers[3];
    timers[1] = tmp - timers[1];
    dotr_rn = dot(rk, rk, op.n);
    if (sqrt(dotr_rn) < 1e-10) {
      timers[1] = tmp - timers[1];
      timers[4] = k;
      return timers;
    }
    beta = dotr_rn / dotr_r;
    tmp = csecond();
    timers[1] = tmp - timers[1];
    timers[2] = tmp - timers[2];
    for (int i = 0; i < op.n; i++) pk[i] = pk[i] * beta / alpha;
    tmp = csecond();
    timers[2] = tmp - timers[2];
    timers[3] = tmp - timers[3];
    vec_add<double>((double *)pk, rk, op.n);
    timers[3] = tmp - timers[3];
  }
  timers[4] = NR_ITER;
  return timers;
}

// vec_print<int>(data->rowInd, op.nz, "rowInd");
// vec_print<int>(data->colInd, op.nz, "colInd");
// vec_print<double>((double*)data->values, op.nz, "values");

/// Execute OpenMP csr

// Warmup
for (int i = 0; i < 100; i++) openmp_op.openmp_csr();

// Run OpenMP csr
openmp_op.timer = csecond();
for (int i = 0; i < NR_ITER; i++) {
  openmp_op.openmp_csr();
  y_out = openmp_op.y;
  openmp_op.y = openmp_op.x;
  openmp_op.x = y_out;
}
openmp_op.timer = csecond() - openmp_op.timer;
printf("openmp_csr: ");
report_results(openmp_op.timer, openmp_op.flops, openmp_op.bytes);

vec_print<double>((double *)y_out, 100, "openmp_op.y");

// Warmup
for (int i = 0; i < 100; i++) mkl_op.mkl_csr();

// Run Mkl csr
mkl_op.timer = csecond();
for (int i = 0; i < NR_ITER; i++) {
  mkl_op.mkl_csr();
  y_out = mkl_op.y;
  mkl_op.y = mkl_op.x;
  mkl_op.x = y_out;
  /*if ( i%10 == 0 ){
       exc_timer = csecond();
       openmp_op.vec_alloc(x);
       exc_timer = csecond() - exc_timer;
  } */
}
mkl_op.timer = csecond() - mkl_op.timer - exc_timer;
printf("mkl_csr: ");
report_results(mkl_op.timer, mkl_op.flops, mkl_op.bytes);
y_out = mkl_op.y_get_copy();
check_result<double>((double *)y_out, (double *)op.y, mkl_op.n);
cpu_timer = mkl_op.timer;

/*
  /// Execute csr5 cuda csr
  SpmvOperator csr5_op(cuSPARSE_op);
  csr5_op.format_convert(SPMV_FORMAT_CSR);
  csr5_op.cuCSR5_init();

  // Warmup
  for (int i = 0; i < 100; i++) csr5_op.cuCSR5_csr();
  hipDeviceSynchronize();

  // Run csr5 cuda
  csr5_op.timer = csecond();
  for (int i = 0; i < NR_ITER; i++) {
    csr5_op.cuCSR5_csr();
    hipDeviceSynchronize();

    //y_out = csr5_op.y;
    //csr5_op.y = csr5_op.x;
    //csr5_op.x = y_out;
    /*if ( i%10 == 0 ){
         exc_timer = csecond();

         exc_timer = csecond() - exc_timer;
    }
  }
  csr5_op.timer = csecond() - csr5_op.timer - exc_timer;
  printf("cuCSR5_csr: ");
  report_results(csr5_op.timer, csr5_op.flops, csr5_op.bytes);
  y_out = csr5_op.y_get_copy();
  check_result<double>((double *)y_out, (double *)op.y, csr5_op.n);

  /// Execute csr5 avx512 csr
  SpmvOperator csr5_op(cuSPARSE_op);
  csr5_op.format_convert(SPMV_FORMAT_CSR);
  csr5_op.mem_convert(SPMV_MEMTYPE_HOST);
  csr5_op.avx512CSR5_init();

  // Warmup
  for (int i = 0; i < 100; i++) csr5_op.avx512CSR5_csr();

  // Run csr5 cuda
  csr5_op.timer = csecond();
  for (int i = 0; i < NR_ITER; i++) {
    csr5_op.avx512CSR5_csr();


    //y_out = csr5_op.y;
    //csr5_op.y = csr5_op.x;
    //csr5_op.x = y_out;
    /*if ( i%10 == 0 ){
         exc_timer = csecond();

         exc_timer = csecond() - exc_timer;
    }
  }
  csr5_op.timer = csecond() - csr5_op.timer - exc_timer;
  printf("avx512CSR5_csr: ");
  report_results(csr5_op.timer, csr5_op.flops, csr5_op.bytes);
vec_init((double*)csr5_op.y, csr5_op.n, 0);
csr5_op.avx512CSR5_csr();
  y_out = csr5_op.y_get_copy();
  check_result<double>((double *)y_out, (double *)op.y, csr5_op.n);
*/

/// Execute cuSPARSE csr
cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);

// Warmup
for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_csr();
hipDeviceSynchronize();

// Run cuSPARSE csr
cuSPARSE_op.timer = csecond();
for (int i = 0; i < NR_ITER; i++) {
  cuSPARSE_op.cuSPARSE_csr();
  hipDeviceSynchronize();
  y_out = cuSPARSE_op.y;
  cuSPARSE_op.y = cuSPARSE_op.x;
  cuSPARSE_op.x = y_out;
  /*if ( i%10 == 0 ){
       exc_timer = csecond();
       cuSPARSE_op.vec_alloc(x);
       exc_timer = csecond() - exc_timer;
  } */
}
cuSPARSE_op.timer = csecond() - cuSPARSE_op.timer - exc_timer;
printf("cuSPARSE_csr: ");
report_results(cuSPARSE_op.timer, cuSPARSE_op.flops, cuSPARSE_op.bytes);
y_out = cuSPARSE_op.y_get_copy();
check_result<double>((double *)y_out, (double *)op.y, openmp_op.n);
gpu_timer = cuSPARSE_op.timer;

/// Execute cuSPARSE hyb
cuSPARSE_op.format_convert(SPMV_FORMAT_HYB);
cuSPARSE_op.vec_alloc(x);
// Warmup
for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_hyb();
hipDeviceSynchronize();

// Run cuSPARSE hyb
cuSPARSE_op.timer = csecond();
for (int i = 0; i < NR_ITER; i++) {
  cuSPARSE_op.cuSPARSE_hyb();
  hipDeviceSynchronize();
  y_out = cuSPARSE_op.y;
  cuSPARSE_op.y = cuSPARSE_op.x;
  cuSPARSE_op.x = y_out;
  /*if ( i%10 == 0 ){
       exc_timer = csecond();
       cuSPARSE_op.vec_alloc(x);
       exc_timer = csecond() - exc_timer;
  } */
}
cuSPARSE_op.timer = csecond() - cuSPARSE_op.timer - exc_timer;
printf("cuSPARSE_hyb: ");
report_results(cuSPARSE_op.timer, cuSPARSE_op.flops, cuSPARSE_op.bytes);
y_out = cuSPARSE_op.y_get_copy();
check_result<double>((double *)y_out, (double *)op.y, openmp_op.n);
