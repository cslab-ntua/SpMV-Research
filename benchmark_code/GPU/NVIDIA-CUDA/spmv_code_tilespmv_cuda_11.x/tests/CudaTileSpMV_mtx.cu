#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <fstream>

#include "gpu_utils.hpp"
#include "spmv_utils.hpp"
#include "cuSPARSE.hpp"
#include "nvem.hpp"

//From cuda 11 - cuSPARSE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "common.h"
#include "mmio_highlevel.h"
#include "csr2tile.h"
#include "tilespmv_cpu.h"
#include "tilespmv_cuda.h"
#define INDEX_DATA_TYPE unsigned char
#define DEBUG_FORMATCOST 0

#include <utils.h>

using namespace std;

#ifndef VALUE_TYPE_AX
#error
#endif

#ifndef VALUE_TYPE_Y
#error
#endif

#ifndef VALUE_TYPE_COMP
#error
#endif

#ifndef NR_ITER
#error
#endif


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

/* definition to expand macro then apply to pragma message */
#define VALUE_TO_STRING(x) #x
#define VALUE(x) VALUE_TO_STRING(x)
#define VAR_NAME_VALUE(var) #var "="  VALUE(var)

/* Some example here */
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_AX))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_Y))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_COMP))

//Add here any supported combinations. CUDA data types I hate you for this. 
hipDataType CUDA_VALUE_TYPE_AX, CUDA_VALUE_TYPE_Y, CUDA_VALUE_TYPE_COMP;
void cpp_compargs_to_cuda_dtype(){
	if (std::is_same<VALUE_TYPE_AX, int8_t>::value) CUDA_VALUE_TYPE_AX = HIP_R_8I;
	else if (std::is_same<VALUE_TYPE_AX, int>::value) CUDA_VALUE_TYPE_AX = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_AX, float>::value) CUDA_VALUE_TYPE_AX = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_AX");
	
	if (std::is_same<VALUE_TYPE_Y, int>::value) CUDA_VALUE_TYPE_Y = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_Y, float>::value) CUDA_VALUE_TYPE_Y = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_Y, double>::value) CUDA_VALUE_TYPE_Y = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_Y");
	
	if (std::is_same<VALUE_TYPE_COMP, int>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_COMP, float>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_COMP, double>::value) CUDA_VALUE_TYPE_COMP = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_COMP");
	cout << "CUDA_VALUE_TYPE_AX: " << CUDA_VALUE_TYPE_AX << ", CUDA_VALUE_TYPE_Y: " << CUDA_VALUE_TYPE_Y << ", CUDA_VALUE_TYPE_COMP: " << CUDA_VALUE_TYPE_COMP << endl;
}
	
int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 3,
	  "Incorrect arguments.\nUsage:\t./Executable logfilename Matrix_name.mtx");
	  
	// Set/Check for device
	int device_id = 0;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *name = argv[2], *outfile = argv[1];
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	
	//int true_nz = op.nz; 
	//op.nz = (op.nz / BLOCK_SIZE) * BLOCK_SIZE;
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf, mem_footprint = %lf MB, mem_range=%s\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bw=%lf, std_bw = %lf, avg_bw_scaled = %lf, std_bw_scaled = %lf\n\t\
	  avg_sc=%lf, std_sc=%lf, avg_sc_scaled = %lf, std_sc_scaled = %lf\
	  \n\t, skew =%lf, avg_num_neighbours =%lf, cross_row_similarity =%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, op.mem_footprint, op.mem_range,
	  op.nz, op.avg_nnz_per_row,  op.std_nnz_per_row, 
	  op.avg_bw,  op.std_bw, op.avg_bw_scaled, op.std_bw_scaled,
	  op.avg_sc,  op.std_sc, op.avg_sc_scaled, op.std_sc_scaled, 
	  op.skew, op.avg_num_neighbours, op.cross_row_similarity);
	  
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_Y *out = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc(x);
    
	op.cuSPARSE_init();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
    VALUE_TYPE_COMP alpha = (VALUE_TYPE_COMP) 1.0;
    VALUE_TYPE_COMP beta = (VALUE_TYPE_COMP) 0.0;
    cout << "alpha: " << alpha << ", beta: " << beta << endl;

    srand(time(NULL));

    cout << " ( " << op.m << ", " << op.n << " ) nnz = " << op.nz << endl;
    
     int err = 0;
    hipError_t err_cuda = hipSuccess;

    // Define pointers of matrix A, vector x and y
    int *d_csrRowPtrA;
    int *d_csrColIdxA;
    VALUE_TYPE_AX *d_csrValA;
    VALUE_TYPE_AX *dX;
    VALUE_TYPE_Y *dY;

    // Matrix A
    CHECK_CUDA(hipMalloc((void **)&d_csrRowPtrA, (op.m+1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_csrColIdxA, op.nz  * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_csrValA,    op.nz  * sizeof(VALUE_TYPE_AX)));

    CHECK_CUDA(hipMemcpy(d_csrRowPtrA, data->rowPtr, (op.m+1) * sizeof(int),   hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColIdxA, data->colInd, op.nz  * sizeof(int),   hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrValA,    data->values,    op.nz  * sizeof(VALUE_TYPE_AX),   hipMemcpyHostToDevice));

    // Vector x
    CHECK_CUDA(hipMalloc((void **)&dX, op.n * sizeof(VALUE_TYPE_AX)));
    CHECK_CUDA(hipMemcpy(dX, x, op.n * sizeof(VALUE_TYPE_AX), hipMemcpyHostToDevice));

    // Vector y
    CHECK_CUDA(hipMalloc((void **)&dY, op.m  * sizeof(VALUE_TYPE_Y)));
    CHECK_CUDA(hipMemcpy(dY, out, op.m * sizeof(VALUE_TYPE_Y), hipMemcpyHostToDevice));

	op.timer = csecond();
    // Get amount of temporary storage needed
    Tile_matrix *matrixA = (Tile_matrix *)malloc(sizeof (Tile_matrix));

    //format conversion

    Tile_create(matrixA, 
                op.m, op.n, op.nz,
                data->rowPtr,
                data->colInd,
                data->values);
	
	int tilenum = matrixA->tilenum;
	int * ptroffset1 = (int *)malloc(sizeof(int) * tilenum);
    int * ptroffset2 = (int *)malloc(sizeof(int) * tilenum);
    memset(ptroffset1, 0, sizeof(int) * tilenum);
    memset(ptroffset2, 0, sizeof(int) * tilenum);

    int rowblkblock = 0;

    unsigned int * blkcoostylerowidx;
    int * blkcoostylerowidx_colstart;
    int * blkcoostylerowidx_colstop;
    
    int tilem = matrixA->tilem;
    int tilen = matrixA->tilen;
    MAT_PTR_TYPE *tile_ptr = matrixA->tile_ptr;
    
        // balance analysis
    int rowblkblock_tmp = 0;
    for (int blki = 0; blki < tilem; blki++)
    {
        int balancenumblk = tile_ptr[blki + 1] - tile_ptr[blki];
        if (balancenumblk <= PREFETCH_SMEM_TH)
            rowblkblock_tmp++;
        else
        {
            rowblkblock_tmp += ceil((double)balancenumblk / (double)PREFETCH_SMEM_TH);
        }
    }
    rowblkblock = rowblkblock_tmp;


    blkcoostylerowidx = (unsigned int *)malloc(sizeof(unsigned int) * rowblkblock);
    unsigned int *blkcoostylerowidx_tmp = blkcoostylerowidx;
    memset(blkcoostylerowidx_tmp, 0, sizeof(unsigned int) * rowblkblock);

    blkcoostylerowidx_colstart = (int *)malloc(sizeof(int) * rowblkblock);
    int *blkcoostylerowidx_colstart_tmp = blkcoostylerowidx_colstart;
    memset(blkcoostylerowidx_colstart_tmp, 0, sizeof(int) * rowblkblock);
    blkcoostylerowidx_colstop = (int *)malloc(sizeof(int) * rowblkblock);
    int *blkcoostylerowidx_colstop_tmp = blkcoostylerowidx_colstop;
    memset(blkcoostylerowidx_colstop_tmp, 0, sizeof(int) * rowblkblock);

    int rowblkblockcnt = 0;
    for (int blki = 0; blki < tilem; blki++)
    {
        int balancenumblk = tile_ptr[blki + 1] - tile_ptr[blki];
        if (balancenumblk <= PREFETCH_SMEM_TH)
        {
            blkcoostylerowidx_tmp[rowblkblockcnt] = blki;
            rowblkblockcnt++;
        }
        else
        {
            int numblklocal = ceil((double)balancenumblk / (double)PREFETCH_SMEM_TH);
            int lenblklocal = ceil((double)balancenumblk / (double)numblklocal);
            for (int iii = 0; iii < numblklocal; iii++)
            {
                blkcoostylerowidx_tmp[rowblkblockcnt] = blki | 0x80000000; // can generate -0
                blkcoostylerowidx_colstart_tmp[rowblkblockcnt] = tile_ptr[blki] + iii * lenblklocal;
                if (iii == numblklocal - 1)
                    blkcoostylerowidx_colstop_tmp[rowblkblockcnt] = tile_ptr[blki] + balancenumblk;
                else
                    blkcoostylerowidx_colstop_tmp[rowblkblockcnt] = tile_ptr[blki] + (iii + 1) * lenblklocal;

                rowblkblockcnt++;
            }
        }
    }
    
    op.timer = csecond() - op.timer;
    cout << "TileSpMV preproc time time cpu = " << op.timer*1000 << " ms." << endl;

	op.timer = csecond();
    int *tile_columnidx = matrixA->tile_columnidx;
    int *tile_nnz = matrixA->tile_nnz;
    char *Format = matrixA->Format;
    int *blknnz = matrixA->blknnz;
    unsigned char *blknnznnz = matrixA->blknnznnz;
    char *tilewidth = matrixA->tilewidth;
    int *csr_offset = matrixA->csr_offset;
    int *csrptr_offset = matrixA->csrptr_offset;
    int *coo_offset = matrixA->coo_offset;
    int *ell_offset = matrixA->ell_offset;
    int *hyb_offset = matrixA->hyb_offset;
    int *hyb_coocount = matrixA->hyb_coocount;
    int *dns_offset = matrixA->dns_offset;
    int *dnsrowptr = matrixA->dnsrowptr;
    int *dnsrow_offset = matrixA->dnsrow_offset;
    int *dnscolptr = matrixA->dnscolptr;
    int *dnscol_offset = matrixA->dnscol_offset;
    int *new_coocount = matrixA->new_coocount;
    VALUE_TYPE_COMP *Blockcsr_Val = matrixA->Blockcsr_Val;
    unsigned char *csr_compressedIdx = matrixA->csr_compressedIdx;
    unsigned char *Blockcsr_Ptr = matrixA->Blockcsr_Ptr;
    VALUE_TYPE_COMP *Blockcoo_Val = matrixA->Blockcoo_Val;
    unsigned char *coo_compressed_Idx = matrixA->coo_compressed_Idx;
    VALUE_TYPE_COMP *Blockell_Val = matrixA->Blockell_Val;
    unsigned char *ell_compressedIdx = matrixA->ell_compressedIdx;
    VALUE_TYPE_COMP *Blockhyb_Val = matrixA->Blockhyb_Val;
    unsigned char *hybIdx = matrixA->hybIdx;
    VALUE_TYPE_COMP *Blockdense_Val = matrixA->Blockdense_Val;
    VALUE_TYPE_COMP *Blockdenserow_Val = matrixA->Blockdenserow_Val;
    char *denserowid = matrixA->denserowid;
    VALUE_TYPE_COMP *Blockdensecol_Val = matrixA->Blockdensecol_Val;
    char *densecolid = matrixA->densecolid;
    int csrsize = matrixA->csrsize;
    int csrptrlen = matrixA->csrptrlen;
    int coosize = matrixA->coosize;
    int ellsize = matrixA->ellsize;
    int hybcoosize = matrixA->hybcoosize;
    int hybellsize = matrixA->hybellsize;
    int dense_size = matrixA->dnssize;
    int denserow_size = matrixA->dnsrowsize;
    int densecol_size = matrixA->dnscolsize;
    int coototal = matrixA->coototal;
    MAT_PTR_TYPE *deferredcoo_ptr = matrixA->deferredcoo_ptr;
    int *deferredcoo_colidx = matrixA->deferredcoo_colidx;
    VALUE_TYPE_COMP *deferredcoo_val = matrixA->deferredcoo_val;

    int csr_csize = csrsize % 2 == 0 ? csrsize / 2 : csrsize / 2 + 1;
    int ell_csize = ellsize % 2 == 0 ? ellsize / 2 : ellsize / 2 + 1;
    int hyb_size = hybellsize % 2 == 0 ? hybellsize / 2 : (hybellsize / 2) + 1;

    // tile matrix

    MAT_PTR_TYPE *d_tile_ptr;
    int *d_tile_columnidx;
    char *d_Format;
    int *d_blknnz;
    unsigned char *d_blknnznnz;

    hipMalloc((void **)&d_tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE));
    hipMalloc((void **)&d_tile_columnidx, tilenum * sizeof(int));
    hipMalloc((void **)&d_Format, tilenum * sizeof(char));
    hipMalloc((void **)&d_blknnz, (tilenum + 1) * sizeof(int));
    hipMalloc((void **)&d_blknnznnz, (tilenum + 1) * sizeof(unsigned char));

    hipMemcpy(d_tile_ptr, tile_ptr, (tilem + 1) * sizeof(MAT_PTR_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_tile_columnidx, tile_columnidx, tilenum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Format, Format, tilenum * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_blknnz, blknnz, (tilenum + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blknnznnz, blknnznnz, (tilenum + 1) * sizeof(unsigned char), hipMemcpyHostToDevice);
	cudaCheckErrors();
 
    // CSR
    unsigned char *d_csr_compressedIdx = (unsigned char *)malloc((csr_csize) * sizeof(unsigned char));
    VALUE_TYPE_COMP *d_Blockcsr_Val;
    unsigned char *d_Blockcsr_Ptr;

    hipMalloc((void **)&d_csr_compressedIdx, (csr_csize) * sizeof(unsigned char));
    hipMalloc((void **)&d_Blockcsr_Val, (csrsize) * sizeof(VALUE_TYPE_COMP));
    hipMalloc((void **)&d_Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char));

    hipMemcpy(d_csr_compressedIdx, csr_compressedIdx, (csr_csize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Val, Blockcsr_Val, (csrsize) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcsr_Ptr, Blockcsr_Ptr, (csrptrlen) * sizeof(unsigned char), hipMemcpyHostToDevice);
	cudaCheckErrors();

    // COO
    unsigned char *d_coo_compressed_Idx;
    VALUE_TYPE_COMP *d_Blockcoo_Val;

    hipMalloc((void **)&d_coo_compressed_Idx, (coosize) * sizeof(unsigned char));
    hipMalloc((void **)&d_Blockcoo_Val, (coosize) * sizeof(VALUE_TYPE_COMP));

    hipMemcpy(d_coo_compressed_Idx, coo_compressed_Idx, (coosize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockcoo_Val, Blockcoo_Val, (coosize) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
    // ELL
    unsigned char *d_ell_compressedIdx;
    VALUE_TYPE_COMP *d_Blockell_Val;

    hipMalloc((void **)&d_ell_compressedIdx, (ell_csize) * sizeof(unsigned char));
    hipMalloc((void **)&d_Blockell_Val, (ellsize) * sizeof(VALUE_TYPE_COMP));

    hipMemcpy(d_ell_compressedIdx, ell_compressedIdx, (ell_csize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockell_Val, Blockell_Val, (ellsize) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
	cudaCheckErrors();

    // HYB
    unsigned char *d_hybIdx;
    char *d_tilewidth;
    VALUE_TYPE_COMP *d_Blockhyb_Val;

    hipMalloc((void **)&d_hybIdx, (hyb_size + hybcoosize) * sizeof(unsigned char));
    hipMalloc((void **)&d_tilewidth, tilenum * sizeof(char));
    hipMalloc((void **)&d_Blockhyb_Val, (hybellsize + hybcoosize) * sizeof(VALUE_TYPE_COMP));

    hipMemcpy(d_hybIdx, hybIdx, (hyb_size + hybcoosize) * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_tilewidth, tilewidth, tilenum * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockhyb_Val, Blockhyb_Val, (hybellsize + hybcoosize) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
    // dense
    VALUE_TYPE_COMP *d_Blockdense_Val;

    hipMalloc((void **)&d_Blockdense_Val, (dense_size) * sizeof(VALUE_TYPE_COMP));

    hipMemcpy(d_Blockdense_Val, Blockdense_Val, (dense_size) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
    // denserow
    int *d_dnsrowptr;
    VALUE_TYPE_COMP *d_Blockdenserow_Val;
    char *d_denserowid;

    hipMalloc((void **)&d_dnsrowptr, (tilenum + 1) * sizeof(int));
    hipMalloc((void **)&d_Blockdenserow_Val, (denserow_size) * sizeof(VALUE_TYPE_COMP));
    hipMalloc((void **)&d_denserowid, dnsrowptr[tilenum] * sizeof(char));

    hipMemcpy(d_dnsrowptr, dnsrowptr, (tilenum + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockdenserow_Val, Blockdenserow_Val, (denserow_size) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
    hipMemcpy(d_denserowid, denserowid, dnsrowptr[tilenum] * sizeof(char), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
    // dense column
    int *d_dnscolptr;
    VALUE_TYPE_COMP *d_Blockdensecol_Val;
    char *d_densecolid;

    hipMalloc((void **)&d_dnscolptr, (tilenum + 1) * sizeof(int));
    hipMalloc((void **)&d_Blockdensecol_Val, (densecol_size) * sizeof(VALUE_TYPE_COMP));
    hipMalloc((void **)&d_densecolid, dnscolptr[tilenum] * sizeof(char));

    hipMemcpy(d_dnscolptr, dnscolptr, (tilenum + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Blockdensecol_Val, Blockdensecol_Val, (densecol_size) * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);
    hipMemcpy(d_densecolid, densecolid, dnscolptr[tilenum] * sizeof(char), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
    unsigned int *d_blkcoostylerowidx;
    int *d_blkcoostylerowidx_colstart;
    int *d_blkcoostylerowidx_colstop;

    hipMalloc((void **)&d_blkcoostylerowidx, rowblkblock * sizeof(unsigned int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstart, rowblkblock * sizeof(int));
    hipMalloc((void **)&d_blkcoostylerowidx_colstop, rowblkblock * sizeof(int));

    hipMemcpy(d_blkcoostylerowidx, blkcoostylerowidx, rowblkblock * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstart, blkcoostylerowidx_colstart, rowblkblock * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_blkcoostylerowidx_colstop, blkcoostylerowidx_colstop, rowblkblock * sizeof(int), hipMemcpyHostToDevice);

    int *d_ptroffset1;
    int *d_ptroffset2;

    hipMalloc((void **)&d_ptroffset1, tilenum * sizeof(int));
    hipMalloc((void **)&d_ptroffset2, tilenum * sizeof(int));
    hipMemcpy(d_ptroffset1, ptroffset1, tilenum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ptroffset2, ptroffset2, tilenum * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckErrors();
	
    // deferred coo
    MAT_PTR_TYPE *d_deferredcoo_ptr;
    int *d_deferredcoo_colidx;
    VALUE_TYPE_COMP *d_deferredcoo_val;

    hipMalloc((void **)&d_deferredcoo_ptr, (op.m + 1) * sizeof(MAT_PTR_TYPE));
    hipMalloc((void **)&d_deferredcoo_colidx, (coototal) * sizeof(int));
    hipMalloc((void **)&d_deferredcoo_val, (coototal) * sizeof(VALUE_TYPE_COMP));

    hipMemcpy(d_deferredcoo_ptr, deferredcoo_ptr, (op.m + 1) * sizeof(MAT_PTR_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_deferredcoo_colidx, deferredcoo_colidx, coototal * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_deferredcoo_val, deferredcoo_val, coototal * sizeof(VALUE_TYPE_COMP), hipMemcpyHostToDevice);

    int *d_coodeferoffset;
    int *d_deferbuf_coooff;
    int *d_deferbuf_dxoff;

    hipMalloc((void **)&d_coodeferoffset, rowblkblock * sizeof(int));
    hipMemset(d_coodeferoffset, 0, rowblkblock * sizeof(int));

    hipMalloc((void **)&d_deferbuf_coooff, rowblkblock * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
    hipMemset(d_deferbuf_coooff, 0, rowblkblock * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
    hipMalloc((void **)&d_deferbuf_dxoff, rowblkblock * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
    hipMemset(d_deferbuf_dxoff, 0, rowblkblock * PREFETCH_SMEM_TH * COO_NNZ_TH * sizeof(int));
	cudaCheckErrors();
	
	op.timer = csecond() - op.timer;
    cout << "TileSpMV transfers and weird stuff = " << op.timer*1000 << " ms." << endl;
    
    int num_threads = WARP_PER_BLOCK * WARP_SIZE;
    int num_blocks = ceil((double)rowblkblock / (double)(num_threads / WARP_SIZE));

    stir_spmv_cuda_kernel_v5<<<num_blocks, num_threads>>>(tilem, tilen, op.m, op.n,
                                                          d_tile_ptr, d_tile_columnidx, d_Format, d_blknnz, d_blknnznnz,
                                                          d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                          d_coo_compressed_Idx, d_Blockcoo_Val,
                                                          d_tilewidth, d_ell_compressedIdx, d_Blockell_Val,
                                                          d_hybIdx, d_Blockhyb_Val,
                                                          d_Blockdense_Val,
                                                          d_dnsrowptr, d_Blockdenserow_Val, d_denserowid,
                                                          d_dnscolptr, d_Blockdensecol_Val, d_densecolid,
                                                          d_ptroffset1, d_ptroffset2,
                                                          rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                          dX, dY, 7, d_coodeferoffset, d_deferbuf_coooff, d_deferbuf_dxoff, 1);
    hipDeviceSynchronize();       
    cudaCheckErrors();                                               
#ifdef TEST

	VALUE_TYPE_Y *out1 = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	spmv_csr<VALUE_TYPE_AX, VALUE_TYPE_Y, VALUE_TYPE_COMP>(data->rowPtr, data->colInd, data->values, x,
		   out1, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");

    hipMemset(dY, 0,  op.m * sizeof(VALUE_TYPE_COMP));  
	fprintf(stdout,"Testing TileSpMV_11...\t");
    // execute SpMV
    stir_spmv_cuda_kernel_v6<<<num_blocks, num_threads>>>(tilem, tilen, op.m, op.n, op.nz,
                                                          d_tile_ptr, d_tile_columnidx, d_Format, d_blknnz, d_blknnznnz,
                                                          d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                          d_coo_compressed_Idx, d_Blockcoo_Val,
                                                          d_tilewidth, d_ell_compressedIdx, d_Blockell_Val,
                                                          d_hybIdx, d_Blockhyb_Val,
                                                          d_Blockdense_Val,
                                                          d_dnsrowptr, d_Blockdenserow_Val, d_denserowid,
                                                          d_dnscolptr, d_Blockdensecol_Val, d_densecolid,
                                                          d_ptroffset1, d_ptroffset2,
                                                          rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                          dX, dY, 7, d_coodeferoffset, d_deferbuf_coooff, d_deferbuf_dxoff, 1);

    
	hipDeviceSynchronize();
    cudaCheckErrors();     
    // device result check
    hipMemcpy(out, dY, op.m * sizeof(VALUE_TYPE_Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	check_result<VALUE_TYPE_Y>((VALUE_TYPE_Y*)out, out1, op.m);
	free(out1);
#endif

    // warm up by running 10000 times
    if (NR_ITER)
    {
        for (int i = 0; i < 10000; i++)
		// execute SpMV
		stir_spmv_cuda_kernel_v6<<<num_blocks, num_threads>>>(tilem, tilen, op.m, op.n, op.nz,
                                                          d_tile_ptr, d_tile_columnidx, d_Format, d_blknnz, d_blknnznnz,
                                                          d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                          d_coo_compressed_Idx, d_Blockcoo_Val,
                                                          d_tilewidth, d_ell_compressedIdx, d_Blockell_Val,
                                                          d_hybIdx, d_Blockhyb_Val,
                                                          d_Blockdense_Val,
                                                          d_dnsrowptr, d_Blockdenserow_Val, d_denserowid,
                                                          d_dnscolptr, d_Blockdensecol_Val, d_densecolid,
                                                          d_ptroffset1, d_ptroffset2,
                                                          rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                          dX, dY, 7, d_coodeferoffset, d_deferbuf_coooff, d_deferbuf_dxoff, 1);


    }

    err_cuda = hipDeviceSynchronize();
    cudaCheckErrors();     
    
	short CUDA_VALUE_TYPE_AX;
	if (std::is_same<VALUE_TYPE_AX, float>::value)  CUDA_VALUE_TYPE_AX = 0;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = 1;
	char powa_filename[256];
	sprintf(powa_filename, "TILE_CUDA_SPMV_11_mtx_dtype-%d.log", CUDA_VALUE_TYPE_AX);
	NvemStartMeasure(device_id, powa_filename, 0); // Set to 1 for NVEM log messages. ;
	op.timer = csecond();

    // time spmv by running NR_ITER times
    for (int i = 0; i < NR_ITER; i++){
		stir_spmv_cuda_kernel_v6<<<num_blocks, num_threads>>>(tilem, tilen, op.m, op.n, op.nz,
                                                          d_tile_ptr, d_tile_columnidx, d_Format, d_blknnz, d_blknnznnz,
                                                          d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                          d_coo_compressed_Idx, d_Blockcoo_Val,
                                                          d_tilewidth, d_ell_compressedIdx, d_Blockell_Val,
                                                          d_hybIdx, d_Blockhyb_Val,
                                                          d_Blockdense_Val,
                                                          d_dnsrowptr, d_Blockdenserow_Val, d_denserowid,
                                                          d_dnscolptr, d_Blockdensecol_Val, d_densecolid,
                                                          d_ptroffset1, d_ptroffset2,
                                                          rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                          dX, dY, 7, d_coodeferoffset, d_deferbuf_coooff, d_deferbuf_dxoff, 1);
    	err_cuda = hipDeviceSynchronize();
    }
	op.timer = (csecond() - op.timer)/NR_ITER;     
    cudaCheckErrors();     
    unsigned int extra_itter = 0;
	if (op.timer*NR_ITER < 1.0){
		extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
    		stir_spmv_cuda_kernel_v6<<<num_blocks, num_threads>>>(tilem, tilen, op.m, op.n, op.nz,
                                                          d_tile_ptr, d_tile_columnidx, d_Format, d_blknnz, d_blknnznnz,
                                                          d_csr_compressedIdx, d_Blockcsr_Val, d_Blockcsr_Ptr,
                                                          d_coo_compressed_Idx, d_Blockcoo_Val,
                                                          d_tilewidth, d_ell_compressedIdx, d_Blockell_Val,
                                                          d_hybIdx, d_Blockhyb_Val,
                                                          d_Blockdense_Val,
                                                          d_dnsrowptr, d_Blockdenserow_Val, d_denserowid,
                                                          d_dnscolptr, d_Blockdensecol_Val, d_densecolid,
                                                          d_ptroffset1, d_ptroffset2,
                                                          rowblkblock, d_blkcoostylerowidx, d_blkcoostylerowidx_colstart, d_blkcoostylerowidx_colstop,
                                                          dX, dY, 7, d_coodeferoffset, d_deferbuf_coooff, d_deferbuf_dxoff, 1);

    		err_cuda = hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
    NvemStats_p nvem_data = NvemStopMeasure(device_id, "Energy measure TileSpMV_11_mtx");
	gflops_s = op.flops*1e-9/op.timer;
	double W_avg = nvem_data->W_avg, J_estimated = nvem_data->J_estimated/(NR_ITER+extra_itter); 
	fprintf(stdout, "TileSpMV_11: t = %lf ms (%lf Gflops/s ). Average Watts = %lf, Estimated Joules = %lf\n", op.timer*1000, gflops_s, W_avg, J_estimated);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.nz << "," << op.density << 
	"," << op.mem_footprint << "," << op.mem_range << "," << op.avg_nnz_per_row << "," << op.std_nnz_per_row <<
	"," << op.avg_bw << "," << op.std_bw <<
	"," << op.avg_bw_scaled << "," << op.std_bw_scaled <<
	"," << op.avg_sc << "," << op.std_sc <<
	"," << op.avg_sc_scaled << "," << op.std_sc_scaled <<
	"," << op.skew << "," << op.avg_num_neighbours << "," << op.cross_row_similarity <<
	"," << "TileSpMV_11" <<  "," << op.timer << "," << gflops_s << "," << W_avg <<  "," << J_estimated << endl;

    CHECK_CUDA(hipFree(d_csrRowPtrA));
    CHECK_CUDA(hipFree(d_csrColIdxA));
    CHECK_CUDA(hipFree(d_csrValA));
    CHECK_CUDA(hipFree(dX));
    CHECK_CUDA(hipFree(dY));
    foutp.close();

    free(x);
    free(out);


    return 0;
}

