#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>

#include "gpu_utils.hpp"
#include "spmv_utils.hpp"
#include "cuSPARSE.hpp"
#include "nvem.hpp"


//From cuda 11 - cuSPARSE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include <cub/device/device_spmv.cuh>
#include <cub/util_allocator.cuh>
#include <cub/iterator/tex_ref_input_iterator.cuh>

#include "sparse_matrix.h"
#include <utils.h>
using namespace hipcub;

using namespace std;

#ifndef VALUE_TYPE_AX
#error
#endif

#ifndef VALUE_TYPE_Y
#error
#endif

#ifndef VALUE_TYPE_COMP
#error
#endif

#ifndef NR_ITER
#error
#endif


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

/* definition to expand macro then apply to pragma message */
#define VALUE_TO_STRING(x) #x
#define VALUE(x) VALUE_TO_STRING(x)
#define VAR_NAME_VALUE(var) #var "="  VALUE(var)

/* Some example here */
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_AX))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_Y))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_COMP))

//Add here any supported combinations. CUDA data types I hate you for this. 
hipDataType CUDA_VALUE_TYPE_AX, CUDA_VALUE_TYPE_Y, CUDA_VALUE_TYPE_COMP;
void cpp_compargs_to_cuda_dtype(){
	if (std::is_same<VALUE_TYPE_AX, int8_t>::value) CUDA_VALUE_TYPE_AX = HIP_R_8I;
	else if (std::is_same<VALUE_TYPE_AX, int>::value) CUDA_VALUE_TYPE_AX = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_AX, float>::value) CUDA_VALUE_TYPE_AX = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_AX");
	
	if (std::is_same<VALUE_TYPE_Y, int>::value) CUDA_VALUE_TYPE_Y = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_Y, float>::value) CUDA_VALUE_TYPE_Y = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_Y, double>::value) CUDA_VALUE_TYPE_Y = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_Y");
	
	if (std::is_same<VALUE_TYPE_COMP, int>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_COMP, float>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_COMP, double>::value) CUDA_VALUE_TYPE_COMP = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_COMP");
	cout << "CUDA_VALUE_TYPE_AX: " << CUDA_VALUE_TYPE_AX << ", CUDA_VALUE_TYPE_Y: " << CUDA_VALUE_TYPE_Y << ", CUDA_VALUE_TYPE_COMP: " << CUDA_VALUE_TYPE_COMP << endl;
}

CachingDeviceAllocator  g_allocator(true);          // Caching allocator for device memory
	
int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 3,
	  "Incorrect arguments.\nUsage:\t./Executable logfilename Matrix_name.mtx");
	  
	// Set/Check for device
	int device_id = 0;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *name = argv[2], *outfile = argv[1];
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf, mem_footprint = %lf MB, mem_range=%s\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bw=%lf, std_bw = %lf, avg_bw_scaled = %lf, std_bw_scaled = %lf\n\t\
	  avg_sc=%lf, std_sc=%lf, avg_sc_scaled = %lf, std_sc_scaled = %lf\
	  \n\t, skew =%lf, avg_num_neighbours =%lf, cross_row_similarity =%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, op.mem_footprint, op.mem_range,
	  op.nz, op.avg_nnz_per_row,  op.std_nnz_per_row, 
	  op.avg_bw,  op.std_bw, op.avg_bw_scaled, op.std_bw_scaled,
	  op.avg_sc,  op.std_sc, op.avg_sc_scaled, op.std_sc_scaled, 
	  op.skew, op.avg_num_neighbours, op.cross_row_similarity);
	  
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_Y *out = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc(x);
    
	op.cuSPARSE_init();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
    VALUE_TYPE_COMP alpha = (VALUE_TYPE_COMP) 1.0;
    VALUE_TYPE_COMP beta = (VALUE_TYPE_COMP) 0.0;
    cout << "alpha: " << alpha << ", beta: " << beta << endl;

    srand(time(NULL));

    cout << " ( " << op.m << ", " << op.n << " ) nnz = " << op.nz << endl;
    
     int err = 0;
    hipError_t err_cuda = hipSuccess;

    // Define pointers of matrix A, vector x and y
    int *d_csrRowPtrA;
    int *d_csrColIdxA;
    VALUE_TYPE_AX *d_csrValA;
    VALUE_TYPE_AX *dX;
    VALUE_TYPE_Y *dY;

    // Matrix A
    CHECK_CUDA(hipMalloc((void **)&d_csrRowPtrA, (op.m+1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_csrColIdxA, op.nz  * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&d_csrValA,    op.nz  * sizeof(VALUE_TYPE_AX)));

    CHECK_CUDA(hipMemcpy(d_csrRowPtrA, data->rowPtr, (op.m+1) * sizeof(int),   hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrColIdxA, data->colInd, op.nz  * sizeof(int),   hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_csrValA,    data->values,    op.nz  * sizeof(VALUE_TYPE_AX),   hipMemcpyHostToDevice));

    // Vector x
    CHECK_CUDA(hipMalloc((void **)&dX, op.n * sizeof(VALUE_TYPE_AX)));
    CHECK_CUDA(hipMemcpy(dX, x, op.n * sizeof(VALUE_TYPE_AX), hipMemcpyHostToDevice));

    // Vector y
    CHECK_CUDA(hipMalloc((void **)&dY, op.m  * sizeof(VALUE_TYPE_Y)));
    CHECK_CUDA(hipMemcpy(dY, out, op.m * sizeof(VALUE_TYPE_Y), hipMemcpyHostToDevice));

/*
	SpmvParams<VALUE_TYPE_AX, int> merge_params;
    merge_params.d_values 			= d_csrValA; 
    merge_params.d_row_end_offsets 	= d_csrRowPtrA; 
    merge_params.d_column_indices 	= d_csrColIdxA; 
    merge_params.d_vector_x 		= dX; 
    merge_params.d_vector_y 		= dY; 
    merge_params.num_rows         	= op.m;
    merge_params.num_cols         	= op.n;
    merge_params.num_nonzeros     	= op.nz;
    merge_params.alpha            	= alpha;
    merge_params.beta             	= beta;
*/   
       // Allocate temporary storage
    size_t temp_storage_bytes = 0;
    void *d_temp_storage = NULL;

	op.timer = csecond();
    // Get amount of temporary storage needed
    DeviceSpmv::CsrMV(d_temp_storage, temp_storage_bytes,
		d_csrValA, d_csrRowPtrA, d_csrColIdxA, dX, dY, op.m, op.n, op.nz,
        (hipStream_t) 0, false);

    // Allocate
    g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);

	op.timer = csecond() - op.timer;
    cout << "Allocated additional dev temp storage time = " << op.timer*1000 << " ms." << endl;

    
#ifdef TEST

	VALUE_TYPE_Y *out1 = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	spmv_csr<VALUE_TYPE_AX, VALUE_TYPE_Y, VALUE_TYPE_COMP>(data->rowPtr, data->colInd, data->values, x,
		   out1, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");

	fprintf(stdout,"Testing Merge_11...\t");
    // execute SpMV
    DeviceSpmv::CsrMV( d_temp_storage, temp_storage_bytes,
		d_csrValA, d_csrRowPtrA, d_csrColIdxA, dX, dY, op.m, op.n, op.nz, (hipStream_t) 0, 1 /*!g_quiet*/);
    
	hipDeviceSynchronize();
	// device result check
    hipMemcpy(out, dY, op.m * sizeof(VALUE_TYPE_Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	check_result<VALUE_TYPE_Y>((VALUE_TYPE_Y*)out, out1, op.m);
	free(out1);
#endif

    // warm up by running 10000 times
    if (NR_ITER)
    {
        for (int i = 0; i < 10000; i++)
		// execute SpMV
		DeviceSpmv::CsrMV( d_temp_storage, temp_storage_bytes,
			d_csrValA, d_csrRowPtrA, d_csrColIdxA, dX, dY, op.m, op.n, op.nz, (hipStream_t) 0, 0 /*!g_quiet*/);
    }

    err_cuda = hipDeviceSynchronize();

	short CUDA_VALUE_TYPE_AX;
	if (std::is_same<VALUE_TYPE_AX, float>::value)  CUDA_VALUE_TYPE_AX = 0;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = 1;
	char powa_filename[256];
	sprintf(powa_filename, "MERGE_CUDA_SPMV_11_mtx_dtype-%d.log", CUDA_VALUE_TYPE_AX);
	NvemStartMeasure(device_id, powa_filename, 0); // Set to 1 for NVEM log messages. ;
	op.timer = csecond();

    // time spmv by running NR_ITER times
    for (int i = 0; i < NR_ITER; i++){
		DeviceSpmv::CsrMV( d_temp_storage, temp_storage_bytes,
			d_csrValA, d_csrRowPtrA, d_csrColIdxA, dX, dY, op.m, op.n, op.nz, (hipStream_t) 0, 0 /*!g_quiet*/);
    	err_cuda = hipDeviceSynchronize();
    }
	op.timer = (csecond() - op.timer)/NR_ITER;
	unsigned int extra_itter = 0;
	if (op.timer*NR_ITER < 1.0){
		extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
        	DeviceSpmv::CsrMV( d_temp_storage, temp_storage_bytes,
				d_csrValA, d_csrRowPtrA, d_csrColIdxA, dX, dY, op.m, op.n, op.nz, (hipStream_t) 0, 0 /*!g_quiet*/);
    		err_cuda = hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
    NvemStats_p nvem_data = NvemStopMeasure(device_id, "Energy measure MERGE_11_mtx");
	gflops_s = op.flops*1e-9/op.timer;
	double W_avg = nvem_data->W_avg, J_estimated = nvem_data->J_estimated/(NR_ITER+extra_itter); 
	fprintf(stdout, "MERGE_11: t = %lf ms (%lf Gflops/s ). Average Watts = %lf, Estimated Joules = %lf\n", op.timer*1000, gflops_s, W_avg, J_estimated);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.nz << "," << op.density << 
	"," << op.mem_footprint << "," << op.mem_range << "," << op.avg_nnz_per_row << "," << op.std_nnz_per_row <<
	"," << op.avg_bw << "," << op.std_bw <<
	"," << op.avg_bw_scaled << "," << op.std_bw_scaled <<
	"," << op.avg_sc << "," << op.std_sc <<
	"," << op.avg_sc_scaled << "," << op.std_sc_scaled <<
	"," << op.skew << "," << op.avg_num_neighbours << "," << op.cross_row_similarity <<
	"," << "Merge_11" <<  "," << op.timer << "," << gflops_s << "," << W_avg <<  "," << J_estimated << endl;

    CHECK_CUDA(hipFree(d_csrRowPtrA));
    CHECK_CUDA(hipFree(d_csrColIdxA));
    CHECK_CUDA(hipFree(d_csrValA));
    CHECK_CUDA(hipFree(dX));
    CHECK_CUDA(hipFree(dY));
    foutp.close();

    free(x);
    free(out);


    return 0;
}

