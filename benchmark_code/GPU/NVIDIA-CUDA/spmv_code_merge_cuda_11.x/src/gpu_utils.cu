#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cassert>
#include <cstdio>
//#include "CSR5.hpp"
//#include "cuSPARSE.hpp"
//#include <numa.h>
#include <unistd.h>

#include "gpu_utils.hpp"

void print_devices() {
  ddebug(" -> print_devices()\n");
  hipDeviceProp_t properties;
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&properties, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", properties.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           properties.memoryClockRate / 1024);
    printf("  Memory Bus Width (bits): %d\n", properties.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) /
               1.0e6);
    if (properties.major >= 3)
      printf("  Unified Memory support: YES\n\n");
    else
      printf("  Unified Memory support: NO\n\n");
  }
  ddebug(" <- print_devices()\n");
}

void cudaCheckErrors() {
  massert(hipGetLastError() == hipSuccess,
          hipGetErrorString(hipGetLastError()));
}

void *gpu_alloc(size_t count) {
  void *ret;
  assert(hipMalloc(&ret, count) == hipSuccess);
  return ret;
}

void gpu_free(void *gpuptr) { assert(hipFree(gpuptr) == hipSuccess); }

void copy_to_gpu(const void *host, void *gpu, size_t count) {
  assert(hipMemcpy(gpu, host, count, hipMemcpyHostToDevice) == hipSuccess);
}

void copy_from_gpu(void *host, const void *gpu, size_t count) {
  assert(hipMemcpy(host, gpu, count, hipMemcpyDeviceToHost) == hipSuccess);
}

void gpu_showMem(char *message) {
  size_t free, total;
  assert(hipMemGetInfo(&free, &total) == hipSuccess);
  printf("showMem(%s): %u free out of %u MB \n", message, free / (1024 * 1024),
         total / (1024 * 1024));
}

void vec_free(void *dataptr, int size, SpmvMemType mem_alloc) {
  ddebug(" -> vec_free(dataptr, mem_alloc)\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_NUMA): {
      massert(false, "vec_free -> mem_alloc default reached");
      //numa_free(dataptr, size);
    } break;
    case (SPMV_MEMTYPE_HOST): {
      // free(dataptr);
      hipHostFree(dataptr);
    } break;
    case (SPMV_MEMTYPE_DEVICE):
    case (SPMV_MEMTYPE_UNIFIED): {
      gpu_free(dataptr);
    } break;
    default:
      massert(false, "vec_free -> mem_alloc default reached");
      break;
  }
  ddebug(" <- vec_free(dataptr, mem_alloc)\n");
}

__global__ void gpu_memCopy(double *destination, double *source, int size) {
  // int *dest=(int *)destination;

  // int *src=(int *)source;

  for (int tid = threadIdx.x; tid < size; tid += blockDim.x)

    destination[tid] = source[tid];
}
