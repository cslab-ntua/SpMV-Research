///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some helpfull functions for SpMV
///

#include <unistd.h>
#include <cassert>
#include <cstdio>
#include "cuSPARSE.hpp"

#include "gpu_utils.hpp"

/// SpmvOperator functions
SpmvOperator::SpmvOperator(int argc, char *argv[], int start_of_matrix_generation_args, int verbose) {
  ddebug(" -> SpmvOperator::SpmvOperator()\n");
  /// Default intialization values for an operator
  mem_bytes = 0;
  mem_bytes += sizeof(SpmvOperator);
  n = m = nz = bytes = flops = bsr_blockDim = 0;
  x = NULL;
  y = NULL;
  mem_alloc = SPMV_MEMTYPE_HOST;
  format_data = NULL;
  format = SPMV_FORMAT_CSR;
  lib_struct = NULL;
  lib = SPMV_NONE;
  mtx_name = "synthetic";
  mtx_generate(argc, argv, start_of_matrix_generation_args, verbose);
  bytes = sizeof(int) * (m + 1) + sizeof(int) * nz + sizeof(VALUE_TYPE_AX) * nz +
          sizeof(VALUE_TYPE_Y) * m + sizeof(VALUE_TYPE_AX) * n;
  flops = 2 * nz;
  ddebug(" <- SpmvOperator::SpmvOperator()\n");
}

SpmvOperator::SpmvOperator(char *matrix_name) {
  ddebug(" -> SpmvOperator::SpmvOperator(matrix_name)\n");
  /// Default intialization values for an operator
  mem_bytes = 0;
  mem_bytes += sizeof(SpmvOperator);
  mtx_name = matrix_name;
  n = m = nz = bytes = flops = bsr_blockDim = 0;
  x = NULL;
  y = NULL;
  mem_alloc = SPMV_MEMTYPE_HOST;
  format_data = NULL;
  format = SPMV_FORMAT_CSR;
  lib_struct = NULL;
  lib = SPMV_NONE;
  mtx_read();
  bytes = sizeof(int) * (m + 1) + sizeof(int) * nz + sizeof(VALUE_TYPE_AX) * nz +
          sizeof(VALUE_TYPE_Y) * m + sizeof(VALUE_TYPE_AX) * n;
  flops = 2 * nz;
  ddebug(" <- SpmvOperator::SpmvOperator(matrix_name)\n");
}

SpmvOperator::SpmvOperator(SpmvOperator &op) {
  ddebug(" -> SpmvOperator::SpmvOperator(copy)\n");
  lib = op.lib;
  // TODO: This switch could be replaced with an actual function
  switch (lib) {
    case (SPMV_LIBRARY_CUSPARSE):
      lib_struct = cuSPARSE_desc();
      debug(
          "SpmvOperator::SpmvOperator(copy) -> Generated new cuSPARSE_desc\n");
      break;
    case (SPMV_NONE):
      debug(
          "SpmvOperator::SpmvOperator(copy) -> warning... copying SPMV_NONE "
          "operator\n");
      break;
    case (SPMV_LIBRARY_OPENMP):
      lib_struct = NULL;
      break;
    default: {
      massert(false,
              "SpmvOperator::SpmvOperator(copy) -> Unreachable lib default "
              "reached\n");
      break;
    }
  }
  bsr_blockDim = op.bsr_blockDim;
  mem_alloc = op.mem_alloc;
  mtx_name = op.mtx_name;
  format = op.format;
  mem_alloc = op.mem_alloc;
  m = op.m;
  n = op.n;
  nz = op.nz;
  density =  op.density;
	//bytes = matrix->mem_footprint;
	avg_nnz_per_row = op.avg_nnz_per_row;
	std_nnz_per_row = op.std_nnz_per_row;
	avg_bw = op.avg_bw;
	std_bw = op.std_bw;
	avg_sc = op.avg_sc;
	std_sc = op.std_sc;
	distribution = op.distribution;
	placement = op.placement;
	skew = op.skew;
	avg_bw_scaled = op.avg_bw_scaled;
	std_bw_scaled = op.std_bw_scaled;
	avg_sc_scaled = op.avg_sc_scaled;
	std_sc_scaled = op.std_sc_scaled;
	avg_num_neighbours = op.avg_num_neighbours;
	cross_row_similarity = op.cross_row_similarity;
	mem_range = op.mem_range; 
	seed = op.seed;
  
  flops = op.flops;
  bytes = op.bytes;
  format_data = op.spmv_data_get_copy();
  debug("SpmvOperator::SpmvOperator(copy) -> Copied format struct\n");

  // TODO: This switch could be replaced with an actual function
  switch (op.mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
    case (SPMV_MEMTYPE_UNIFIED):
      vec_alloc(op.x);
      debug("SpmvOperator::SpmvOperator(copy) -> Copied x vector\n");
      break;
    case (SPMV_MEMTYPE_DEVICE):
      massert(0,
              "SpmvOperator::SpmvOperator(copy) -> Copy from device vector not "
              "supported\n");
      break;
    default:
      massert(0,
              "SpmvOperator::SpmvOperator(copy) -> Unreachable mem_alloc "
              "default reached\n");
      break;
  }
  ddebug(" <- SpmvOperator::SpmvOperator(copy)\n");
}

SpmvOperator::SpmvOperator(SpmvOperator &op, int start, int end, int mode) {
  ddebug(" -> SpmvOperator::SpmvOperator(copy_op,start,end)\n");
  massert(start >= 0 && start < op.nz,
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> start is not "
          "within accepted limits");
  massert(end > 0 && end <= op.nz,
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> end is not within "
          "accepted limits");
  massert(end > start,
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> end is not greater "
          "than start");
  op.format_convert(SPMV_FORMAT_COO);
  lib = op.lib;
  // TODO: This switch could be replaced with an actual function
  switch (lib) {
    case (SPMV_LIBRARY_CUSPARSE):
      lib_struct = cuSPARSE_desc();
      debug(
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> Generated new "
          "cuSPARSE_desc\n");
      break;
    case (SPMV_NONE):
      debug(
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> warning... copying "
          "SPMV_NONE "
          "operator\n");
      break;
    case (SPMV_LIBRARY_OPENMP):
      lib_struct = NULL;
      break;
    default: {
      massert(false,
              "SpmvOperator::SpmvOperator(copy_op,start,end) -> Unreachable "
              "lib default "
              "reached\n");
      break;
    }
  }
  bsr_blockDim = op.bsr_blockDim;
  mem_alloc = op.mem_alloc;
  mtx_name = op.mtx_name;
  format = op.format;
  mem_alloc = op.mem_alloc;
  // TODO: All the splitting mechanism will be defined here
  m = n = op.m;
  nz = end - start;
  
	distribution = op.distribution;
	placement = op.placement;
	avg_num_neighbours = 0;
	cross_row_similarity = 0;
	mem_range = op.mem_range; 
	skew = 0;
	avg_bw_scaled = 0;
	std_bw_scaled = 0;
	avg_sc_scaled = 0;
	std_sc_scaled = 0;
	seed = op.seed;
	density = 0; 
	avg_nnz_per_row = 0;
	std_nnz_per_row = 0;
	avg_bw = 0;
	std_bw = 0;
	avg_sc = 0;
	std_sc = 0;
	
  format_data = op.spmv_data_get_subcopy(&start, &nz, mode);
  /// FIXME:EXP
  /*
  if (mode == 0) n = ((SpmvCooData *)format_data)->rowInd[nz - 1] + 1;
  else  if (mode == 1) n = m - ((SpmvCooData *)op.format_data)->rowInd[op.nz -
  nz];

    bytes = 2 * sizeof(int) * n + 1 * sizeof(int) * nz + 2 * sizeof(double) * nz
  +
            2 * sizeof(double) * n;
    /// FLOPS
    flops = 2 * nz;
  */

  debug(
      "SpmvOperator::SpmvOperator(copy_op,start,end) -> Copied format "
      "struct\n");

  // TODO: This switch could be replaced with an actual function
  switch (op.mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
    case (SPMV_MEMTYPE_UNIFIED):
      vec_alloc(op.x);
      debug(
          "SpmvOperator::SpmvOperator(copy_op,start,end) -> Copied x vector\n");
      break;
    case (SPMV_MEMTYPE_DEVICE):
      massert(0,
              "SpmvOperator::SpmvOperator(copy_op,start,end) -> Copy from "
              "device vector not "
              "supported\n");
      break;
    default:
      massert(0,
              "SpmvOperator::SpmvOperator(copy_op,start,end) -> Unreachable "
              "mem_alloc "
              "default reached\n");
      break;
  }
  if (mode == 0)
    n = ((SpmvCooData *)format_data)->rowInd[nz - 1] + 1;
  else if (mode == 1)
    n = m - ((SpmvCooData *)op.format_data)->rowInd[op.nz - nz];

  bytes = sizeof(int) * (m + 1) + sizeof(int) * nz + sizeof(VALUE_TYPE_AX) * nz +
          sizeof(VALUE_TYPE_Y) * m + sizeof(VALUE_TYPE_AX) * n;
  /// FLOPS
  flops = 2 * nz;

  ddebug(" <- SpmvOperator::SpmvOperator(copy_op,start,end)\n");
}

SpmvOperator::~SpmvOperator() {
  ddebug(" -> SpmvOperator::~SpmvOperator()\n");
  vec_free(x, n * sizeof(VALUE_TYPE_AX), mem_alloc);
  vec_free(y, m * sizeof(VALUE_TYPE_Y), mem_alloc);
  spmv_free();
  free(lib_struct);
  debug("SpmvOperator::~SpmvOperator -> Operator successfully destroyed\n");
  ddebug(" <- SpmvOperator::~SpmvOperator()\n");
}

void SpmvOperator::spmv_free_host() {
  ddebug(" -> SpmvOperator::spmv_free_host()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      free(data->rowInd);
      free(data->colInd);
      free(data->values);
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      free(data->rowPtr);
      free(data->colInd);
      free(data->values);
    } break;
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      free(data->rowPtr);
      free(data->colInd);
      free(data->values);
    } break;
    case (SPMV_FORMAT_HYB): {
      massert(false,
              "SpmvOperator::Spmv_free_host -> Unsupported format = "
              "SPMV_FORMAT_HYB");
    } break;
    default:
      massert(false, "SpmvOperator::Spmv_free_host -> format default reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_free_host()\n");
}

void SpmvOperator::spmv_free_numa() {
  ddebug(" -> SpmvOperator::spmv_free_numa()\n");
	massert(false, "SpmvOperator::spmv_free_numa -> No numa please");
/*
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      vec_free(data->rowInd, nz * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->colInd, nz * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->values, nz * sizeof(double), SPMV_MEMTYPE_NUMA);
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      vec_free(data->rowPtr, (n + 1) * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->colInd, nz * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->values, nz * sizeof(double), SPMV_MEMTYPE_NUMA);
    } break;
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      const int nb = (n + data->blockDim - 1) / data->blockDim;
      const int mb = (m + data->blockDim - 1) / data->blockDim;
      vec_free(data->rowPtr, (nb + 1) * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->colInd, data->nnzb * sizeof(int), SPMV_MEMTYPE_NUMA);
      vec_free(data->values,
               (data->blockDim * data->blockDim) * data->nnzb * sizeof(double),
               SPMV_MEMTYPE_NUMA);
    } break;
    case (SPMV_FORMAT_HYB): {
      massert(false,
              "SpmvOperator::spmv_free_numa -> Unsupported format = "
              "SPMV_FORMAT_HYB");
    } break;
    default:
      massert(false, "SpmvOperator::spmv_free_numa -> format default reached");
      break;
  }
*/
  ddebug(" <- SpmvOperator::spmv_free_numa()\n");
}

void SpmvOperator::spmv_free_device() {
  ddebug(" -> SpmvOperator::spmv_free_device()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      gpu_free(data->rowInd);
      gpu_free(data->colInd);
      gpu_free(data->values);
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      gpu_free(data->rowPtr);
      gpu_free(data->colInd);
      gpu_free(data->values);
    } break;
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      gpu_free(data->rowPtr);
      gpu_free(data->colInd);
      gpu_free(data->values);
    } break;
    case (SPMV_FORMAT_HYB): {
	  massert(false, "SpmvOperator::Spmv_free_device -> No SPMV_FORMAT_HYB free()");
    } break;
    case (SPMV_FORMAT_CSR5): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      gpu_free(data->rowPtr);
      gpu_free(data->colInd);
      gpu_free(data->values);
      switch (lib) {
        case (SPMV_LIBRARY_CSR5_AVX512):
	  massert(false, "SpmvOperator::Spmv_free_device -> No aCSR5_free()");
          //aCSR5_free((aCSR5_wrap *)lib_struct);
          break;
        case (SPMV_LIBRARY_CSR5_CUDA):
	  massert(false, "SpmvOperator::Spmv_free_device -> No cuCSR5_free()");
          //cuCSR5_free((cuCSR5_wrap *)lib_struct);
          break;
        default:
          massert(false,
                  "SpmvOperator::Spmv_free_device -> CSR5 lib default reached "
                  "- something is wrong");
          break;
      }

    } break;
    default:
      massert(false,
              "SpmvOperator::Spmv_free_device -> format default reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_free_device()\n");
}

void SpmvOperator::spmv_free() {
  ddebug(" -> SpmvOperator::spmv_free()\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST): {
      spmv_free_host();
    } break;
    case (SPMV_MEMTYPE_NUMA): {
      spmv_free_numa();
    } break;
    case (SPMV_MEMTYPE_DEVICE):
    case (SPMV_MEMTYPE_UNIFIED): {
      spmv_free_device();
    } break;
    default:
      massert(false, "SpmvOperator::Spmv_free -> mem_alloc default reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_free()\n");
}

// FIXME: Deprecated function for spliting, must change completely
SpmvOperator **split_nz(SpmvOperator *op, int div) {
  massert(0, "split_nz -> Not implemented\n");
}
/*
SpmvOperator **split_nz(SpmvOperator *op, int div) {
        massert(op->format == SPMV_FORMAT_COO, "split_nz -> Only coo format
supported\n");
  SpmvOperator **split_nz =
      (SpmvOperator **)malloc(div * sizeof(SpmvOperator *));
  const int nze = op->nz / div;
  printf("Initializing spliting with div= %d -> nze= %d\n", div, nze);
  for (int i = 0; i < div; i++) {
    split_nz[i] = new SpmvOperator(*op);
    split_nz[i]->nz = nze;
    split_nz[i]->bytes = split_nz[i]->bytes / div;
    split_nz[i]->flops = 2 * split_nz[i]->nz;
    if (i == div - 1) {
      split_nz[i]->nz = nze + op->nz % div;
      split_nz[i]->flops = 2 * split_nz[i]->nz;
    }
    SpmvCooData *tmp_data = (SpmvCooData *)split_nz[i]->format_data;
    SpmvCooData *cp_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
    hipMallocManaged(&cp_data->rowInd, split_nz[i]->nz * sizeof(int));
    hipMallocManaged(&cp_data->colInd, split_nz[i]->nz * sizeof(int));
    hipMallocManaged(&cp_data->values, split_nz[i]->nz * sizeof(double));

    vec_copy_int(cp_data->rowInd, &(((int *)tmp_data->rowInd)[nze * i]),
                 split_nz[i]->nz, 0);
    vec_copy_int(cp_data->colInd, &(((int *)tmp_data->colInd)[nze * i]),
                 split_nz[i]->nz, 0);
    /// TODO: Update this for other value_types
    vec_copy(cp_data->values, &(((double *)tmp_data->values)[nze * i]),
             split_nz[i]->nz, 0, split_nz[i]->value_type);
    //SpmvFree(split_nz[i]->format_struct, split_nz[i]->mem_alloc);
    split_nz[i]->format_data = cp_data;
  }
  return split_nz;
}
*/

void SpmvOperator::vec_alloc(VALUE_TYPE_AX *x) {
  ddebug(" -> SpmvOperator::vec_alloc(x)\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      vec_alloc_host(x);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      vec_alloc_device(x);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      vec_alloc_uni(x);
      break;
    case (SPMV_MEMTYPE_NUMA):
      vec_alloc_numa(x);
      break;
    default:
      massert(0,
              "SpmvOperator::vec_alloc -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  ddebug(" <- SpmvOperator::vec_alloc(x)\n");
}

void SpmvOperator::vec_alloc_numa(VALUE_TYPE_AX *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_numa(x_in)\n");
        massert(false, "SpmvOperator::vec_alloc_numa -> No numa please");
  ddebug(" <- SpmvOperator::vec_alloc_numa(x_in)\n");
}

void SpmvOperator::vec_alloc_host(VALUE_TYPE_AX *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_host(x_in)\n");
  VALUE_TYPE_AX *x_tmp;
  VALUE_TYPE_Y *y_tmp;

      /// Allocate vectors x,y
      hipHostAlloc(&x_tmp, n * sizeof(VALUE_TYPE_AX), hipHostMallocDefault);
      hipHostAlloc(&y_tmp, m * sizeof(VALUE_TYPE_Y), hipHostMallocDefault);
      /// Initialize vector x to x_in (y was calloc'ed)
      vec_copy<VALUE_TYPE_AX>(x_tmp, x_in, n, 0);
      for (int i = 0; i < m; i++) y_tmp[i] = 0;


  massert(x_tmp && y_tmp,
          "SpmvOperator::vec_alloc_host -> Vector Alloc failed");

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_host -> previous x and y allocated, "
        "deleting...\n");
    vec_free(x, n * sizeof(VALUE_TYPE_AX), mem_alloc);
    vec_free(y, m * sizeof(VALUE_TYPE_Y), mem_alloc);
  } else mem_bytes += n * sizeof(VALUE_TYPE_AX) + m * sizeof(VALUE_TYPE_Y);


  x = x_tmp;
  y = y_tmp;
  ddebug(" <- SpmvOperator::vec_alloc_host(x_in)\n");
}

void SpmvOperator::vec_alloc_uni(VALUE_TYPE_AX *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_uni(x)\n");
  VALUE_TYPE_AX *x_tmp;
  VALUE_TYPE_Y *y_tmp;

  /// Allocate vectors x,y
  hipMallocManaged(&x_tmp, n * sizeof(VALUE_TYPE_AX));
  hipMallocManaged(&y_tmp, m * sizeof(VALUE_TYPE_Y));
  /// Initialize vectors (x,y) to (x_in,0)
  vec_copy<VALUE_TYPE_AX>(x_tmp, x_in, n, 0);
  for (int i = 0; i < m; i++) y_tmp[i] = 0;


  massert(x_tmp && y_tmp, "SpmvOperator::vec_alloc_uni -> Vector Alloc failed");

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_uni -> previous x and y allocated, "
        "deleting...\n");
    vec_free(x, n * sizeof(VALUE_TYPE_AX), mem_alloc);
    vec_free(y, m * sizeof(VALUE_TYPE_Y), mem_alloc);

  } else {
        mem_bytes += n * sizeof(VALUE_TYPE_AX) + m * sizeof(VALUE_TYPE_Y);
        gpu_mem_bytes += n * sizeof(VALUE_TYPE_AX) + m * sizeof(VALUE_TYPE_Y);

  }
  x = x_tmp;
  y = y_tmp;
  ddebug(" <- SpmvOperator::vec_alloc_uni(x)\n");
}

void SpmvOperator::vec_alloc_device(VALUE_TYPE_AX *x_in) {
  ddebug(" -> SpmvOperator::vec_alloc_device(x)\n");
  VALUE_TYPE_AX *x_tmp;
  VALUE_TYPE_Y *y_tmp;

	  /// Allocate vectors x,y
	  x_tmp = (VALUE_TYPE_AX *)gpu_alloc(n * sizeof(VALUE_TYPE_AX));
	  y_tmp = (VALUE_TYPE_Y *)gpu_alloc(m * sizeof(VALUE_TYPE_Y));
	  /// Initialize vectors (x,y) to (x_in,0)
	  copy_to_gpu(x_in, x_tmp, n * sizeof(VALUE_TYPE_AX));
	  hipMemset(y_tmp, 0, m * sizeof(VALUE_TYPE_Y));

  cudaCheckErrors();

  /// Free previous vectors
  if (x && y) {
    debug(
        "SpmvOperator::vec_alloc_device -> previous x and y allocated, "
        "deleting...");
    vec_free(x, n * sizeof(VALUE_TYPE_AX), mem_alloc);
    vec_free(y, m * sizeof(VALUE_TYPE_Y), mem_alloc);
  } else gpu_mem_bytes += n * sizeof(VALUE_TYPE_AX) + m * sizeof(VALUE_TYPE_Y);
  x = x_tmp;
  y = y_tmp;
  ddebug(" <- SpmvOperator::vec_alloc_device(x)\n");
}

void *SpmvOperator::spmv_data_get_copy() {
  ddebug(" -> SpmvOperator::spmv_data_get_copy()\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      return spmv_data_copy_host();
    case (SPMV_MEMTYPE_DEVICE):
      return spmv_data_copy_device();
    case (SPMV_MEMTYPE_UNIFIED):
      return spmv_data_copy_uni();
    default:
      massert(0,
              "SpmvOperator::spmv_data_copy -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  return NULL;
}

void *SpmvOperator::spmv_data_copy_uni() {
  ddebug(" -> SpmvOperator::spmv_data_get_copy_uni()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      SpmvCooData *cp_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
      if (data->rowInd && data->colInd && data->values) {
        hipMallocManaged(&cp_data->rowInd, nz * sizeof(int));
        hipMallocManaged(&cp_data->colInd, nz * sizeof(int));

        vec_copy<int>(cp_data->rowInd, data->rowInd, nz, 0);
        vec_copy<int>(cp_data->colInd, data->colInd, nz, 0);

        hipMallocManaged(&cp_data->values, nz * sizeof(VALUE_TYPE_AX));
        vec_copy<VALUE_TYPE_AX>(cp_data->values, data->values, nz,0);
      } else
        debug(
            "SpmvOperator::spmv_data_copy_uni -> warning... empty Spmv struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      SpmvCsrData *cp_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));
      if (data->rowPtr && data->colInd && data->values) {
        hipMallocManaged(&cp_data->rowPtr, (m + 1) * sizeof(int));
        hipMallocManaged(&cp_data->colInd, nz * sizeof(int));

        vec_copy<int>(cp_data->rowPtr, data->rowPtr, m + 1, 0);
        vec_copy<int>(cp_data->colInd, data->colInd, nz, 0);

        hipMallocManaged(&cp_data->values, nz * sizeof(VALUE_TYPE_AX));
        vec_copy<VALUE_TYPE_AX>(cp_data->values, data->values, nz,0);

      } else
        debug(
            "SpmvOperator::spmv_data_copy_uni -> warning... empty Spmv struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_BSR): {
      SpmvBsrData *data = (SpmvBsrData *)format_data;
      SpmvBsrData *cp_data = (SpmvBsrData *)malloc(sizeof(SpmvBsrData));
      const int nb = (n + data->blockDim - 1) / data->blockDim;
      const int mb = (m + data->blockDim - 1) / data->blockDim;
      if (data->rowPtr && data->colInd && data->values) {
        hipMallocManaged(&cp_data->rowPtr, (nb + 1) * sizeof(int));
        hipMallocManaged(&cp_data->colInd, data->nnzb * sizeof(int));

        vec_copy<int>(cp_data->rowPtr, data->rowPtr, nb + 1, 0);
        vec_copy<int>(cp_data->colInd, data->colInd, data->nnzb, 0);

        hipMallocManaged(&cp_data->values,(data->blockDim * data->blockDim) * data->nnzb * sizeof(VALUE_TYPE_AX));
        vec_copy<VALUE_TYPE_AX>(cp_data->values, data->values,(data->blockDim * data->blockDim) * data->nnzb, 0);


        cp_data->nnzb = data->nnzb;
        cp_data->blockDim = data->blockDim;
      } else
        debug(
            "SpmvOperator::spmv_data_copy_uni -> warning... empty Spmv struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_DIA):
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_ELL):
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> SPMV_FORMAT_ELL not "
              "implemented");
    case (SPMV_FORMAT_HYB):
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> SPMV_FORMAT_HYB not "
              "implemented");
    default:
      massert(0,
              "SpmvOperator::spmv_data_copy_uni -> Unreacheable format default "
              "reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_data_get_copy_uni()\n");
  return NULL;
}

void *SpmvOperator::spmv_data_copy_device() {
  ddebug(" -> SpmvOperator::spmv_data_get_copy_device()\n");
  massert(0, "SpmvOperator::spmv_data_copy_device -> Not implemented");
  ddebug(" <- SpmvOperator::spmv_data_get_copy_device()\n");
  return NULL;
}

void *SpmvOperator::spmv_data_get_subcopy(int *start, int *nzc, int mode) {
  ddebug(" -> SpmvOperator::spmv_data_get_subcopy()\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      return spmv_data_subcopy_host(start, nzc, mode);
    case (SPMV_MEMTYPE_DEVICE):
      return spmv_data_subcopy_device(start, nzc, mode);
    case (SPMV_MEMTYPE_UNIFIED):
      return spmv_data_subcopy_uni(start, nzc, mode);
    default:
      massert(0,
              "SpmvOperator::spmv_data_subcopy -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  return NULL;
}

void *SpmvOperator::spmv_data_subcopy_uni(int *start, int *nzc, int mode) {
  ddebug(" -> SpmvOperator::spmv_data_subcopy_uni()\n");
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      SpmvCooData *cp_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
      if (data->rowInd && data->colInd && data->values) {
        if (mode == 0)
          while (data->rowInd[*nzc - 1] == data->rowInd[*nzc]) (*nzc)++;
        else if (mode == 1)
          while (data->rowInd[*start - 1] == data->rowInd[*start]) {
            (*start)++;
            (*nzc)--;
          }
        else
          massert(false,
                  "SpmvOperator::spmv_data_subcopy_uni -> unsupported subcopy "
                  "mode");
        hipMallocManaged(&cp_data->rowInd, *nzc * sizeof(int));
        hipMallocManaged(&cp_data->colInd, *nzc * sizeof(int));

        vec_copy<int>(cp_data->rowInd, &(data->rowInd[*start]), *nzc, 0);
        vec_copy<int>(cp_data->colInd, &(data->colInd[*start]), *nzc, 0);

        if (mode == 1) {
          for (int i = 1; i < *nzc; i++)
            cp_data->rowInd[i] = cp_data->rowInd[i] - cp_data->rowInd[0];
          cp_data->rowInd[0] = 0;
        }
        hipMallocManaged(&cp_data->values, *nzc * sizeof(VALUE_TYPE_AX));
        vec_copy<VALUE_TYPE_AX>(cp_data->values,&(data->values)[*start], *nzc, 0);


      } else
        debug(
            "SpmvOperator::spmv_data_subcopy_uni -> warning... empty Spmv "
            "struct, "
            "copied nothing\n");
      ddebug(" <- SpmvOperator::spmv_data_subcopy_uni()\n");
      return cp_data;
    }
    case (SPMV_FORMAT_CSR):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_BSR):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_DIA):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_DIA not "
              "implemented");
    case (SPMV_FORMAT_ELL):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_ELL not "
              "implemented");
    case (SPMV_FORMAT_HYB):
      massert(0,
              "SpmvOperator::spmv_data_subcopy_uni -> SPMV_FORMAT_HYB not "
              "implemented");
    default:
      massert(
          0,
          "SpmvOperator::spmv_data_subcopy_uni -> Unreacheable format default "
          "reached");
      break;
  }
  ddebug(" <- SpmvOperator::spmv_data_subcopy_uni()\n");
  return NULL;
}

void *SpmvOperator::spmv_data_subcopy_device(int *start, int *nzc, int mode) {
  ddebug(" -> SpmvOperator::spmv_data_subcopy_device()\n");
  massert(0, "SpmvOperator::spmv_data_subcopy_device -> Not implemented");
  ddebug(" <- SpmvOperator::spmv_data_subcopy_device()\n");
  return NULL;
}

void SpmvOperator::mem_convert(SpmvMemType target_mem) {
  ddebug(" -> SpmvOperator::mem_convert(target_mem)\n");
  switch (target_mem) {
    case (SPMV_MEMTYPE_HOST):
      mem_convert_host();
      break;
    case (SPMV_MEMTYPE_DEVICE):
      mem_convert_device();
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      mem_convert_uni();
      break;
    case (SPMV_MEMTYPE_NUMA):
      mem_convert_numa();
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  ddebug(" <- SpmvOperator::mem_convert(target_mem)\n");
}

void SpmvOperator::mem_convert_uni() {
  ddebug(" -> SpmvOperator::mem_convert_uni()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      newptr = spmv_data_copy_uni();
      spmv_free_host();
      format_data = newptr;
      vec_alloc_uni(x);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      debug(
          "SpmvOperator::mem_convert_uni -> Data already in unified memory\n");
      break;
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_uni -> warning... "
          "SpmvOperator::spmv_data_copy_uni from device is not properly "
          "tested\n");
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_uni -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_UNIFIED;
  ddebug(" <- SpmvOperator::mem_convert_uni()\n");
}

void SpmvOperator::mem_convert_host() {
  ddebug(" -> SpmvOperator::mem_convert_host()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      debug("SpmvOperator::mem_convert_host -> Data already in host memory\n");
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      newptr = spmv_data_copy_host();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_host(x);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_host -> warning... "
          "SpmvOperator::spmv_data_copy_uni from device is not properly "
          "tested\n");
      /// Convert to unified from device
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      /// Convert to host from unified
      newptr = spmv_data_copy_host();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_host(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_host -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_HOST;
  ddebug(" <- SpmvOperator::mem_convert_host()\n");
}

void SpmvOperator::mem_convert_numa() {
  ddebug(" -> SpmvOperator::mem_convert_numa()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_NUMA):
      debug("SpmvOperator::mem_convert_numa -> Data already in numa memory\n");
      break;
    case (SPMV_MEMTYPE_HOST):
      newptr = spmv_data_copy_numa();
      spmv_free_host();
      format_data = newptr;
      vec_alloc_numa(x);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      newptr = spmv_data_copy_numa();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_numa(x);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_numa -> warning... "
          "SpmvOperator::spmv_data_copy_uni from device is not properly "
          "tested\n");
      /// Convert to unified from device
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      /// Convert to host from unified
      newptr = spmv_data_copy_numa();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_numa(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_numa -> Unreachable mem_alloc default "
              "reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_NUMA;
  ddebug(" <- SpmvOperator::mem_convert_numa()\n");
}

void SpmvOperator::mem_convert_device() {
  ddebug(" -> SpmvOperator::mem_convert_device()\n");
  void *newptr = NULL;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_DEVICE):
      debug(
          "SpmvOperator::mem_convert_device -> Data already in device memory");
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      newptr = spmv_data_copy_device();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_device(x);
    case (SPMV_MEMTYPE_HOST):
      /// Convert to unified from host
      newptr = spmv_data_copy_uni();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_uni(x);
      /// Convert to device from unified
      debug(
          "SpmvOperator::mem_convert_device -> warning... "
          "SpmvOperator::spmv_data_copy_device from unified is not properly "
          "tested");
      newptr = spmv_data_copy_device();
      spmv_free_device();
      format_data = newptr;
      vec_alloc_device(x);
      break;
    default:
      massert(0,
              "SpmvOperator::mem_convert_device -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  mem_alloc = SPMV_MEMTYPE_DEVICE;
  ddebug(" <- SpmvOperator::mem_convert_device()\n");
}

void SpmvOperator::format_convert(SpmvFormat target_format) {
  ddebug(" -> SpmvOperator::format_convert(target_format)\n");
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
      format_convert_host(target_format);
      break;
    case (SPMV_MEMTYPE_DEVICE):
      format_convert_device(target_format);
      break;
    case (SPMV_MEMTYPE_UNIFIED):
      format_convert_uni(target_format);
      break;
    default:
      massert(0,
              "SpmvOperator::format_convert_type -> Unreachable mem_alloc "
              "default reached");
      break;
  }
  ddebug(" <- SpmvOperator::format_convert(target_format)\n");
}

void SpmvOperator::format_convert_uni(SpmvFormat target_format) {
  ddebug(" -> SpmvOperator::format_convert_uni(target_format)\n");
  switch (target_format) {
    case (SPMV_FORMAT_COO): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          debug(
              "SpmvOperator::format_convert_uni -> Struct already in the "
              "correct format\n");
          break;
        case (SPMV_FORMAT_CSR):
          format_convert_uni_csr2coo();
          break;
        case (SPMV_FORMAT_BSR):
          format_convert_uni_bsr2csr();
          format_convert_uni_csr2coo();
          break;
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          format_convert_uni_hyb2csr();
          format_convert_uni_csr2coo();
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    case (SPMV_FORMAT_CSR): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          format_convert_uni_coo2csr();
          break;
        case (SPMV_FORMAT_CSR):
          debug(
              "SpmvOperator::format_convert_uni -> Struct already in the "
              "correct format");
          break;
        case (SPMV_FORMAT_BSR):
          format_convert_uni_bsr2csr();
          break;
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          format_convert_uni_hyb2csr();
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    case (SPMV_FORMAT_BSR): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          format_convert_uni_coo2csr();
          format_convert_uni_csr2bsr();
          break;
        case (SPMV_FORMAT_CSR):
          format_convert_uni_csr2bsr();
          break;
        case (SPMV_FORMAT_BSR):
          if (bsr_blockDim != ((SpmvBsrData *)format_data)->blockDim)
            format_convert_uni_bsr2bsr();
          else
            debug(
                "SpmvOperator::format_convert_uni -> Already in the correct "
                "bsr format");
          break;
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          format_convert_uni_hyb2csr();
          format_convert_uni_csr2bsr();
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    case (SPMV_FORMAT_DIA):
      massert(0,
              "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA target not "
              "supported");
      break;
    case (SPMV_FORMAT_ELL):
      massert(0,
              "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL target not "
              "supported");
      break;
    case (SPMV_FORMAT_HYB): {
      switch (format) {
        case (SPMV_FORMAT_COO):
          format_convert_uni_coo2csr();
          format_convert_uni_csr2hyb();
          break;
        case (SPMV_FORMAT_CSR):
          format_convert_uni_csr2hyb();
          break;
        case (SPMV_FORMAT_BSR):
          format_convert_uni_bsr2csr();
          format_convert_uni_csr2hyb();
        case (SPMV_FORMAT_DIA):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_DIA "
                  "source not supported");
          break;
        case (SPMV_FORMAT_ELL):
          massert(0,
                  "SpmvOperator::format_convert_uni -> SPMV_FORMAT_ELL source "
                  "not supported");
          break;
        case (SPMV_FORMAT_HYB):
          debug(
              "SpmvOperator::format_convert_uni -> Struct already in the "
              "correct format");
          break;
        default:
          massert(0,
                  "SpmvOperator::format_convert_uni -> Unreacheable source "
                  "format default reached");
          break;
      }
    } break;
    default:
      massert(0,
              "SpmvOperator::format_convert_uni -> Unreacheable target format "
              "default reached");
      break;
  }
  ddebug(" <- SpmvOperator::format_convert_uni(target_format)\n");
}

void SpmvOperator::format_convert_device(SpmvFormat target_format) {
  ddebug(" -> SpmvOperator::format_convert_device(target_format)\n");
  massert(0, "SpmvOperator::format_convert_device -> Not Implemented");
  ddebug(" <- SpmvOperator::format_convert_device(target_format)\n");
}

VALUE_TYPE_Y *SpmvOperator::y_get_copy() {
  ddebug(" -> SpmvOperator::y_get_copy()\n");
  VALUE_TYPE_Y *out;
  switch (mem_alloc) {
    case (SPMV_MEMTYPE_HOST):
    case (SPMV_MEMTYPE_NUMA):
    case (SPMV_MEMTYPE_UNIFIED):
		out = (VALUE_TYPE_Y *) malloc(m * sizeof(VALUE_TYPE_Y));
		vec_copy<VALUE_TYPE_Y>(out, y, m, 0);
		break;
    case (SPMV_MEMTYPE_DEVICE):
      out = (VALUE_TYPE_Y *)malloc(m * sizeof(VALUE_TYPE_Y));
      copy_from_gpu(out, y, m * sizeof(VALUE_TYPE_Y));
      cudaCheckErrors();
      break;
    default:
      massert(0, "SpmvOperator::y_get_copy op->mem_alloc type unsupported");
  }
  ddebug(" <- SpmvOperator::y_get_copy()\n");
  return out;
}

void SpmvOperator::free_lib_struct() {
  ddebug(" -> SpmvOperator::free_lib_struct()\n");
  switch (lib) {
    case (SPMV_NONE):
      debug(
          "SpmvOperator::free_lib_struct -> Tried to free SPMV_NONE "
          "lib_struct");
      break;
    case (SPMV_LIBRARY_CUSPARSE):
      cuSPARSE_free((cuSPARSE_wrap *)lib_struct);
      break;
    default:
      massert(
          0,
          "SpmvOperator::free_lib_struct -> lib unreachable default reached");
  }
  ddebug(" <- SpmvOperator::free_lib_struct()\n");
}

int SpmvOperator::count_transactions() {
  ddebug(" -> SpmvOperator::count_transactions()\n");
  massert(format_data != NULL, "count_transactions -> No format struct");
  int ctr = 0, *exists = (int *)calloc(n, sizeof(int));
  switch (format) {
    case (SPMV_FORMAT_COO): {
      SpmvCooData *data = (SpmvCooData *)format_data;
      for (int i = 0; i < nz; i++) exists[data->colInd[i]] = 1;
    } break;
    case (SPMV_FORMAT_CSR): {
      SpmvCsrData *data = (SpmvCsrData *)format_data;
      for (int i = 0; i < nz; i++) exists[data->colInd[i]] = 1;
    } break;
    default:
      free(exists);
      massert(false, "count_transactions -> Unsupported SpMV format");
      break;
  }
  for (int i = 0; i < n; i++)
    if (exists[i]) ctr++;
  //free(exists);
  ddebug(" <- SpmvOperator::count_transactions()\n");
  return ctr;
}
