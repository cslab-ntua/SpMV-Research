///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Nvidia CuSPARSE wrapper for benchmarking
///

#include <cstdio>
#include "cuSPARSE.hpp"

cuSPARSE_wrap *cuSPARSE_desc() {
  ddebug(" -> cuSPARSE_desc()\n");
  cuSPARSE_wrap *tmp = (cuSPARSE_wrap *)malloc(sizeof(cuSPARSE_wrap));
  tmp->target_mem = SPMV_MEMTYPE_HOST;
  int double_device = -1, unified_device = -1, nDevices = 0;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&tmp->properties, i);
    if (tmp->properties.major >= 2) double_device = i;
    if (tmp->properties.major >= 3) unified_device = i;
    if ((unified_device + 1) && (double_device + 1)) break;
  }
  massert(nDevices, "cuSPARSE_desc: No CUDA device found");
  if (unified_device + 1)
    tmp->target_mem = SPMV_MEMTYPE_UNIFIED;
  else if (double_device < 0)
    massert(0, "cuSPARSE_desc: Device does not support double values");
  else
    tmp->target_mem = SPMV_MEMTYPE_DEVICE;
  /// TODO: Add multiple gpu support?
  hipSetDevice(unified_device);
  // printf("Using Device: %d\n", unified_device);
  hipStreamCreate(&tmp->stream);
  hipsparseCreate(&tmp->handle);
  hipsparseSetStream(tmp->handle, tmp->stream);
  hipsparseCreateMatDescr(&tmp->descA);
  hipsparseSetMatType(tmp->descA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(tmp->descA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseCreateMatDescr(&tmp->descB);
  hipsparseSetMatType(tmp->descB, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(tmp->descB, HIPSPARSE_INDEX_BASE_ZERO);
  tmp->dir = HIPSPARSE_DIRECTION_COLUMN;
  hipDeviceSynchronize();
  ddebug("  <- cuSPARSE_desc()\n");
  return tmp;
}

void SpmvOperator::cuSPARSE_check_compatibility() {
  ddebug(" -> SpmvOperator::cuSPARSE_check_compatibility()\n");
  if (0)
    massert(0,
            "SpmvOperator::cuSPARSE_check_compatibility -> SpmvOperator not "
            "compatible");
  ddebug(" <- SpmvOperator::cuSPARSE_check_compatibility()\n");
}

void SpmvOperator::cuSPARSE_init() {
  ddebug(" -> SpmvOperator::cuSPARSE_init()\n");
  cuSPARSE_check_compatibility();
  free_lib_struct();
  lib_struct = cuSPARSE_desc();
  lib = SPMV_LIBRARY_CUSPARSE;
  mem_convert(((cuSPARSE_wrap *)lib_struct)->target_mem);
  ddebug(" <- SpmvOperator::cuSPARSE_init()\n");
}

void cuSPARSE_free(cuSPARSE_wrap *tmp) {
  ddebug(" -> cuSPARSE_free(wrapper)\n");
  hipStreamDestroy(tmp->stream);
  hipsparseDestroy(tmp->handle);
  hipsparseDestroyMatDescr(tmp->descA);
  hipsparseDestroyMatDescr(tmp->descB);
  hipDeviceSynchronize();
  free(tmp);
  ddebug(" <- cuSPARSE_free(wrapper)\n");
}

void SpmvOperator::format_convert_uni_coo2csr() {
  ddebug(" -> SpmvOperator::format_convert_uni_coo2csr()\n");
  SpmvCooData *coo_data = (SpmvCooData *)format_data;
  SpmvCsrData *csr_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));
  csr_data->colInd = coo_data->colInd;
  csr_data->values = coo_data->values;
  hipMallocManaged(&csr_data->rowPtr, (m + 1) * sizeof(int));

  timer = csecond();
  hipsparseXcoo2csr(((cuSPARSE_wrap *)lib_struct)->handle, coo_data->rowInd, nz,
                   m, csr_data->rowPtr, HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  timer = csecond() - timer;

  gpu_free(coo_data->rowInd);
  cudaCheckErrors();
  format_data = csr_data;
  format = SPMV_FORMAT_CSR;
  ddebug(" <- SpmvOperator::format_convert_uni_coo2csr()\n");
}

void SpmvOperator::format_convert_uni_csr2hyb() {
  ddebug(" -> SpmvOperator::format_convert_uni_csr2hyb()\n");
  massert(false,"SpmvOperator::format_convert_uni_csr2hyb -> Not available/implemented for cuda11.0");
  ddebug(" <- SpmvOperator::format_convert_uni_csr2hyb()\n");
}

void SpmvOperator::format_convert_uni_csr2bsr() {
  ddebug(" -> SpmvOperator::format_convert_uni_csr2bsr()\n");
  SpmvCsrData *csr_data = (SpmvCsrData *)format_data;
  SpmvBsrData *bsr_data = (SpmvBsrData *)malloc(sizeof(SpmvBsrData));
  const int nb = (n + bsr_blockDim - 1) / bsr_blockDim;
  const int mb = (m + bsr_blockDim - 1) / bsr_blockDim;
  bsr_data->blockDim = bsr_blockDim;
  timer = csecond();
  hipMallocManaged(&bsr_data->rowPtr, (mb + 1) * sizeof(int));

  hipsparseXcsr2bsrNnz(
      ((cuSPARSE_wrap *)lib_struct)->handle, ((cuSPARSE_wrap *)lib_struct)->dir,
      m, n, ((cuSPARSE_wrap *)lib_struct)->descA, csr_data->rowPtr,
      csr_data->colInd, bsr_data->blockDim,
      ((cuSPARSE_wrap *)lib_struct)->descB, bsr_data->rowPtr, &bsr_data->nnzb);
  hipDeviceSynchronize();
  hipMallocManaged(&bsr_data->colInd, bsr_data->nnzb * sizeof(int));


      hipMallocManaged(&bsr_data->values,
                        (bsr_data->blockDim * bsr_data->blockDim) *
                            bsr_data->nnzb * sizeof(VALUE_TYPE_AX));
#if VALUE_TYPE_AX == float
      hipsparseScsr2bsr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, m, n,
          ((cuSPARSE_wrap *)lib_struct)->descA, (float *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd, bsr_data->blockDim,
          ((cuSPARSE_wrap *)lib_struct)->descB, (float *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd);
#elif VALUE_TYPE_AX == double    
      hipsparseDcsr2bsr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, m, n,
          ((cuSPARSE_wrap *)lib_struct)->descA, (double *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd, bsr_data->blockDim,
          ((cuSPARSE_wrap *)lib_struct)->descB, (double *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd);
#else
		massert(0, "SpmvOperator::format_convert_uni_csr2bsr(): Unsupported VALUE_TYPE_AX");
#endif

  hipDeviceSynchronize();
  cudaCheckErrors();
  timer = csecond() - timer;
  spmv_free();
  cudaCheckErrors();
  format_data = bsr_data;
  format = SPMV_FORMAT_BSR;

  /// BSR vector padding TODO: This is kind of a cheat, maybe n, m should change
  /// too and vec_alloc_uni used for this
  VALUE_TYPE_AX *xp;
  VALUE_TYPE_Y *yp;

      hipMallocManaged(&xp, nb * bsr_data->blockDim * sizeof(VALUE_TYPE_AX));
      hipMallocManaged(&yp, mb * bsr_data->blockDim * sizeof(VALUE_TYPE_Y));
      vec_copy<VALUE_TYPE_AX>((VALUE_TYPE_AX *)xp, (VALUE_TYPE_AX *)x, m, mb * bsr_data->blockDim - m);
      for (int i = 0; i < m; i++) ((VALUE_TYPE_Y *)yp)[i] = 0;
 
  massert(xp && yp,
          "SpmvOperator::format_convert_uni_csr2bsr -> Padded Vector Unified "
          "Alloc failed");
  gpu_free(x);
  gpu_free(y);
  x = xp;
  y = yp;
  ddebug(" <- SpmvOperator::format_convert_uni_csr2bsr()\n");
}

void SpmvOperator::format_convert_uni_csr2coo() {
  ddebug(" -> SpmvOperator::format_convert_uni_csr2coo()\n");
  SpmvCsrData *csr_data = (SpmvCsrData *)format_data;
  SpmvCooData *coo_data = (SpmvCooData *)malloc(sizeof(SpmvCooData));
  coo_data->colInd = csr_data->colInd;
  coo_data->values = csr_data->values;
  hipMallocManaged(&coo_data->rowInd, nz * sizeof(int));

  timer = csecond();
  hipsparseXcsr2coo(((cuSPARSE_wrap *)lib_struct)->handle, csr_data->rowPtr, nz,
                   m, coo_data->rowInd, HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  timer = csecond() - timer;

  gpu_free(csr_data->rowPtr);
  cudaCheckErrors();
  format_data = coo_data;
  format = SPMV_FORMAT_COO;
  ddebug(" <- SpmvOperator::format_convert_uni_csr2coo()\n");
}

void SpmvOperator::format_convert_uni_bsr2csr() {
  ddebug(" -> SpmvOperator::format_convert_uni_bsr2csr()\n");
massert(false, "SpmvOperator::format_convert_uni_bsr2csr -> Implementation is not correct");
  SpmvBsrData *bsr_data = (SpmvBsrData *)format_data;
  SpmvCsrData *csr_data = (SpmvCsrData *)malloc(sizeof(SpmvCsrData));
  const int nb = (n + bsr_data->blockDim - 1) / bsr_data->blockDim;
  const int mb = (m + bsr_data->blockDim - 1) / bsr_data->blockDim;

  timer = csecond();
  hipMallocManaged(&csr_data->rowPtr, (m + 1) * sizeof(int));
  hipMallocManaged(&csr_data->colInd, nz * sizeof(int));

      hipMallocManaged(&csr_data->values,
                        nz * sizeof(VALUE_TYPE_AX));
                        
#if VALUE_TYPE_AX == float
      hipsparseSbsr2csr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, 
          ((cuSPARSE_wrap *)lib_struct)->descA, (float *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd, bsr_data->blockDim, 
          ((cuSPARSE_wrap *)lib_struct)->descB, (float *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd);
#elif VALUE_TYPE_AX == double    
      hipsparseDbsr2csr(
          ((cuSPARSE_wrap *)lib_struct)->handle,
          ((cuSPARSE_wrap *)lib_struct)->dir, mb, nb, 
          ((cuSPARSE_wrap *)lib_struct)->descA, (double *)bsr_data->values,
          bsr_data->rowPtr, bsr_data->colInd, bsr_data->blockDim, 
          ((cuSPARSE_wrap *)lib_struct)->descB, (double *)csr_data->values,
          csr_data->rowPtr, csr_data->colInd);
#else
		massert(0, "SpmvOperator::format_convert_uni_csr2bsr(): Unsupported VALUE_TYPE_AX");
#endif

  hipDeviceSynchronize();
  timer = csecond() - timer;
  spmv_free();
  cudaCheckErrors();
  format_data = csr_data;
  format = SPMV_FORMAT_CSR;


  ddebug(" <- SpmvOperator::format_convert_uni_bsr2csr()\n");
}

void SpmvOperator::format_convert_uni_bsr2bsr() {
  ddebug(" -> SpmvOperator::format_convert_uni_bsr2bsr()\n");
  massert(false,"SpmvOperator::format_convert_uni_bsr2bsr -> Not available/implemented for cuda11.0");
  ddebug(" <- SpmvOperator::format_convert_uni_bsr2bsr()\n");
}

void SpmvOperator::format_convert_uni_hyb2csr() {
  ddebug(" -> SpmvOperator::format_convert_uni_hyb2csr()\n");
  massert(false,"SpmvOperator::format_convert_uni_hyb2csr -> Not available/implemented for cuda11.0");
  ddebug(" <- SpmvOperator::format_convert_uni_hyb2csr()\n");
}

void SpmvOperator::cuSPARSE_csr() {
  ddebug(" -> SpmvOperator::cuSPARSE_csr()\n");
  massert(false,"SpmvOperator::cuSPARSE_csr -> Not available/implemented for cuda11.0");
  ddebug(" <- SpmvOperator::cuSPARSE_csr()\n");
}

void SpmvOperator::cuSPARSE_hyb() {
  ddebug(" -> SpmvOperator::cuSPARSE_hyb()\n");
  massert(false,"SpmvOperator::cuSPARSE_hyb -> Not available/implemented for cuda11.0");
  ddebug(" <- SpmvOperator::cuSPARSE_hyb()\n");
}

void SpmvOperator::cuSPARSE_bsr() {
  ddebug(" -> SpmvOperator::cuSPARSE_bsr()\n");
  massert(false,"SpmvOperator::cuSPARSE_bsr -> Not available/implemented for cuda11.0");
  ddebug(" <- SpmvOperator::cuSPARSE_bsr()\n");
}
