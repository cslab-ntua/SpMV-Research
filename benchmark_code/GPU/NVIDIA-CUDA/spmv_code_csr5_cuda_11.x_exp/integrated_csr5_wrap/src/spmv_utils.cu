#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some helpfull functions for SpMV
///

#include <time.h>

#include <stdio.h>
#include <stdlib.h>

#include <iostream>
using namespace std;

#include "mmio.h"
#include "spmv_utils.hpp"
//#include "mkl.hpp"

void ddebug(const char *message) {
#ifdef DDEBUG
#define DEBUG
  printf("%s", message);
#endif
}

void debug(const char *message) {
#ifdef DEBUG
  printf("%s", message);
#endif
}

double csecond(void) {
  ddebug(" -> csecond()\n");
  struct timespec tms;

  if (clock_gettime(CLOCK_REALTIME, &tms)) {
    return (0.0);
  }
  /// seconds, multiplied with 1 million
  int64_t micros = tms.tv_sec * 1000000;
  /// Add full microseconds
  micros += tms.tv_nsec / 1000;
  /// round up if necessary
  if (tms.tv_nsec % 1000 >= 500) {
    ++micros;
  }
  ddebug(" <- csecond()\n");
  return ((double)micros / 1000000.0);
}

void massert(bool condi, const char *msg) {
  if (!condi) {
    printf("Error: %s\n", msg);
    exit(1);
  }
}

void spmv_csr(int *csrPtr, int *csrCol, double *csrVal, double *x, double *ys,
              int n) {
  ddebug(" -> spmv_csr()\n");
  int i, j;
  for (i = 0; i < n; ++i) {
    register double yi = 0;
    for (j = csrPtr[i]; j < csrPtr[i + 1]; j++) yi += csrVal[j] * x[csrCol[j]];
    ys[i] = yi;
  }
  ddebug(" <- spmv_csr()\n");
}

void spmv_coo(int *csrInd, int *csrCol, double *csrVal, double *x, double *ys,
              int nz) {
  int i;
  for (i = 0; i < nz; i++) ys[csrInd[i]] += csrVal[i] * x[csrCol[i]];
}

void spmv_coo_f(int *csrInd, int *csrCol, float *csrVal, float *x, float *ys,
              int nz) {
  int i;
  for (i = 0; i < nz; i++) ys[csrInd[i]] += csrVal[i] * x[csrCol[i]];
}

void report_results(double timer, int flops, size_t bytes) {
  double time = timer / NUM_RUN;
  double Gflops = flops / (time * 1.e9);
  double Gbytes = bytes / (time * 1.e9);
  fprintf(stderr,"%lf ms ( %.2lf Gflops/s %.2lf Gbytes/s)", 1000.0 * time, Gflops,
         Gbytes);
}

void report_bandwidth(double timer, size_t bytes) {
  double time = timer / NUM_RUN;
  double Gbytes = bytes / (time * 1.e9);
  printf("%lf ms ( %.2lf Gbytes/s)", 1000.0 * time, Gbytes);
}

double min_elem(double *arr, int size, int *pos) {
  ddebug(" -> min_elem()\n");
  double minimum = arr[0];
  *pos = 0;
  for (int i = 1; i < size; i++)
    if (arr[i] < minimum) {
      minimum = arr[i];
      *pos = i;
    }
  ddebug(" <- min_elem()\n");
  return minimum;
}

void vec_init(double *v, int n, double val) {
  ddebug(" -> vec_init(v, n, val)\n");
  int i;
  for (i = 0; i < n; ++i) {
    v[i] = val;
  }
  ddebug(" <- vec_init(v, n, val)\n");
}

void vec_init_rand(double *v, int n, int np) {
  ddebug(" -> vec_init_rand(v, n, np)\n");
  srand48(42);  // should only be called once
  int i;
  for (i = 0; i < n; ++i) {
    v[i] = (double)drand48();
  }
  for (i = n; i < n + np; ++i) {
    v[i] = 0.0;
  }
  ddebug(" <- vec_init_rand(v, n, np)\n");
}

double serial_dot(double *v1, double *v2, int size) {
  register double sum = 0;
  for (int i = 0; i < size; i++) sum += v1[i] * v2[i];
  return sum;
}

double serial_dot_r2(double *v1, int size) {
  register double sum = 0;
  for (int i = 0; i < size; i++) sum += v1[i] * v1[i];
  return sum;
}

bool breakdown(double inner, double *v, double *w, int size) {
  ddebug(" -> breakdown(inner,v,w,size)\n");

  /// This was copied from miniFE
  // This is code that was copied from Aztec, and originally written
  // by my(his) hero, Ray Tuminaro.
  //
  // Assuming that inner = <v,w> (inner product of v and w),
  // v and w are considered orthogonal if
  //  |inner| < 100 * ||v||_2 * ||w||_2 * epsilon

  bool tmp = 0;
  double vnorm = sqrt(serial_dot(v, v, size));
  double wnorm = sqrt(serial_dot(w, w, size));
  tmp = abs(inner) <=
        100 * vnorm * wnorm * std::numeric_limits<double>::epsilon();
  if (tmp) debug("breakdown(inner,v,w,size) -> Returns breakdown\n");
  ddebug(" <- breakdown(inner,v,w,size)\n");
  return tmp;
}

void serial_waxpby(double *dest, double a, double *x, double b, double *y,
                   int size) {
  for (int i = 0; i < size; i++) dest[i] = a * x[i] + b * y[i];
}

void serial_daxpby(double a, double *x, double b, double *y, int size) {
  for (int i = 0; i < size; i++) y[i] = a * x[i] + b * y[i];
}

int isArraySorted(int* s, int n) {
  int a = 1, i = 0;

  while (a == 1  && i < n - 1) {
	if (s[i] > s[i+1]) a = 0;
    	i++;
  }

  if (a == 1)
    return 1;
  else
    return 0;
}

void mergeSortAux(int *X, int *Y, VALUE_TYPE *Z, int n, int *tmp_X, int *tmp_Y, VALUE_TYPE *tmp_Z){
   int i = 0;
   int j = n/2;
   int ti = 0;

   while (i<n/2 && j<n) {
      if (X[i] < X[j]) {
         tmp_X[ti] = X[i];
         tmp_Y[ti] = Y[i];
         tmp_Z[ti] = Z[i];
         ti++; i++;
      } else {
         tmp_X[ti] = X[j];
         tmp_Y[ti] = Y[j];
         tmp_Z[ti] = Z[j];
         ti++; j++;
      }
   }
   while (i<n/2) { /* finish up lower half */
      tmp_X[ti] = X[i];
      tmp_Y[ti] = Y[i];
      tmp_Z[ti] = Z[i];
      ti++; i++;
   }
   while (j<n) { /* finish up upper half */
      tmp_X[ti] = X[j];
      tmp_Y[ti] = Y[j];
      tmp_Z[ti] = Z[j];
      ti++; j++;
   }
   memcpy(X, tmp_X, n*sizeof(int));
   memcpy(Y, tmp_Y, n*sizeof(int));
   memcpy(Z, tmp_Z, n*sizeof(int));
} 

void mergeSort(int *X, int *Y, VALUE_TYPE *Z, int n, int *tmp_X, int *tmp_Y, VALUE_TYPE *tmp_Z)
{
   if (n < 2) return;

   #pragma omp task shared(X) if (n > TASK_SIZE)
   mergeSort(X, Y, Z, n/2, tmp_X, tmp_Y, tmp_Z);

   #pragma omp task shared(X) if (n > TASK_SIZE)
   mergeSort(X+(n/2), Y+(n/2), Z+(n/2), n-(n/2), tmp_X + n/2, tmp_Y + n/2, tmp_Z + n/2);

   #pragma omp taskwait
   mergeSortAux(X, Y, Z, n, tmp_X, tmp_Y, tmp_Z);
}

SpmvCsrData* mtx_read_csr(char* filename){

    int m, n, nnzA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;
    
	// read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;
    // load matrix
    if ((f = fopen(filename, "r")) == NULL)
        exit(1);

    if (mm_read_banner(f, &matcode) != 0)
    {
        cout << "Could not process Matrix Market banner." << endl;
        exit(2);
    }

    if ( mm_is_complex( matcode ) )
    {
        cout <<"Sorry, data type 'COMPLEX' is not supported. " << endl;
        exit(3);
    }

    if ( mm_is_pattern( matcode ) )  { isPattern = 1; /*cout << "type = Pattern" << endl;*/ }
    if ( mm_is_real ( matcode) )     { isReal = 1; /*cout << "type = real" << endl;*/ }
    if ( mm_is_integer ( matcode ) ) { isInteger = 1; /*cout << "type = integer" << endl;*/ }

    /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
        exit(4);

    if ( mm_is_symmetric( matcode ) || mm_is_hermitian( matcode ) )
    {
        isSymmetric = 1;
        //cout << "symmetric = true" << endl;
    }
    else
    {
        //cout << "symmetric = false" << endl;
    }

    int *csrRowPtrA_counter = (int *)malloc((m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    int *csrRowIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    int *csrColIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    VALUE_TYPE *csrValA_tmp    = (VALUE_TYPE *)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (int i = 0; i < nnzA_mtx_report; i++)
    {
        int idxi, idxj;
        double fval;
        int ival;

        if (isReal)
            fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
        else if (isInteger)
        {
            fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
            fval = ival;
        }
        else if (isPattern)
        {
            fscanf(f, "%d %d\n", &idxi, &idxj);
            fval = 1.0;
        }

        // adjust from 1-based to 0-based
        idxi--;
        idxj--;

        csrRowPtrA_counter[idxi]++;
        csrRowIdxA_tmp[i] = idxi;
        csrColIdxA_tmp[i] = idxj;
        csrValA_tmp[i] = fval;
    }

    if (f != stdin)
        fclose(f);

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
        }
    }

    // exclusive scan for csrRowPtrA_counter
    int old_val, new_val;

    old_val = csrRowPtrA_counter[0];
    csrRowPtrA_counter[0] = 0;
    for (int i = 1; i <= m; i++)
    {
        new_val = csrRowPtrA_counter[i];
        csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i-1];
        old_val = new_val;
    }

    nnzA = csrRowPtrA_counter[m];
    csrRowPtrA = (int *)malloc((m+1) * sizeof(int));
    memcpy(csrRowPtrA, csrRowPtrA_counter, (m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    csrColIdxA = (int *)malloc(nnzA * sizeof(int));
    csrValA    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

                offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
                csrColIdxA[offset] = csrRowIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
            }
            else
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
            }
        }
    }
    else
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
            csrColIdxA[offset] = csrColIdxA_tmp[i];
            csrValA[offset] = csrValA_tmp[i];
            csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
        }
    }

    // free tmp space
    free(csrColIdxA_tmp);
    free(csrValA_tmp);
    free(csrRowIdxA_tmp);
    free(csrRowPtrA_counter);
    
    SpmvCsrData* csr_output = (SpmvCsrData *) malloc(sizeof(SpmvCsrData));
    csr_output->m = m;
	csr_output->n = n;
	csr_output->nz = nnzA;
  	csr_output->rowPtr = csrRowPtrA;
  	csr_output->colInd = csrColIdxA;
  	csr_output->values = csrValA;
  	
  	return csr_output;
}

SpmvCooData* mtx_read_coo(char* mtx_name){
  int ret_code, nz1, *I, *J, ctr;
  VALUE_TYPE *val;
  MM_typecode matcode;
  FILE *f;
  int i;

  if ((f = fopen(mtx_name, "r")) == NULL)
    massert(0, "mtx_read_coo -> Failed to open mtx file");


  if (mm_read_banner(f, &matcode) != 0)
    massert(0,
            "mtx_read_coo -> Could not process Matrix Market "
            "banner");

  //  This is how one can screen matrix types if their application
  //  only supports a subset of the Matrix Market data types. 

  massert(mm_is_valid(matcode),
          "mtx_read_coo -> mm_is_valid(matcode) returned false");
  massert(!mm_is_complex(matcode),
          "mtx_read_coo -> Complex Matrices not supported");
  massert(mm_is_sparse(matcode),
          "mtx_read_coo -> Dense Matrices not supported");

  /* find out size of sparse matrix .... */

  SpmvCooData *data = (SpmvCooData *)malloc(sizeof(SpmvCooData));

  if ((ret_code = mm_read_mtx_crd_size(f, &data->m, &data->n, &nz1)) != 0)
    massert(0,"mtx_read_coo -> Error in finding size of mtx matrix");

  massert(data->n == data->m, "mtx_read_coo -> Only square Matrices supported in this version");

  I = (int *)malloc(nz1 * sizeof(int));
  J = (int *)malloc(nz1 * sizeof(int));
  val = (VALUE_TYPE *)malloc(nz1 * sizeof(VALUE_TYPE));
  data->nz = nz1;

  /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
  /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
  /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

  for (i = 0; i < nz1; i++) {
    if (mm_is_pattern(matcode)) {
      fscanf(f, "%d %d\n", &(I[i]), &(J[i]));
      val[i] = 1.0;
    } else
      fscanf(f, "%d %d %lf\n", &(I[i]), &(J[i]), &(val[i]));
    if (mm_is_symmetric(matcode) && (I[i] != J[i])) data->nz++;
  }

  data->rowInd = (int *)malloc(data->nz * sizeof(int));
  data->colInd = (int *)malloc(data->nz * sizeof(int));
  data->values = malloc(data->nz * sizeof(VALUE_TYPE));
  //mem_bytes += (data->nz) * sizeof(double) + (2 * data->nz) * sizeof(int);
  VALUE_TYPE *values = (VALUE_TYPE *)data->values;

  ctr = nz1;
  for (i = 0; i < nz1; i++) {
    data->rowInd[i] = I[i];
    data->colInd[i] = J[i];
    values[i] = val[i];
    data->rowInd[i]--; /* adjust from 1-based to 0-based */
    data->colInd[i]--;
    if (mm_is_symmetric(matcode) && (data->rowInd[i] != data->colInd[i])) {
      data->rowInd[ctr] = data->colInd[i];
      data->colInd[ctr] = data->rowInd[i];
      values[ctr] = values[i];
      ctr++;
    }
  }

  if (f != stdin) fclose(f);

  int *tmp_X = (int *)malloc(data->nz * sizeof(int));
  int *tmp_Y = (int *)malloc(data->nz * sizeof(int));
  VALUE_TYPE *tmp_Z = (VALUE_TYPE *)malloc(data->nz * sizeof(VALUE_TYPE));

  if (!isArraySorted(data->rowInd, data->nz)) mergeSort(data->rowInd, data->colInd, values, data->nz - 1, tmp_X, tmp_Y, tmp_Z);
  ctr = 0;
  for (i = 1; i < data->nz; i++)
    if (data->rowInd[i] > data->rowInd[i - 1]) {
      if (!isArraySorted(&(data->colInd[ctr]), i - ctr)) mergeSort(&(data->colInd[ctr]), &(data->rowInd[ctr]), &(values[ctr]),
                i - 1 - ctr, tmp_X, tmp_Y, tmp_Z);
      ctr = i;
    }
  if (!isArraySorted(&(data->colInd[ctr]), i - ctr)) mergeSort(&(data->colInd[ctr]), &(data->rowInd[ctr]), &(values[ctr]),
            i - 1 - ctr, tmp_X, tmp_Y, tmp_Z);
  free(I);
  free(J);
  free(val);
  // vec_print<int>(data->rowInd, data->nz, "rowInd");
  // vec_print<int>(data->colInd, data->nz, "colInd");
  // vec_print<double>((double*)data->values, data->nz, "values");
  massert(data->rowInd && data->colInd && data->values,
          "mtx_read_coo -> Format Struct Alloc failed");
  ddebug(" <- mtx_read_coo()\n");
  return data;
}

SpmvCsrData* sortedcoo2csrhost(SpmvCooData* sorted_coo_input)
{
	SpmvCsrData* csr_output = (SpmvCsrData *) malloc(sizeof(SpmvCsrData));
	csr_output->m = sorted_coo_input->m;
	csr_output->n = sorted_coo_input->n;
	csr_output->nz = sorted_coo_input->nz;
  	csr_output->rowPtr = (int *)malloc((csr_output->m + 1) * sizeof(int));
  	csr_output->colInd = (int *)malloc(csr_output->nz * sizeof(int));
  	csr_output->values = malloc(csr_output->nz * sizeof(VALUE_TYPE));

	memcpy(csr_output->colInd, sorted_coo_input->colInd, csr_output->nz * sizeof(int));
	memcpy(csr_output->values, sorted_coo_input->values, csr_output->nz * sizeof(VALUE_TYPE));
	for (int i = 0; i < csr_output->nz; i++)
		csr_output->rowPtr[sorted_coo_input->colInd[i] + 1]++;
	for (int i = 0; i < csr_output->m; i++)
    		csr_output->rowPtr[i + 1] += csr_output->rowPtr[i];

	return csr_output;
}

