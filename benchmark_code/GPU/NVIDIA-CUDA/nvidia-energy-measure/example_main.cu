#include "nvem.hpp"
#include "nvem_helpers.hpp"
#include "hipblas.h"

#define ITER 10000
int main(int args, char* argv[]){
	char* f_out = "Power_data.txt";
	
	int dev_id, verbose = 1; 
	// Uncomment to run example in other than default device
	//hipSetDevice(1);
	hipGetDevice(&dev_id);
	

	// Do stuff in GPU with dev_id
	int N = 1<<22; 
	double *x, *y;
	x = (double*) malloc (N* sizeof(double));
	for (int i = 0; i < N; i++) x[i] = 1.0;
	y = (double*) calloc (N, sizeof(double));
	

	double *d_x, *d_y; 
	double timer = csecond();
	NvemStartMeasure(dev_id, f_out, verbose);
	hipMalloc((void**)&d_x, N*sizeof(double));
	hipMalloc((void**)&d_y, N*sizeof(double));
	hipDeviceSynchronize();	
	NvemStats_p nvem_data_init = NvemStopMeasure(dev_id, "Vector Init GPU");
	
	massert(HIPBLAS_STATUS_SUCCESS == cublasInit(), "cublasInit() failed");
	//Warmup - TODO: Energy measurements after warmup for a 'hot' GPU state. 
	for (int itt = 0; itt< ITER; itt++) hipblasDaxpy(N, 2.0, d_x, 1, d_y, 1);
	hipDeviceSynchronize();
	
	NvemStartMeasure(dev_id, f_out, verbose);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasSetVector(N, sizeof(double), x, 1, d_x, 1), "hipblasSetVector(x) failed");
	massert(HIPBLAS_STATUS_SUCCESS == hipblasSetVector(N, sizeof(double), y, 1, d_y, 1), "hipblasSetVector(y) failed");
	hipDeviceSynchronize();	
	NvemStats_p nvem_data_h2d = NvemStopMeasure(dev_id, " H2D x,y");
	
	NvemStartMeasure(dev_id, f_out, verbose);
	// Perform ITER SAXPYs on 100M elements
	for (int itt = 0; itt< ITER; itt++) hipblasDaxpy(N, 2.0, d_x, 1, d_y, 1);
	hipDeviceSynchronize();	
	NvemStats_p nvem_data_exec = NvemStopMeasure(dev_id, "Daxpy execution");
	
	NvemStartMeasure(dev_id, f_out, verbose);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasGetVector(N, sizeof(double), d_y, 1, y, 1), "hipblasGetVector(y) failed");
	massert(HIPBLAS_STATUS_SUCCESS == cublasShutdown(), "cublasShutdown() failed");
	hipDeviceSynchronize();	
	NvemStats_p nvem_data_d2h = NvemStopMeasure(dev_id, "D2H y");
	timer = csecond() - timer;
	fprintf(stdout, "Stuff executed for total of %lf s\n", timer);

	return 1;
}
