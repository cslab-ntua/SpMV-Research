#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include "nvpl_sparse.h"
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/nvpl_sparse_util.h"
#ifdef __cplusplus
}
#endif

extern int prefetch_distance;

#if DOUBLE == 0
	#define ValueTypeCuda  NVPL_SPARSE_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  NVPL_SPARSE_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	nvpl_sparse_handle_t     handle = NULL;
	nvpl_sparse_sp_mat_descr_t matA;
    nvpl_sparse_spmv_descr_t   mv_descr;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * z = NULL;
	nvpl_sparse_dn_vec_descr_t vecX;
	nvpl_sparse_dn_vec_descr_t vecY;
	nvpl_sparse_dn_vec_descr_t vecZ;

	int max_persistent_l2_cache;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_persistent_l2_cache, cudaDevAttrMaxPersistingL2CacheSize, 0));
		// printf("max_smem_per_block=%d\n", max_smem_per_block);
		// printf("multiproc_count=%d\n", multiproc_count);
		// printf("max_threads_per_block=%d\n", max_threads_per_block);
		// printf("warp_size=%d\n", warp_size);
		// printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		gpuNVPLSparseErrorCheck(nvpl_sparse_create(&handle));
	    gpuNVPLSparseErrorCheck(nvpl_sparse_spmv_create_descr(&mv_descr));

		// Create sparse matrix A in CSR format
		gpuNVPLSparseErrorCheck(nvpl_sparse_create_csr(&matA, m, n, nnz, ia, ja, a, NVPL_SPARSE_INDEX_32I, NVPL_SPARSE_INDEX_32I, NVPL_SPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		// destroy matrix/vector descriptors
		gpuNVPLSparseErrorCheck(nvpl_sparse_destroy_sp_mat(matA));
		gpuNVPLSparseErrorCheck(nvpl_sparse_destroy_dn_vec(vecX));
		gpuNVPLSparseErrorCheck(nvpl_sparse_destroy_dn_vec(vecY));
		gpuNVPLSparseErrorCheck(nvpl_sparse_destroy_dn_vec(vecZ));
		gpuNVPLSparseErrorCheck(nvpl_sparse_spmv_destroy_descr(mv_descr));
		gpuNVPLSparseErrorCheck(nvpl_sparse_destroy(handle));

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "NVPL_SPARSE_CSR";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		// Create dense vector X
		gpuNVPLSparseErrorCheck(nvpl_sparse_create_dn_vec(&csr->vecX, csr->n, x, ValueTypeCuda));

		// Create dense vector y
		gpuNVPLSparseErrorCheck(nvpl_sparse_create_dn_vec(&csr->vecY, csr->m, y, ValueTypeCuda));

		// Create dense vector z
		gpuNVPLSparseErrorCheck(nvpl_sparse_create_dn_vec(&csr->vecZ, csr->m, y, ValueTypeCuda));

		// Allocate an external buffer if needed
		void* dBuffer = NULL;
		gpuNVPLSparseErrorCheck(nvpl_sparse_spmv_buffer_size(csr->handle, NVPL_SPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, csr->vecZ, ValueTypeCuda, NVPL_SPARSE_SPMV_ALG_DEFAULT, csr->mv_descr, &csr->bufferSize));
		dBuffer = malloc(csr->bufferSize);
		gpuNVPLSparseErrorCheck(nvpl_sparse_spmv_analysis(csr->handle, NVPL_SPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, csr->vecZ, ValueTypeCuda, NVPL_SPARSE_SPMV_ALG_DEFAULT, csr->mv_descr, dBuffer));
	}

	gpuNVPLSparseErrorCheck(nvpl_sparse_spmv(csr->handle, NVPL_SPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, csr->vecZ, ValueTypeCuda, NVPL_SPARSE_SPMV_ALG_DEFAULT, csr->mv_descr));
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

