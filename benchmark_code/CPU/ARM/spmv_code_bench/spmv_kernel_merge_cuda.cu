#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <map>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <fstream>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

// #define HIPCUB_STDERR
// using namespace hipcub;

// #include "merge/cub/util_allocator.cuh"
// #include "merge/cub/device/device_spmv.cuh" /// PROBLIMA
// #include "merge/cub/iterator/tex_ref_input_iterator.cuh"

// #include "merge/sparse_matrix.h"
// #include "merge/utils.h"



double * thread_time_compute, * thread_time_barrier;

#ifndef NUM_THREADS
#define NUM_THREADS 1024
#endif

// #ifndef NUM_STREAMS
// #define NUM_STREAMS 1
// #endif

#ifndef TIME_IT
#define TIME_IT 1
#endif

struct MERGEArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;
	
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	MERGEArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpy(ia_d, ia, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpy(ja_d, ja, nnz * sizeof(*ja_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpy(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));

			float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));

			printf("(CUDA) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a);
		}
	}

	~MERGEArrays()
	{
		free(a);
		free(ia);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(MERGEArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
MERGEArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct MERGEArrays * csr = new MERGEArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "MERGE_CUDA");
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= MERGE Custom
//==========================================================================================================================================



void
compute_csr(MERGEArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	if (csr->x == NULL)
	{
		csr->x = x;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(csr->x_d, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}
	}


	gpuCudaErrorCheck(hipDeviceSynchronize());
	
	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y));
		gpuCudaErrorCheck(hipMemcpy(csr->y, csr->y_d, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
MERGEArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
MERGEArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

