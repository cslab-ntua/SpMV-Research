#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cusparse_util.h"
#ifdef __cplusplus
}
#endif

extern int prefetch_distance;

#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct COOArrays : Matrix_Format
{
	INT_T * rowind;      // the usual rowptr (of size m+1)
	INT_T * colind;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * rowind_d;
	INT_T * colind_d;
	ValueType * a_d;

	INT_T * rowind_h;
	INT_T * colind_h;
	ValueType * a_h;

	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	hipsparseDnVecDescr_t vecX;
	hipsparseDnVecDescr_t vecY;

	hipStream_t stream;
	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipEvent_t startEvent_memcpy_rowind;
	hipEvent_t endEvent_memcpy_rowind;
	hipEvent_t startEvent_memcpy_colind;
	hipEvent_t endEvent_memcpy_colind;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_create_matA;
	hipEvent_t endEvent_create_matA;
	hipEvent_t startEvent_spmv_buffersize;
	hipEvent_t endEvent_spmv_buffersize;

	hipEvent_t startEvent_create_vecX;
	hipEvent_t endEvent_create_vecX;
	hipEvent_t startEvent_create_vecY;
	hipEvent_t endEvent_create_vecY;

	int max_persistent_l2_cache;

	COOArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), colind(ja), a(a)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_persistent_l2_cache, cudaDevAttrMaxPersistingL2CacheSize, 0));
		// printf("max_smem_per_block=%d\n", max_smem_per_block);
		// printf("multiproc_count=%d\n", multiproc_count);
		// printf("max_threads_per_block=%d\n", max_threads_per_block);
		// printf("warp_size=%d\n", warp_size);
		// printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		gpuCudaErrorCheck(hipMalloc(&rowind_d, nnz * sizeof(*rowind_d)));
		gpuCudaErrorCheck(hipMalloc(&colind_d, nnz * sizeof(*colind_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));
		gpuCusparseErrorCheck(hipsparseCreate(&handle));
		gpuCusparseErrorCheck(hipsparseSetStream(handle, stream));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_spmv_buffersize));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		rowind = (typeof(rowind)) malloc(nnz * sizeof(*rowind));
		#pragma omp parallel
		{
			long i, j, j_s, j_e;
			#pragma omp for
			for (i=0;i<nnz;i++)
			{
				rowind[i] = 0;
			}
			#pragma omp for
			for (i=0;i<m;i++)
			{
				j_s = ia[i];
				j_e = ia[i+1];
				for (j=j_s;j<j_e;j++)
					rowind[j] = i;
			}
		}

		gpuCudaErrorCheck(hipHostMalloc(&rowind_h, nnz * sizeof(*rowind_h)));
		gpuCudaErrorCheck(hipHostMalloc(&colind_h, nnz * sizeof(*colind_h)));
		gpuCudaErrorCheck(hipHostMalloc(&a_h, nnz * sizeof(*a_h)));
		gpuCudaErrorCheck(hipHostMalloc(&x_h, n * sizeof(*x_h)));
		gpuCudaErrorCheck(hipHostMalloc(&y_h, m * sizeof(*y_h)));

		memcpy(rowind_h, rowind, nnz * sizeof(*rowind_h));
		memcpy(colind_h, colind, nnz * sizeof(*colind_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_rowind));
		gpuCudaErrorCheck(hipMemcpyAsync(rowind_d, rowind_h, nnz * sizeof(*rowind_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_rowind));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_colind));
		gpuCudaErrorCheck(hipMemcpyAsync(colind_d, colind_h, nnz * sizeof(*colind_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_colind));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		// Create sparse matrix A in COO format
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_create_matA));
		gpuCusparseErrorCheck(hipsparseCreateCoo(&matA, m, n, nnz, rowind_d, colind_d, a_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_create_matA));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_create_matA));

			float memcpyTime_cuda_rowind, memcpyTime_cuda_colind, memcpyTime_cuda_a, create_matA_Time;//memcpyTime_cuda_thread_i_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_rowind, startEvent_memcpy_rowind, endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_colind, startEvent_memcpy_colind, endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_matA_Time, startEvent_create_matA, endEvent_create_matA));
			printf("(CUDA) Memcpy rowind time = %.4lf ms, colind time = %.4lf ms, a time = %.4lf ms, matA time = %.4lf ms\n", memcpyTime_cuda_rowind, memcpyTime_cuda_colind, memcpyTime_cuda_a, create_matA_Time);
		}
	}

	~COOArrays()
	{
		free(a);
		free(rowind);
		free(colind);

		// destroy matrix/vector descriptors
		gpuCusparseErrorCheck(hipsparseDestroySpMat(matA));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecX));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecY));
		gpuCusparseErrorCheck(hipsparseDestroy(handle));
		gpuCudaErrorCheck(hipStreamDestroy(stream));

		gpuCudaErrorCheck(hipFree(rowind_d));
		gpuCudaErrorCheck(hipFree(colind_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(dBuffer));

		gpuCudaErrorCheck(hipHostFree(rowind_h));
		gpuCudaErrorCheck(hipHostFree(colind_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_spmv_buffersize));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_coo(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);


void
COOArrays::spmv(ValueType * x, ValueType * y)
{
	compute_coo(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct COOArrays * coo = new COOArrays(row_ptr, col_ind, values, m, n, nnz);
	coo->mem_footprint = nnz * (sizeof(ValueType) + 2 * sizeof(INT_T));
	coo->format_name = (char *) "CUSPARSE_COO";
	return coo;
}


//==========================================================================================================================================
//= COO Custom
//==========================================================================================================================================


void
compute_coo(COOArrays * restrict coo, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (coo->x == NULL)
	{
		coo->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_x, coo->stream));
		memcpy(coo->x_h, x, coo->n * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(coo->x_d, coo->x_h, coo->n * sizeof(*coo->x_d), hipMemcpyHostToDevice, coo->stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_x, coo->stream));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_x, coo->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		// Create dense vector X
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_create_vecX));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecX, coo->n, coo->x_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_create_vecX));

		// Create dense vector y
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_create_vecY));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecY, coo->m, coo->y_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_create_vecY));

		// Allocate an external buffer if needed
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_spmv_buffersize));
		gpuCusparseErrorCheck(hipsparseSpMV_bufferSize(coo->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA, coo->vecX, &beta, coo->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &coo->bufferSize));
		gpuCudaErrorCheck(hipMalloc(&coo->dBuffer, coo->bufferSize));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_spmv_buffersize));
		// printf("SpMV_bufferSize = %zu bytes\n", coo->bufferSize, coo->bufferSize); // size of the workspace that is needed by hipsparseSpMV()

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_spmv_buffersize));
			float create_vecX_time, create_vecY_time, spmv_buffersize_time;
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecX_time, coo->startEvent_create_vecX, coo->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecY_time, coo->startEvent_create_vecY, coo->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventElapsedTime(&spmv_buffersize_time, coo->startEvent_spmv_buffersize, coo->endEvent_spmv_buffersize));
			printf("(CUDA) Create vecX time = %.4lf ms, vecY time = %.4lf ms, spmv_buffersize time = %.4lf (SpMV_bufferSize = %zu)\n", create_vecX_time, create_vecY_time, spmv_buffersize_time, coo->bufferSize);
		}

		#ifdef PERSISTENT_L2_PREFETCH
			int x_d_size = coo->n * sizeof(*coo->x);
			gpuCudaErrorCheck(cudaCtxResetPersistingL2Cache()); // This needs to happen every time before running kernel for 1st time for a matrix...
			if(x_d_size < coo->max_persistent_l2_cache){
				hipLaunchAttributeValue attribute;
				auto &window = attribute.accessPolicyWindow;
				window.base_ptr = coo->x_d;
				window.num_bytes = x_d_size;
				window.hitRatio = 1.0;
				window.hitProp = hipAccessPropertyPersisting;
				window.missProp = hipAccessPropertyStreaming;
				gpuCudaErrorCheck(hipStreamSetAttribute(coo->stream, hipLaunchAttributeAccessPolicyWindow, &attribute));
			}
		#endif
	}

	gpuCusparseErrorCheck(hipsparseSpMV(coo->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA, coo->vecX, &beta, coo->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, coo->dBuffer));
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (coo->y == NULL)
	{
		coo->y = y;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_y, coo->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(coo->y_h, coo->y_d, coo->m * sizeof(*coo->y_d), hipMemcpyDeviceToHost, coo->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(coo->stream));
		memcpy(y, coo->y_h, coo->m * sizeof(ValueType));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_y, coo->stream));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_y, coo->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
COOArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
COOArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

