#include <stdlib.h>
#include <stdio.h>
#include <omp.h>
#include <iostream>
#include <cmath>

#include "dasp/dasp_f64.h"

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	// #include "macros/macrolib.h"
	#include "time_it.h"
	// #include "parallel_util.h"
	// #include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct DASPArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	// DASP specific
	INT_T * order_rid;
	double threshold = 0.75;
	int block_longest = 256;

	int *short_rid_1, *short_rid_2, *short_rid_3, *short_rid_4, *long_rid, *zero_rid;
	int *ridA;

	// INT_T is int
	INT_T *rptA, *long_rpt;
	INT_T *long_rpt_new;
	ValueType *val_by_warp;
	int *rid_by_warp;

	ValueType *short_val, *long_val, *reg_val, *irreg_val;
	int *short_cid, *long_cid, *reg_cid, *irreg_cid;
	INT_T *blockPtr, *irreg_rpt;

	int BlockNum_all;
	int ThreadNum_all;
	int sumBlockNum;

	int rowloop;
	int row_long = 0, row_block = 0, row_zero = 0;
	int blocknum;
	int common_13, short_row_1 = 0, short_row_3 = 0, short_row_2 = 0, short_row_4 = 0, short_row_34;
	int offset_reg, offset_short1, offset_short13, offset_short34, offset_short22;
	INT_T fill0_nnz_short13, fill0_nnz_short34;

	// DASP specific - device buffers
	// init cuda data of long part
	ValueType *long_val_d, *val_by_warp_d;
	int *long_ptr_warp_d;
	int *long_cid_d; 

	// init cuda data of short part
	ValueType *short_val_d;
	int *short_cid_d;

	// init cuda data of reg & irreg part
	ValueType *reg_val_d, *irreg_val_d;
	int *block_ptr_d, *irreg_rpt_d;
	int *reg_cid_d, *irreg_cid_d;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_long_val;
	hipEvent_t endEvent_memcpy_long_val;
	hipEvent_t startEvent_memcpy_val_by_warp;
	hipEvent_t endEvent_memcpy_val_by_warp;
	hipEvent_t startEvent_memcpy_long_ptr_warp;
	hipEvent_t endEvent_memcpy_long_ptr_warp;
	hipEvent_t startEvent_memcpy_long_cid;
	hipEvent_t endEvent_memcpy_long_cid;
	hipEvent_t startEvent_memcpy_short_val;
	hipEvent_t endEvent_memcpy_short_val;
	hipEvent_t startEvent_memcpy_short_cid;
	hipEvent_t endEvent_memcpy_short_cid;
	hipEvent_t startEvent_memcpy_reg_val;
	hipEvent_t endEvent_memcpy_reg_val;
	hipEvent_t startEvent_memcpy_irreg_val;
	hipEvent_t endEvent_memcpy_irreg_val;
	hipEvent_t startEvent_memcpy_block_ptr;
	hipEvent_t endEvent_memcpy_block_ptr;
	hipEvent_t startEvent_memcpy_irreg_rpt;
	hipEvent_t endEvent_memcpy_irreg_rpt;
	hipEvent_t startEvent_memcpy_reg_cid;
	hipEvent_t endEvent_memcpy_reg_cid;
	hipEvent_t startEvent_memcpy_irreg_cid;
	hipEvent_t endEvent_memcpy_irreg_cid;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	DASPArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		/************************************************************************************************************************************************************************/
		/*************************************************************************** DASP preprocessing **************************************************************************/
		/************************************************************************************************************************************************************************/
		order_rid = (typeof(order_rid))malloc(m * sizeof(*order_rid));

		// three parts: short row (1 & 3 & 2 & 4), long row, row-block (regular（origin & fill0） & irregular)
		INT_T nnz_short, nnz_long, origin_nnz_reg, fill0_nnz_reg, nnz_irreg;

		// block_longest = -1;
		// for(int i=0; i<m; i++){
		// 	int row_len = ia[i + 1] - ia[i];
		// 	if(row_len>block_longest)
		// 		block_longest=row_len;
		// }

		// get the short part data
		// (short_val, short_cid)
		for (int i = 0; i < m; i ++)
		{
			int row_len = ia[i + 1] - ia[i];
			if (row_len == 1)
			{
				short_row_1 ++;
			}
			else if (row_len == 3)
			{
				short_row_3 ++;
			}
			else if (row_len == 2)
			{
				short_row_2 ++;
			}
			else if (row_len == 0)
			{
				row_zero ++;
			}
			else if (row_len == 4)
			{
				short_row_4 ++;
			}
			// else if (row_len >= warpNum_long * loopNum_long * MMA_M * MMA_K)
			else if (row_len >= block_longest)
			{
				row_long ++;
			}
			else
			{
				row_block ++;
			}
		}

		if (row_block < 59990) rowloop = 1;
		else if (row_block >= 59990 && row_block < 400000) rowloop = 2;
		else rowloop = 4;

		short_rid_1 = (typeof(short_rid_1))malloc(short_row_1 * sizeof(*short_rid_1));
		short_rid_2 = (typeof(short_rid_2))malloc(short_row_2 * sizeof(*short_rid_2));
		short_rid_3 = (typeof(short_rid_3))malloc(short_row_3 * sizeof(*short_rid_3));
		short_rid_4 = (typeof(short_rid_4))malloc(short_row_4 * sizeof(*short_rid_4));
		long_rid = (typeof(long_rid))malloc(row_long * sizeof(*long_rid));
		zero_rid = (typeof(zero_rid))malloc(row_zero * sizeof(*zero_rid));
		ridA = (typeof(ridA))malloc(row_block * sizeof(*ridA));

		INT_T *rptA = (typeof(rptA))malloc((row_block + 1) * sizeof(*rptA));
		memset(rptA, 0, sizeof(INT_T) * (row_block + 1));
		INT_T *long_rpt = (typeof(long_rpt))malloc((row_long + 1) * sizeof(*long_rpt));
		memset(long_rpt, 0, sizeof(INT_T) * (row_long + 1));

		int short_row_flag1 = 0, short_row_flag3 = 0, short_row_flag2 = 0, short_row_flag4 = 0;
		int row_long_flag = 0, flag0 = 0, row_block_flag = 0;
		for (int i = 0; i < m; i ++)
		{
			int row_len = ia[i + 1] - ia[i];
			if (row_len == 1)
			{
				short_rid_1[short_row_flag1] = i;
				short_row_flag1 ++;
			}
			else if (row_len == 3)
			{
				short_rid_3[short_row_flag3] = i;
				short_row_flag3 ++;
			}
			else if (row_len == 2)
			{
				short_rid_2[short_row_flag2] = i;
				short_row_flag2 ++;
			}
			else if (row_len == 0)
			{
				zero_rid[flag0] = i;
				flag0 ++;
			}
			else if (row_len == 4)
			{
				short_rid_4[short_row_flag4] = i;
				short_row_flag4 ++;
			}
			// else if (row_len >= warpNum_long * loopNum_long * MMA_M * MMA_K)
			else if (row_len >= block_longest)
			{
				long_rpt[row_long_flag] = row_len;
				long_rid[row_long_flag] = i;
				row_long_flag ++;
			}
			else
			{
				rptA[row_block_flag] = row_len;
				ridA[row_block_flag] = i;
				row_block_flag ++;
			}
		} 
		nnz_short = short_row_1 + short_row_3 * 3 + short_row_2 * 2 + short_row_4 * 4;
	 
		common_13 = short_row_1 < short_row_3 ? short_row_1 : short_row_3;
		if (common_13 / BlockSize >= 16)
		{
			common_13 = BlockSize * (common_13 / BlockSize);
			short_row_1 = short_row_1 - common_13;
			short_row_3 = short_row_3 - common_13;
		}
		else
		{
			common_13 = 0;
		}

		int short_block13 = (common_13 + BlockSize - 1) / BlockSize;  
		int half_short_row_2 = (short_row_2 + 1) / 2;
		int short_block22 = (half_short_row_2 + BlockSize - 1) / BlockSize;
		short_row_34 = short_row_3 + short_row_4;
		int short_block34 = (short_row_34 + BlockSize - 1) / BlockSize;

		int block13_per_threadblock = warpNum_short * groupNum * 2;
		int block22_per_threadblock = warpNum_short * groupNum * 2;
		int block34_per_threadblock = warpNum_short * groupNum * loopNum_short;

		int threadblock13 = (short_block13 + block13_per_threadblock - 1) / block13_per_threadblock;
		int threadblock22 = (short_block22 + block22_per_threadblock - 1) / block22_per_threadblock;
		int threadblock34 = (short_block34 + block34_per_threadblock - 1) / block34_per_threadblock;

		fill0_nnz_short13 = threadblock13 * block13_per_threadblock * MMA_M * MMA_K;
		fill0_nnz_short34 = threadblock34 * block34_per_threadblock * MMA_M * MMA_K;
		INT_T fill0_nnz_short22 = threadblock22 * block22_per_threadblock * MMA_M * MMA_K;
		INT_T fill0_nnz_short = short_row_1 + fill0_nnz_short13 + fill0_nnz_short34 + fill0_nnz_short22;
		short_val = (typeof(short_val))malloc(fill0_nnz_short * sizeof(*short_val));
		short_cid = (typeof(short_cid))malloc(fill0_nnz_short * sizeof(*short_cid));
		memset(short_val, 0.0, sizeof(ValueType) * fill0_nnz_short);
		memset(short_cid, 0, sizeof(int) * fill0_nnz_short);

		int super_group = 1 + threadblock13 + threadblock34 + threadblock22;
		INT_T *superX_ptr = (typeof(superX_ptr))malloc((super_group + 1) * sizeof(*superX_ptr));
		
		for (int i = 0; i < short_row_1; i ++)
		{
			int cur_row = short_rid_1[i];
			short_val[i] = a[ia[cur_row]];
			short_cid[i] = ja[ia[cur_row]];
		}

		for (int i = 0; i < short_block13; i ++)
		{
			ValueType *cur_short_val = short_val + short_row_1 + i * MMA_M * MMA_K;
			int *cur_short_cid = short_cid + short_row_1 + i * MMA_M * MMA_K;

			for (int j = 0; j < BlockSize && i * BlockSize + j < common_13; j ++)
			{
				int cur_row_1 = short_rid_1[short_row_1 + i * BlockSize + j];
				int cur_row_3 = short_rid_3[i * BlockSize + j];
				cur_short_val[j * MMA_K] = a[ia[cur_row_1]];
				cur_short_cid[j * MMA_K] = ja[ia[cur_row_1]];
				cur_short_val[j * MMA_K + 1] = a[ia[cur_row_3]];
				cur_short_val[j * MMA_K + 2] = a[ia[cur_row_3] + 1];
				cur_short_val[j * MMA_K + 3] = a[ia[cur_row_3] + 2];
				cur_short_cid[j * MMA_K + 1] = ja[ia[cur_row_3]];
				cur_short_cid[j * MMA_K + 2] = ja[ia[cur_row_3] + 1];
				cur_short_cid[j * MMA_K + 3] = ja[ia[cur_row_3] + 2];
			}
		}

		for (int i = 0; i < short_row_3; i ++)
		{
			// ValueType *cur_short_val = short_val + short_row_1 + short_block13 * MMA_M * MMA_K + i * MMA_K;
			// int *cur_short_cid = short_cid + short_row_1 + short_block13 * MMA_M * MMA_K + i * MMA_K;
			ValueType *cur_short_val = short_val + short_row_1 + fill0_nnz_short13 + i * MMA_K;
			int *cur_short_cid = short_cid + short_row_1 + fill0_nnz_short13 + i * MMA_K;
			
			int cur_row = short_rid_3[common_13 + i];

			cur_short_val[0] = a[ia[cur_row]];
			cur_short_val[1] = a[ia[cur_row] + 1]; 
			cur_short_val[2] = a[ia[cur_row] + 2]; 
			cur_short_cid[0] = ja[ia[cur_row]];
			cur_short_cid[1] = ja[ia[cur_row] + 1]; 
			cur_short_cid[2] = ja[ia[cur_row] + 2]; 
		}

		for (int i = 0; i < short_row_4; i ++)
		{
			ValueType *cur_short_val = short_val + short_row_1 + fill0_nnz_short13 + (short_row_3 + i) * MMA_K;
			int *cur_short_cid = short_cid + short_row_1 + fill0_nnz_short13 + (short_row_3 + i) * MMA_K;
			
			int cur_row = short_rid_4[i];

			cur_short_val[0] = a[ia[cur_row]];
			cur_short_val[1] = a[ia[cur_row] + 1]; 
			cur_short_val[2] = a[ia[cur_row] + 2]; 
			cur_short_val[3] = a[ia[cur_row] + 3]; 
			cur_short_cid[0] = ja[ia[cur_row]];
			cur_short_cid[1] = ja[ia[cur_row] + 1]; 
			cur_short_cid[2] = ja[ia[cur_row] + 2]; 
			cur_short_cid[3] = ja[ia[cur_row] + 3]; 
		}

		for (int i = 0; i < short_block22; i ++)
		{
			ValueType *cur_short_val = short_val + short_row_1 + fill0_nnz_short13 + fill0_nnz_short34 + i * MMA_M * MMA_K;
			int *cur_short_cid = short_cid + short_row_1 + fill0_nnz_short13 + fill0_nnz_short34 + i * MMA_M * MMA_K;

			for (int j = 0; j < BlockSize * 2 && (i * BlockSize * 2 + j) < short_row_2; j ++)
			{
				int cur_row = short_rid_2[i * BlockSize * 2 + j];
				cur_short_val[j % BlockSize * MMA_K + (j / BlockSize) * 2] = a[ia[cur_row]];
				cur_short_val[j % BlockSize * MMA_K + (j / BlockSize) * 2 + 1] = a[ia[cur_row] + 1];
				cur_short_cid[j % BlockSize * MMA_K + (j / BlockSize) * 2] = ja[ia[cur_row]];
				cur_short_cid[j % BlockSize * MMA_K + (j / BlockSize) * 2 + 1] = ja[ia[cur_row] + 1];
			}
		}

		int *short_cid_temp = (typeof(short_cid_temp))malloc(fill0_nnz_short * sizeof(*short_cid_temp));
		memcpy(short_cid_temp, short_cid, sizeof(int) * fill0_nnz_short);

		quick_sort_key(short_cid_temp, short_row_1);
		int nnzr = short_row_1 > 0 ? 1 : 0;
		for (int i = 1; i < short_row_1; i ++)
		{
			nnzr += short_cid_temp[i] != short_cid_temp[i - 1] ? 1 : 0;
		}
		superX_ptr[0] = nnzr;

		INT_T *cur_superX_ptr = superX_ptr + 1;
		for (int i = 0; i < threadblock13; i++)
		{
			int *cur_short_cid_temp = short_cid_temp + short_row_1 + i * block13_per_threadblock * MMA_M * MMA_K;
			int len = block13_per_threadblock * MMA_M * MMA_K;
			quick_sort_key(cur_short_cid_temp, len);
			int nnzcid = len > 0 ? 1 : 0;
			for (int j = 1; j < len; j ++)
			{
				nnzcid += cur_short_cid_temp[j] != cur_short_cid_temp[j - 1] ? 1 : 0;
			}
			cur_superX_ptr[i] = nnzcid;
		}

		cur_superX_ptr = superX_ptr + 1 + threadblock13;
		for (int i = 0; i < threadblock34; i++)
		{
			int *cur_short_cid_temp = short_cid_temp + short_row_1 + fill0_nnz_short13 + i * block34_per_threadblock * MMA_M * MMA_K;
			int len = block34_per_threadblock * MMA_M * MMA_K;
			quick_sort_key(cur_short_cid_temp, len);
			int nnzcid = len > 0 ? 1 : 0;
			for (int j = 1; j < len; j ++)
			{
				nnzcid += cur_short_cid_temp[j] != cur_short_cid_temp[j - 1] ? 1 : 0;
			}
			cur_superX_ptr[i] = nnzcid;
		}

		cur_superX_ptr = superX_ptr + 1 + threadblock13 + threadblock34;
		for (int i = 0; i < threadblock22; i++)
		{
			int *cur_short_cid_temp = short_cid_temp + short_row_1 + fill0_nnz_short13 + fill0_nnz_short34 + i * block22_per_threadblock * MMA_M * MMA_K;
			int len = block22_per_threadblock * MMA_M * MMA_K;
			quick_sort_key(cur_short_cid_temp, len);
			int nnzcid = len > 0 ? 1 : 0;
			for (int j = 1; j < len; j ++)
			{
				nnzcid += cur_short_cid_temp[j] != cur_short_cid_temp[j - 1] ? 1 : 0;
			}
			cur_superX_ptr[i] = nnzcid;
		}
		exclusive_scan(superX_ptr, super_group + 1);
		INT_T nnz_superX = superX_ptr[super_group];

		int new_cid_len = short_row_1 + threadblock13 * block13_per_threadblock * MMA_M * MMA_K / 4 + \
										threadblock34 * block34_per_threadblock * MMA_M * MMA_K / 4 + \
										threadblock22 * block22_per_threadblock * MMA_M * MMA_K / 4;

		int *short_cid_new = (typeof(short_cid_new))malloc(new_cid_len * sizeof(*short_cid_new));

		int *superX_cid = (typeof(superX_cid))malloc(nnz_superX * sizeof(*superX_cid));
		int flag = 0;
		if (short_row_1)
		{
			superX_cid[0] = short_cid_temp[0];
			flag ++;
		}
		for (int j = 1; j < short_row_1; j ++)
		{
			if (short_cid_temp[j] != short_cid_temp[j - 1])
			{
				superX_cid[flag] = short_cid_temp[j];
				flag ++;
			}
		}
		if (flag != superX_ptr[1]) printf("flag1 = %d, len = %d\n", flag, superX_ptr[1]);
		for (int i = 0; i < short_row_1; i ++)
		{
			short_cid_new[i] = BinarySearch(superX_cid, superX_ptr[1], short_cid[i]);
		}

		cur_superX_ptr = superX_ptr + 1;
		for (int i = 0; i < threadblock13; i ++)
		{
			int *cur_short_cid_temp = short_cid_temp + short_row_1 + i * block13_per_threadblock * MMA_M * MMA_K;
			int len = block13_per_threadblock * MMA_M * MMA_K;
			int *cur_superX_cid = superX_cid + cur_superX_ptr[i];
			int xlen = cur_superX_ptr[i + 1] - cur_superX_ptr[i];
			int flag_cid = 0;
			if (len)
			{
				cur_superX_cid[0] = cur_short_cid_temp[0];
				flag_cid ++;
			}
			else
			{
				continue;
			}
			for (int j = 1; j < len; j ++)
			{
				if (cur_short_cid_temp[j] != cur_short_cid_temp[j - 1])
				{
					cur_superX_cid[flag_cid] = cur_short_cid_temp[j];
					flag_cid ++;
				}
			}
			if (flag_cid != xlen) printf("flag13 = %d, len = %d\n", flag_cid, xlen);

			int *cur_short_cid_new = short_cid_new + short_row_1 + i * (block13_per_threadblock * MMA_M * MMA_K / 4);
			int *cur_short_cid = short_cid + short_row_1 + i * block13_per_threadblock * MMA_M * MMA_K;
			for (int j = 0; j < len; j ++)
			{
				// cur_short_cid_new[j] = BinarySearch(cur_superX_cid, xlen, cur_short_cid[j]);
				SET_8_BIT(cur_short_cid_new[j / 4], BinarySearch(cur_superX_cid, xlen, cur_short_cid[j]), j % 4);
			}
		}

		cur_superX_ptr = superX_ptr + 1 + threadblock13;
		for (int i = 0; i < threadblock34; i ++)
		{
			int *cur_short_cid_temp = short_cid_temp + short_row_1 + fill0_nnz_short13 + i * block34_per_threadblock * MMA_M * MMA_K;
			int len = block34_per_threadblock * MMA_M * MMA_K;
			int *cur_superX_cid = superX_cid + cur_superX_ptr[i];
			int xlen = cur_superX_ptr[i + 1] - cur_superX_ptr[i];
			int flag_cid = 0;
			if (len)
			{
				cur_superX_cid[0] = cur_short_cid_temp[0];
				flag_cid ++;
			}
			else
			{
				continue;
			}
			for (int j = 1; j < len; j ++)
			{
				if (cur_short_cid_temp[j] != cur_short_cid_temp[j - 1])
				{
					cur_superX_cid[flag_cid] = cur_short_cid_temp[j];
					flag_cid ++;
				}
			}
			if (flag_cid != xlen) printf("flag34 = %d, len = %d\n", flag_cid, xlen);

			int *cur_short_cid_new = short_cid_new + short_row_1 + fill0_nnz_short13 / 4 + i * (block34_per_threadblock * MMA_M * MMA_K / 4);
			int *cur_short_cid = short_cid + short_row_1 + fill0_nnz_short13 + i * block34_per_threadblock * MMA_M * MMA_K;
			for (int j = 0; j < len; j ++)
			{
				// cur_short_cid_new[j] = BinarySearch(cur_superX_cid, xlen, cur_short_cid[j]);
				SET_8_BIT(cur_short_cid_new[j / 4], BinarySearch(cur_superX_cid, xlen, cur_short_cid[j]), j % 4);
			}
		}

		cur_superX_ptr = superX_ptr + 1 + threadblock13 + threadblock34;
		for (int i = 0; i < threadblock22; i ++)
		{
			int *cur_short_cid_temp = short_cid_temp + short_row_1 + fill0_nnz_short13 + fill0_nnz_short34 + i * block22_per_threadblock * MMA_M * MMA_K;
			int len = block22_per_threadblock * MMA_M * MMA_K;
			int *cur_superX_cid = superX_cid + cur_superX_ptr[i];
			int xlen = cur_superX_ptr[i + 1] - cur_superX_ptr[i];
			int flag_cid = 0;
			if (len)
			{
				cur_superX_cid[0] = cur_short_cid_temp[0];
				flag_cid ++;
			}
			else
			{
				continue;
			}
			for (int j = 1; j < len; j ++)
			{
				if (cur_short_cid_temp[j] != cur_short_cid_temp[j - 1])
				{
					cur_superX_cid[flag_cid] = cur_short_cid_temp[j];
					flag_cid ++;
				}
			}
			if (flag_cid != xlen) printf("flag22 = %d, len = %d\n", flag_cid, xlen);

			int *cur_short_cid_new = short_cid_new + short_row_1 + (fill0_nnz_short13 + fill0_nnz_short34) / 4 + i * (block22_per_threadblock * MMA_M * MMA_K / 4);
			int *cur_short_cid = short_cid + short_row_1 + fill0_nnz_short13 + fill0_nnz_short34 + i * block22_per_threadblock * MMA_M * MMA_K;
			for (int j = 0; j < len; j ++)
			{
				// cur_short_cid_new[j] = BinarySearch(cur_superX_cid, xlen, cur_short_cid[j]);
				SET_8_BIT(cur_short_cid_new[j / 4], BinarySearch(cur_superX_cid, xlen, cur_short_cid[j]), j % 4);
			}
		}
		free(superX_ptr);
		free(superX_cid);
		free(short_cid_temp);
		free(short_cid_new);

		// resort except rows
		radix_sort(rptA, ridA, row_block);

		// get the data except short row part
		// (rptA, cidA, valA)
		exclusive_scan(rptA, row_block + 1);
		exclusive_scan(long_rpt, row_long + 1);
		// nnz_row_block = rptA[row_block];
		nnz_long = long_rpt[row_long];

		//record the sort order
		memcpy(order_rid, long_rid, sizeof(int) * row_long);
		memcpy(order_rid + row_long, ridA, sizeof(int) * row_block);
		memcpy(order_rid + row_long + row_block, short_rid_1, sizeof(int) * short_row_1);
		for (int i = 0; i < short_block13; i ++)
		{
			int *cur_order_rid = order_rid + row_long + row_block + short_row_1 + i * BlockSize * 2;

			for (int j = 0; j < BlockSize; j ++)
			{
				cur_order_rid[j] = short_rid_1[short_row_1 + i * BlockSize + j];
				cur_order_rid[BlockSize + j] = short_rid_3[i * BlockSize + j];
			}
		}
		memcpy(order_rid + row_long + row_block + short_row_1 + common_13 * 2, short_rid_3 + common_13, sizeof(int) * short_row_3);
		memcpy(order_rid + row_long + row_block + short_row_1 + common_13 * 2 + short_row_3, short_rid_4, sizeof(int) * short_row_4);
		memcpy(order_rid + row_long + row_block + short_row_1 + common_13 * 2 + short_row_3 + short_row_4, short_rid_2, sizeof(int) * short_row_2);
		memcpy(order_rid + row_long + row_block + short_row_1 + common_13 * 2 + short_row_3 + short_row_4 + short_row_2, zero_rid, sizeof(int) * row_zero);

		int short_row = short_row_1 + common_13 * 2 + short_row_34 + short_row_2;
		int offset_short_row = row_long + row_block;

		ValueType *short3_val = (typeof(short3_val))malloc(nnz_short * sizeof(*short3_val));
		int *short3_cid = (typeof(short3_cid))malloc(nnz_short * sizeof(*short3_cid));
		INT_T *short3_rpt = (typeof(short3_rpt))malloc((short_row + 1) * sizeof(*short3_rpt));

		for (int i = 0; i < short_row; i ++)
		{
			int idx = order_rid[offset_short_row + i];
			short3_rpt[i] = ia[idx + 1] - ia[idx];
		}
		exclusive_scan(short3_rpt, short_row + 1);

		for (int i = 0; i < short_row; i ++)
		{
			int idx = order_rid[offset_short_row + i];
			memcpy(short3_val + short3_rpt[i], a + ia[idx], sizeof(ValueType) * (ia[idx + 1] - ia[idx]));
			memcpy(short3_cid + short3_rpt[i], ja + ia[idx], sizeof(int) * (ia[idx + 1] - ia[idx]));
		}
		free(short3_val);
		free(short3_cid);
		free(short3_rpt);

		// get the long part data
		INT_T *long_rpt_new = (typeof(long_rpt_new))malloc((row_long + 1) * sizeof(*long_rpt_new));
		memset(long_rpt_new, 0, sizeof(INT_T) * (row_long + 1));
		int warp_number = 0;
		for (int i = 0; i < row_long; i ++)
		{
			int nnz_num = long_rpt[i + 1] - long_rpt[i];
			int cur_warp_num = (nnz_num + MMA_M * MMA_K * loopNum_long - 1) / (MMA_M * MMA_K * loopNum_long);
			// warp_number += cur_warp_num;
			long_rpt_new[i] = cur_warp_num;
		}
		exclusive_scan(long_rpt_new, row_long + 1);
		warp_number = long_rpt_new[row_long];

		int BlockNum_long = (warp_number + warpNum_long - 1) / warpNum_long;
		int fill0_nnz_long = BlockNum_long * warpNum_long * loopNum_long * MMA_M * MMA_K;
		warp_number = BlockNum_long * warpNum_long;
		val_by_warp = (typeof(val_by_warp))malloc(warp_number * sizeof(*val_by_warp));
		rid_by_warp = (typeof(rid_by_warp))malloc(warp_number * sizeof(*rid_by_warp));
		long_val = (typeof(long_val))malloc(fill0_nnz_long * sizeof(*long_val));
		memset(long_val, 0.0, sizeof(ValueType) * fill0_nnz_long);
		long_cid = (typeof(long_cid))malloc(fill0_nnz_long * sizeof(*long_cid));
		memset(long_cid, 0, sizeof(int) * fill0_nnz_long);

		// int count_warp = 0;
		for (int i = 0; i < row_long; i ++)
		{
			ValueType *cur_val = long_val + long_rpt_new[i] * loopNum_long * MMA_M * MMA_K;
			int *cur_cid = long_cid + long_rpt_new[i] * loopNum_long * MMA_M * MMA_K;
			int real_rid = long_rid[i];
			for (int j = 0; j < long_rpt[i + 1] - long_rpt[i]; j ++)
			{
				cur_val[j] = a[ia[real_rid] + j];
				cur_cid[j] = ja[ia[real_rid] + j];
			}

			for (int j = long_rpt_new[i]; j < long_rpt_new[i + 1]; j ++)
			{
				rid_by_warp[j] = i;
			}
		}

		// preprocessing the row-block part : divide that into regular part and irregular part  
		blocknum = (row_block + BlockSize - 1) / BlockSize;
		blocknum = ((blocknum + rowloop * 4 - 1) / (rowloop * 4)) * rowloop * 4;
		blockPtr = (typeof(blockPtr))malloc((blocknum + 1) * sizeof(*blockPtr));
		memset(blockPtr, 0, sizeof(INT_T) * (blocknum + 1));

		irreg_rpt = (typeof(irreg_rpt))malloc((row_block + 1) * sizeof(*irreg_rpt));
		memset(irreg_rpt, 0, sizeof(INT_T) * (row_block + 1));

		#pragma omp parallel for
		for (int i = 0; i < blocknum; i++)
		{
			int row_start = i * BlockSize;
			int row_end = (i + 1) * BlockSize >= row_block ? row_block : (i + 1) * BlockSize;
			int k = 1;
			while(1)
			{
				int block_nnz = 0;
				for (int cur_row = row_start; cur_row < row_end; cur_row++)
				{
					int row_len = rptA[cur_row + 1] - rptA[cur_row];
					if (row_len / MMA_K >= k) block_nnz += MMA_K;
					else if(row_len / MMA_K == k - 1) block_nnz += row_len % MMA_K;
				}
				
				if (block_nnz >= threshold * MMA_K * MMA_M)
				{
					blockPtr[i] += MMA_K * MMA_M;
				}
				else
				{
					for (int cur_row = row_start; cur_row < row_end; cur_row++ )
					{
						int row_len = rptA[cur_row + 1] - rptA[cur_row];
						irreg_rpt[cur_row] = row_len - (k - 1) * MMA_K > 0 ? row_len - (k - 1) * MMA_K : 0;
					}
					break;
				}
				k++;
			}
		}
		
		exclusive_scan(blockPtr, blocknum + 1);
		exclusive_scan(irreg_rpt, row_block + 1);
		
		// int offset_row_block = row_long;
		fill0_nnz_reg = blockPtr[blocknum];
		nnz_irreg = irreg_rpt[row_block];
		origin_nnz_reg = nnz - nnz_irreg - nnz_long - nnz_short;

		// get the row-block part data---irregular part
		irreg_val = (typeof(irreg_val))malloc(nnz_irreg * sizeof(*irreg_val));
		irreg_cid = (typeof(irreg_cid))malloc(nnz_irreg * sizeof(*irreg_cid));
		for (int i = 0; i < row_block; i ++)
		{
			int cur_rid = ridA[i];
			int irreg_offset = irreg_rpt[i];
			int irreg_len = irreg_rpt[i + 1] - irreg_offset;
			for (int j = 0; j < irreg_len; j ++)
			{
				irreg_val[irreg_offset + j] = a[ia[cur_rid + 1] - irreg_len + j];
				irreg_cid[irreg_offset + j] = ja[ia[cur_rid + 1] - irreg_len + j];
			}
		}

		// get the row_block part data---regular part
		reg_val = (typeof(reg_val))malloc(fill0_nnz_reg * sizeof(*reg_val));
		reg_cid = (typeof(reg_cid))malloc(fill0_nnz_reg * sizeof(*reg_cid));

		for (int bid = 0; bid < blocknum; bid ++)
		{
			int nnz_block = (blockPtr[bid + 1] - blockPtr[bid]);
			int blocklen = nnz_block / BlockSize;

			for (int rowid = bid * BlockSize; rowid < (bid + 1) * BlockSize; rowid ++)
			{
				int regA_start = blockPtr[bid] + blocklen * (rowid - bid * BlockSize);
				if (rowid < row_block)
				{
					int real_id = ridA[rowid];
					int A_start = ia[real_id];
					int row_len = ia[real_id + 1] - A_start;
					for (int i = 0; i < blocklen; i ++)
					{
						reg_val[regA_start + i] = i < row_len ? a[A_start + i] : 0.0;
						reg_cid[regA_start + i] = i < row_len ? ja[A_start + i] : 0;
					}
				}
				else
				{
					for (int i = 0; i < blocklen; i ++)
					{
						reg_val[regA_start + i] = 0.0;
						reg_cid[regA_start + i] = 0;
					}
				}

			}

			ValueType *temp_val = (typeof(temp_val))malloc(nnz_block * sizeof(*temp_val));
			int *temp_cid = (typeof(temp_cid))malloc(nnz_block * sizeof(*temp_cid));
			ValueType *cur_val = reg_val + blockPtr[bid];
			int *cur_cid = reg_cid + blockPtr[bid];

			for (int i = 0; i < nnz_block; i ++)
			{
				int new_id = ((i % blocklen) / MMA_K) * BlockSize * MMA_K + (i / blocklen) * MMA_K + i % MMA_K;
				temp_val[new_id] = cur_val[i];
				temp_cid[new_id] = cur_cid[i];
			}
			memcpy(cur_val, temp_val, sizeof(ValueType) * nnz_block);
			memcpy(cur_cid, temp_cid, sizeof(int) * nnz_block);
			free(temp_val);
			free(temp_cid);
		}

		long fill0_nnz = fill0_nnz_short + fill0_nnz_long + nnz_irreg + fill0_nnz_reg;
		double rate_fill0 = (double)(fill0_nnz - nnz) / nnz;

		int BlockNum = (blocknum + rowloop * 4 - 1) / (rowloop * 4);
		int ThreadNum_short = warpNum_short * WARP_SIZE;
		int BlockNum_short_1 = (short_row_1 + ThreadNum_short - 1) / ThreadNum_short;
		int BlockNum_short = BlockNum_short_1 + threadblock13 + threadblock34 + threadblock22;

		offset_reg = BlockNum_long;
		offset_short1 = offset_reg + BlockNum;
		offset_short13 = offset_short1 + BlockNum_short_1;
		offset_short34 = offset_short13 + threadblock13;
		offset_short22 = offset_short34 + threadblock34;

		BlockNum_all = BlockNum_long + BlockNum + BlockNum_short;
		ThreadNum_all = 4 * WARP_SIZE;

		sumBlockNum = (row_long + 3) / 4;

		/************************************************************************************************************************************************************************/
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCudaErrorCheck(hipMalloc(&long_ptr_warp_d, (row_long + 1) * sizeof(*long_ptr_warp_d)));
		gpuCudaErrorCheck(hipMalloc(&long_cid_d, fill0_nnz_long * sizeof(*long_cid_d)));
		gpuCudaErrorCheck(hipMalloc(&long_val_d, fill0_nnz_long * sizeof(*long_val_d)));
		gpuCudaErrorCheck(hipMalloc(&val_by_warp_d, warp_number * sizeof(*val_by_warp_d)));
		gpuCudaErrorCheck(hipMemset(val_by_warp_d, 0, warp_number * sizeof(*val_by_warp_d)));

		gpuCudaErrorCheck(hipMalloc(&short_cid_d, fill0_nnz_short * sizeof(*short_cid_d)));
		gpuCudaErrorCheck(hipMalloc(&short_val_d, fill0_nnz_short * sizeof(*short_val_d)));
		
		gpuCudaErrorCheck(hipMalloc(&block_ptr_d, (blocknum + 1) * sizeof(*block_ptr_d)));
		gpuCudaErrorCheck(hipMalloc(&reg_cid_d, fill0_nnz_reg * sizeof(*reg_cid_d)));
		gpuCudaErrorCheck(hipMalloc(&reg_val_d, fill0_nnz_reg * sizeof(*reg_val_d)));
		
		gpuCudaErrorCheck(hipMalloc(&irreg_rpt_d, (row_block + 1) * sizeof(*irreg_rpt_d)));
		gpuCudaErrorCheck(hipMalloc(&irreg_cid_d, nnz_irreg * sizeof(*irreg_cid_d)));
		gpuCudaErrorCheck(hipMalloc(&irreg_val_d, nnz_irreg * sizeof(*irreg_val_d)));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_long_val));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_long_val));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_val_by_warp));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_val_by_warp));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_long_ptr_warp));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_long_ptr_warp));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_long_cid));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_long_cid));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_short_val));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_short_val));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_short_cid));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_short_cid));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_reg_val));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_reg_val));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_irreg_val));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_irreg_val));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_block_ptr));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_block_ptr));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_irreg_rpt));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_irreg_rpt));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_reg_cid));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_reg_cid));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_irreg_cid));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_irreg_cid));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}
		
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_long_ptr_warp));
		gpuCudaErrorCheck(hipMemcpy(long_ptr_warp_d, long_rpt_new, (row_long + 1) * sizeof(*long_ptr_warp_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_long_ptr_warp));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_long_cid));
		gpuCudaErrorCheck(hipMemcpy(long_cid_d, long_cid, fill0_nnz_long * sizeof(*long_cid_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_long_cid));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_long_val));
		gpuCudaErrorCheck(hipMemcpy(long_val_d, long_val, fill0_nnz_long * sizeof(*long_val_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_long_val));
		// for(int i=0;i<fill0_nnz_long;i++) printf("%d = %.2lf\n", long_cid[i], long_val[i]);
		// for(int i=0;i<(row_long+1);i++) printf("%d\n", long_rpt_new[i]);

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_short_cid));
		gpuCudaErrorCheck(hipMemcpy(short_cid_d, short_cid, fill0_nnz_short * sizeof(*short_cid_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_short_cid));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_short_val));
		gpuCudaErrorCheck(hipMemcpy(short_val_d, short_val, fill0_nnz_short * sizeof(*short_val_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_short_val));
		// for(int i=0;i<fill0_nnz_short;i++) printf("%d = %.2lf\n", short_cid[i], short_val[i]);

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_block_ptr));
		gpuCudaErrorCheck(hipMemcpy(block_ptr_d, blockPtr, (blocknum + 1) * sizeof(*block_ptr_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_block_ptr));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_reg_cid));
		gpuCudaErrorCheck(hipMemcpy(reg_cid_d, reg_cid, fill0_nnz_reg * sizeof(*reg_cid_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_reg_cid));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_reg_val));
		gpuCudaErrorCheck(hipMemcpy(reg_val_d, reg_val, fill0_nnz_reg * sizeof(*reg_val_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_reg_val));
		// for(int i=0;i<(blocknum + 1);i++) printf("%d\n", blockPtr[i]);
		// for(int i=0;i<fill0_nnz_reg;i++) printf("%d = %.2lf\n", reg_cid[i], reg_val[i]);

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_irreg_rpt));
		gpuCudaErrorCheck(hipMemcpy(irreg_rpt_d, irreg_rpt, (row_block + 1) * sizeof(*irreg_rpt_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_irreg_rpt));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_irreg_cid));
		gpuCudaErrorCheck(hipMemcpy(irreg_cid_d, irreg_cid, nnz_irreg * sizeof(*irreg_cid_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_irreg_cid));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_irreg_val));
		gpuCudaErrorCheck(hipMemcpy(irreg_val_d, irreg_val, nnz_irreg * sizeof(*irreg_val_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_irreg_val));
		// for(int i=0;i<(row_block + 1);i++) printf("%d\n", irreg_rpt[i]);
		// for(int i=0;i<nnz_irreg;i++) printf("%d = %.2lf\n", irreg_cid[i], irreg_val[i]);

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_long_ptr_warp));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_long_cid));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_long_val));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_short_cid));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_short_val));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_block_ptr));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_reg_cid));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_reg_val));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_irreg_rpt));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_irreg_cid));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_irreg_val));
			float memcpyTime_cuda_long_ptr_warp, memcpyTime_cuda_long_cid, memcpyTime_cuda_long_val, memcpyTime_cuda_short_cid, memcpyTime_cuda_short_val, memcpyTime_cuda_block_ptr, memcpyTime_cuda_reg_cid, memcpyTime_cuda_reg_val, memcpyTime_cuda_irreg_rpt, memcpyTime_cuda_irreg_cid, memcpyTime_cuda_irreg_val;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_long_ptr_warp, startEvent_memcpy_long_ptr_warp, endEvent_memcpy_long_ptr_warp));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_long_cid, startEvent_memcpy_long_cid, endEvent_memcpy_long_cid));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_long_val, startEvent_memcpy_long_val, endEvent_memcpy_long_val));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_short_cid, startEvent_memcpy_short_cid, endEvent_memcpy_short_cid));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_short_val, startEvent_memcpy_short_val, endEvent_memcpy_short_val));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_block_ptr, startEvent_memcpy_block_ptr, endEvent_memcpy_block_ptr));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_reg_cid, startEvent_memcpy_reg_cid, endEvent_memcpy_reg_cid));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_reg_val, startEvent_memcpy_reg_val, endEvent_memcpy_reg_val));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_irreg_rpt, startEvent_memcpy_irreg_rpt, endEvent_memcpy_irreg_rpt));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_irreg_cid, startEvent_memcpy_irreg_cid, endEvent_memcpy_irreg_cid));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_irreg_val, startEvent_memcpy_irreg_val, endEvent_memcpy_irreg_val));
			printf("(CUDA) Memcpy long_ptr_warp time = %.4lf ms, long_cid time = %.4lf ms, long_val time = %.4lf ms, short_cid time = %.4lf ms, short_val time = %.4lf ms, block_ptr time = %.4lf ms, reg_cid time = %.4lf ms, reg_val time = %.4lf ms, irreg_rpt time = %.4lf ms, irreg_cid time = %.4lf ms, irreg_val time = %.4lf ms\n", memcpyTime_cuda_long_ptr_warp, memcpyTime_cuda_long_cid, memcpyTime_cuda_long_val, memcpyTime_cuda_short_cid, memcpyTime_cuda_short_val, memcpyTime_cuda_block_ptr, memcpyTime_cuda_reg_cid, memcpyTime_cuda_reg_val, memcpyTime_cuda_irreg_rpt, memcpyTime_cuda_irreg_cid, memcpyTime_cuda_irreg_val);
		}
		printf("row_long = %d, row_block = %d, blocknum = %d, short_row_1 = %d, common_13 = %d, short_row_34 = %d, short_row_2 = %d, offset_reg = %d, offset_short1 = %d, offset_short13 = %d, offset_short34 = %d, offset_short22 = %d, fill0_nnz_short13 = %d, fill0_nnz_short34 = %d\n", row_long, row_block, blocknum, short_row_1, common_13, short_row_34, short_row_2, offset_reg, offset_short1, offset_short13, offset_short34, offset_short22, fill0_nnz_short13, fill0_nnz_short34);
	}

	~DASPArrays()
	{
		free(ia);
		free(ja);
		free(a);
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipFree(long_ptr_warp_d));
		gpuCudaErrorCheck(hipFree(long_cid_d));
		gpuCudaErrorCheck(hipFree(long_val_d));
		gpuCudaErrorCheck(hipFree(val_by_warp_d));

		gpuCudaErrorCheck(hipFree(short_val_d));
		gpuCudaErrorCheck(hipFree(short_cid_d));
		
		gpuCudaErrorCheck(hipFree(block_ptr_d));
		gpuCudaErrorCheck(hipFree(reg_cid_d));
		gpuCudaErrorCheck(hipFree(reg_val_d));
		
		gpuCudaErrorCheck(hipFree(irreg_rpt_d));
		gpuCudaErrorCheck(hipFree(irreg_cid_d));
		gpuCudaErrorCheck(hipFree(irreg_val_d));

		// DASP specific
		free(order_rid);
		free(short_rid_1);
		free(short_rid_2);
		free(short_rid_3);
		free(short_rid_4);
		free(long_rid);
		free(zero_rid);
		free(ridA);

		free(rptA);
		free(long_rpt);

		free(short_val);
		free(short_cid);

		free(long_cid);
		free(long_val);
		free(long_rpt_new);
		free(val_by_warp);
		free(rid_by_warp);

		free(reg_val);
		free(reg_cid);
		free(blockPtr);

		free(irreg_rpt);
		free(irreg_cid);
		free(irreg_val);
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_dasp(DASPArrays * dasp, ValueType * x , ValueType * y);


void
DASPArrays::spmv(ValueType * x, ValueType * y)
{
	compute_dasp(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{

	struct DASPArrays * dasp = new DASPArrays(row_ptr, col_ind, values, m, n, nnz);
	//dasp->mem_footprint = ; TODO!
	dasp->format_name = (char *) "DASP_CUDA";
	return dasp;
}


__host__ void
compute_dasp(DASPArrays * dasp, ValueType * x , ValueType * y)
{
	if (dasp->x == NULL)
	{
		dasp->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(dasp->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(dasp->x_d, x, dasp->n * sizeof(ValueType), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(dasp->endEvent_memcpy_x));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(dasp->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, dasp->startEvent_memcpy_x, dasp->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}
	}

	hipMemset(dasp->y_d, 0, dasp->m * sizeof(dasp->y_d));

	int carveout = 0;
	gpuCudaErrorCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(dasp_spmv2<1>), hipFuncAttributePreferredSharedMemoryCarveout, carveout));
	gpuCudaErrorCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(dasp_spmv2<2>), hipFuncAttributePreferredSharedMemoryCarveout, carveout));
	gpuCudaErrorCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(dasp_spmv2<4>), hipFuncAttributePreferredSharedMemoryCarveout, carveout));

	if (dasp->rowloop == 1) {
		dasp_spmv2<1><<<dasp->BlockNum_all, dasp->ThreadNum_all>>>(
			dasp->x_d, dasp->y_d,
			dasp->long_val_d, dasp->long_cid_d, dasp->val_by_warp_d, dasp->long_ptr_warp_d, dasp->row_long,
			dasp->reg_val_d, dasp->reg_cid_d, dasp->block_ptr_d, dasp->row_block, dasp->blocknum,
			dasp->irreg_val_d, dasp->irreg_cid_d, dasp->irreg_rpt_d,
			dasp->short_val_d, dasp->short_cid_d, dasp->short_row_1, dasp->common_13, dasp->short_row_34, dasp->short_row_2,
			dasp->offset_reg, dasp->offset_short1, dasp->offset_short13, dasp->offset_short34, dasp->offset_short22,
			dasp->fill0_nnz_short13, dasp->fill0_nnz_short34);
	}
	else if (dasp->rowloop == 2) {
		dasp_spmv2<2><<<dasp->BlockNum_all, dasp->ThreadNum_all>>>(
			dasp->x_d, dasp->y_d,
			dasp->long_val_d, dasp->long_cid_d, dasp->val_by_warp_d, dasp->long_ptr_warp_d, dasp->row_long,
			dasp->reg_val_d, dasp->reg_cid_d, dasp->block_ptr_d, dasp->row_block, dasp->blocknum,
			dasp->irreg_val_d, dasp->irreg_cid_d, dasp->irreg_rpt_d,
			dasp->short_val_d, dasp->short_cid_d, dasp->short_row_1, dasp->common_13, dasp->short_row_34, dasp->short_row_2,
			dasp->offset_reg, dasp->offset_short1, dasp->offset_short13, dasp->offset_short34, dasp->offset_short22,
			dasp->fill0_nnz_short13, dasp->fill0_nnz_short34);
	}
	else {
		dasp_spmv2<4><<<dasp->BlockNum_all, dasp->ThreadNum_all>>>(
			dasp->x_d, dasp->y_d,
			dasp->long_val_d, dasp->long_cid_d, dasp->val_by_warp_d, dasp->long_ptr_warp_d, dasp->row_long,
			dasp->reg_val_d, dasp->reg_cid_d, dasp->block_ptr_d, dasp->row_block, dasp->blocknum,
			dasp->irreg_val_d, dasp->irreg_cid_d, dasp->irreg_rpt_d,
			dasp->short_val_d, dasp->short_cid_d, dasp->short_row_1, dasp->common_13, dasp->short_row_34, dasp->short_row_2,
			dasp->offset_reg, dasp->offset_short1, dasp->offset_short13, dasp->offset_short34, dasp->offset_short22,
			dasp->fill0_nnz_short13, dasp->fill0_nnz_short34);
	}
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());
	if(dasp->row_long)
		longPart_sum<<<dasp->sumBlockNum, dasp->ThreadNum_all>>>(dasp->long_ptr_warp_d, dasp->val_by_warp_d, dasp->y_d, dasp->row_long);
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (dasp->y == NULL)
	{
		dasp->y = y;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(dasp->startEvent_memcpy_y));
		ValueType *y_reordered = (typeof(y_reordered))malloc(dasp->m * sizeof(*y_reordered));
		gpuCudaErrorCheck(hipMemcpy(y_reordered, dasp->y_d, dasp->m * sizeof(ValueType), hipMemcpyDeviceToHost));		
		// Need to perform reordering to result, apart from Copying it back to CPU
		for(INT_T i=0; i<dasp->m; i++)
			dasp->y[dasp->order_rid[i]] = y_reordered[i];
		free(y_reordered);
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(dasp->endEvent_memcpy_y));
		
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(dasp->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, dasp->startEvent_memcpy_y, dasp->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}

	}

}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
DASPArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
DASPArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

