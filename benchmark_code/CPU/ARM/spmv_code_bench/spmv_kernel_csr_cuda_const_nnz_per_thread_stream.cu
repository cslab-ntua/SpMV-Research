#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cublas_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

using namespace cooperative_groups;

#ifndef NNZ_PER_THREAD
#define NNZ_PER_THREAD  6
#endif

#ifndef BLOCK_SIZE
// #define BLOCK_SIZE  32
// #define BLOCK_SIZE  64
// #define BLOCK_SIZE  128
// #define BLOCK_SIZE  256
// #define BLOCK_SIZE  512
#define BLOCK_SIZE  1024
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 1
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

#ifndef TIME_IT2
#define TIME_IT2 0
#endif


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;      // the usual rowptr (of size m+1)
	INT_T * ia[NUM_STREAMS];      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * thread_block_i_s[NUM_STREAMS];
	INT_T * thread_block_i_e[NUM_STREAMS];
	INT_T * thread_block_j_s[NUM_STREAMS];
	INT_T * thread_block_j_e[NUM_STREAMS];
	INT_T * row_ptr_stream[NUM_STREAMS];

	INT_T * row_ptr_h[NUM_STREAMS];
	INT_T * ia_h[NUM_STREAMS];
	INT_T * ja_h[NUM_STREAMS];
	ValueType * a_h[NUM_STREAMS];
	INT_T * thread_block_i_s_h[NUM_STREAMS];
	INT_T * thread_block_i_e_h[NUM_STREAMS];
	INT_T * thread_block_j_s_h[NUM_STREAMS];
	INT_T * thread_block_j_e_h[NUM_STREAMS];

	INT_T * row_ptr_d[NUM_STREAMS];
	INT_T * ia_d[NUM_STREAMS];
	INT_T * ja_d[NUM_STREAMS];
	ValueType * a_d[NUM_STREAMS];
	INT_T * thread_block_i_s_d[NUM_STREAMS];
	INT_T * thread_block_i_e_d[NUM_STREAMS];
	INT_T * thread_block_j_s_d[NUM_STREAMS];
	INT_T * thread_block_j_e_d[NUM_STREAMS];

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h[NUM_STREAMS];
	ValueType * y_h[NUM_STREAMS];
	// ValueType * y_h2;
	// ValueType * y_h_final;
	ValueType * x_d[NUM_STREAMS];
	// ValueType * y_d[NUM_STREAMS];
	ValueType * y_d2;
	ValueType * y_d_reduction;

	hipStream_t stream[NUM_STREAMS];
	INT_T n_stream[NUM_STREAMS];
	INT_T nnz_stream[NUM_STREAMS];

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution[NUM_STREAMS];
	hipEvent_t endEvent_execution[NUM_STREAMS];
	float execution_time[NUM_STREAMS];
	int iterations;
	
	hipEvent_t startEvent_memcpy_row_ptr[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_row_ptr[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_block_i_s[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_block_i_s[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_block_i_e[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_block_i_e[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_block_j_s[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_block_j_s[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_block_j_e[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_block_j_e[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipblasHandle_t handle;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x, max_num_threads;
	int nnz_per_thread;
	int num_threads[NUM_STREAMS];
	int block_size;
	int num_blocks[NUM_STREAMS];
	int num_streams;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0));
		max_num_threads = max_threads_per_multiproc * multiproc_count;
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_num_threads=%d\n", max_num_threads);

		block_size = BLOCK_SIZE;
		num_streams = NUM_STREAMS;

		/********************************************************************************************************/
		printf("/********************************************************************************************************/\n");
		// Convert CSR representation ton CSC
		INT_T * row_indices; //for CSC format
		INT_T * row_idx;
		INT_T * col_ptr;
		ValueType * val_c;

		row_indices = (typeof(row_indices)) malloc(nnz * sizeof(*row_indices));
		row_idx = (typeof(row_idx)) malloc(nnz * sizeof(*row_idx));
		col_ptr = (typeof(col_ptr)) malloc((n+1) * sizeof(*col_ptr));
		val_c = (typeof(val_c)) malloc(nnz * sizeof(*val_c));

		double time = time_it(1,
			csr_row_indices(row_ptr, ja, m, n, nnz, &row_indices);
			coo_to_csc(row_indices, ja, a, m, n, nnz, row_idx, col_ptr, val_c, 1);
			free(row_indices);
		);
		printf("time coo_to_csc = %g ms\n", time*1e3);

		INT_T *local_stream_j_s = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_s));
		INT_T *local_stream_j_e = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_e));
		double time_balance = time_it(1,
			for (int i=0;i<num_streams;i++)
				loop_partitioner_balance_prefix_sums(num_streams, i, col_ptr, n, nnz, &local_stream_j_s[i], &local_stream_j_e[i]);
		);

		int cnt=0, cnt2=0;
		for(int i=0; i<num_streams; i++){
			nnz_stream[i] = col_ptr[local_stream_j_e[i]] - col_ptr[local_stream_j_s[i]];
			n_stream[i] = local_stream_j_e[i] - local_stream_j_s[i];
			printf("local_stream[%d] = %d - %d (%d cols) (%d nnz)\n", i, local_stream_j_s[i], local_stream_j_e[i], n_stream[i], nnz_stream[i]);

			cnt  += nnz_stream[i];
			cnt2 += n_stream[i];
		}

		printf("balance time (col) = %g ms\n", time_balance*1e3);

		INT_T * row_idx_stream[num_streams];
		INT_T * col_ptr_stream[num_streams];
		ValueType * val_c_stream[num_streams];
		
		double time_memcpy_stream_locals = time_it(1,
		for(int i=0; i<num_streams; i++){
			col_ptr_stream[i] = (INT_T *) malloc((n_stream[i]+1) * sizeof(INT_T));
			row_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_c_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			memcpy(col_ptr_stream[i], col_ptr + local_stream_j_s[i], (n_stream[i] + 1) * sizeof(INT_T));
			// col_ptr needs to be fixed, so that it will start from 0 again...
			for(int j=0; j<n_stream[i]+1; j++)
				col_ptr_stream[i][j] -= col_ptr[local_stream_j_s[i]];
			memcpy(row_idx_stream[i], row_idx + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(INT_T));
			memcpy(val_c_stream[i], val_c + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy_stream_locals = %lf ms\n", time_memcpy_stream_locals*1e3);
		free(local_stream_j_s);
		free(local_stream_j_e);

		INT_T * row_ptr_stream[num_streams];
		INT_T * col_idx_stream[num_streams];
		ValueType * val_stream[num_streams];

		for(int i=0; i<num_streams; i++){
			INT_T * col_indices;
			csc_col_indices(row_idx_stream[i], col_ptr_stream[i], m, n_stream[i], nnz_stream[i], &col_indices);

			row_ptr_stream[i] = (INT_T *) malloc((m+1) * sizeof(INT_T));
			col_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			coo_to_csr(row_idx_stream[i], col_indices, val_c_stream[i], m, n_stream[i], nnz_stream[i], row_ptr_stream[i], col_idx_stream[i], val_stream[i], 1, 0);
			free(col_indices);
		}

		for(int i=0; i<num_streams; i++){
			free(row_idx_stream[i]);
			free(col_ptr_stream[i]);
			free(val_c_stream[i]);
		}
		free(row_idx);
		free(col_ptr);
		free(val_c);

		// for(int i=0; i<num_streams; i++)
		// 	printf("Stream %d: %d columns, %d nnz\n", i, n_stream[i], nnz_stream[i]);

		printf("/********************************************************************************************************/\n");
		/********************************************************************************************************/


		for(int i=0; i<num_streams; i++){
			num_threads[i] = (nnz_stream[i] + NNZ_PER_THREAD - 1) / NNZ_PER_THREAD;

			num_threads[i] = ((num_threads[i] + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;

			num_blocks[i] = num_threads[i] / BLOCK_SIZE;

			printf("Stream %d: %d columns, %d nnz\tnum_threads=%d, block_size=%d, num_blocks=%d\n", i, n_stream[i], nnz_stream[i], num_threads[i], BLOCK_SIZE, num_blocks[i]);
			thread_block_i_s[i] = (INT_T *) malloc(num_blocks[i] * sizeof(INT_T));
			thread_block_i_e[i] = (INT_T *) malloc(num_blocks[i] * sizeof(INT_T));
			thread_block_j_s[i] = (INT_T *) malloc(num_blocks[i] * sizeof(INT_T));
			thread_block_j_e[i] = (INT_T *) malloc(num_blocks[i] * sizeof(INT_T));
			// double time_balance = time_it(1,
			long lower_boundary;
			// for (i=0;i<num_blocks;i++)
			// {
				// loop_partitioner_balance_iterations(num_blocks, i, 0, nnz, &thread_block_j_s[i], &thread_block_j_e[i]);
				// macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				// thread_block_i_s[i] = lower_boundary;
			// }
			long nnz_per_block = block_size * NNZ_PER_THREAD;
			for (int k=0;k<num_blocks[i];k++)
			{
				thread_block_j_s[i][k] = nnz_per_block * k;
				thread_block_j_e[i][k] = nnz_per_block * (k+ 1);
				if (thread_block_j_s[i][k] > nnz_stream[i])
					thread_block_j_s[i][k] = nnz_stream[i];
				if (thread_block_j_e[i][k] > nnz_stream[i])
					thread_block_j_e[i][k] = nnz_stream[i];
				macros_binary_search(row_ptr_stream[i], 0, m, thread_block_j_s[i][k], &lower_boundary, NULL);           // Index boundaries are inclusive.
				thread_block_i_s[i][k] = lower_boundary;
			}
			for (int k=0;k<num_blocks[i];k++)
			{
				if (k == num_blocks[i] - 1)   // If we calculate each thread's boundaries individually some empty rows might be unassigned.
					thread_block_i_e[i][k] = m;
				else
					thread_block_i_e[i][k] = thread_block_i_s[i][k+1] + 1;
				if ((thread_block_j_s[i][k] >= row_ptr_stream[i][thread_block_i_e[i][k]]) || (thread_block_j_s[i][k] < row_ptr_stream[i][thread_block_i_s[i][k]]))
					error("bad binary search of row start: i=%d j:[%d, %d] j=%d", thread_block_i_s[i][k], row_ptr_stream[i][thread_block_i_s[i][k]], row_ptr_stream[i][thread_block_i_e[i][k]], thread_block_j_s[i][k]);
			}
			// );
			// printf("Stream %d: balance time = %g\n", i, time_balance);

			ia[i] = (INT_T*) malloc(nnz_stream[i] * sizeof(INT_T));
			_Pragma("omp parallel")
			{
				long k, j;
				_Pragma("omp for")
				for (k=0;k<m;k++)
				{
					for (j=row_ptr_stream[i][k];j<row_ptr_stream[i][k+1];j++)
					{
						ia[i][j] = k;
					}
				}
			}

			_Pragma("omp parallel")
			{
				long k, j;
				_Pragma("omp for")
				for (j=0;j<nnz_stream[i];j+=32*NNZ_PER_THREAD)
				{
					long j_e = j + 32*NNZ_PER_THREAD;
					if (j_e > nnz_stream[i])
						j_e = nnz_stream[i];
					if (ia[i][j] == ia[i][j_e-1])
					{
						for (k=j;k<j_e;k++)
						{
							col_idx_stream[i][k] = col_idx_stream[i][k] | 0x80000000;
						}
					}
				}
			}
		}

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipMalloc(&row_ptr_d[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ia_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ja_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&a_d[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&thread_block_i_s_d[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&thread_block_i_e_d[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&thread_block_j_s_d[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&thread_block_j_e_d[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&x_d[i], n_stream[i] * sizeof(ValueType)));
			// gpuCudaErrorCheck(hipMalloc(&y_d[i], m * sizeof(ValueType)));
		}
		gpuCudaErrorCheck(hipMalloc(&y_d2, m * num_streams * sizeof(ValueType)));
		gpuCudaErrorCheck(hipMalloc(&y_d_reduction, m * sizeof(ValueType)));
		gpuCublasErrorCheck(hipblasCreate(&handle));

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipHostMalloc(&row_ptr_h[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ia_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ja_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&a_h[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_s_h[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_e_h[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_block_j_s_h[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_block_j_e_h[i], num_blocks[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&x_h[i], n_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&y_h[i], m * sizeof(ValueType)));
		}
		// gpuCudaErrorCheck(hipHostMalloc(&y_h2, m * num_streams * sizeof(ValueType)));
		// gpuCudaErrorCheck(hipHostMalloc(&y_h_final, m * sizeof(ValueType)));

		double time_memcpy = time_it(1,
		for(int i=0; i<num_streams; i++){
			memcpy(row_ptr_h[i], row_ptr_stream[i], (m + 1) * sizeof(INT_T));
			memcpy(ia_h[i], ia[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(ja_h[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(a_h[i], val_stream[i], nnz_stream[i] * sizeof(ValueType));
			memcpy(thread_block_i_s_h[i], thread_block_i_s[i], num_blocks[i] * sizeof(INT_T));
			memcpy(thread_block_i_e_h[i], thread_block_i_e[i], num_blocks[i] * sizeof(INT_T));
			memcpy(thread_block_j_s_h[i], thread_block_j_s[i], num_blocks[i] * sizeof(INT_T));
			memcpy(thread_block_j_e_h[i], thread_block_j_e[i], num_blocks[i] * sizeof(INT_T));
		}
		);
		printf("time_memcpy (row_ptr_h, ia_h, ja_h, a_h) = %lf ms\n", time_memcpy*1e3);

		// cuda events for timing measurements
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipStreamCreate(&stream[i]));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_execution[i]));
		}
		iterations=0;
		gpuCublasErrorCheck(hipblasSetStream(handle, stream[0]));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_row_ptr[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_row_ptr[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_j_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_j_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_j_e[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_j_e[i]));

				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		for(int i=0; i<num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_row_ptr[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(row_ptr_d[i], row_ptr_h[i], (m+1) * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_row_ptr[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ia_d[i], ia_h[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ja_d[i], ja_h[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(a_d[i], a_h[i], nnz_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_s[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_s_d[i], thread_block_i_s_h[i], num_blocks[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_s[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_e[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_e_d[i], thread_block_i_e_h[i], num_blocks[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_e[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_j_s[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_block_j_s_d[i], thread_block_j_s_h[i], num_blocks[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_j_s[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_j_e[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_block_j_e_d[i], thread_block_j_e_h[i], num_blocks[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_j_e[i], stream[i]));
		}

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(stream[i]));
				float memcpyTime_cuda_row_ptr, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e, memcpyTime_cuda_thread_block_j_s, memcpyTime_cuda_thread_block_j_e;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_row_ptr, startEvent_memcpy_row_ptr[i], endEvent_memcpy_row_ptr[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia[i], endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja[i], endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a[i], endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_s, startEvent_memcpy_thread_block_i_s[i], endEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_e, startEvent_memcpy_thread_block_i_e[i], endEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_j_s, startEvent_memcpy_thread_block_j_s[i], endEvent_memcpy_thread_block_j_s[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_j_e, startEvent_memcpy_thread_block_j_e[i], endEvent_memcpy_thread_block_j_e[i]));
				printf("(CUDA) (stream %d) Memcpy row_ptr time = %.4lf ms, ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, thread_block_i_s time = %.4lf, thread_block_i_e time = %.4lf, thread_block_j_s time = %.4lf, thread_block_j_e time = %.4lf\n", i, memcpyTime_cuda_row_ptr, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e, memcpyTime_cuda_thread_block_j_s, memcpyTime_cuda_thread_block_j_e);
			}
		}
		for(int i=0; i<num_streams; i++){
			_Pragma("omp parallel")
			{
				long j;
				_Pragma("omp for")
				for (j=0;j<nnz_stream[i];j++)
				{
					ja_h[i][j] = ja_h[i][j] & 0x7FFFFFFF;
				}
			}
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		for(int i=0; i<num_streams; i++){
			free(thread_block_i_s[i]);
			free(thread_block_i_e[i]);
			free(thread_block_j_s[i]);
			free(thread_block_j_e[i]);

			gpuCudaErrorCheck(hipFree(ia_d[i]));
			gpuCudaErrorCheck(hipFree(row_ptr_d[i]));
			gpuCudaErrorCheck(hipFree(ja_d[i]));
			gpuCudaErrorCheck(hipFree(a_d[i]));
			gpuCudaErrorCheck(hipFree(x_d[i]));
			// gpuCudaErrorCheck(hipFree(y_d[i]));
			gpuCudaErrorCheck(hipFree(thread_block_i_s_d[i]));
			gpuCudaErrorCheck(hipFree(thread_block_i_e_d[i]));
			gpuCudaErrorCheck(hipFree(thread_block_j_s_d[i]));
			gpuCudaErrorCheck(hipFree(thread_block_j_e_d[i]));

			gpuCudaErrorCheck(hipHostFree(ia_h[i]));
			gpuCudaErrorCheck(hipHostFree(row_ptr_h[i]));
			gpuCudaErrorCheck(hipHostFree(ja_h[i]));
			gpuCudaErrorCheck(hipHostFree(a_h[i]));
			gpuCudaErrorCheck(hipHostFree(x_h[i]));
			gpuCudaErrorCheck(hipHostFree(y_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_block_i_s_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_block_i_e_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_block_j_s_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_block_j_e_h[i]));

			gpuCudaErrorCheck(hipStreamDestroy(stream[i]));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_execution[i]));
		}
		gpuCudaErrorCheck(hipFree(y_d2));
		gpuCudaErrorCheck(hipFree(y_d_reduction));
		gpuCublasErrorCheck(hipblasDestroy(handle));
		// gpuCudaErrorCheck(hipHostFree(y_h2));
		// gpuCudaErrorCheck(hipHostFree(y_h_final));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_row_ptr[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_row_ptr[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_j_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_j_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_j_e[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_j_e[i]));
			}
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_constant_nnz_per_thread_b%d_nnz%d_s%d", BLOCK_SIZE, NNZ_PER_THREAD, csr->num_streams);
	csr->format_name = format_name;
	/*if(0){
		for(int i=0; i<csr->num_streams; i++){
			char matrix_part[100];
			sprintf(matrix_part, "Stream%d", i);
			csr_matrix_features_validation(matrix_part, csr->row_ptr_h[i], csr->ja_h[i], csr->m, csr->n_stream[i], csr->nnz_stream[i]);
			char file_fig[100];
			sprintf(file_fig, "figures/Stream%d", i);
			long num_pixels = 4096;
			long num_pixels_x = (csr->n_stream[i] < num_pixels) ? csr->n_stream[i] : num_pixels;
			long num_pixels_y = (csr->m < num_pixels) ? csr->m : num_pixels;
			if(csr->m!=csr->n_stream[i]) {
				double ratio = csr->n_stream[i]*1.0 / csr->m;
				// if((ratio>16.0) || (ratio<(1/16.0)))
				if(ratio>16.0)
					ratio=16.0;
				if(ratio < (1/16.0))
					ratio=1/16.0;
				// in order to keep both below 1024
				if(ratio>1) // n > m
					num_pixels_y = (1/ratio) * num_pixels_x;
				else // m > n
					num_pixels_x = ratio * num_pixels_y;
				printf("Stream %d: ratio %lf\n", i, ratio);
			}

			csr_plot(file_fig, csr->row_ptr_h[i], csr->ja_h[i], csr->a_h[i], csr->m, csr->n_stream[i], csr->nnz_stream[i], 0, num_pixels_x, num_pixels_y);
		}
	}*/
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	int row = ia_buf[tidb];
	int k;
	for (k=1;k<BLOCK_SIZE;k*=2)
	{
		if ((tidb & (2*k-1)) == k-1)
		{
			ValueType val = val_buf[tidb];
			if (row == ia_buf[tidb+k])
			{
				val_buf[tidb+k] += val;
				// val_buf[tidb] = 0;
			}
			else
			{
				atomicAdd(&y[row], val);
				// y[row] += val;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[BLOCK_SIZE-1]], val_buf[BLOCK_SIZE-1]);
} */


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	int k;
	INT_T row = ia_buf[tidb];
	for (k=1;k<BLOCK_SIZE;k*=2)
	{
		if ((tidb & (2*k-1)) == 0)
		{
			INT_T row_next = ia_buf[tidb+k];
			ValueType val_next = val_buf[tidb+k];
			if (row == row_next)
			{
				val_buf[tidb] += val_next;
			}
			else
			{
				atomicAdd(&y[row], val_buf[tidb]);
				val_buf[tidb] = val_next;
				ia_buf[tidb] = row_next;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[0]], val_buf[0]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int row = ia_buf[tidl];
	ValueType val;
	int k;
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		// val = val_buf[tidl];
		// if ((tidl & (2*k-1)) == k-1)
		// {
			// if (tidl >= k && row == ia_buf[tidl-k])
			// {
				// val_buf[tidl-k] += val;
				// val = 0;
			// }
		// }
		// g.sync();
		// if ((tidl & (2*k-1)) == k-1 && val != 0)
		// {
			// if (row == ia_buf[tidl+k])
			// {
				// val_buf[tidl+k] += val;
			// }
			// else
			// {
				// atomicAdd(&y[row], val);
			// }
		// }
		// g.sync();
		val = val_buf[tidl];
		if ((tidl & (2*k-1)) == k-1)
		{
			if (row == ia_buf[tidl+k])
			{
				val_buf[tidl+k] += val;
			}
			else
			{
				atomicAdd(&y[row], val);
			}
		}
		g.sync();
	}
}
inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &ia_buf[tidb_div*32], &val_buf[tidb_div*32], y);
	// __syncthreads();
	// if (tidb_mod == 31)
	// {
		// ia_buf[tidb_mod] = ia_buf[tidb];
		// val_buf[tidb_mod] = val_buf[tidb];
	// }
	// __syncthreads();
	// if (tidb_div == 0)
		// reduce_warp(tile32, ia_buf, val_buf, y);
	// if (tidb == 0)
		// atomicAdd(&y[ia_buf[31]], val_buf[31]);
	if (tidb_mod == 31)
		atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * row_ptr, ValueType * val_ptr, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	INT_T row = *row_ptr;
	ValueType val = *val_ptr;
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_next;
		ValueType val_next;
		row_next = __shfl_sync(0xffffffff, row, tidl+k);
		val_next = __shfl_sync(0xffffffff, val, tidl+k);
		if ((tidl & (2*k-1)) == 0)
		{
			if (row == row_next)
			{
				val += val_next;
			}
			else
			{
				atomicAdd(&y[row], val);
				val = val_next;
				row = row_next;
			}
		}
		g.sync();
	}
	*row_ptr = row;
	*val_ptr = val;
	// if (tidl == 0)
		// atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &row, &val, y);
	if (tidb_mod == 0)
		atomicAdd(&y[row], val);
	// extern __shared__ char sm[];
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[32 * sizeof(ValueType)];
	// if (tidb_mod == 0)
	// {
		// ia_buf[tidb_div] = row;
		// val_buf[tidb_div] = val;
	// }
	// __syncthreads();
	// if (tidb_div == 0)
	// {
		// row = ia_buf[tidb];
		// val = val_buf[tidb];
		// reduce_warp(tile32, &row, &val, y);
	// }
	// if (tidb == 0)
		// atomicAdd(&y[row], val);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_prev;
		ValueType val_prev;
		row_prev = __shfl_sync(0xffffffff, row, tidl-k);
		val_prev = __shfl_sync(0xffffffff, val, tidl-k);
		if ((tidl & (2*k-1)) == 2*k-1)
		{
			if (row == row_prev)
			{
				val += val_prev;
			}
			else
			{
				atomicAdd(&y[row_prev], val_prev);
			}
		}
		g.sync();
	}
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
} */


// Threads may only read data from another thread which is actively participating in the __shfl_sync() command.
// If the target thread is inactive, the retrieved value is undefined.
template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	const int tidl_one_hot = 1 << tidl;
	int flag;
	INT_T row_prev;
	ValueType val_prev;
	flag = 0xaaaaaaaa; // 10101010101010101010101010101010
	row_prev = g.shfl_up(row, 1); // __shfl_sync(flag, row, tidl-1);
	val_prev = g.shfl_up(val, 1); // __shfl_sync(flag, val, tidl-1);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x88888888; // 10001000100010001000100010001000
	row_prev = g.shfl_up(row, 2); // __shfl_sync(flag, row, tidl-2);
	val_prev = g.shfl_up(val, 2); // __shfl_sync(flag, val, tidl-2);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80808080; // 10000000100000001000000010000000
	row_prev = g.shfl_up(row, 4); // __shfl_sync(flag, row, tidl-4);
	val_prev = g.shfl_up(val, 4); // __shfl_sync(flag, val, tidl-4);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80008000; // 10000000000000001000000000000000
	row_prev = g.shfl_up(row, 8); // __shfl_sync(flag, row, tidl-8);
	val_prev = g.shfl_up(val, 8); // __shfl_sync(flag, val, tidl-8);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80000000; // 10000000000000000000000000000000
	row_prev = g.shfl_up(row, 16); // __shfl_sync(flag, row, tidl-16);
	val_prev = g.shfl_up(val, 16); // __shfl_sync(flag, val, tidl-16);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	g.sync();
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
}


__device__ void spmv_last_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[BLOCK_SIZE * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = block_id * nnz_per_block + tidb * NNZ_PER_THREAD;
	j_e = j_s + NNZ_PER_THREAD;
	if (j_e > nnz)
		j_e = nnz;
	k = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_s >= row_ptr[k])
		{
			i_s = k + 1;
		}
		else
		{
			i_e = k;
		}
		k = (i_e + i_s) / 2;
	}
	i = i_s - 1;
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_s;j<j_e;j++)
	{
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
		}
		// sum += a[j] * x[ja[j] & 0x7FFFFFFF];
		sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
	}
	reduce_block(i, sum, y);
}


template <typename group_t>
__device__ ValueType reduce_warp_single_line(group_t g, ValueType val, ValueType * restrict y)
{
	// Use XOR mode to perform butterfly reduction
	for (int i=g.size()/2; i>=1; i/=2)
	{
		val += g.shfl_xor(val, i); // __shfl_xor_sync(0xffffffff, val, i, g.size());   // 'sum' is same on all threads
		// val += __shfl_down_sync(0xffffffff, val, i, g.size());   // Only thread 0 has the total sum.
	}
	return val;
}


template <typename group_t>
__device__ void spmv_warp_single_row(group_t g, int i, int j_s, int j_e, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int j;
	double sum = 0;
	for (j=j_s;j<j_e;j++)
	{
		sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
	}
	sum = reduce_warp_single_line(g, sum, y);
	if (tidl == 0)
		atomicAdd(&y[i], sum);
}


template <typename group_t>
__device__ void spmv_full_warp(group_t g, int one_line, int i_s, int j_s, int j_e, INT_T * row_ptr, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	[[gnu::unused]] int i, j, k, l, p;
	int ptr_next;
	i = i_s;
	ptr_next = row_ptr[i_s+1];
	for (j=j_s;j<j_e;j++)   // Find the row of the last nnz.
	{
		if (j >= ptr_next)
		{
			i++;
			break;
		}
	}
	double sum = 0;
	// int i_w_s, i_w_e;
	// i_w_s = __shfl_sync(0xffffffff, i_s, 0);
	// i_w_e = __shfl_sync(0xffffffff, i, 31);
	i = i_s;
	// if (i_w_e != i_w_s)
	if (one_line)
	{
		spmv_warp_single_row(g, i_s, j_s, j_e, ja, a, x, y);
	}
	else
	{
		ptr_next = row_ptr[i+1];
		k = 0;
		for (j=j_s;j<j_e;j++)
		{
			if (j >= ptr_next)
			{
				atomicAdd(&y[i], sum);
				sum = 0;
				while (j >= ptr_next)
				{
					i++;
					ptr_next = row_ptr[i+1];
				}
				k++;
			}
			// sum += a[j] * x[ja[j] & 0x7FFFFFFF];
			sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
		}
		reduce_warp(g, i, sum, y);
	}
}


__device__ void spmv_full_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	// const int tidb = threadIdx.x;
	const int tidw = threadIdx.x % 32;
	const int warp_id = threadIdx.x / 32;
	const int block_id = blockIdx.x;
	const int nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[BLOCK_SIZE * sizeof(ValueType)];
	[[gnu::unused]] int i_s, i_e, j, j_s, j_e, j_w_s, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	// i_s = 0;
	// i_e = m;
	j_w_s = block_id * nnz_per_block + warp_id * NNZ_PER_THREAD * 32;
	j_s = j_w_s + tidw * NNZ_PER_THREAD;
	j_e = j_s + NNZ_PER_THREAD;
	k = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_s >= row_ptr[k])
		{
			i_s = k + 1;
		}
		else
		{
			i_e = k;
		}
		k = (i_e + i_s) / 2;
	}
	i_s--;
	int one_line = (ja[j_s] & 0x80000000) ? 1 : 0;
	// int one_line = 0;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	spmv_full_warp(tile32, one_line, i_s, j_s, j_e, row_ptr, ja, a, x, y);
}


__global__ void gpu_kernel_spmv_row_indices_continuous(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	int grid_size = gridDim.x;
	int block_id = blockIdx.x;
	if (block_id == grid_size - 1)
		spmv_last_block(thread_block_i_s, thread_block_i_e, thread_block_j_s, thread_block_j_e, row_ptr, ia, ja, a, m, n, nnz, x, y);
	else
		spmv_full_block(thread_block_i_s, thread_block_i_e, row_ptr, ia, ja, a, m, n, nnz, x, y);
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	dim3 block_dims(BLOCK_SIZE);
	dim3 grid_dims[csr->num_streams];
	for(int i=0; i<csr->num_streams; i++)
		grid_dims[i] = dim3(csr->num_blocks[i]);
	// long shared_mem_size = BLOCK_SIZE * (sizeof(ValueType));
	// long shared_mem_size = BLOCK_SIZE * (sizeof(ValueType) + sizeof(INT_T));
	long shared_mem_size = 0;

	if (csr->x == NULL)
	{
		for(int i=0; i<csr->num_streams; i++)
			printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims[i].x, grid_dims[i].y, grid_dims[i].z, block_dims.x, block_dims.y, block_dims.z);
		csr->x = x;
		int offset = 0;
		for(int i=0; i<csr->num_streams; i++){
			memcpy(csr->x_h[i], x + offset, csr->n_stream[i] * sizeof(ValueType));
			offset += csr->n_stream[i];
		}

		for(int i=0; i<csr->num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x[i], csr->stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d[i], csr->x_h[i], csr->n_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x[i], csr->stream[i]));
		}

		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

		if(TIME_IT){
			for(int i=0; i<csr->num_streams; i++){
				float memcpyTime_cuda;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x[i], csr->endEvent_memcpy_x[i]));
				printf("(CUDA) (stream %d) Memcpy x time = %.4lf ms\n", i, memcpyTime_cuda);
			}
		}
	}

	if(TIME_IT2){
		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution[i], csr->stream[i]));
	}

	hipMemset(csr->y_d2, 0, csr->m * csr->num_streams * sizeof(csr->y_d2));
	// hipFuncCachePreferNone:   no preference for shared memory or L1 (default);
	// hipFuncCachePreferShared: prefer larger shared memory and smaller L1 cache;
	// hipFuncCachePreferL1:     prefer larger L1 cache and smaller shared memory;
	gpuCudaErrorCheck(hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_kernel_spmv_row_indices_continuous), hipFuncCachePreferL1));
	for(int i=0; i<csr->num_streams; i++){
		// if(TIME_IT2){
		// 	gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution[i], csr->stream[i]));
		// }
		gpu_kernel_spmv_row_indices_continuous<<<grid_dims[i], block_dims, shared_mem_size, csr->stream[i]>>>(csr->thread_block_i_s_d[i], csr->thread_block_i_e_d[i], csr->thread_block_j_s_d[i], csr->thread_block_j_e_d[i], csr->row_ptr_d[i], csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->m, csr->n_stream[i], csr->nnz_stream[i], csr->x_d[i], csr->y_d2 + i*csr->m);
		// if(TIME_IT2){
		// 	gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution[i], csr->stream[i]));
		// 	gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_execution[i]));
		// 	float curr_execution_time;
		// 	gpuCudaErrorCheck(hipEventElapsedTime(&curr_execution_time, csr->startEvent_execution[i], csr->endEvent_execution[i]));
		// 	csr->execution_time[i] += curr_execution_time;
		// }
		// printf("arxi %d\n", i);
		// gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));
		// printf("telos %d\n", i);
	}

	gpuCudaErrorCheck(hipPeekAtLastError());
	// for(int i=0; i<csr->num_streams; i++)
	// 	gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if(TIME_IT2){
		for(int i=0; i<csr->num_streams; i++){
			gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution[i], csr->stream[i]));
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_execution[i]));
			float curr_execution_time;
			gpuCudaErrorCheck(hipEventElapsedTime(&curr_execution_time, csr->startEvent_execution[i], csr->endEvent_execution[i]));
			csr->execution_time[i] += curr_execution_time;	
		}
	}
	csr->iterations++;

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream[0]));

		ValueType *ones_host, *ones_device;
		gpuCudaErrorCheck(hipHostMalloc(&ones_host, csr->num_streams * sizeof(ValueType)));
		for (int i=0; i<csr->num_streams; i++) ones_host[i] = 1.0;
		gpuCudaErrorCheck(hipMalloc(&ones_device, csr->num_streams * sizeof(ValueType)));	
		gpuCudaErrorCheck(hipMemcpyAsync(ones_device, ones_host, csr->num_streams * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[0]));
	
		ValueType  alpha = 1.0, beta = 0.0;
		gpuCublasErrorCheck(hipblasDgemv(csr->handle, HIPBLAS_OP_N, csr->m, csr->num_streams, &alpha, csr->y_d2, csr->m, ones_device, 1, &beta, csr->y_d_reduction, 1));

		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y, csr->y_d_reduction, csr->m * sizeof(csr->y), hipMemcpyDeviceToHost, csr->stream[0]));

		gpuCudaErrorCheck(hipHostFree(ones_host));
		gpuCudaErrorCheck(hipFree(ones_device));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream[0]));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[0]));
		if(TIME_IT){
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
	#ifdef PRINT_STATISTICS
	if(TIME_IT2){
		iterations = 0;
		for(int i=0; i<num_streams; i++)
			execution_time[i]=0.0;
	}
	#endif
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	#ifdef PRINT_STATISTICS
	if(TIME_IT2){
		printf("--------\n");
		for(int i=0; i<num_streams; i++){
			double gflops = 2.0 * nnz_stream[i] / execution_time[i] / 1e6 * iterations;
			printf("Stream %d: %lf ms (GFLOPs = %.4lf)\n", i, execution_time[i], gflops);
		}
		printf("--------\n");
	}
	#endif
	return 0;
}

