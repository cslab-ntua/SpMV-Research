#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cusparse_util.h"
#ifdef __cplusplus
}
#endif

extern int prefetch_distance;

#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct CSCArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;

	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	hipsparseDnVecDescr_t vecX;
	hipsparseDnVecDescr_t vecY;

	hipStream_t stream;
	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_create_matA;
	hipEvent_t endEvent_create_matA;
	hipEvent_t startEvent_spmv_buffersize;
	hipEvent_t endEvent_spmv_buffersize;
	hipEvent_t startEvent_spmv_preprocess;
	hipEvent_t endEvent_spmv_preprocess;

	hipEvent_t startEvent_create_vecX;
	hipEvent_t endEvent_create_vecX;
	hipEvent_t startEvent_create_vecY;
	hipEvent_t endEvent_create_vecY;

	int max_persistent_l2_cache;

	CSCArrays(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz) : Matrix_Format(m, n, nnz)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_persistent_l2_cache, cudaDevAttrMaxPersistingL2CacheSize, 0));
		// printf("max_smem_per_block=%d\n", max_smem_per_block);
		// printf("multiproc_count=%d\n", multiproc_count);
		// printf("max_threads_per_block=%d\n", max_threads_per_block);
		// printf("warp_size=%d\n", warp_size);
		// printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		ia = (INT_T *) malloc(nnz * sizeof(INT_T));
		ja = (INT_T *) malloc((n+1) * sizeof(INT_T));
		a = (ValueType *) malloc(nnz * sizeof(ValueType));

		gpuCudaErrorCheck(hipMalloc(&ia_d, nnz * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, (n+1) * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));
		gpuCusparseErrorCheck(hipsparseCreate(&handle));
		gpuCusparseErrorCheck(hipsparseSetStream(handle, stream));

		size_t csc_bufferSize = 0;
		void* csc_dBuffer    = NULL;
		gpuCusparseErrorCheck(hipsparseCsr2cscEx2_bufferSize(handle, m, n, nnz, values, row_ptr, col_ind, a, ja, ia, ValueTypeCuda, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, &csc_bufferSize));
		gpuCudaErrorCheck(hipMalloc(&csc_dBuffer, csc_bufferSize));
		gpuCusparseErrorCheck(hipsparseCsr2cscEx2(handle, m, n, nnz, values, row_ptr, col_ind, a, ja, ia, ValueTypeCuda, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, csc_dBuffer));
		gpuCudaErrorCheck(hipFree(csc_dBuffer));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_spmv_preprocess));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_spmv_preprocess));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		gpuCudaErrorCheck(hipHostMalloc(&ia_h, nnz * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc(&ja_h, (n+1) * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc(&a_h, nnz * sizeof(*a_h)));
		gpuCudaErrorCheck(hipHostMalloc(&x_h, n * sizeof(*x_h)));
		gpuCudaErrorCheck(hipHostMalloc(&y_h, m * sizeof(*y_h)));

		memcpy(ia_h, ia, nnz * sizeof(*ia_h));
		memcpy(ja_h, ja, (n+1) * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia, nnz * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja, (n+1) * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		// Create sparse matrix A in CSC format
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_create_matA));
		gpuCusparseErrorCheck(hipsparseCreateCsc(&matA, m, n, nnz, ja_d, ia_d, a_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_create_matA));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_create_matA));

			float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, create_matA_Time;//memcpyTime_cuda_thread_i_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_matA_Time, startEvent_create_matA, endEvent_create_matA));
			printf("(CUDA) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, matA time = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, create_matA_Time);
		}
	}

	~CSCArrays()
	{
		free(a);
		free(ia);
		free(ja);

		// destroy matrix/vector descriptors
		gpuCusparseErrorCheck(hipsparseDestroySpMat(matA));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecX));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecY));
		gpuCusparseErrorCheck(hipsparseDestroy(handle));
		gpuCudaErrorCheck(hipStreamDestroy(stream));

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(dBuffer));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_spmv_preprocess));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_spmv_preprocess));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csc(CSCArrays * restrict csc, ValueType * restrict x , ValueType * restrict y);


void
CSCArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csc(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSCArrays * csc = new CSCArrays(row_ptr, col_ind, values, m, n, nnz);
	csc->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (n+1) * sizeof(INT_T);
	csc->format_name = (char *) "CUSPARSE_CSC";
	return csc;
}


//==========================================================================================================================================
//= CSC Custom
//==========================================================================================================================================


void
compute_csc(CSCArrays * restrict csc, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (csc->x == NULL)
	{
		csc->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->startEvent_memcpy_x, csc->stream));
		memcpy(csc->x_h, x, csc->n * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csc->x_d, csc->x_h, csc->n * sizeof(*csc->x_d), hipMemcpyHostToDevice, csc->stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->endEvent_memcpy_x, csc->stream));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csc->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csc->startEvent_memcpy_x, csc->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		// Create dense vector X
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->startEvent_create_vecX));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&csc->vecX, csc->n, csc->x_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->endEvent_create_vecX));

		// Create dense vector y
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->startEvent_create_vecY));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&csc->vecY, csc->m, csc->y_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->endEvent_create_vecY));

		// Allocate an external buffer if needed
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->startEvent_spmv_buffersize));
		gpuCusparseErrorCheck(hipsparseSpMV_bufferSize(csc->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csc->matA, csc->vecX, &beta, csc->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &csc->bufferSize));
		gpuCudaErrorCheck(hipMalloc(&csc->dBuffer, csc->bufferSize));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->endEvent_spmv_buffersize));
		// printf("SpMV_bufferSize = %zu bytes\n", csc->bufferSize, csc->bufferSize); // size of the workspace that is needed by hipsparseSpMV()

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->startEvent_spmv_preprocess));
		gpuCusparseErrorCheck(hipsparseSpMV_preprocess(csc->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csc->matA, csc->vecX, &beta, csc->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, csc->dBuffer));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->endEvent_spmv_preprocess));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csc->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventSynchronize(csc->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventSynchronize(csc->endEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventSynchronize(csc->endEvent_spmv_preprocess));
			float create_vecX_time, create_vecY_time, spmv_buffersize_time, spmv_preprocess_time;
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecX_time, csc->startEvent_create_vecX, csc->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecY_time, csc->startEvent_create_vecY, csc->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventElapsedTime(&spmv_buffersize_time, csc->startEvent_spmv_buffersize, csc->endEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventElapsedTime(&spmv_preprocess_time, csc->startEvent_spmv_preprocess, csc->endEvent_spmv_preprocess));
			printf("(CUDA) Create vecX time = %.4lf ms, vecY time = %.4lf ms, spmv_buffersize time = %.4lf (SpMV_bufferSize = %zu), spmv_preprocess time = %.4lf\n", create_vecX_time, create_vecY_time, spmv_buffersize_time, csc->bufferSize, spmv_preprocess_time);
		}

		#ifdef PERSISTENT_L2_PREFETCH
			int x_d_size = csc->n * sizeof(*csc->x);
			gpuCudaErrorCheck(cudaCtxResetPersistingL2Cache()); // This needs to happen every time before running kernel for 1st time for a matrix...
			if(x_d_size < csc->max_persistent_l2_cache){
				hipLaunchAttributeValue attribute;
				auto &window = attribute.accessPolicyWindow;
				window.base_ptr = csc->x_d;
				window.num_bytes = x_d_size;
				window.hitRatio = 1.0;
				window.hitProp = hipAccessPropertyPersisting;
				window.missProp = hipAccessPropertyStreaming;
				gpuCudaErrorCheck(hipStreamSetAttribute(csc->stream, hipLaunchAttributeAccessPolicyWindow, &attribute));
			}
		#endif
	}

	gpuCusparseErrorCheck(hipsparseSpMV(csc->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csc->matA, csc->vecX, &beta, csc->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, csc->dBuffer));
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csc->y == NULL)
	{
		csc->y = y;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->startEvent_memcpy_y, csc->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csc->y_h, csc->y_d, csc->m * sizeof(*csc->y_d), hipMemcpyDeviceToHost, csc->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csc->stream));
		memcpy(y, csc->y_h, csc->m * sizeof(ValueType));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csc->endEvent_memcpy_y, csc->stream));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csc->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csc->startEvent_memcpy_y, csc->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSCArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSCArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

