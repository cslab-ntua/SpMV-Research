#include <stdlib.h>
#include <stdio.h>
#include <omp.h>
#include <iostream>
#include <cmath>

#include "csr5_cuda/anonymouslib_cuda.h"

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

using namespace std;


struct CSR5Arrays : Matrix_Format
{
	anonymouslibHandle<int, unsigned int, ValueType> * A;

	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	CSR5Arrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		// Matrix A
		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(INT_T)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz  * sizeof(INT_T)));
		gpuCudaErrorCheck(hipMalloc(&a_d,  nnz  * sizeof(ValueType)));

		gpuCudaErrorCheck(hipMemcpy(ia_d, ia, (m+1) * sizeof(INT_T),   hipMemcpyHostToDevice));
		gpuCudaErrorCheck(hipMemcpy(ja_d, ja, nnz * sizeof(INT_T),     hipMemcpyHostToDevice));
		gpuCudaErrorCheck(hipMemcpy(a_d,   a, nnz * sizeof(ValueType), hipMemcpyHostToDevice));

		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

	}

	~CSR5Arrays()
	{
		free(ia);
		free(ja);
		free(a);
		A->destroy();
		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr5(CSR5Arrays * csr5, ValueType * x , ValueType * y);


void
CSR5Arrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr5(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSR5Arrays * csr5 = new CSR5Arrays(row_ptr, col_ind, values, m, n, nnz);
	csr5->format_name = (char *) "CSR5_CUDA";

	csr5->A = new anonymouslibHandle<int, unsigned int, ValueType>(m, n);
	csr5->A->inputCSR(nnz, csr5->ia_d, csr5->ja_d, csr5->a_d);

	int sigma = ANONYMOUSLIB_AUTO_TUNED_SIGMA;         // defined in common_cuda.h
	// int sigma = nnz / (8*ANONYMOUSLIB_CSR5_OMEGA);
	csr5->A->setSigma(sigma);

	csr5->A->asCSR5();

	return csr5;
}


void
compute_csr5(CSR5Arrays * csr5, ValueType * x , ValueType * y)
{
	if (csr5->x == NULL)
	{
		csr5->x = x;
		gpuCudaErrorCheck(hipMemcpy(csr5->x_d, x, csr5->n * sizeof(ValueType), hipMemcpyHostToDevice));
		csr5->A->setX(csr5->x_d);
	}

	ValueType alpha = 1.0;
	csr5->A->spmv(alpha, csr5->y_d);
	gpuCudaErrorCheck(hipDeviceSynchronize());
	
	if (csr5->y == NULL)
	{
		csr5->y = y;
		gpuCudaErrorCheck(hipMemcpy(csr5->y, csr5->y_d, csr5->m * sizeof(ValueType), hipMemcpyDeviceToHost));
	}

}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSR5Arrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSR5Arrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

