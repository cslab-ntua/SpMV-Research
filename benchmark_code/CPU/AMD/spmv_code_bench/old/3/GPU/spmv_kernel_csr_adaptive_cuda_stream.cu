#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cublas_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

#ifndef MULTIBLOCK_SIZE
#define MULTIBLOCK_SIZE 4
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 1
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

#ifndef TIME_IT2
#define TIME_IT2 1
#endif

INT_T spmv_csr_adaptive_rowblocks(INT_T *row_ptr, INT_T m, INT_T *row_blocks)
{
	row_blocks[0] = 0; 
	INT_T sum = 0; 
	INT_T last_i = 0; 
	INT_T cnt = 1;
	for (INT_T i = 1; i < m; i++) {
		// Count non-zeroes in this row 
		sum += row_ptr[i] - row_ptr[i-1];
		if (sum == BLOCK_SIZE){
			// This row fills up LOCAL_SIZE 
			last_i = i;
			row_blocks[cnt++] = i;
			sum = 0;
		}
		else if (sum > BLOCK_SIZE){
			if (i - last_i > 1) {
				// This extra row will not fit 
				row_blocks[cnt++] = i - 1;
				i--;
			}
			else if (i - last_i == 1){
				// This one row is too large
				row_blocks[cnt++] = i;
			}
			last_i = i;
			sum = 0;
		}
	}
	//  fill remaining positions of row_blocks until cnt % MULTIBLOCK_SIZE equals zero
	while (cnt % MULTIBLOCK_SIZE != 0)
		row_blocks[cnt++] = m;
	row_blocks[cnt++] = m;
	return cnt;
}

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * row_blocks[NUM_STREAMS];
	INT_T row_blocks_cnt[NUM_STREAMS];

	INT_T * ia_h[NUM_STREAMS];
	INT_T * ja_h[NUM_STREAMS];
	ValueType * a_h[NUM_STREAMS];

	INT_T * row_blocks_h[NUM_STREAMS];

	INT_T * ia_d[NUM_STREAMS];
	INT_T * ja_d[NUM_STREAMS];
	ValueType * a_d[NUM_STREAMS];

	INT_T * row_blocks_d[NUM_STREAMS];

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h[NUM_STREAMS];
	ValueType * y_h[NUM_STREAMS];
	// ValueType * y_h2;
	// ValueType * y_h_final;
	ValueType * x_d[NUM_STREAMS];
	// ValueType * y_d[NUM_STREAMS];
	ValueType * y_d2;
	ValueType * y_d_reduction;

	hipStream_t stream[NUM_STREAMS];
	INT_T n_stream[NUM_STREAMS];
	INT_T nnz_stream[NUM_STREAMS];

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution[NUM_STREAMS];
	hipEvent_t endEvent_execution[NUM_STREAMS];
	float execution_time[NUM_STREAMS];
	int iterations;
	
	hipEvent_t startEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_row_blocks[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_row_blocks[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_a[NUM_STREAMS];

	hipEvent_t startEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipblasHandle_t handle;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, block_size, block_size2, max_threads_per_multiproc;
	int num_streams;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		block_size = BLOCK_SIZE;
		block_size2 = MULTIBLOCK_SIZE;
		num_streams = NUM_STREAMS;

		/********************************************************************************************************/
		printf("/********************************************************************************************************/\n");
		// Convert CSR representation ton CSC
		INT_T * row_indices; //for CSC format
		INT_T * row_idx;
		INT_T * col_ptr;
		ValueType * val_c;

		row_indices = (typeof(row_indices)) malloc(nnz * sizeof(*row_indices));
		row_idx = (typeof(row_idx)) malloc(nnz * sizeof(*row_idx));
		col_ptr = (typeof(col_ptr)) malloc((n+1) * sizeof(*col_ptr));
		val_c = (typeof(val_c)) malloc(nnz * sizeof(*val_c));

		double time = time_it(1,
			csr_row_indices(ia, ja, m, n, nnz, &row_indices);
			coo_to_csc(row_indices, ja, a, m, n, nnz, row_idx, col_ptr, val_c, 1);
			free(row_indices);
		);
		printf("time coo_to_csc = %g ms\n", time*1e3);

		INT_T *local_stream_j_s = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_s));
		INT_T *local_stream_j_e = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_e));
		double time_balance = time_it(1,
			for (int i=0;i<num_streams;i++)
				loop_partitioner_balance_prefix_sums(num_streams, i, col_ptr, n, nnz, &local_stream_j_s[i], &local_stream_j_e[i]);
		);

		int cnt=0, cnt2=0;
		for(int i=0; i<num_streams; i++){
			nnz_stream[i] = col_ptr[local_stream_j_e[i]] - col_ptr[local_stream_j_s[i]];
			n_stream[i] = local_stream_j_e[i] - local_stream_j_s[i];
			printf("local_stream[%d] = %d - %d (%d cols) (%d nnz)\n", i, local_stream_j_s[i], local_stream_j_e[i], n_stream[i], nnz_stream[i]);

			cnt  += nnz_stream[i];
			cnt2 += n_stream[i];
		}

		printf("balance time (col) = %g ms\n", time_balance*1e3);

		INT_T * row_idx_stream[num_streams];
		INT_T * col_ptr_stream[num_streams];
		ValueType * val_c_stream[num_streams];
		
		double time_memcpy_stream_locals = time_it(1,
		for(int i=0; i<num_streams; i++){
			col_ptr_stream[i] = (INT_T *) malloc((n_stream[i]+1) * sizeof(INT_T));
			row_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_c_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			memcpy(col_ptr_stream[i], col_ptr + local_stream_j_s[i], (n_stream[i] + 1) * sizeof(INT_T));
			// col_ptr needs to be fixed, so that it will start from 0 again...
			for(int j=0; j<n_stream[i]+1; j++)
				col_ptr_stream[i][j] -= col_ptr[local_stream_j_s[i]];
			memcpy(row_idx_stream[i], row_idx + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(INT_T));
			memcpy(val_c_stream[i], val_c + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy_stream_locals = %lf ms\n", time_memcpy_stream_locals*1e3);
		free(local_stream_j_s);
		free(local_stream_j_e);

		INT_T * row_ptr_stream[num_streams];
		INT_T * col_idx_stream[num_streams];
		ValueType * val_stream[num_streams];

		for(int i=0; i<num_streams; i++){
			INT_T * col_indices;
			csc_col_indices(row_idx_stream[i], col_ptr_stream[i], m, n_stream[i], nnz_stream[i], &col_indices);

			row_ptr_stream[i] = (INT_T *) malloc((m+1) * sizeof(INT_T));
			col_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			coo_to_csr(row_idx_stream[i], col_indices, val_c_stream[i], m, n_stream[i], nnz_stream[i], row_ptr_stream[i], col_idx_stream[i], val_stream[i], 1, 0);
			free(col_indices);
		}

		for(int i=0; i<num_streams; i++){
			free(row_idx_stream[i]);
			free(col_ptr_stream[i]);
			free(val_c_stream[i]);
		}
		free(row_idx);
		free(col_ptr);
		free(val_c);

		// for(int i=0; i<num_streams; i++)
		// 	printf("Stream %d: %d columns, %d nnz\n", i, n_stream[i], nnz_stream[i]);

		printf("/********************************************************************************************************/\n");
		/********************************************************************************************************/

		for(int i=0; i<num_streams; i++){
			row_blocks[i] = (INT_T *) malloc(m * sizeof(INT_T));
			row_blocks_cnt[i] = spmv_csr_adaptive_rowblocks(row_ptr_stream[i], m, row_blocks[i]);
			printf("Stream %d: %d columns, %d nnz, %d row_blocks ( %.0lf nnz/row_block )\n", i, n_stream[i], nnz_stream[i], row_blocks_cnt[i], nnz_stream[i]*1.0/row_blocks_cnt[i]);
		}

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipMalloc(&ia_d[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&row_blocks_d[i], row_blocks_cnt[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ja_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&a_d[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&x_d[i], n_stream[i] * sizeof(ValueType)));
			// gpuCudaErrorCheck(hipMalloc(&y_d[i], m * sizeof(ValueType)));
		}
		gpuCudaErrorCheck(hipMalloc(&y_d2, m * num_streams * sizeof(ValueType)));
		gpuCudaErrorCheck(hipMalloc(&y_d_reduction, m * sizeof(ValueType)));
		gpuCublasErrorCheck(hipblasCreate(&handle));

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipHostMalloc(&ia_h[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&row_blocks_h[i], row_blocks_cnt[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ja_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&a_h[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&x_h[i], n_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&y_h[i], m * sizeof(ValueType)));
		}
		// gpuCudaErrorCheck(hipHostMalloc(&y_h2, m * num_streams * sizeof(ValueType)));
		// gpuCudaErrorCheck(hipHostMalloc(&y_h_final, m * sizeof(ValueType)));

		double time_memcpy = time_it(1,
		for(int i=0; i<num_streams; i++){
			memcpy(ia_h[i], row_ptr_stream[i], (m + 1) * sizeof(INT_T));
			memcpy(row_blocks_h[i], row_blocks[i], row_blocks_cnt[i] * sizeof(INT_T));
			memcpy(ja_h[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(a_h[i], val_stream[i], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy (ia_h, row_blocks_h, ja_h, a_h) = %lf ms\n", time_memcpy*1e3);

		// cuda events for timing measurements
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipStreamCreate(&stream[i]));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_execution[i]));
			execution_time[i] = 0.0;
		}
		iterations=0;
		gpuCublasErrorCheck(hipblasSetStream(handle, stream[0]));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_row_blocks[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_row_blocks[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a[i]));

				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		for(int i=0; i<num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ia_d[i], ia_h[i], (m+1) * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_row_blocks[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(row_blocks_d[i], row_blocks_h[i], row_blocks_cnt[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_row_blocks[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ja_d[i], ja_h[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(a_d[i], a_h[i], nnz_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a[i], stream[i]));
		}

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(stream[i]));
				float memcpyTime_cuda_ia, memcpyTime_cuda_row_blocks, memcpyTime_cuda_ja, memcpyTime_cuda_a;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia[i], endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_row_blocks, startEvent_memcpy_row_blocks[i], endEvent_memcpy_row_blocks[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja[i], endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a[i], endEvent_memcpy_a[i]));
				printf("(CUDA) (stream %d) Memcpy ia time = %.4lf ms, row_blocks time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms\n", i, memcpyTime_cuda_ia, memcpyTime_cuda_row_blocks, memcpyTime_cuda_ja, memcpyTime_cuda_a);
			}
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		for(int i=0; i<num_streams; i++){
			free(row_blocks[i]);

			gpuCudaErrorCheck(hipFree(ia_d[i]));
			gpuCudaErrorCheck(hipFree(row_blocks_d[i]));
			gpuCudaErrorCheck(hipFree(ja_d[i]));
			gpuCudaErrorCheck(hipFree(a_d[i]));
			gpuCudaErrorCheck(hipFree(x_d[i]));
			// gpuCudaErrorCheck(hipFree(y_d[i]));

			gpuCudaErrorCheck(hipHostFree(ia_h[i]));
			gpuCudaErrorCheck(hipHostFree(row_blocks_h[i]));
			gpuCudaErrorCheck(hipHostFree(ja_h[i]));
			gpuCudaErrorCheck(hipHostFree(a_h[i]));
			gpuCudaErrorCheck(hipHostFree(x_h[i]));
			gpuCudaErrorCheck(hipHostFree(y_h[i]));

			gpuCudaErrorCheck(hipStreamDestroy(stream[i]));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_execution[i]));
		}
		gpuCudaErrorCheck(hipFree(y_d2));
		gpuCudaErrorCheck(hipFree(y_d_reduction));
		gpuCublasErrorCheck(hipblasDestroy(handle));
		// gpuCudaErrorCheck(hipHostFree(y_h2));
		// gpuCudaErrorCheck(hipHostFree(y_h_final));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_row_blocks[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_row_blocks[i]));
			}
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz, int symmetric)
{
	if (symmetric)
		error("symmetric matrices not supported by this format, expand symmetry");
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_ADAPTIVE_b%d_%d_s%d", csr->block_size, csr->block_size2, csr->num_streams);

	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================

/*__global__ void gpu_kernel_csr_adaptive(INT_T * ia, INT_T * ja, ValueType * a, INT_T * row_blocks, ValueType * restrict x, ValueType * restrict y)
{
	INT_T startRow = row_blocks[blockIdx.x];
	INT_T nextStartRow = row_blocks[blockIdx.x + 1];
	INT_T num_rows = nextStartRow - startRow;
	INT_T i = threadIdx.x;
	__shared__ volatile ValueType LDS[BLOCK_SIZE];
	
	// If the block consists of more than one row then run CSR Stream
	if (num_rows > 1) {
		// how many nonzeros does this rowblock hold?
		// they will be less than the BLOCK_SIZE (the size of LDS)
		int nnz = ia[nextStartRow] - ia[startRow];
		int col_offset = ia[startRow];

		// Each thread writes to shared memory the result of multiplication for one nonzero
		// However, if there are less nonzeros than the block size, some threads will not be utilized
		if (i < nnz)
			LDS[i] = a[col_offset + i] * x[ja[col_offset + i]];
 		// After all positions of LDS have been filled, proceed. 
		__syncthreads();
		
		// Threads that fall within a range sum up the partial results
		// Thread0 of the block will be assigned with the first row of the thread block (startRow+0) and then the next row will be (startRow+BLOCK_SIZE) etc...
		// How many rows per thread depends on how few nonzeros this specific block can hold...
		for (int k = startRow + i; k < nextStartRow; k += BLOCK_SIZE){
			ValueType temp = 0;
			// Sum partial results that this row (k) has results in LDS
			for (INT_T j = (ia[k] - col_offset); j < (ia[k + 1] - col_offset); j++)
				temp = temp + LDS[j];
			// And finally store result in the output y vector.
			y[k] = temp;
		}
	}
	// If the block consists of only one row then run CSR Vector
	else {
		// Thread ID in warp
		INT_T ia_Start = ia[startRow];
		INT_T ia_End   = ia[nextStartRow];
		ValueType sum  = 0;

		// Use all threads in a warp to accumulate multiplied elements
		// Due to the fact that each for loop starts from "ia_Start" + some i (the index inside the thread block) 
		// LDS will be filled with all partial results from this specific row
		// It may be underutilized, considering the fact that this row will consist of less than BLOCK_SIZE elements
		for (INT_T j = ia_Start + i; j < ia_End; j += BLOCK_SIZE){
			INT_T col = ja[j];
			sum += a[j] * x[col];
		}
		// write partial sum at position i (index in thread block) in the LDS array
		LDS[i] = sum;
		__syncthreads();

		// Reduce partial sums
		// reduce results as in 
		// (BS/2 sums)  LDS[i] = LDS[i] + LDS[i + BS/2];, LDS[i+1] = LDS[i+1] + LDS[i+1 + BS/2];
		// (BS/4 sums)  LDS[i] = LDS[i] + LDS[i + BS/4]
		// ...
		// (1 sum)      LDS[i] = LDS[i] + LDS[i+1]; and then finish
		for (int stride = BLOCK_SIZE >> 1; stride > 0; stride >>= 1) {
			__syncthreads();
			if (i < stride)
				LDS[i] += LDS[i + stride]; 
		}
		// Write result
		if (i == 0){
			y[startRow] = LDS[i];
		}
	}
}*/


__global__ void gpu_kernel_csr_adaptive(INT_T * ia, INT_T * ja, ValueType * a, INT_T * row_blocks, ValueType * restrict x, ValueType * restrict y)
{
	__shared__ volatile ValueType LDS[MULTIBLOCK_SIZE][BLOCK_SIZE];
	INT_T i = threadIdx.x;

	INT_T startRow[MULTIBLOCK_SIZE];
	INT_T nextStartRow[MULTIBLOCK_SIZE];
	INT_T num_rows[MULTIBLOCK_SIZE];

	for(int kk = 0; kk < MULTIBLOCK_SIZE; kk++){
		startRow[kk]     = row_blocks[blockIdx.x*MULTIBLOCK_SIZE + kk];
		nextStartRow[kk] = row_blocks[blockIdx.x*MULTIBLOCK_SIZE + kk + 1];
		num_rows[kk]     = nextStartRow[kk] - startRow[kk];
	}

	for(int kk = 0; kk < MULTIBLOCK_SIZE; kk++){
		// If the block consists of more than one row then run CSR Stream
		if (num_rows[kk] > 1) {
			// how many nonzeros does this rowblock hold?
			// they will be less than the BLOCK_SIZE (the size of LDS)
			int nnz = ia[nextStartRow[kk]] - ia[startRow[kk]];
			int col_offset = ia[startRow[kk]];

			// Each thread writes to shared memory the result of multiplication for one nonzero
			if (i < nnz)
				LDS[kk][i] = a[col_offset + i] * x[ja[col_offset + i]];
	 		// After all positions of LDS have been filled, proceed. 
			__syncthreads();
			
			// Threads that fall within a range sum up the partial results
			// Thread0 of the block will be assigned with the first row of the thread block (startRow+0) and then the next row will be (startRow+BLOCK_SIZE) etc...
			// How many rows per thread depends on how few nonzeros this specific block can hold...
			for (int k = startRow[kk] + i; k < nextStartRow[kk]; k += BLOCK_SIZE){
				ValueType temp = 0;
				// Sum partial results that this row (k) has results in LDS
				for (INT_T j = (ia[k] - col_offset); j < (ia[k + 1] - col_offset); j++)
					temp = temp + LDS[kk][j];
				// And finally store result in the output y vector.
				y[k] = temp;
			}
		}
		// If the block consists of only one row then run CSR Vector
		else if(num_rows[kk] == 1) {
			// Thread ID in warp
			INT_T ia_Start = ia[startRow[kk]];
			INT_T ia_End   = ia[nextStartRow[kk]];
			ValueType sum  = 0;

			// Use all threads in a warp to accumulate multiplied elements
			// Due to the fact that each for loop starts from "ia_Start" + some i (the index inside the thread block) 
			// LDS will be filled with all partial results from this specific row
			// It may be underutilized, considering the fact that this row will consist of less than BLOCK_SIZE elements
			for (INT_T j = ia_Start + i; j < ia_End; j += BLOCK_SIZE){
				INT_T col = ja[j];
				sum = __fma_rn(a[j], x[col], sum); // sum += a[j] * x[col];
			}
			// write partial sum at position i (index in thread block) in the LDS array
			LDS[kk][i] = sum;
			__syncthreads();

			// Reduce partial sums
			// reduce results as in 
			// (BS/2 sums)  LDS[i] = LDS[i] + LDS[i + BS/2];, LDS[i+1] = LDS[i+1] + LDS[i+1 + BS/2];
			// (BS/4 sums)  LDS[i] = LDS[i] + LDS[i + BS/4]
			// ...
			// (1 sum)      LDS[i] = LDS[i] + LDS[i+1]; and then finish
			for (int stride = BLOCK_SIZE >> 1; stride > 0; stride >>= 1) {
				__syncthreads();
				if (i < stride)
					LDS[kk][i] += LDS[kk][i + stride]; 
			}
			// Write result
			if (i == 0){
				y[startRow[kk]] = LDS[kk][i];
			}
		}
	}
}

__global__ void gpu_kernel_csr_adaptive_local2048(INT_T * ia, INT_T * ja, ValueType * a, INT_T * row_blocks, ValueType * restrict x, ValueType * restrict y)
{
	__shared__ volatile ValueType LDS[MULTIBLOCK_SIZE][BLOCK_SIZE];
	INT_T i = threadIdx.x;
	__shared__ volatile ValueType x_local[2048];
	// instruct each thread of block to fetch some values of x to x_local
	for(int j=i; j<2048; j+=BLOCK_SIZE)
		x_local[j] = x[j];

	INT_T startRow[MULTIBLOCK_SIZE];
	INT_T nextStartRow[MULTIBLOCK_SIZE];
	INT_T num_rows[MULTIBLOCK_SIZE];

	for(int kk = 0; kk < MULTIBLOCK_SIZE; kk++){
		startRow[kk]     = row_blocks[blockIdx.x*MULTIBLOCK_SIZE + kk];
		nextStartRow[kk] = row_blocks[blockIdx.x*MULTIBLOCK_SIZE + kk + 1];
		num_rows[kk]     = nextStartRow[kk] - startRow[kk];
	}

	for(int kk = 0; kk < MULTIBLOCK_SIZE; kk++){
		// If the block consists of more than one row then run CSR Stream
		if (num_rows[kk] > 1) {
			// how many nonzeros does this rowblock hold?
			// they will be less than the BLOCK_SIZE (the size of LDS)
			int nnz = ia[nextStartRow[kk]] - ia[startRow[kk]];
			int col_offset = ia[startRow[kk]];

			// Each thread writes to shared memory the result of multiplication for one nonzero
			if (i < nnz)
				LDS[kk][i] = a[col_offset + i] * x_local[ja[col_offset + i]];
	 		// After all positions of LDS have been filled, proceed. 
			__syncthreads();
			
			// Threads that fall within a range sum up the partial results
			// Thread0 of the block will be assigned with the first row of the thread block (startRow+0) and then the next row will be (startRow+BLOCK_SIZE) etc...
			// How many rows per thread depends on how few nonzeros this specific block can hold...
			for (int k = startRow[kk] + i; k < nextStartRow[kk]; k += BLOCK_SIZE){
				ValueType temp = 0;
				// Sum partial results that this row (k) has results in LDS
				for (INT_T j = (ia[k] - col_offset); j < (ia[k + 1] - col_offset); j++)
					temp = temp + LDS[kk][j];
				// And finally store result in the output y vector.
				y[k] = temp;
			}
		}
		// If the block consists of only one row then run CSR Vector
		else if(num_rows[kk] == 1) {
			// Thread ID in warp
			INT_T ia_Start = ia[startRow[kk]];
			INT_T ia_End   = ia[nextStartRow[kk]];
			ValueType sum  = 0;

			// Use all threads in a warp to accumulate multiplied elements
			// Due to the fact that each for loop starts from "ia_Start" + some i (the index inside the thread block) 
			// LDS will be filled with all partial results from this specific row
			// It may be underutilized, considering the fact that this row will consist of less than BLOCK_SIZE elements
			for (INT_T j = ia_Start + i; j < ia_End; j += BLOCK_SIZE){
				INT_T col = ja[j];
				sum += a[j] * x_local[col];
			}
			// write partial sum at position i (index in thread block) in the LDS array
			LDS[kk][i] = sum;
			__syncthreads();

			// Reduce partial sums
			// reduce results as in 
			// (BS/2 sums)  LDS[i] = LDS[i] + LDS[i + BS/2];, LDS[i+1] = LDS[i+1] + LDS[i+1 + BS/2];
			// (BS/4 sums)  LDS[i] = LDS[i] + LDS[i + BS/4]
			// ...
			// (1 sum)      LDS[i] = LDS[i] + LDS[i+1]; and then finish
			for (int stride = BLOCK_SIZE >> 1; stride > 0; stride >>= 1) {
				__syncthreads();
				if (i < stride)
					LDS[kk][i] += LDS[kk][i + stride]; 
			}
			// Write result
			if (i == 0){
				y[startRow[kk]] = LDS[kk][i];
			}
		}
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	dim3 block_dims(csr->block_size);
	dim3 grid_dims[csr->num_streams];
		// dim3 grid_dims(csr->row_blocks_cnt-1);
	// dim3 grid_dims(ceil((csr->row_blocks_cnt-1)/MULTIBLOCK_SIZE));

	for(int i=0; i<csr->num_streams; i++)
		grid_dims[i] = dim3(ceil((csr->row_blocks_cnt[i]-1)/MULTIBLOCK_SIZE));

	if (csr->x == NULL)
	{
		for(int i=0; i<csr->num_streams; i++)
			printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims[i].x, grid_dims[i].y, grid_dims[i].z, block_dims.x, block_dims.y, block_dims.z);
		csr->x = x;
		int offset = 0;
		for(int i=0; i<csr->num_streams; i++){
			memcpy(csr->x_h[i], x + offset, csr->n_stream[i] * sizeof(ValueType));
			offset += csr->n_stream[i];
		}

		for(int i=0; i<csr->num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x[i], csr->stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d[i], csr->x_h[i], csr->n_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x[i], csr->stream[i]));
		}

		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

		if(TIME_IT){
			for(int i=0; i<csr->num_streams; i++){
				float memcpyTime_cuda;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x[i], csr->endEvent_memcpy_x[i]));
				printf("(CUDA) (stream %d) Memcpy x time = %.4lf ms\n", i, memcpyTime_cuda);
			}
		}
	}

	if(TIME_IT2){
		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution[i], csr->stream[i]));
	}

	for(int i=0; i<csr->num_streams; i++){
		gpu_kernel_csr_adaptive<<<grid_dims[i], block_dims, 0, csr->stream[i]>>>(csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->row_blocks_d[i], csr->x_d[i], csr->y_d2 + i*csr->m);
	}

	gpuCudaErrorCheck(hipPeekAtLastError());
	for(int i=0; i<csr->num_streams; i++)
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

	if(TIME_IT2){
		for(int i=0; i<csr->num_streams; i++){
			gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution[i], csr->stream[i]));
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_execution[i]));
			float curr_execution_time;
			gpuCudaErrorCheck(hipEventElapsedTime(&curr_execution_time, csr->startEvent_execution[i], csr->endEvent_execution[i]));
			csr->execution_time[i] += curr_execution_time;	
		}
	}
	csr->iterations++;

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream[0]));

		ValueType *ones_host, *ones_device;
		gpuCudaErrorCheck(hipHostMalloc(&ones_host, csr->num_streams * sizeof(ValueType)));
		for (int i=0; i<csr->num_streams; i++) ones_host[i] = 1.0;
		gpuCudaErrorCheck(hipMalloc(&ones_device, csr->num_streams * sizeof(ValueType)));	
		gpuCudaErrorCheck(hipMemcpyAsync(ones_device, ones_host, csr->num_streams * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[0]));
	
		ValueType  alpha = 1.0, beta = 0.0;
		gpuCublasErrorCheck(hipblasDgemv(csr->handle, HIPBLAS_OP_N, csr->m, csr->num_streams, &alpha, csr->y_d2, csr->m, ones_device, 1, &beta, csr->y_d_reduction, 1));

		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y, csr->y_d_reduction, csr->m * sizeof(csr->y), hipMemcpyDeviceToHost, csr->stream[0]));

		gpuCudaErrorCheck(hipHostFree(ones_host));
		gpuCudaErrorCheck(hipFree(ones_device));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream[0]));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[0]));
		if(TIME_IT){
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
	#ifdef PRINT_STATISTICS
	if(TIME_IT2){
		iterations = 0;
		for(int i=0; i<num_streams; i++)
			execution_time[i]=0.0;
	}
	#endif
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	#ifdef PRINT_STATISTICS
	if(TIME_IT2){		
		printf("--------\n");
		for(int i=0; i<num_streams; i++){
			double gflops = 2.0 * nnz_stream[i] / execution_time[i] / 1e6 * iterations;
			printf("Stream %d: %lf ms (GFLOPs = %.4lf)\n", i, execution_time[i], gflops);
		}
		printf("--------\n");
	}
	#endif
	return 0;
}

