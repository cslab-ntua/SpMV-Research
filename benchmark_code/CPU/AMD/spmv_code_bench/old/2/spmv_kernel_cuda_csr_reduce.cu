#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


INT_T * thread_block_i_s = NULL;
INT_T * thread_block_i_e = NULL;

INT_T * thread_block_j_s = NULL;
INT_T * thread_block_j_e = NULL;


INT_T * thread_block_i_s_dev = NULL;
INT_T * thread_block_i_e_dev = NULL;

INT_T * thread_block_j_s_dev = NULL;
INT_T * thread_block_j_e_dev = NULL;


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

void
cuda_push_duplicate_base(void ** dst_ptr, void * src, long bytes)
{
	hipMalloc(dst_ptr, bytes);
	hipMemcpy(*((char **) dst_ptr), src, bytes, hipMemcpyHostToDevice);
}
#define cuda_push_duplicate(dst_ptr, src, bytes) cuda_push_duplicate_base((void **) dst_ptr, src, bytes)


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;
	INT_T * ia;
	INT_T * ja;
	ValueType * a;

	INT_T * row_ptr_dev;
	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	ValueType * multres_dev;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x, max_num_threads;
	int num_threads;
	int block_size;
	int num_blocks;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		double time_balance;
		long i;

		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0);
		max_num_threads = max_threads_per_multiproc * multiproc_count;
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_num_threads=%d\n", max_num_threads);

		// block_size = 32;
		// block_size = 64;
		// block_size = 128;
		// block_size = 256;
		// block_size = 512;
		block_size = 1024;

		// num_threads = 128;
		// num_threads = 1ULL << 10;
		// num_threads = 1ULL << 12;
		// num_threads = 1ULL << 13;
		// num_threads = 1ULL << 14;
		// num_threads = 1ULL << 15;
		// num_threads = 1ULL << 16;
		// num_threads = 1ULL << 17;
		// num_threads = 1ULL << 20;
		// num_threads = 1ULL << 21;
		// num_threads = 1ULL << 22;
		// num_threads = 1ULL << 23;
		// num_threads = 1ULL << 24;
		// num_threads = nnz / 2;
		// num_threads = nnz / 3;
		num_threads = nnz / 4;
		// num_threads = nnz / 5;
		// num_threads = nnz / 8;
		// num_threads = nnz / 16;
		// num_threads = m;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		thread_block_j_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_s));
		thread_block_j_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_e));
		time_balance = time_it(1,
			for (i=0;i<num_blocks;i++)
			{

				// loop_partitioner_balance_iterations(num_blocks, i, 0, m, &thread_block_i_s[i], &thread_block_i_e[i]);
				// loop_partitioner_balance_prefix_sums(num_blocks, i, row_ptr, m, nnz, &thread_block_i_s[i], &thread_block_i_e[i]);
				// thread_block_j_s[i] = row_ptr[thread_block_i_s[i]];
				// thread_block_j_e[i] = row_ptr[thread_block_i_e[i]];

				long lower_boundary;
				loop_partitioner_balance_iterations(num_blocks, i, 0, nnz, &thread_block_j_s[i], &thread_block_j_e[i]);
				macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				thread_block_i_s[i] = lower_boundary;
			}
			for (i=0;i<num_blocks;i++)
			{
				if (i == num_blocks - 1)   // If we calculate each thread's boundaries individually some empty rows might be unassigned.
					thread_block_i_e[i] = m;
				else
					thread_block_i_e[i] = thread_block_i_s[i+1] + 1;
				if ((thread_block_j_s[i] >= row_ptr[thread_block_i_e[i]]) || (thread_block_j_s[i] < row_ptr[thread_block_i_s[i]]))
					error("bad binary search of row start: i=%d j:[%d, %d] j=%d", thread_block_i_s[i], row_ptr[thread_block_i_s[i]], row_ptr[thread_block_i_e[i]], thread_block_j_s[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		ia = (typeof(ia)) malloc(nnz * sizeof(*ia));
		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (i=0;i<m;i++)
			{
				for (j=row_ptr[i];j<row_ptr[i+1];j++)
				{
					ia[j] = i;
				}
			}
		}

		cuda_push_duplicate(&row_ptr_dev, row_ptr, (m+1) * sizeof(*row_ptr_dev));
		cuda_push_duplicate(&ia_dev, ia, nnz * sizeof(*ia_dev));
		cuda_push_duplicate(&ja_dev, ja, nnz * sizeof(*ja_dev));
		cuda_push_duplicate(&a_dev, a, nnz * sizeof(*a_dev));
		hipMalloc(&multres_dev, nnz * sizeof(*y_dev));

		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));

		cuda_push_duplicate(&thread_block_i_s_dev, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_dev));
		cuda_push_duplicate(&thread_block_i_e_dev, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_dev));
		cuda_push_duplicate(&thread_block_j_s_dev, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_dev));
		cuda_push_duplicate(&thread_block_j_e_dev, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_dev));

	}

	~CSRArrays()
	{
		free(a);
		free(row_ptr);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);

		hipFree(row_ptr_dev);
		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz, int symmetric)
{
	if (symmetric)
		error("symmetric matrices not supported by this format, expand symmetry");
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Custom_CSR_CUDA_reduce";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


// __device__ int add(int a, int b)
// {
	// return a + b;
// }


__global__ void gpu_kernel_spmv_row_indices(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	extern __shared__ char sm[];
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	ValueType * val_buf = (typeof(val_buf)) sm;
	INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	INT_T * ja_rel;
	ValueType * a_rel;
	// int i, i_s, i_e, j, j_s, j_e, k;
	int i, j, j_s, j_e;
	// i_s = thread_block_i_s[block_id];
	// i_e = thread_block_i_e[block_id];
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	int j_e_div = j_e - ((j_e-j_s) % block_size);
	for (j=j_s;j<j_e_div;j+=block_size)
	{
		ia_buf[tidb] = ia[j+tidb];
		ja_rel = &ja[j];
		a_rel = &a[j];
		val_buf[tidb] = a_rel[tidb] * x[ja_rel[tidb]];
		__syncthreads();
		for (i=1;i<block_size;i*=2)
		{
			if ((tidb & (2*i-1)) == i-1)
			{
				if (ia_buf[tidb] == ia_buf[tidb+i])
					val_buf[tidb+i] += val_buf[tidb];
				else
					y[ia_buf[tidb]] += val_buf[tidb];
			}
			__syncthreads();
		}
		if (tidb == 0)
			y[ia_buf[block_size-1]] += val_buf[block_size-1];
		__syncthreads();
	}
	if (tidb == 0)
	{
		for (j=j_e_div;j<j_e;j++)
		{
			y[ia[j]] += a[j] * x[ja[j]];
		}
	}
}


__global__ void gpu_kernel_spmv_gather_multiply(INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * multres)
{
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	int j, j_s, j_e;
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	int j_e_div = j_e - ((j_e-j_s) % block_size);
	for (j=j_s;j<j_e_div;j+=block_size)
		multres[j+tidb] = a[j+tidb] * x[ja[j+tidb]];
	j = j_e_div + tidb;
	if (j < j_e)
		multres[j] = a[j] * x[ja[j]];
}


__global__ void gpu_kernel_spmv_row_indices_atomics(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	extern __shared__ char sm[];
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	ValueType * val_buf = (typeof(val_buf)) sm;
	INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	INT_T * ja_rel;
	ValueType * a_rel;
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	int j_e_div = j_e - ((j_e-j_s) % block_size);
	i = i_s;
	for (j=j_s;j<j_e_div;j+=block_size)
	{

		// if (tidb == 0)
		// {
			// for (l=j;l<j+block_size;l++)
			// {
				// while (l >= row_ptr[i+1])
					// i++;
				// ia_buf[l-j] = i;
			// }
		// }

		// l = j + tidb;
		// while (l >= row_ptr[i+1])
			// i++;
		// ia_buf[l-j] = i;
		ia_buf[tidb] = ia[j+tidb];

		ja_rel = &ja[j];
		a_rel = &a[j];
		val_buf[tidb] = a_rel[tidb] * x[ja_rel[tidb]];
		// atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
		__syncthreads();
		i = ia_buf[block_size - 1];
		for (k=1;k<block_size;k*=2)
		{
			if ((tidb & (2*k-1)) == k-1)
			{
				ValueType val = val_buf[tidb];
				int row = ia_buf[tidb];
				if (row == ia_buf[tidb+k])
				{
					val_buf[tidb+k] += val;
					// val_buf[tidb] = 0;
				}
				else
				{
					atomicAdd(&y[row], val);
				}
			}
			__syncthreads();
		}
		if (tidb == 0)
			atomicAdd(&y[ia_buf[block_size-1]], val_buf[block_size-1]);
		// if (val_buf[tidb] != 0)
			// atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
		__syncthreads();
	}
	/* if (tidb == 0)
	{
		for (j=j_e_div;j<j_e;j++)
		{
			// y[ia[j]] += a[j] * x[ja[j]];
			// atomicAdd(&y[ia[j]], a[j] * x[ja[j]]);

			while (j >= row_ptr[i+1])
				i++;
			atomicAdd(&y[i], a[j] * x[ja[j]]);
		}
	} */
	j = j_e_div + tidb;
	if (j < j_e)
	{
		// while (j >= row_ptr[i+1])
			// i++;
		// atomicAdd(&y[i], a[j] * x[ja[j]]);
		atomicAdd(&y[ia[j]], a[j] * x[ja[j]]);
	}
}


__global__ void gpu_kernel_spmv_row_indices_continuous(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	extern __shared__ char sm[];
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	ValueType * val_buf = (typeof(val_buf)) sm;
	INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	INT_T * ja_rel;
	ValueType * a_rel;
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	int total_j = j_e - j_s;
	int j_per_t = total_j / block_size;
	int mod = total_j % block_size;
	int j_l_s, j_l_e;
	j_l_s = j_s + tidb * (total_j / block_size);
	j_l_e = j_l_s + (total_j / block_size);
	if (tidb < mod)
	{
		j_l_s += tidb;
		j_l_e += tidb + 1;
	}
	else
	{
		j_l_s += mod;
		j_l_e += mod;
	}
	// int m = (i_e + i_s) / 2;
	// while (i_s < i_e)
	// {
		// if (j_l_s >= row_ptr[m])
		// {
			// i_s = m + 1;
		// }
		// else
		// {
			// i_e = m;
		// }
		// m = (i_e + i_s) / 2;
	// }
	// i = i_s - 1;
	i = ia[j_l_s];
	// if (tidb == block_size-1)
	// {
		// if (j_l_e != j_e)
		// {
			// printf("wrong");
		// }
	// }
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_l_s;j<j_l_e;j++)
	{
		// if (ia[j] != i)
		// {
			// atomicAdd(&y[i], sum);
			// sum = 0;
			// i = ia[j];
		// }
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			// y[i] += sum;
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
			// i = ia[j];
		}
		// sum += a[j] * x[ja[j]];
		sum = __fma_rn(a[j], x[ja[j]], sum);
	}
	// if (j_l_s < j_l_e)
		// atomicAdd(&y[i], sum);
	val_buf[tidb] = sum;
	ia_buf[tidb] = i;
	__syncthreads();
	for (k=1;k<block_size;k*=2)
	{
		if ((tidb & (2*k-1)) == k-1)
		{
			ValueType val = val_buf[tidb];
			int row = ia_buf[tidb];
			if (row == ia_buf[tidb+k])
			{
				val_buf[tidb+k] += val;
				// val_buf[tidb] = 0;
			}
			else
			{
				atomicAdd(&y[row], val);
				// y[row] += val;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[block_size-1]], val_buf[block_size-1]);
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);
	// long shared_mem_size = block_size * (sizeof(ValueType));
	long shared_mem_size = block_size * (sizeof(ValueType) + sizeof(INT_T));

	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
	}

	hipMemset(csr->y_dev, 0, csr->m * sizeof(csr->y_dev));

	// gpu_kernel_spmv_gather_multiply<<<grid_dims, block_dims>>>(thread_block_j_s_dev, thread_block_j_e_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->multres_dev);
	// gpu_kernel_spmv_row_indices<<<grid_dims, block_dims, shared_mem_size>>>(thread_block_i_s_dev, thread_block_i_e_dev, thread_block_j_s_dev, thread_block_j_e_dev, csr->row_ptr_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);
	// gpu_kernel_spmv_row_indices_atomics<<<grid_dims, block_dims, shared_mem_size>>>(thread_block_i_s_dev, thread_block_i_e_dev, thread_block_j_s_dev, thread_block_j_e_dev, csr->row_ptr_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);
	gpu_kernel_spmv_row_indices_continuous<<<grid_dims, block_dims, shared_mem_size>>>(thread_block_i_s_dev, thread_block_i_e_dev, thread_block_j_s_dev, thread_block_j_e_dev, csr->row_ptr_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);

	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("hipDeviceSynchronize: %s\n", hipGetErrorString(err));
	err = hipGetLastError();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));

	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}

	// exit(0);
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

