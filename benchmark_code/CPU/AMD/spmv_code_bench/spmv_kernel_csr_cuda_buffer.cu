#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef NUM_THREADS
#define NUM_THREADS 1024
#endif

// int row_cluster_size = 256;
// int row_cluster_size = 192;
// int row_cluster_size = 128;
// int row_cluster_size = 64;
// int row_cluster_size = 32;
// int row_cluster_size = 16;
// int row_cluster_size = 8;
#ifndef ROW_CLUSTER_SIZE
#define ROW_CLUSTER_SIZE 4
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	unsigned char * rel_row_idx;
	INT_T * thread_block_i_s = NULL;
	INT_T * thread_block_i_e = NULL;

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;
	unsigned char * rel_row_idx_d;
	INT_T * thread_block_i_s_d;
	INT_T * thread_block_i_e_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;
	unsigned char * rel_row_idx_h;
	INT_T * thread_block_i_s_h;
	INT_T * thread_block_i_e_h;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	ValueType * x_h = NULL;
	ValueType * y_h = NULL;

	hipStream_t stream;
	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;
	
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;
	hipEvent_t startEvent_memcpy_thread_block_i_s;
	hipEvent_t endEvent_memcpy_thread_block_i_s;
	hipEvent_t startEvent_memcpy_thread_block_i_e;
	hipEvent_t endEvent_memcpy_thread_block_i_e;
	hipEvent_t startEvent_memcpy_rel_row_idx;
	hipEvent_t endEvent_memcpy_rel_row_idx;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_persistent_l2_cache, max_block_dim_x;
	int num_threads;
	int block_size;
	int num_blocks;

	int row_cluster_size;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		double time_balance;
		long i;

		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_persistent_l2_cache, cudaDevAttrMaxPersistingL2CacheSize, 0));
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_persistent_l2_cache=%d\n", max_persistent_l2_cache);

		// block_size = warp_size / 2;
		block_size = warp_size;
		// block_size = warp_size * 2;
		// block_size = warp_size * 4;
		row_cluster_size = ROW_CLUSTER_SIZE;

		// num_threads = 128;
		// num_threads = 1ULL << 10;
		// num_threads = 3584;
		// num_threads = 1ULL << 12;
		// num_threads = 1ULL << 13;
		// num_threads = 1ULL << 14;
		// num_threads = 1ULL << 15;
		// num_threads = 1ULL << 16;
		// num_threads = 1ULL << 17;
		// num_threads = 1ULL << 21;
		// num_threads = 1ULL << 22;
		num_threads = NUM_THREADS;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		time_balance = time_it(1,
			for (i=0; i<num_blocks; i++)
			{
				// loop_partitioner_balance_iterations(num_blocks, i, 0, m, &thread_block_i_s[i], &thread_block_i_e[i]);
				loop_partitioner_balance_prefix_sums(num_blocks, i, ia, m, nnz, &thread_block_i_s[i], &thread_block_i_e[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		rel_row_idx = (typeof(rel_row_idx)) malloc(nnz * sizeof(*rel_row_idx));
		#pragma omp parallel
		{
			long i, i_s, i_e, i_rel, j, k;
			#pragma omp for
			for (k=0;k<num_blocks;k++)
			{
				i_s = thread_block_i_s[k];
				i_e = thread_block_i_e[k];
				for (i=i_s;i<i_e;i++)
				{
					i_rel = (i - i_s) % row_cluster_size;
					for (j=ia[i];j<ia[i+1];j++)
					{
						rel_row_idx[j] = i_rel;
					}
				}
			}
		}

		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&rel_row_idx_d, nnz * sizeof(*rel_row_idx_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_i_s_d, num_blocks * sizeof(*thread_block_i_s_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_i_e_d, num_blocks * sizeof(*thread_block_i_e_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_rel_row_idx));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_rel_row_idx));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_e));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		gpuCudaErrorCheck(hipHostMalloc(&ia_h, (m+1) * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc(&ja_h, nnz * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc(&a_h, nnz * sizeof(*a_h)));
		gpuCudaErrorCheck(hipHostMalloc(&rel_row_idx_h, nnz * sizeof(*rel_row_idx_h)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_s_h, num_blocks * sizeof(*thread_block_i_s_h)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_e_h, num_blocks * sizeof(*thread_block_i_e_h)));
		gpuCudaErrorCheck(hipHostMalloc(&x_h, n * sizeof(*x_h)));
		gpuCudaErrorCheck(hipHostMalloc(&y_h, m * sizeof(*y_h)));

		memcpy(ia_h, ia, (m+1) * sizeof(*ia_h));
		memcpy(ja_h, ja, nnz * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));
		memcpy(rel_row_idx_h, rel_row_idx, nnz * sizeof(*rel_row_idx_h));
		memcpy(thread_block_i_s_h, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_h));
		memcpy(thread_block_i_e_h, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_h));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_rel_row_idx));
		gpuCudaErrorCheck(hipMemcpyAsync(rel_row_idx_d, rel_row_idx_h, nnz * sizeof(*rel_row_idx_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_rel_row_idx));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_s));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_s_d, thread_block_i_s_h, num_blocks * sizeof(*thread_block_i_s_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_s));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_e));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_e_d, thread_block_i_e_h, num_blocks * sizeof(*thread_block_i_e_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_e));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_rel_row_idx));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_i_e));

			float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_rel_row_idx, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_rel_row_idx, startEvent_memcpy_rel_row_idx, endEvent_memcpy_rel_row_idx));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_s, startEvent_memcpy_thread_block_i_s, endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_e, startEvent_memcpy_thread_block_i_e, endEvent_memcpy_thread_block_i_e));
			printf("(CUDA) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, rel_row_idx time = %.4lf ms, thread_block_s = %.4lf ms, thread_block_e = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_rel_row_idx, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e);
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		free(rel_row_idx);
		free(thread_block_i_s);
		free(thread_block_i_e);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(rel_row_idx_d));
		gpuCudaErrorCheck(hipFree(thread_block_i_s_d));
		gpuCudaErrorCheck(hipFree(thread_block_i_e_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(rel_row_idx_h));
		gpuCudaErrorCheck(hipHostFree(thread_block_i_s_h));
		gpuCudaErrorCheck(hipHostFree(thread_block_i_e_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));

		gpuCudaErrorCheck(hipStreamDestroy(stream));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_rel_row_idx));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_rel_row_idx));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_e));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_BUFFER_t%d_rc_%d", csr->num_threads, csr->row_cluster_size);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


__global__ void gpu_kernel_csr_basic(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y, unsigned char * rel_row_idx, int row_cluster_size)
{
	extern __shared__ ValueType sdata[];
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	long i, i_s, i_e, i_rel, i_rel_e, j, j_s, j_e, k;
	ValueType sum;
	for (i=0;i<row_cluster_size;i++)
		sdata[i*block_size + tidb] = 0;
	__syncthreads();
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	// printf("%d,%d: bs=%d , bid=%d , %ld %ld\n", tidg, tidb, block_size, block_id, i_s, i_e);
	for (k=i_s;k<i_e;k+=row_cluster_size)
	{
		i_rel_e = k + row_cluster_size > i_e ? i_e - k : row_cluster_size;
		j_s = ia[k];
		j_e = ia[k+i_rel_e];
		for (j=j_s+tidb;j<j_e;j+=block_size)
		{
			i_rel = rel_row_idx[j];
			sdata[i_rel*block_size + tidb] += a[j] * x[ja[j]];
		}

		__syncthreads();

		for (i_rel=tidb;i_rel<i_rel_e;i_rel+=block_size)
		{
			sum = 0;
			for (j=0;j<block_size;j++)
			{
				sum += sdata[i_rel*block_size + j];
				sdata[i_rel*block_size + j] = 0;
			}
			y[k + i_rel] = sum;
		}

		// for (i_rel=0;i_rel<i_rel_e;i_rel++)
		// {
			// sum = 0;
			// for (j=1;j<block_size;j*=2)
			// {
				// if (tidb % (2*j) == 0)
				// {
					// sdata[i_rel*block_size + tidb] += sdata[i_rel*block_size + tidb + j];
					// sdata[i_rel*block_size + tidb + j] = 0;
				// }
				// __syncthreads();
			// }
			// if (tidb == 0)
			// {
				// y[k + i_rel] = sdata[i_rel*block_size];
				// sdata[i_rel*block_size] = 0;
			// }
		// }

		__syncthreads();
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);

	if (csr->x == NULL)
	{
		printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);
		csr->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x, csr->stream));
		memcpy(csr->x_h, x, csr->n * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x, csr->stream));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		#ifdef PERSISTENT_L2_PREFETCH
			int x_d_size = csr->n * sizeof(*csr->x);
			gpuCudaErrorCheck(cudaCtxResetPersistingL2Cache()); // This needs to happen every time before running kernel for 1st time for a matrix...
			if(x_d_size < csr->max_persistent_l2_cache){
				hipLaunchAttributeValue attribute;
				auto &window = attribute.accessPolicyWindow;
				window.base_ptr = csr->x_d;
				window.num_bytes = x_d_size;
				window.hitRatio = 1.0;
				window.hitProp = hipAccessPropertyPersisting;
				window.missProp = hipAccessPropertyStreaming;
				gpuCudaErrorCheck(hipStreamSetAttribute(csr->stream, hipLaunchAttributeAccessPolicyWindow, &attribute));
			}
		#endif
	}

	gpu_kernel_csr_basic<<<grid_dims, block_dims, (csr->row_cluster_size*block_size*sizeof(ValueType)), csr->stream>>>(csr->thread_block_i_s_d, csr->thread_block_i_e_d, csr->ia_d, csr->ja_d, csr->a_d, csr->x_d, csr->y_d, csr->rel_row_idx_d, csr->row_cluster_size);
	// gpu_kernel_csr_flat<<<grid_dims, block_dims>>>(thread_block_i_s_d, thread_block_i_e_d, csr->ia_d, csr->ja_d, csr->a_d, csr->x_d, csr->y_d);
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * sizeof(*csr->y_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * sizeof(ValueType));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

