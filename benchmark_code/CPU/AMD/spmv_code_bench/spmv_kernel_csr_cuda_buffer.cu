#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


INT_T * thread_block_i_s = NULL;
INT_T * thread_block_i_e = NULL;

INT_T * thread_i_s_dev = NULL;
INT_T * thread_i_e_dev = NULL;


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;


struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;

	unsigned char * rel_row_idx;
	unsigned char * rel_row_idx_dev;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x;
	int num_threads;
	int block_size;
	int num_blocks;

	// int row_cluster_size = 256;
	// int row_cluster_size = 192;
	// int row_cluster_size = 128;
	// int row_cluster_size = 64;
	// int row_cluster_size = 32;
	// int row_cluster_size = 16;
	// int row_cluster_size = 8;
	int row_cluster_size = 4;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		double time_balance;
		long i;

		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0);
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);

		// block_size = warp_size / 2;
		block_size = warp_size;
		// block_size = warp_size * 2;

		// num_threads = 128;
		// num_threads = 1ULL << 10;
		// num_threads = 3584;
		// num_threads = 1ULL << 12;
		// num_threads = 1ULL << 13;
		// num_threads = 1ULL << 14;
		// num_threads = 1ULL << 15;
		// num_threads = 1ULL << 16;
		// num_threads = 1ULL << 17;
		num_threads = 1ULL << 21;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		time_balance = time_it(1,
			for (i=0;i<num_blocks;i++)
			{
				// loop_partitioner_balance_iterations(num_blocks, i, 0, m, &thread_block_i_s[i], &thread_block_i_e[i]);
				loop_partitioner_balance_prefix_sums(num_blocks, i, ia, m, nnz, &thread_block_i_s[i], &thread_block_i_e[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		rel_row_idx = (typeof(rel_row_idx)) malloc(nnz * sizeof(*rel_row_idx));
		#pragma omp parallel
		{
			long i, i_s, i_e, i_rel, j, k;
			#pragma omp for
			for (k=0;k<num_blocks;k++)
			{
				i_s = thread_block_i_s[k];
				i_e = thread_block_i_e[k];
				for (i=i_s;i<i_e;i++)
				{
					i_rel = (i - i_s) % row_cluster_size;
					for (j=ia[i];j<ia[i+1];j++)
					{
						rel_row_idx[j] = i_rel;
					}
				}
			}
		}


		hipMalloc(&ia_dev, (m+1) * sizeof(*ia_dev));
		hipMemcpy(ia_dev, ia, (m+1) * sizeof(*ia_dev), hipMemcpyHostToDevice);

		hipMalloc(&ja_dev, nnz * sizeof(*ja_dev));
		hipMemcpy(ja_dev, ja, nnz * sizeof(*ja_dev), hipMemcpyHostToDevice);

		hipMalloc(&a_dev, nnz * sizeof(*a_dev));
		hipMemcpy(a_dev, a, nnz * sizeof(*a_dev), hipMemcpyHostToDevice);

		hipMalloc(&rel_row_idx_dev, nnz * sizeof(*rel_row_idx_dev));
		hipMemcpy(rel_row_idx_dev, rel_row_idx, nnz * sizeof(*rel_row_idx_dev), hipMemcpyHostToDevice);

		hipMalloc(&x_dev, n * sizeof(*x_dev));

		hipMalloc(&y_dev, m * sizeof(*y_dev));

		hipMalloc(&thread_i_s_dev, num_blocks * sizeof(*thread_i_s_dev));
		hipMemcpy(thread_i_s_dev, thread_block_i_s, num_blocks * sizeof(*thread_i_s_dev), hipMemcpyHostToDevice);

		hipMalloc(&thread_i_e_dev, num_blocks * sizeof(*thread_i_e_dev));
		hipMemcpy(thread_i_e_dev, thread_block_i_e, num_blocks * sizeof(*thread_i_e_dev), hipMemcpyHostToDevice);

	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);
		free(rel_row_idx);

		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Custom_CSR_CUDA";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


// __device__ int add(int a, int b)
// {
	// return a + b;
// }


__global__ void gpu_kernel_csr_basic(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y, unsigned char * rel_row_idx, int row_cluster_size)
{
	extern __shared__ ValueType sdata[];
	int tgid = cuda_get_thread_num();
	int tbid = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	long i, i_s, i_e, i_rel, i_rel_e, j, j_s, j_e, k;
	ValueType sum;
	for (i=0;i<row_cluster_size;i++)
		sdata[i*block_size + tbid] = 0;
	__syncthreads();
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	// printf("%d,%d: bs=%d , bid=%d , %ld %ld\n", tgid, tbid, block_size, block_id, i_s, i_e);
	for (k=i_s;k<i_e;k+=row_cluster_size)
	{
		i_rel_e = k + row_cluster_size > i_e ? i_e - k : row_cluster_size;
		j_s = ia[k];
		j_e = ia[k+i_rel_e];
		for (j=j_s+tbid;j<j_e;j+=block_size)
		{
			i_rel = rel_row_idx[j];
			sdata[i_rel*block_size + tbid] += a[j] * x[ja[j]];
		}

		__syncthreads();

		for (i_rel=tbid;i_rel<i_rel_e;i_rel+=block_size)
		{
			sum = 0;
			for (j=0;j<block_size;j++)
			{
				sum += sdata[i_rel*block_size + j];
				sdata[i_rel*block_size + j] = 0;
			}
			y[k + i_rel] = sum;
		}

		// for (i_rel=0;i_rel<i_rel_e;i_rel++)
		// {
			// sum = 0;
			// for (j=1;j<block_size;j*=2)
			// {
				// if (tbid % (2*j) == 0)
				// {
					// sdata[i_rel*block_size + tbid] += sdata[i_rel*block_size + tbid + j];
					// sdata[i_rel*block_size + tbid + j] = 0;
				// }
				// __syncthreads();
			// }
			// if (tbid == 0)
			// {
				// y[k + i_rel] = sdata[i_rel*block_size];
				// sdata[i_rel*block_size] = 0;
			// }
		// }

		__syncthreads();
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);
	// long shared_mem_size = block_size * sizeof(*C_dev);

	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
	}

	gpu_kernel_csr_basic<<<grid_dims, block_dims, (csr->row_cluster_size*block_size*sizeof(ValueType))>>>(thread_i_s_dev, thread_i_e_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev, csr->rel_row_idx_dev, csr->row_cluster_size);
	// gpu_kernel_csr_flat<<<grid_dims, block_dims>>>(thread_i_s_dev, thread_i_e_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);

	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));
	err = hipGetLastError();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));

	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}

	// exit(0);
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

