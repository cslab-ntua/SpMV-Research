#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;
	
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, block_size, max_threads_per_multiproc;
	// int num_streams;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		block_size = BLOCK_SIZE;

		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpy(ia_d, ia, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpy(ja_d, ja, nnz * sizeof(*ja_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpy(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));

			float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			printf("(CUDA) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a);
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_VECTOR_b%d", csr->block_size);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================

__global__ void gpu_kernel_csr_vector(INT_T * ia, INT_T * ja, ValueType * a, INT_T m, int block_size, int warp_size, ValueType * restrict x, ValueType * restrict y)
{
	// Thread ID in block
	INT_T t = threadIdx.x;

	// Thread ID in warp
	INT_T lane = t & (warp_size-1);

	// Number of warps per block
	INT_T warpsPerBlock = blockDim.x / warp_size;

	// One row per warp
	INT_T row = (blockIdx.x * warpsPerBlock) + (t / warp_size);

	__shared__ volatile ValueType LDS[BLOCK_SIZE];

	if (row < m){
		INT_T rowStart = ia[row];
		INT_T rowEnd = ia[row+1];
		ValueType sum = 0;

		// Use all threads in a warp accumulate multiplied elements
		for (INT_T j = rowStart + lane; j < rowEnd; j += warp_size){
			INT_T col = ja[j];
			sum += a[j] * x[col];
		}
		LDS[t] = sum;
		__syncthreads();
	
		// Reduce partial sums
		if (lane < 16) LDS[t] += LDS[t + 16];
		if (lane <  8) LDS[t] += LDS[t + 8];
		if (lane <  4) LDS[t] += LDS[t + 4];
		if (lane <  2) LDS[t] += LDS[t + 2];
		if (lane <  1) LDS[t] += LDS[t + 1];
		__syncthreads();
		
		// Write result
		if (lane == 0){
			y[row] = LDS[t];
		}
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	dim3 block_dims(csr->block_size);
	dim3 grid_dims(ceil(csr->m/((float)csr->block_size/csr->warp_size)));
	// printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);

	if (csr->x == NULL)
	{
		csr->x = x;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(csr->x_d, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}
	}

	gpu_kernel_csr_vector<<<grid_dims, block_dims>>>(csr->ia_d, csr->ja_d, csr->a_d, csr->m, csr->block_size, csr->warp_size, csr->x_d, csr->y_d);
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y));
		gpuCudaErrorCheck(hipMemcpy(csr->y, csr->y_d, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

