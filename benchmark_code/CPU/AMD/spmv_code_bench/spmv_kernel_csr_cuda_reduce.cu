#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


INT_T * thread_block_i_s = NULL;
INT_T * thread_block_i_e = NULL;

INT_T * thread_block_j_s = NULL;
INT_T * thread_block_j_e = NULL;


INT_T * thread_block_i_s_dev = NULL;
INT_T * thread_block_i_e_dev = NULL;

INT_T * thread_block_j_s_dev = NULL;
INT_T * thread_block_j_e_dev = NULL;


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

void
cuda_push_duplicate_base(void ** dst_ptr, void * src, long bytes)
{
	hipMalloc(dst_ptr, bytes);
	hipMemcpy(*((char **) dst_ptr), src, bytes, hipMemcpyHostToDevice);
}
#define cuda_push_duplicate(dst_ptr, src, bytes) cuda_push_duplicate_base((void **) dst_ptr, src, bytes)


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;
	INT_T * ia;
	INT_T * ja;
	ValueType * a;

	INT_T * row_ptr_dev;
	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x;
	int num_threads;
	int block_size;
	int num_blocks;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		double time_balance;
		long i;

		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0);
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);

		// block_size = 32;
		// block_size = 64;
		block_size = 128;
		// block_size = 256;
		// block_size = 512;

		// num_threads = 128;
		// num_threads = 1ULL << 10;
		// num_threads = 1ULL << 12;
		// num_threads = 1ULL << 13;
		// num_threads = 1ULL << 14;
		// num_threads = 1ULL << 15;
		// num_threads = 1ULL << 16;
		// num_threads = 1ULL << 17;
		num_threads = 1ULL << 21;
		// num_threads = 1ULL << 24;
		// num_threads = nnz / 32;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		thread_block_j_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_s));
		thread_block_j_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_e));
		time_balance = time_it(1,
			for (i=0;i<num_blocks;i++)
			{
				// loop_partitioner_balance_iterations(num_blocks, i, 0, m, &thread_block_i_s[i], &thread_block_i_e[i]);
				loop_partitioner_balance_prefix_sums(num_blocks, i, row_ptr, m, nnz, &thread_block_i_s[i], &thread_block_i_e[i]);
				thread_block_j_s[i] = row_ptr[thread_block_i_s[i]];
				thread_block_j_e[i] = row_ptr[thread_block_i_e[i]];
			}
		);
		printf("balance time = %g\n", time_balance);

		ia = (typeof(ia)) malloc(nnz * sizeof(*ia));
		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (i=0;i<m;i++)
			{
				for (j=row_ptr[i];j<row_ptr[i+1];j++)
				{
					ia[j] = i;
				}
			}
		}

		cuda_push_duplicate(&row_ptr_dev, row_ptr, (m+1) * sizeof(*row_ptr_dev));
		cuda_push_duplicate(&ia_dev, ia, nnz * sizeof(*ia_dev));
		cuda_push_duplicate(&ja_dev, ja, nnz * sizeof(*ja_dev));
		cuda_push_duplicate(&a_dev, a, nnz * sizeof(*a_dev));

		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));

		cuda_push_duplicate(&thread_block_i_s_dev, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_dev));
		cuda_push_duplicate(&thread_block_i_e_dev, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_dev));
		cuda_push_duplicate(&thread_block_j_s_dev, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_dev));
		cuda_push_duplicate(&thread_block_j_e_dev, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_dev));

	}

	~CSRArrays()
	{
		free(a);
		free(row_ptr);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);

		hipFree(row_ptr_dev);
		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Custom_CSR_CUDA_reduce";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


// __device__ int add(int a, int b)
// {
	// return a + b;
// }


__global__ void gpu_kernel_spmv(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	extern __shared__ ValueType val_buf[];
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	INT_T * ia_rel;
	INT_T * ja_rel;
	ValueType * a_rel;
	int i, i_s, i_e, j, j_s, j_e, k;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	int j_e_div = j_e - ((j_e-j_s) % block_size);
	for (j=j_s;j<j_e_div;j+=block_size)
	{
		ia_rel = &ia[j];
		ja_rel = &ja[j];
		a_rel = &a[j];
		val_buf[tidb] = a_rel[tidb] * x[ja_rel[tidb]];
		__syncthreads();
		for (i=1;i<block_size;i*=2)
		{
			if ((tidb & (2*i-1)) == i-1)
			{
				if (ia_rel[tidb] == ia_rel[tidb+i])
					val_buf[tidb+i] += val_buf[tidb];
				else
					y[ia_rel[tidb]] += val_buf[tidb];
			}
			__syncthreads();
		}
		if (tidb == 0)
			y[ia_rel[block_size-1]] += val_buf[block_size-1];
		__syncthreads();
	}
	if (tidb == 0)
	{
		for (j=j_e_div;j<j_e;j++)
		{
			y[ia[j]] += a[j] * x[ja[j]];
		}
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);
	long shared_mem_size = block_size * (sizeof(ValueType));

	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
	}

	hipMemset(csr->y_dev, 0, csr->m * sizeof(csr->y_dev));

	gpu_kernel_spmv<<<grid_dims, block_dims, shared_mem_size>>>(thread_block_i_s_dev, thread_block_i_e_dev, thread_block_j_s_dev, thread_block_j_e_dev, csr->row_ptr_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);

	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("hipDeviceSynchronize: %s\n", hipGetErrorString(err));
	err = hipGetLastError();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));

	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}

	// exit(0);
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

