#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cublas_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 1
#endif

#ifndef TIME_IT
#define TIME_IT 1
#endif

#ifndef VERIFIED
#define VERIFIED 1
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	INT_T * ia_h[NUM_STREAMS];
	INT_T * ja_h[NUM_STREAMS];
	ValueType * a_h[NUM_STREAMS];

	INT_T * ia_d[NUM_STREAMS];
	INT_T * ja_d[NUM_STREAMS];
	ValueType * a_d[NUM_STREAMS];

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h[NUM_STREAMS];
	ValueType * y_h[NUM_STREAMS];
	ValueType * x_d[NUM_STREAMS];
	// ValueType * y_d[NUM_STREAMS];
	ValueType * y_d2;
	ValueType * y_d_reduction;

	hipStream_t stream[NUM_STREAMS];
	INT_T n_stream[NUM_STREAMS];
	INT_T nnz_stream[NUM_STREAMS];

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution[NUM_STREAMS];
	hipEvent_t endEvent_execution[NUM_STREAMS];
	
	hipEvent_t startEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_a[NUM_STREAMS];

	hipEvent_t startEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipblasHandle_t handle;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, block_size, max_threads_per_multiproc;
	int num_streams;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		block_size = BLOCK_SIZE;
		num_streams = NUM_STREAMS;

		/********************************************************************************************************/
		printf("/********************************************************************************************************/\n");
		// Convert CSR representation ton CSC
		INT_T * row_indices; //for CSC format
		INT_T * row_idx;
		INT_T * col_ptr;
		ValueType * val_c;

		row_indices = (typeof(row_indices)) malloc(nnz * sizeof(*row_indices));
		row_idx = (typeof(row_idx)) malloc(nnz * sizeof(*row_idx));
		col_ptr = (typeof(col_ptr)) malloc((n+1) * sizeof(*col_ptr));
		val_c = (typeof(val_c)) malloc(nnz * sizeof(*val_c));

		double time = time_it(1,
			csr_row_indices(ia, ja, m, n, nnz, &row_indices);
			coo_to_csc(row_indices, ja, a, m, n, nnz, row_idx, col_ptr, val_c, 1);
			free(row_indices);
		);
		printf("time coo_to_csc = %g ms\n", time*1e3);

		INT_T *local_stream_j_s = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_s));
		INT_T *local_stream_j_e = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_e));
		double time_balance = time_it(1,
			for (int i=0;i<num_streams;i++)
				loop_partitioner_balance_prefix_sums(num_streams, i, col_ptr, n, nnz, &local_stream_j_s[i], &local_stream_j_e[i]);
		);

		int cnt=0, cnt2=0;
		for(int i=0; i<num_streams; i++){
			nnz_stream[i] = col_ptr[local_stream_j_e[i]] - col_ptr[local_stream_j_s[i]];
			n_stream[i] = local_stream_j_e[i] - local_stream_j_s[i];
			// printf("local_stream[%d] = %d - %d (%d cols) (%d nnz)\n", i, local_stream_j_s[i], local_stream_j_e[i], n_stream[i], nnz_stream[i]);

			cnt  += nnz_stream[i];
			cnt2 += n_stream[i];
		}

		printf("balance time (col) = %g ms\n", time_balance*1e3);

		INT_T * row_idx_stream[num_streams];
		INT_T * col_ptr_stream[num_streams];
		ValueType * val_c_stream[num_streams];
		
		double time_memcpy_stream_locals = time_it(1,
		for(int i=0; i<num_streams; i++){
			col_ptr_stream[i] = (INT_T *) malloc((n_stream[i]+1) * sizeof(INT_T));
			row_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_c_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			memcpy(col_ptr_stream[i], col_ptr + local_stream_j_s[i], (n_stream[i] + 1) * sizeof(INT_T));
			// col_ptr needs to be fixed, so that it will start from 0 again...
			for(int j=0; j<n_stream[i]+1; j++)
				col_ptr_stream[i][j] -= col_ptr[local_stream_j_s[i]];
			memcpy(row_idx_stream[i], row_idx + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(INT_T));
			memcpy(val_c_stream[i], val_c + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy_stream_locals = %lf ms\n", time_memcpy_stream_locals*1e3);
		free(local_stream_j_s);
		free(local_stream_j_e);

		INT_T * row_ptr_stream[num_streams];
		INT_T * col_idx_stream[num_streams];
		ValueType * val_stream[num_streams];

		for(int i=0; i<num_streams; i++){
			INT_T * col_indices;
			csc_col_indices(row_idx_stream[i], col_ptr_stream[i], m, n_stream[i], nnz_stream[i], &col_indices);

			row_ptr_stream[i] = (INT_T *) malloc((m+1) * sizeof(INT_T));
			col_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			coo_to_csr(row_idx_stream[i], col_indices, val_c_stream[i], m, n_stream[i], nnz_stream[i], row_ptr_stream[i], col_idx_stream[i], val_stream[i], 1, 0);
			free(col_indices);
		}

		for(int i=0; i<num_streams; i++){
			free(row_idx_stream[i]);
			free(col_ptr_stream[i]);
			free(val_c_stream[i]);
		}
		free(row_idx);
		free(col_ptr);
		free(val_c);

		printf("/********************************************************************************************************/\n");
		/********************************************************************************************************/

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipMalloc(&ia_d[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ja_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&a_d[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&x_d[i], n_stream[i] * sizeof(ValueType)));
			// gpuCudaErrorCheck(hipMalloc(&y_d[i], m * sizeof(ValueType)));
		}
		gpuCudaErrorCheck(hipMalloc(&y_d2, m * num_streams * sizeof(ValueType)));
		gpuCudaErrorCheck(hipMalloc(&y_d_reduction, m * sizeof(ValueType)));
		gpuCublasErrorCheck(hipblasCreate(&handle));

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipHostMalloc(&ia_h[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ja_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&a_h[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&x_h[i], n_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&y_h[i], m * sizeof(ValueType)));
		}

		double time_memcpy = time_it(1,
		for(int i=0; i<num_streams; i++){
			memcpy(ia_h[i], row_ptr_stream[i], (m + 1) * sizeof(INT_T));
			memcpy(ja_h[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(a_h[i], val_stream[i], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy (ia_h, ja_h, a_h) = %lf ms\n", time_memcpy*1e3);

		// cuda events for timing measurements
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipStreamCreate(&stream[i]));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_execution[i]));
		}
		gpuCublasErrorCheck(hipblasSetStream(handle, stream[0]));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a[i]));

				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		for(int i=0; i<num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ia_d[i], row_ptr_stream[i], (m+1) * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ja_d[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(a_d[i], val_stream[i], nnz_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a[i], stream[i]));
		}

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(stream[i]));
				float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia[i], endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja[i], endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a[i], endEvent_memcpy_a[i]));
				printf("(CUDA) (stream %d) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms\n", i, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a);
			}
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipFree(ia_d[i]));
			gpuCudaErrorCheck(hipFree(ja_d[i]));
			gpuCudaErrorCheck(hipFree(a_d[i]));
			gpuCudaErrorCheck(hipFree(x_d[i]));
			// gpuCudaErrorCheck(hipFree(y_d[i]));

			gpuCudaErrorCheck(hipHostFree(ia_h[i]));
			gpuCudaErrorCheck(hipHostFree(ja_h[i]));
			gpuCudaErrorCheck(hipHostFree(a_h[i]));
			gpuCudaErrorCheck(hipHostFree(x_h[i]));
			gpuCudaErrorCheck(hipHostFree(y_h[i]));

			gpuCudaErrorCheck(hipStreamDestroy(stream[i]));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_execution[i]));
		}
		gpuCudaErrorCheck(hipFree(y_d2));
		gpuCudaErrorCheck(hipFree(y_d_reduction));
		gpuCublasErrorCheck(hipblasDestroy(handle));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a[i]));
			}
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_VECTOR_b%d_s%d", csr->block_size, csr->num_streams);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================

__global__ void gpu_kernel_csr_vector(INT_T * ia, INT_T * ja, ValueType * a, INT_T m, int block_size, int warp_size, ValueType * restrict x, ValueType * restrict y)
{
	// Thread ID in block
	INT_T t = threadIdx.x;

	// Thread ID in warp
	INT_T lane = t & (warp_size-1);

	// Number of warps per block
	INT_T warpsPerBlock = blockDim.x / warp_size;

	// One row per warp
	INT_T row = (blockIdx.x * warpsPerBlock) + (t / warp_size);

	__shared__ volatile ValueType LDS[BLOCK_SIZE];

	if (row < m){
		INT_T rowStart = ia[row];
		INT_T rowEnd = ia[row+1];
		ValueType sum = 0;

		// Use all threads in a warp accumulate multiplied elements
		for (INT_T j = rowStart + lane; j < rowEnd; j += warp_size){
			INT_T col = ja[j];
			sum += a[j] * x[col];
		}
		LDS[t] = sum;
		__syncthreads();
	
		// Reduce partial sums
		if (lane < 16) LDS[t] += LDS[t + 16];
		if (lane <  8) LDS[t] += LDS[t + 8];
		if (lane <  4) LDS[t] += LDS[t + 4];
		if (lane <  2) LDS[t] += LDS[t + 2];
		if (lane <  1) LDS[t] += LDS[t + 1];
		__syncthreads();
	
		// Write result
		if (lane == 0){
			y[row] = LDS[t];
		}
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	dim3 block_dims(csr->block_size);
	dim3 grid_dims(ceil(csr->m/((float)csr->block_size/csr->warp_size)));
	printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);

	if (csr->x == NULL)
	{
		csr->x = x;
		int offset = 0;
		for(int i=0; i<csr->num_streams; i++){
			memcpy(csr->x_h[i], x + offset, csr->n_stream[i] * sizeof(ValueType));
			offset += csr->n_stream[i];
		}

		for(int i=0; i<csr->num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x[i], csr->stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d[i], csr->x_h[i], csr->n_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x[i], csr->stream[i]));
		}

		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

		if(TIME_IT){
			for(int i=0; i<csr->num_streams; i++){
				float memcpyTime_cuda;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x[i], csr->endEvent_memcpy_x[i]));
				printf("(CUDA) (stream %d) Memcpy x time = %.4lf ms\n", i, memcpyTime_cuda);
			}
		}
	}

	for(int i=0; i<csr->num_streams; i++)
		gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution[i], csr->stream[i]));

	int num_loops = 128;
	double time_execution = time_it(1,
		for(int k=0;k<num_loops;k++){
			for(int i=0; i<csr->num_streams; i++){
				// gpu_kernel_csr_vector<<<grid_dims, block_dims, 0, csr->stream[i]>>>(csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->m, csr->block_size, csr->warp_size, csr->x_d[i], csr->y_d[i]);
				gpu_kernel_csr_vector<<<grid_dims, block_dims, 0, csr->stream[i]>>>(csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->m, csr->block_size, csr->warp_size, csr->x_d[i], csr->y_d2 + i*csr->m);
			}
			gpuCudaErrorCheck(hipPeekAtLastError());
			gpuCudaErrorCheck(hipDeviceSynchronize());
		}
	);

	double gflops = csr->nnz / time_execution * num_loops * 2 * 1e-9;
	printf("(DGAL timing) Execution time = %.4lf ms (%.4lf GFLOPS @ %d block_size and %d streams for %.2lf MB workload)\n", time_execution*1e3, gflops, csr->block_size, csr->num_streams, csr->mem_footprint/(1024*1024.0));

	for(int i=0; i<csr->num_streams; i++)
		gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution[i], csr->stream[i]));

	for(int i=0; i<csr->num_streams; i++)
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

	if(TIME_IT){
		for(int i=0; i<csr->num_streams; i++){
			float executionTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&executionTime_cuda, csr->startEvent_execution[i], csr->endEvent_execution[i]));

			double gflops_cuda = csr->nnz_stream[i] / executionTime_cuda * num_loops * 2 * 1e-6;
			double mem_footprint = (csr->nnz_stream[i] * (sizeof(ValueType) + sizeof(INT_T)) + (csr->m+1) * sizeof(INT_T))/(1024*1024.0);
			printf("(CUDA) (stream %d) Execution time = %.4lf ms (%.4lf GFLOPS for %.2lf MB workload)\n", i, executionTime_cuda, gflops_cuda, mem_footprint);
		}
	}

	if (csr->y == NULL)
	{
		csr->y = y;

		// for(int i=0; i<csr->num_streams; i++){
		// 	if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y[i], csr->stream[i]));
		// 	gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h[i], csr->y_d[i], csr->m * sizeof(ValueType), hipMemcpyDeviceToHost, csr->stream[i]));
		// 	if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y[i], csr->stream[i]));
		// }

		// if(TIME_IT){
		// 	for(int i=0; i<csr->num_streams; i++){
		// 		gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y[i]));
		// 		float memcpyTime_cuda;
		// 		gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y[i], csr->endEvent_memcpy_y[i]));
		// 		printf("(CUDA) (stream %d) Memcpy y time = %.4lf ms\n", i, memcpyTime_cuda);
		// 	}
		// }

		// // for(int i=0; i<csr->num_streams; i++)
		// // 	memcpy(y, csr->y_h[i], csr->m * sizeof(ValueType));
		// double time_y_reduction = time_it(1,
		// 	for(int i=0; i<csr->m; i++){
		// 		csr->y_h_final[i] = 0;
		// 		for(int k=0; k<csr->num_streams; k++)
		// 			csr->y_h_final[i] += csr->y_h[k][i];
		// 	}
		// 	memcpy(y, csr->y_h_final, csr->m * sizeof(ValueType));
		// );
		// printf("time_y_reduction = %.4lf ms\n", time_y_reduction);

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream[0]));

		ValueType *ones_host, *ones_device;

		gpuCudaErrorCheck(hipHostMalloc(&ones_host, csr->num_streams * sizeof(ValueType)));
		for (int i=0; i<csr->num_streams; i++) ones_host[i] = 1.0;
		gpuCudaErrorCheck(hipMalloc(&ones_device, csr->num_streams * sizeof(ValueType)));	
		gpuCudaErrorCheck(hipMemcpyAsync(ones_device, ones_host, csr->num_streams * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[0]));
	
		ValueType  alpha = 1.0, beta = 0.0;
		gpuCublasErrorCheck(hipblasDgemv(csr->handle, HIPBLAS_OP_N, csr->m, csr->num_streams, &alpha, csr->y_d2, csr->m, ones_device, 1, &beta, csr->y_d_reduction, 1));

		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y, csr->y_d_reduction, csr->m * sizeof(csr->y), hipMemcpyDeviceToHost, csr->stream[0]));

		gpuCudaErrorCheck(hipHostFree(ones_host));
		gpuCudaErrorCheck(hipFree(ones_device));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream[0]));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[0]));
		if(TIME_IT){
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

