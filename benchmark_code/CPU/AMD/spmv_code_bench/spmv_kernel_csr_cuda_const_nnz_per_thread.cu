#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


using namespace cooperative_groups;

#ifndef NNZ_PER_THREAD
#define NNZ_PER_THREAD  6
#endif

#ifndef BLOCK_SIZE
// #define BLOCK_SIZE  32
// #define BLOCK_SIZE  64
// #define BLOCK_SIZE  128
// #define BLOCK_SIZE  256
// #define BLOCK_SIZE  512
#define BLOCK_SIZE  1024
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

void
cuda_push_duplicate_base(void ** dst_ptr, void * src, long bytes)
{
	hipMalloc(dst_ptr, bytes);
	hipMemcpy(*((char **) dst_ptr), src, bytes, hipMemcpyHostToDevice);
}
#define cuda_push_duplicate(dst_ptr, src, bytes) cuda_push_duplicate_base((void **) dst_ptr, src, bytes)


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;
	INT_T * ia;
	INT_T * ja;
	ValueType * a;
	INT_T * thread_block_i_s = NULL;
	INT_T * thread_block_i_e = NULL;
	INT_T * thread_block_j_s = NULL;
	INT_T * thread_block_j_e = NULL;

	INT_T * row_ptr_d;
	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;
	INT_T * thread_block_i_s_d = NULL;
	INT_T * thread_block_i_e_d = NULL;
	INT_T * thread_block_j_s_d = NULL;
	INT_T * thread_block_j_e_d = NULL;

	INT_T * row_ptr_h;
	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;
	INT_T * thread_block_i_s_h = NULL;
	INT_T * thread_block_i_e_h = NULL;
	INT_T * thread_block_j_s_h = NULL;
	INT_T * thread_block_j_e_h = NULL;

	// ValueType * multres_d;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	hipStream_t stream;
	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_row_ptr;
	hipEvent_t endEvent_memcpy_row_ptr;
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;
	hipEvent_t startEvent_memcpy_thread_block_i_s;
	hipEvent_t endEvent_memcpy_thread_block_i_s;
	hipEvent_t startEvent_memcpy_thread_block_i_e;
	hipEvent_t endEvent_memcpy_thread_block_i_e;
	hipEvent_t startEvent_memcpy_thread_block_j_s;
	hipEvent_t endEvent_memcpy_thread_block_j_s;
	hipEvent_t startEvent_memcpy_thread_block_j_e;
	hipEvent_t endEvent_memcpy_thread_block_j_e;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x, max_persistent_l2_cache, max_num_threads;
	int num_threads;
	int block_size;
	int num_blocks;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		double time_balance;
		long i;

		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_persistent_l2_cache, cudaDevAttrMaxPersistingL2CacheSize, 0));
		max_num_threads = max_threads_per_multiproc * multiproc_count;
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_persistent_l2_cache=%d\n", max_persistent_l2_cache);
		printf("max_num_threads=%d\n", max_num_threads);

		block_size = BLOCK_SIZE;

		num_threads = (nnz + NNZ_PER_THREAD - 1) / NNZ_PER_THREAD;

		num_threads = ((num_threads + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;

		num_blocks = num_threads / BLOCK_SIZE;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, BLOCK_SIZE, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		thread_block_j_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_s));
		thread_block_j_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_e));
		time_balance = time_it(1,
			long lower_boundary;
			// for (i=0;i<num_blocks;i++)
			// {
				// loop_partitioner_balance_iterations(num_blocks, i, 0, nnz, &thread_block_j_s[i], &thread_block_j_e[i]);
				// macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				// thread_block_i_s[i] = lower_boundary;
			// }
			long nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
			for (i=0;i<num_blocks;i++)
			{
				thread_block_j_s[i] = nnz_per_block * i;
				thread_block_j_e[i] = nnz_per_block * (i+ 1);
				if (thread_block_j_s[i] > nnz)
					thread_block_j_s[i] = nnz;
				if (thread_block_j_e[i] > nnz)
					thread_block_j_e[i] = nnz;
				macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				thread_block_i_s[i] = lower_boundary;
			}
			for (i=0;i<num_blocks;i++)
			{
				if (i == num_blocks - 1)   // If we calculate each thread's boundaries individually some empty rows might be unassigned.
					thread_block_i_e[i] = m;
				else
					thread_block_i_e[i] = thread_block_i_s[i+1] + 1;
				if ((thread_block_j_s[i] >= row_ptr[thread_block_i_e[i]]) || (thread_block_j_s[i] < row_ptr[thread_block_i_s[i]]))
					error("bad binary search of row start: i=%d j:[%d, %d] j=%d", thread_block_i_s[i], row_ptr[thread_block_i_s[i]], row_ptr[thread_block_i_e[i]], thread_block_j_s[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		ia = (typeof(ia)) malloc(nnz * sizeof(*ia));
		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (i=0;i<m;i++)
			{
				for (j=row_ptr[i];j<row_ptr[i+1];j++)
				{
					ia[j] = i;
				}
			}
		}

		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (j=0;j<nnz;j+=32*NNZ_PER_THREAD)
			{
				long j_e = j + 32*NNZ_PER_THREAD;
				if (j_e > nnz)
					j_e = nnz;
				if (ia[j] == ia[j_e-1])
				{
					for (i=j;i<j_e;i++)
					{
						ja[i] = ja[i] | 0x80000000;
					}
				}
			}
		}

		// cuda_push_duplicate(&row_ptr_d, row_ptr, (m+1) * sizeof(*row_ptr_d));
		// cuda_push_duplicate(&ia_d, ia, nnz * sizeof(*ia_d));
		// cuda_push_duplicate(&ja_d, ja, nnz * sizeof(*ja_d));
		// cuda_push_duplicate(&a_d, a, nnz * sizeof(*a_d));
		// hipMalloc(&multres_d, nnz * sizeof(*y_d));

		// cuda_push_duplicate(&thread_block_i_s_d, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_d));
		// cuda_push_duplicate(&thread_block_i_e_d, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_d));
		// cuda_push_duplicate(&thread_block_j_s_d, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_d));
		// cuda_push_duplicate(&thread_block_j_e_d, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_d));

		gpuCudaErrorCheck(hipMalloc(&row_ptr_d, (m+1) * sizeof(*row_ptr_d)));
		gpuCudaErrorCheck(hipMalloc(&ia_d, nnz * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_i_s_d, num_blocks * sizeof(*thread_block_i_s_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_i_e_d, num_blocks * sizeof(*thread_block_i_e_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_j_s_d, num_blocks * sizeof(*thread_block_j_s_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_j_e_d, num_blocks * sizeof(*thread_block_j_e_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_j_e));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_j_e));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		gpuCudaErrorCheck(hipHostMalloc(&row_ptr_h, (m+1) * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&ia_h, nnz * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&ja_h, nnz * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&a_h, nnz * sizeof(ValueType)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_s_h, num_blocks * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_e_h, num_blocks * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_block_j_s_h, num_blocks * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_block_j_e_h, num_blocks * sizeof(INT_T)));
		gpuCudaErrorCheck(hipHostMalloc(&x_h, n * sizeof(ValueType)));
		gpuCudaErrorCheck(hipHostMalloc(&y_h, m * sizeof(ValueType)));

		memcpy(row_ptr_h, row_ptr, (m + 1) * sizeof(INT_T));
		memcpy(ia_h, ia, nnz * sizeof(INT_T));
		memcpy(ja_h, ja, nnz * sizeof(INT_T));
		memcpy(a_h, a, nnz * sizeof(ValueType));
		memcpy(thread_block_i_s_h, thread_block_i_s, num_blocks * sizeof(INT_T));
		memcpy(thread_block_i_e_h, thread_block_i_e, num_blocks * sizeof(INT_T));
		memcpy(thread_block_j_s_h, thread_block_j_s, num_blocks * sizeof(INT_T));
		memcpy(thread_block_j_e_h, thread_block_j_e, num_blocks * sizeof(INT_T));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_row_ptr, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(row_ptr_d, row_ptr_h, (m+1) * sizeof(*row_ptr_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_row_ptr, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, nnz * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_s, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_s_d, thread_block_i_s_h, num_blocks * sizeof(*thread_block_i_s_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_s, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_e, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_e_d, thread_block_i_e_h, num_blocks * sizeof(*thread_block_i_e_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_e, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_j_s, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_block_j_s_d, thread_block_j_s_h, num_blocks * sizeof(*thread_block_j_s_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_j_s, stream));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_j_e, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_block_j_e_d, thread_block_j_e_h, num_blocks * sizeof(*thread_block_j_e_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_j_e, stream));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_j_e));

			float memcpyTime_cuda_row_ptr, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e, memcpyTime_cuda_thread_block_j_s, memcpyTime_cuda_thread_block_j_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_row_ptr, startEvent_memcpy_row_ptr, endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_s, startEvent_memcpy_thread_block_i_s, endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_e, startEvent_memcpy_thread_block_i_e, endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_j_s, startEvent_memcpy_thread_block_j_s, endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_j_e, startEvent_memcpy_thread_block_j_e, endEvent_memcpy_thread_block_j_e));
			printf("(CUDA) Memcpy row_ptr time = %.4lf ms, ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, thread_block_i_s time = %.4lf, thread_block_i_e time = %.4lf, thread_block_j_s time = %.4lf, thread_block_j_e time = %.4lf\n", memcpyTime_cuda_row_ptr, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e, memcpyTime_cuda_thread_block_j_s, memcpyTime_cuda_thread_block_j_e);
		}
		gpuCudaErrorCheck(hipStreamSynchronize(stream));

		_Pragma("omp parallel")
		{
			long j;
			_Pragma("omp for")
			for (j=0;j<nnz;j++)
			{
				ja[j] = ja[j] & 0x7FFFFFFF;
			}
		}
	}

	~CSRArrays()
	{
		free(a);
		free(row_ptr);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);
		free(thread_block_j_s);
		free(thread_block_j_e);

		gpuCudaErrorCheck(hipFree(row_ptr_d));
		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		// gpuCudaErrorCheck(hipFree(multres_d));
		gpuCudaErrorCheck(hipFree(thread_block_i_s_d));
		gpuCudaErrorCheck(hipFree(thread_block_i_e_d));
		gpuCudaErrorCheck(hipFree(thread_block_j_s_d));
		gpuCudaErrorCheck(hipFree(thread_block_j_e_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipHostFree(row_ptr_h));
		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		// gpuCudaErrorCheck(hipHostFree(multres_h));
		gpuCudaErrorCheck(hipHostFree(thread_block_i_s_h));
		gpuCudaErrorCheck(hipHostFree(thread_block_i_e_h));
		gpuCudaErrorCheck(hipHostFree(thread_block_j_s_h));
		gpuCudaErrorCheck(hipHostFree(thread_block_j_e_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));

		gpuCudaErrorCheck(hipStreamDestroy(stream));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_j_e));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_j_e));
		}
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_constant_nnz_per_thread_b%d_nnz%d", BLOCK_SIZE, NNZ_PER_THREAD);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	int row = ia_buf[tidb];
	int k;
	for (k=1;k<BLOCK_SIZE;k*=2)
	{
		if ((tidb & (2*k-1)) == k-1)
		{
			ValueType val = val_buf[tidb];
			if (row == ia_buf[tidb+k])
			{
				val_buf[tidb+k] += val;
				// val_buf[tidb] = 0;
			}
			else
			{
				atomicAdd(&y[row], val);
				// y[row] += val;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[BLOCK_SIZE-1]], val_buf[BLOCK_SIZE-1]);
} */


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	int k;
	INT_T row = ia_buf[tidb];
	for (k=1;k<BLOCK_SIZE;k*=2)
	{
		if ((tidb & (2*k-1)) == 0)
		{
			INT_T row_next = ia_buf[tidb+k];
			ValueType val_next = val_buf[tidb+k];
			if (row == row_next)
			{
				val_buf[tidb] += val_next;
			}
			else
			{
				atomicAdd(&y[row], val_buf[tidb]);
				val_buf[tidb] = val_next;
				ia_buf[tidb] = row_next;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[0]], val_buf[0]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int row = ia_buf[tidl];
	ValueType val;
	int k;
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		// val = val_buf[tidl];
		// if ((tidl & (2*k-1)) == k-1)
		// {
			// if (tidl >= k && row == ia_buf[tidl-k])
			// {
				// val_buf[tidl-k] += val;
				// val = 0;
			// }
		// }
		// g.sync();
		// if ((tidl & (2*k-1)) == k-1 && val != 0)
		// {
			// if (row == ia_buf[tidl+k])
			// {
				// val_buf[tidl+k] += val;
			// }
			// else
			// {
				// atomicAdd(&y[row], val);
			// }
		// }
		// g.sync();
		val = val_buf[tidl];
		if ((tidl & (2*k-1)) == k-1)
		{
			if (row == ia_buf[tidl+k])
			{
				val_buf[tidl+k] += val;
			}
			else
			{
				atomicAdd(&y[row], val);
			}
		}
		g.sync();
	}
}
inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &ia_buf[tidb_div*32], &val_buf[tidb_div*32], y);
	// __syncthreads();
	// if (tidb_mod == 31)
	// {
		// ia_buf[tidb_mod] = ia_buf[tidb];
		// val_buf[tidb_mod] = val_buf[tidb];
	// }
	// __syncthreads();
	// if (tidb_div == 0)
		// reduce_warp(tile32, ia_buf, val_buf, y);
	// if (tidb == 0)
		// atomicAdd(&y[ia_buf[31]], val_buf[31]);
	if (tidb_mod == 31)
		atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * row_ptr, ValueType * val_ptr, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	INT_T row = *row_ptr;
	ValueType val = *val_ptr;
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_next;
		ValueType val_next;
		row_next = __shfl_sync(0xffffffff, row, tidl+k);
		val_next = __shfl_sync(0xffffffff, val, tidl+k);
		if ((tidl & (2*k-1)) == 0)
		{
			if (row == row_next)
			{
				val += val_next;
			}
			else
			{
				atomicAdd(&y[row], val);
				val = val_next;
				row = row_next;
			}
		}
		g.sync();
	}
	*row_ptr = row;
	*val_ptr = val;
	// if (tidl == 0)
		// atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &row, &val, y);
	if (tidb_mod == 0)
		atomicAdd(&y[row], val);
	// extern __shared__ char sm[];
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[32 * sizeof(ValueType)];
	// if (tidb_mod == 0)
	// {
		// ia_buf[tidb_div] = row;
		// val_buf[tidb_div] = val;
	// }
	// __syncthreads();
	// if (tidb_div == 0)
	// {
		// row = ia_buf[tidb];
		// val = val_buf[tidb];
		// reduce_warp(tile32, &row, &val, y);
	// }
	// if (tidb == 0)
		// atomicAdd(&y[row], val);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_prev;
		ValueType val_prev;
		row_prev = __shfl_sync(0xffffffff, row, tidl-k);
		val_prev = __shfl_sync(0xffffffff, val, tidl-k);
		if ((tidl & (2*k-1)) == 2*k-1)
		{
			if (row == row_prev)
			{
				val += val_prev;
			}
			else
			{
				atomicAdd(&y[row_prev], val_prev);
			}
		}
		g.sync();
	}
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
} */


// Threads may only read data from another thread which is actively participating in the __shfl_sync() command.
// If the target thread is inactive, the retrieved value is undefined.
template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	const int tidl_one_hot = 1 << tidl;
	int flag;
	INT_T row_prev;
	ValueType val_prev;
	flag = 0xaaaaaaaa; // 10101010101010101010101010101010
	row_prev = __shfl_sync(flag, row, tidl-1);
	val_prev = __shfl_sync(flag, val, tidl-1);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x88888888; // 10001000100010001000100010001000
	row_prev = __shfl_sync(flag, row, tidl-2);
	val_prev = __shfl_sync(flag, val, tidl-2);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80808080; // 10000000100000001000000010000000
	row_prev = __shfl_sync(flag, row, tidl-4);
	val_prev = __shfl_sync(flag, val, tidl-4);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80008000; // 10000000000000001000000000000000
	row_prev = __shfl_sync(flag, row, tidl-8);
	val_prev = __shfl_sync(flag, val, tidl-8);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80000000; // 10000000000000000000000000000000
	row_prev = __shfl_sync(flag, row, tidl-16);
	val_prev = __shfl_sync(flag, val, tidl-16);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	g.sync();
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
}


__device__ void spmv_last_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[BLOCK_SIZE * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = block_id * nnz_per_block + tidb * NNZ_PER_THREAD;
	j_e = j_s + NNZ_PER_THREAD;
	if (j_e > nnz)
		j_e = nnz;
	k = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_s >= row_ptr[k])
		{
			i_s = k + 1;
		}
		else
		{
			i_e = k;
		}
		k = (i_e + i_s) / 2;
	}
	i = i_s - 1;
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_s;j<j_e;j++)
	{
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
		}
		// sum += a[j] * x[ja[j] & 0x7FFFFFFF];
		sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
	}
	reduce_block(i, sum, y);
}


template <typename group_t>
__device__ ValueType reduce_warp_single_line(group_t g, ValueType val, ValueType * restrict y)
{
	// Use XOR mode to perform butterfly reduction
	for (int i=g.size()/2; i>=1; i/=2)
	{
		val += __shfl_xor_sync(0xffffffff, val, i, g.size());   // 'sum' is same on all threads
		// val += __shfl_down_sync(0xffffffff, val, i, g.size());   // Only thread 0 has the total sum.
	}
	return val;
}


template <typename group_t>
__device__ void spmv_warp_single_row(group_t g, int i, int j_s, int j_e, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int j;
	double sum = 0;
	for (j=j_s;j<j_e;j++)
	{
		sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
	}
	sum = reduce_warp_single_line(g, sum, y);
	if (tidl == 0)
		atomicAdd(&y[i], sum);
}


template <typename group_t>
__device__ void spmv_full_warp(group_t g, int one_line, int i_s, int j_s, int j_e, INT_T * row_ptr, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	[[gnu::unused]] int i, j, k, l, p;
	int ptr_next;
	i = i_s;
	ptr_next = row_ptr[i_s+1];
	for (j=j_s;j<j_e;j++)   // Find the row of the last nnz.
	{
		if (j >= ptr_next)
		{
			i++;
			break;
		}
	}
	double sum = 0;
	// int i_w_s, i_w_e;
	// i_w_s = __shfl_sync(0xffffffff, i_s, 0);
	// i_w_e = __shfl_sync(0xffffffff, i, 31);
	i = i_s;
	// if (i_w_e != i_w_s)
	if (one_line)
	{
		spmv_warp_single_row(g, i_s, j_s, j_e, ja, a, x, y);
	}
	else
	{
		ptr_next = row_ptr[i+1];
		k = 0;
		for (j=j_s;j<j_e;j++)
		{
			if (j >= ptr_next)
			{
				atomicAdd(&y[i], sum);
				sum = 0;
				while (j >= ptr_next)
				{
					i++;
					ptr_next = row_ptr[i+1];
				}
				k++;
			}
			// sum += a[j] * x[ja[j] & 0x7FFFFFFF];
			sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
		}
		reduce_warp(g, i, sum, y);
	}
}


__device__ void spmv_full_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	// const int tidb = threadIdx.x;
	const int tidw = threadIdx.x % 32;
	const int warp_id = threadIdx.x / 32;
	const int block_id = blockIdx.x;
	const int nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[BLOCK_SIZE * sizeof(ValueType)];
	[[gnu::unused]] int i_s, i_e, j, j_s, j_e, j_w_s, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	// i_s = 0;
	// i_e = m;
	j_w_s = block_id * nnz_per_block + warp_id * NNZ_PER_THREAD * 32;
	j_s = j_w_s + tidw * NNZ_PER_THREAD;
	j_e = j_s + NNZ_PER_THREAD;
	k = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_s >= row_ptr[k])
		{
			i_s = k + 1;
		}
		else
		{
			i_e = k;
		}
		k = (i_e + i_s) / 2;
	}
	i_s--;
	int one_line = (ja[j_s] & 0x80000000) ? 1 : 0;
	// int one_line = 0;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	spmv_full_warp(tile32, one_line, i_s, j_s, j_e, row_ptr, ja, a, x, y);
}


__global__ void gpu_kernel_spmv_row_indices_continuous(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	int grid_size = gridDim.x;
	int block_id = blockIdx.x;
	if (block_id == grid_size - 1)
		spmv_last_block(thread_block_i_s, thread_block_i_e, thread_block_j_s, thread_block_j_e, row_ptr, ia, ja, a, m, n, nnz, x, y);
	else
		spmv_full_block(thread_block_i_s, thread_block_i_e, row_ptr, ia, ja, a, m, n, nnz, x, y);
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(BLOCK_SIZE);
	dim3 grid_dims(num_blocks);
	// long shared_mem_size = BLOCK_SIZE * (sizeof(ValueType));
	// long shared_mem_size = BLOCK_SIZE * (sizeof(ValueType) + sizeof(INT_T));
	long shared_mem_size = 0;

	if (csr->x == NULL)
	{
		printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);
		csr->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x, csr->stream));
		memcpy(csr->x_h, x, csr->n * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice, csr->stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		#ifdef PERSISTENT_L2_PREFETCH
			int x_d_size = csr->n * sizeof(*csr->x);
			gpuCudaErrorCheck(cudaCtxResetPersistingL2Cache()); // This needs to happen every time before running kernel for 1st time for a matrix...
			if(x_d_size < csr->max_persistent_l2_cache){
				hipLaunchAttributeValue attribute;
				auto &window = attribute.accessPolicyWindow;
				window.base_ptr = csr->x_d;
				window.num_bytes = x_d_size;
				window.hitRatio = 1.0;
				window.hitProp = hipAccessPropertyPersisting;
				window.missProp = hipAccessPropertyStreaming;
				gpuCudaErrorCheck(hipStreamSetAttribute(csr->stream, hipLaunchAttributeAccessPolicyWindow, &attribute));
			}
		#endif
	}

	hipMemsetAsync(csr->y_d, 0, csr->m * sizeof(csr->y_d), csr->stream);

	// hipFuncCachePreferNone:   no preference for shared memory or L1 (default);
	// hipFuncCachePreferShared: prefer larger shared memory and smaller L1 cache;
	// hipFuncCachePreferL1:     prefer larger L1 cache and smaller shared memory;
	gpuCudaErrorCheck(hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_kernel_spmv_row_indices_continuous), hipFuncCachePreferL1));
	gpu_kernel_spmv_row_indices_continuous<<<grid_dims, block_dims, shared_mem_size, csr->stream>>>(csr->thread_block_i_s_d, csr->thread_block_i_e_d, csr->thread_block_j_s_d, csr->thread_block_j_e_d, csr->row_ptr_d, csr->ia_d, csr->ja_d, csr->a_d, csr->m, csr->n, csr->nnz, csr->x_d, csr->y_d);
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());
	// gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * sizeof(ValueType));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

