#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


using namespace cooperative_groups;


#define NNZ_PER_THREAD  6

// #define BLOCK_SIZE  32
// #define BLOCK_SIZE  64
// #define BLOCK_SIZE  128
// #define BLOCK_SIZE  256
// #define BLOCK_SIZE  512
#define BLOCK_SIZE  1024


INT_T * thread_block_i_s = NULL;
INT_T * thread_block_i_e = NULL;

INT_T * thread_block_j_s = NULL;
INT_T * thread_block_j_e = NULL;


INT_T * thread_block_i_s_dev = NULL;
INT_T * thread_block_i_e_dev = NULL;

INT_T * thread_block_j_s_dev = NULL;
INT_T * thread_block_j_e_dev = NULL;


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

void
cuda_push_duplicate_base(void ** dst_ptr, void * src, long bytes)
{
	hipMalloc(dst_ptr, bytes);
	hipMemcpy(*((char **) dst_ptr), src, bytes, hipMemcpyHostToDevice);
}
#define cuda_push_duplicate(dst_ptr, src, bytes) cuda_push_duplicate_base((void **) dst_ptr, src, bytes)


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;
	INT_T * ia;
	INT_T * ja;
	ValueType * a;

	INT_T * row_ptr_dev;
	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	ValueType * multres_dev;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x, max_num_threads;
	int num_threads;
	int block_size;
	int num_blocks;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		double time_balance;
		long i;

		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0);
		max_num_threads = max_threads_per_multiproc * multiproc_count;
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_num_threads=%d\n", max_num_threads);

		block_size = BLOCK_SIZE;

		num_threads = (nnz + NNZ_PER_THREAD - 1) / NNZ_PER_THREAD;

		num_threads = ((num_threads + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;

		num_blocks = num_threads / BLOCK_SIZE;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, BLOCK_SIZE, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		thread_block_j_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_s));
		thread_block_j_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_e));
		time_balance = time_it(1,
			long lower_boundary;
			// for (i=0;i<num_blocks;i++)
			// {
				// loop_partitioner_balance_iterations(num_blocks, i, 0, nnz, &thread_block_j_s[i], &thread_block_j_e[i]);
				// macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				// thread_block_i_s[i] = lower_boundary;
			// }
			long nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
			for (i=0;i<num_blocks;i++)
			{
				thread_block_j_s[i] = nnz_per_block * i;
				thread_block_j_e[i] = nnz_per_block * (i+ 1);
				if (thread_block_j_s[i] > nnz)
					thread_block_j_s[i] = nnz;
				if (thread_block_j_e[i] > nnz)
					thread_block_j_e[i] = nnz;
				macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				thread_block_i_s[i] = lower_boundary;
			}
			for (i=0;i<num_blocks;i++)
			{
				if (i == num_blocks - 1)   // If we calculate each thread's boundaries individually some empty rows might be unassigned.
					thread_block_i_e[i] = m;
				else
					thread_block_i_e[i] = thread_block_i_s[i+1] + 1;
				if ((thread_block_j_s[i] >= row_ptr[thread_block_i_e[i]]) || (thread_block_j_s[i] < row_ptr[thread_block_i_s[i]]))
					error("bad binary search of row start: i=%d j:[%d, %d] j=%d", thread_block_i_s[i], row_ptr[thread_block_i_s[i]], row_ptr[thread_block_i_e[i]], thread_block_j_s[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		ia = (typeof(ia)) malloc(nnz * sizeof(*ia));
		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (i=0;i<m;i++)
			{
				for (j=row_ptr[i];j<row_ptr[i+1];j++)
				{
					ia[j] = i;
				}
			}
		}

		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (j=0;j<nnz;j+=32*NNZ_PER_THREAD)
			{
				long j_e = j + 32*NNZ_PER_THREAD;
				if (j_e > nnz)
					j_e = nnz;
				if (ia[j] == ia[j_e-1])
				{
					for (i=j;i<j_e;i++)
					{
						ja[i] = ja[i] | 0x80000000;
					}
				}
			}
		}

		cuda_push_duplicate(&row_ptr_dev, row_ptr, (m+1) * sizeof(*row_ptr_dev));
		cuda_push_duplicate(&ia_dev, ia, nnz * sizeof(*ia_dev));
		cuda_push_duplicate(&ja_dev, ja, nnz * sizeof(*ja_dev));
		cuda_push_duplicate(&a_dev, a, nnz * sizeof(*a_dev));
		hipMalloc(&multres_dev, nnz * sizeof(*y_dev));

		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));

		cuda_push_duplicate(&thread_block_i_s_dev, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_dev));
		cuda_push_duplicate(&thread_block_i_e_dev, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_dev));
		cuda_push_duplicate(&thread_block_j_s_dev, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_dev));
		cuda_push_duplicate(&thread_block_j_e_dev, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_dev));

		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (j=0;j<nnz;j++)
			{
				ja[j] = ja[j] & 0x7FFFFFFF;
			}
		}

	}

	~CSRArrays()
	{
		free(a);
		free(row_ptr);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);

		hipFree(row_ptr_dev);
		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Custom_CSR_CUDA_reduce";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	int row = ia_buf[tidb];
	int k;
	for (k=1;k<BLOCK_SIZE;k*=2)
	{
		if ((tidb & (2*k-1)) == k-1)
		{
			ValueType val = val_buf[tidb];
			if (row == ia_buf[tidb+k])
			{
				val_buf[tidb+k] += val;
				// val_buf[tidb] = 0;
			}
			else
			{
				atomicAdd(&y[row], val);
				// y[row] += val;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[BLOCK_SIZE-1]], val_buf[BLOCK_SIZE-1]);
} */


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	int k;
	INT_T row = ia_buf[tidb];
	for (k=1;k<BLOCK_SIZE;k*=2)
	{
		if ((tidb & (2*k-1)) == 0)
		{
			INT_T row_next = ia_buf[tidb+k];
			ValueType val_next = val_buf[tidb+k];
			if (row == row_next)
			{
				val_buf[tidb] += val_next;
			}
			else
			{
				atomicAdd(&y[row], val_buf[tidb]);
				val_buf[tidb] = val_next;
				ia_buf[tidb] = row_next;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[0]], val_buf[0]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int row = ia_buf[tidl];
	ValueType val;
	int k;
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		// val = val_buf[tidl];
		// if ((tidl & (2*k-1)) == k-1)
		// {
			// if (tidl >= k && row == ia_buf[tidl-k])
			// {
				// val_buf[tidl-k] += val;
				// val = 0;
			// }
		// }
		// g.sync();
		// if ((tidl & (2*k-1)) == k-1 && val != 0)
		// {
			// if (row == ia_buf[tidl+k])
			// {
				// val_buf[tidl+k] += val;
			// }
			// else
			// {
				// atomicAdd(&y[row], val);
			// }
		// }
		// g.sync();
		val = val_buf[tidl];
		if ((tidl & (2*k-1)) == k-1)
		{
			if (row == ia_buf[tidl+k])
			{
				val_buf[tidl+k] += val;
			}
			else
			{
				atomicAdd(&y[row], val);
			}
		}
		g.sync();
	}
}
inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &ia_buf[tidb_div*32], &val_buf[tidb_div*32], y);
	// __syncthreads();
	// if (tidb_mod == 31)
	// {
		// ia_buf[tidb_mod] = ia_buf[tidb];
		// val_buf[tidb_mod] = val_buf[tidb];
	// }
	// __syncthreads();
	// if (tidb_div == 0)
		// reduce_warp(tile32, ia_buf, val_buf, y);
	// if (tidb == 0)
		// atomicAdd(&y[ia_buf[31]], val_buf[31]);
	if (tidb_mod == 31)
		atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * row_ptr, ValueType * val_ptr, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	INT_T row = *row_ptr;
	ValueType val = *val_ptr;
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_next;
		ValueType val_next;
		row_next = __shfl_sync(0xffffffff, row, tidl+k);
		val_next = __shfl_sync(0xffffffff, val, tidl+k);
		if ((tidl & (2*k-1)) == 0)
		{
			if (row == row_next)
			{
				val += val_next;
			}
			else
			{
				atomicAdd(&y[row], val);
				val = val_next;
				row = row_next;
			}
		}
		g.sync();
	}
	*row_ptr = row;
	*val_ptr = val;
	// if (tidl == 0)
		// atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &row, &val, y);
	if (tidb_mod == 0)
		atomicAdd(&y[row], val);
	// extern __shared__ char sm[];
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[32 * sizeof(ValueType)];
	// if (tidb_mod == 0)
	// {
		// ia_buf[tidb_div] = row;
		// val_buf[tidb_div] = val;
	// }
	// __syncthreads();
	// if (tidb_div == 0)
	// {
		// row = ia_buf[tidb];
		// val = val_buf[tidb];
		// reduce_warp(tile32, &row, &val, y);
	// }
	// if (tidb == 0)
		// atomicAdd(&y[row], val);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_prev;
		ValueType val_prev;
		row_prev = __shfl_sync(0xffffffff, row, tidl-k);
		val_prev = __shfl_sync(0xffffffff, val, tidl-k);
		if ((tidl & (2*k-1)) == 2*k-1)
		{
			if (row == row_prev)
			{
				val += val_prev;
			}
			else
			{
				atomicAdd(&y[row_prev], val_prev);
			}
		}
		g.sync();
	}
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
} */


// Threads may only read data from another thread which is actively participating in the __shfl_sync() command.
// If the target thread is inactive, the retrieved value is undefined.
template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	const int tidl_one_hot = 1 << tidl;
	int flag;
	INT_T row_prev;
	ValueType val_prev;
	flag = 0xaaaaaaaa;
	row_prev = __shfl_sync(flag, row, tidl-1);
	val_prev = __shfl_sync(flag, val, tidl-1);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x88888888;
	row_prev = __shfl_sync(flag, row, tidl-2);
	val_prev = __shfl_sync(flag, val, tidl-2);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80808080;
	row_prev = __shfl_sync(flag, row, tidl-4);
	val_prev = __shfl_sync(flag, val, tidl-4);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80008000;
	row_prev = __shfl_sync(flag, row, tidl-8);
	val_prev = __shfl_sync(flag, val, tidl-8);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80000000;
	row_prev = __shfl_sync(flag, row, tidl-16);
	val_prev = __shfl_sync(flag, val, tidl-16);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	g.sync();
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
}


__device__ void spmv_last_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[BLOCK_SIZE * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = block_id * nnz_per_block + tidb * NNZ_PER_THREAD;
	j_e = j_s + NNZ_PER_THREAD;
	if (j_e > nnz)
		j_e = nnz;
	k = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_s >= row_ptr[k])
		{
			i_s = k + 1;
		}
		else
		{
			i_e = k;
		}
		k = (i_e + i_s) / 2;
	}
	i = i_s - 1;
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_s;j<j_e;j++)
	{
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
		}
		// sum += a[j] * x[ja[j] & 0x7FFFFFFF];
		sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
	}
	reduce_block(i, sum, y);
}


template <typename group_t>
__device__ ValueType reduce_warp_single_line(group_t g, ValueType val, ValueType * restrict y) {
	// Use XOR mode to perform butterfly reduction
	for (int i=g.size()/2; i>=1; i/=2)
	{
		val += __shfl_xor_sync(0xffffffff, val, i, g.size());   // 'sum' is same on all threads
		// val += __shfl_down_sync(0xffffffff, val, i, g.size());   // Only thread 0 has the total sum.
	}
	return val;
}


template <typename group_t>
__device__ void spmv_warp_single_row(group_t g, int i, int j_s, int j_e, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int j;
	double sum = 0;
	for (j=j_s;j<j_e;j++)
	{
		sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
	}
	sum = reduce_warp_single_line(g, sum, y);
	if (tidl == 0)
		atomicAdd(&y[i], sum);
}


template <typename group_t>
__device__ void spmv_full_warp(group_t g, int one_line, int i_s, int j_s, int j_e, INT_T * row_ptr, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	[[gnu::unused]] int i, j, k, l, p;
	int ptr_next;
	i = i_s;
	ptr_next = row_ptr[i_s+1];
	for (j=j_s;j<j_e;j++)   // Find the row of the last nnz.
	{
		if (j >= ptr_next)
		{
			i++;
			break;
		}
	}
	double sum = 0;
	// int i_w_s, i_w_e;
	// i_w_s = __shfl_sync(0xffffffff, i_s, 0);
	// i_w_e = __shfl_sync(0xffffffff, i, 31);
	i = i_s;
	// if (i_w_e != i_w_s)
	if (one_line)
	{
		spmv_warp_single_row(g, i_s, j_s, j_e, ja, a, x, y);
	}
	else
	{
		ptr_next = row_ptr[i+1];
		k = 0;
		for (j=j_s;j<j_e;j++)
		{
			if (j >= ptr_next)
			{
				atomicAdd(&y[i], sum);
				sum = 0;
				while (j >= ptr_next)
				{
					i++;
					ptr_next = row_ptr[i+1];
				}
				k++;
			}
			// sum += a[j] * x[ja[j] & 0x7FFFFFFF];
			sum = __fma_rn(a[j], x[ja[j] & 0x7FFFFFFF], sum);
		}
		reduce_warp(g, i, sum, y);
	}
}


__device__ void spmv_full_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int tidw = threadIdx.x % 32;
	const int warp_id = threadIdx.x / 32;
	const int block_id = blockIdx.x;
	const int nnz_per_block = BLOCK_SIZE * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[BLOCK_SIZE * sizeof(ValueType)];
	[[gnu::unused]] int i_s, i_e, j, j_s, j_e, j_w_s, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	// i_s = 0;
	// i_e = m;
	j_w_s = block_id * nnz_per_block + warp_id * NNZ_PER_THREAD * 32;
	j_s = j_w_s + tidw * NNZ_PER_THREAD;
	j_e = j_s + NNZ_PER_THREAD;
	k = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_s >= row_ptr[k])
		{
			i_s = k + 1;
		}
		else
		{
			i_e = k;
		}
		k = (i_e + i_s) / 2;
	}
	i_s--;
	int one_line = (ja[j_s] & 0x80000000) ? 1 : 0;
	// int one_line = 0;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	spmv_full_warp(tile32, one_line, i_s, j_s, j_e, row_ptr, ja, a, x, y);
}


__global__ void gpu_kernel_spmv_row_indices_continuous(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz, ValueType * restrict x, ValueType * restrict y)
{
	int grid_size = gridDim.x;
	int block_id = blockIdx.x;
	if (block_id == grid_size - 1)
		spmv_last_block(thread_block_i_s, thread_block_i_e, thread_block_j_s, thread_block_j_e, row_ptr, ia, ja, a, m, n, nnz, x, y);
	else
		spmv_full_block(thread_block_i_s, thread_block_i_e, row_ptr, ia, ja, a, m, n, nnz, x, y);
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(BLOCK_SIZE);
	dim3 grid_dims(num_blocks);
	// long shared_mem_size = BLOCK_SIZE * (sizeof(ValueType));
	// long shared_mem_size = BLOCK_SIZE * (sizeof(ValueType) + sizeof(INT_T));
	long shared_mem_size = 0;
	hipError_t err;

	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
	}

	hipMemset(csr->y_dev, 0, csr->m * sizeof(csr->y_dev));

	// hipFuncCachePreferNone:   no preference for shared memory or L1 (default);
	// hipFuncCachePreferShared: prefer larger shared memory and smaller L1 cache;
	// hipFuncCachePreferL1:     prefer larger L1 cache and smaller shared memory;
	err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_kernel_spmv_row_indices_continuous), hipFuncCachePreferL1);
	if (err != hipSuccess)
		error("hipFuncSetCacheConfig: %s\n", hipGetErrorString(err));

	gpu_kernel_spmv_row_indices_continuous<<<grid_dims, block_dims, shared_mem_size>>>(thread_block_i_s_dev, thread_block_i_e_dev, thread_block_j_s_dev, thread_block_j_e_dev, csr->row_ptr_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->m, csr->n, csr->nnz, csr->x_dev, csr->y_dev);

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("hipDeviceSynchronize: %s\n", hipGetErrorString(err));
	err = hipGetLastError();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));

	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}

	// exit(0);
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

