#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


using namespace cooperative_groups;

#ifndef NNZ_PER_THREAD
#define NNZ_PER_THREAD 4
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

void
cuda_push_duplicate_base(void ** dst_ptr, void * src, long bytes)
{
	hipMalloc(dst_ptr, bytes);
	hipMemcpy(*((char **) dst_ptr), src, bytes, hipMemcpyHostToDevice);
}
#define cuda_push_duplicate(dst_ptr, src, bytes) cuda_push_duplicate_base((void **) dst_ptr, src, bytes)


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;
	INT_T * ia;
	INT_T * ja;
	ValueType * a;
	INT_T * thread_block_i_s = NULL;
	INT_T * thread_block_i_e = NULL;
	INT_T * thread_block_j_s = NULL;
	INT_T * thread_block_j_e = NULL;

	INT_T * row_ptr_d;
	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;
	INT_T * thread_block_i_s_d = NULL;
	INT_T * thread_block_i_e_d = NULL;
	INT_T * thread_block_j_s_d = NULL;
	INT_T * thread_block_j_e_d = NULL;

	// ValueType * multres_d;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_row_ptr;
	hipEvent_t endEvent_memcpy_row_ptr;
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;
	hipEvent_t startEvent_memcpy_thread_block_i_s;
	hipEvent_t endEvent_memcpy_thread_block_i_s;
	hipEvent_t startEvent_memcpy_thread_block_i_e;
	hipEvent_t endEvent_memcpy_thread_block_i_e;
	hipEvent_t startEvent_memcpy_thread_block_j_s;
	hipEvent_t endEvent_memcpy_thread_block_j_s;
	hipEvent_t startEvent_memcpy_thread_block_j_e;
	hipEvent_t endEvent_memcpy_thread_block_j_e;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x, max_num_threads;
	int nnz_per_thread;
	int num_threads;
	int block_size;
	int num_blocks;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		double time_balance;
		long i;

		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0));
		max_num_threads = max_threads_per_multiproc * multiproc_count;
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_num_threads=%d\n", max_num_threads);

		// block_size = 32;
		// block_size = 64;
		// block_size = 128;
		// block_size = 256;
		// block_size = 512;
		block_size = 1024;

		nnz_per_thread = NNZ_PER_THREAD;

		num_threads = (nnz + NNZ_PER_THREAD - 1) / NNZ_PER_THREAD;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		thread_block_j_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_s));
		thread_block_j_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_e));
		time_balance = time_it(1,
			long lower_boundary;
			// for (i=0;i<num_blocks;i++)
			// {
				// loop_partitioner_balance_iterations(num_blocks, i, 0, nnz, &thread_block_j_s[i], &thread_block_j_e[i]);
				// macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				// thread_block_i_s[i] = lower_boundary;
			// }
			long nnz_per_block = block_size * NNZ_PER_THREAD;
			for (i=0;i<num_blocks;i++)
			{
				thread_block_j_s[i] = nnz_per_block * i;
				thread_block_j_e[i] = nnz_per_block * (i+ 1);
				if (thread_block_j_s[i] > nnz)
					thread_block_j_s[i] = nnz;
				if (thread_block_j_e[i] > nnz)
					thread_block_j_e[i] = nnz;
				macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				thread_block_i_s[i] = lower_boundary;
			}
			for (i=0;i<num_blocks;i++)
			{
				if (i == num_blocks - 1)   // If we calculate each thread's boundaries individually some empty rows might be unassigned.
					thread_block_i_e[i] = m;
				else
					thread_block_i_e[i] = thread_block_i_s[i+1] + 1;
				if ((thread_block_j_s[i] >= row_ptr[thread_block_i_e[i]]) || (thread_block_j_s[i] < row_ptr[thread_block_i_s[i]]))
					error("bad binary search of row start: i=%d j:[%d, %d] j=%d", thread_block_i_s[i], row_ptr[thread_block_i_s[i]], row_ptr[thread_block_i_e[i]], thread_block_j_s[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		ia = (typeof(ia)) malloc(nnz * sizeof(*ia));
		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (i=0;i<m;i++)
			{
				for (j=row_ptr[i];j<row_ptr[i+1];j++)
				{
					ia[j] = i;
				}
			}
		}

		// cuda_push_duplicate(&row_ptr_d, row_ptr, (m+1) * sizeof(*row_ptr_d));
		// cuda_push_duplicate(&ia_d, ia, nnz * sizeof(*ia_d));
		// cuda_push_duplicate(&ja_d, ja, nnz * sizeof(*ja_d));
		// cuda_push_duplicate(&a_d, a, nnz * sizeof(*a_d));
		// hipMalloc(&multres_d, nnz * sizeof(*y_d));

		// cuda_push_duplicate(&thread_block_i_s_d, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_d));
		// cuda_push_duplicate(&thread_block_i_e_d, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_d));
		// cuda_push_duplicate(&thread_block_j_s_d, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_d));
		// cuda_push_duplicate(&thread_block_j_e_d, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_d));

		gpuCudaErrorCheck(hipMalloc(&row_ptr_d, (m+1) * sizeof(*row_ptr_d)));
		gpuCudaErrorCheck(hipMalloc(&ia_d, nnz * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_i_s_d, num_blocks * sizeof(*thread_block_i_s_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_i_e_d, num_blocks * sizeof(*thread_block_i_e_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_j_s_d, num_blocks * sizeof(*thread_block_j_s_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_block_j_e_d, num_blocks * sizeof(*thread_block_j_e_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_j_e));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_j_e));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_row_ptr));
		gpuCudaErrorCheck(hipMemcpy(row_ptr_d, row_ptr, (m+1) * sizeof(*row_ptr_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_row_ptr));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpy(ia_d, ia, nnz * sizeof(*ia_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpy(ja_d, ja, nnz * sizeof(*ja_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpy(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_s));
		gpuCudaErrorCheck(hipMemcpy(thread_block_i_s_d, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_s));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_e));
		gpuCudaErrorCheck(hipMemcpy(thread_block_i_e_d, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_e));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_j_s));
		gpuCudaErrorCheck(hipMemcpy(thread_block_j_s_d, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_j_s));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_j_e));
		gpuCudaErrorCheck(hipMemcpy(thread_block_j_e_d, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_j_e));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_block_j_e));

			float memcpyTime_cuda_row_ptr, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e, memcpyTime_cuda_thread_block_j_s, memcpyTime_cuda_thread_block_j_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_row_ptr, startEvent_memcpy_row_ptr, endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_s, startEvent_memcpy_thread_block_i_s, endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_e, startEvent_memcpy_thread_block_i_e, endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_j_s, startEvent_memcpy_thread_block_j_s, endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_j_e, startEvent_memcpy_thread_block_j_e, endEvent_memcpy_thread_block_j_e));
			printf("(CUDA) Memcpy row_ptr time = %.4lf ms, ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, thread_block_i_s time = %.4lf, thread_block_i_e time = %.4lf, thread_block_j_s time = %.4lf, thread_block_j_e time = %.4lf\n", memcpyTime_cuda_row_ptr, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e, memcpyTime_cuda_thread_block_j_s, memcpyTime_cuda_thread_block_j_e);
		}
	}

	~CSRArrays()
	{
		free(a);
		free(row_ptr);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);
		free(thread_block_j_s);
		free(thread_block_j_e);

		gpuCudaErrorCheck(hipFree(row_ptr_d));
		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		// gpuCudaErrorCheck(hipFree(multres_d));
		gpuCudaErrorCheck(hipFree(thread_block_i_s_d));
		gpuCudaErrorCheck(hipFree(thread_block_i_e_d));
		gpuCudaErrorCheck(hipFree(thread_block_j_s_d));
		gpuCudaErrorCheck(hipFree(thread_block_j_e_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_row_ptr));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_s));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_e));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_j_s));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_j_e));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_j_e));
		}
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_constant_nnz_per_thread_nnz%d", csr->nnz_per_thread);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int block_size = blockDim.x;
	int row = ia_buf[tidb];
	int k;
	for (k=1;k<block_size;k*=2)
	{
		if ((tidb & (2*k-1)) == k-1)
		{
			ValueType val = val_buf[tidb];
			if (row == ia_buf[tidb+k])
			{
				val_buf[tidb+k] += val;
				// val_buf[tidb] = 0;
			}
			else
			{
				atomicAdd(&y[row], val);
				// y[row] += val;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[block_size-1]], val_buf[block_size-1]);
} */


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int block_size = blockDim.x;
	int k;
	INT_T row = ia_buf[tidb];
	for (k=1;k<block_size;k*=2)
	{
		if ((tidb & (2*k-1)) == 0)
		{
			INT_T row_next = ia_buf[tidb+k];
			ValueType val_next = val_buf[tidb+k];
			if (row == row_next)
			{
				val_buf[tidb] += val_next;
			}
			else
			{
				atomicAdd(&y[row], val_buf[tidb]);
				val_buf[tidb] = val_next;
				ia_buf[tidb] = row_next;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[0]], val_buf[0]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int row = ia_buf[tidl];
	ValueType val;
	int k;
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		// val = val_buf[tidl];
		// if ((tidl & (2*k-1)) == k-1)
		// {
			// if (tidl >= k && row == ia_buf[tidl-k])
			// {
				// val_buf[tidl-k] += val;
				// val = 0;
			// }
		// }
		// g.sync();
		// if ((tidl & (2*k-1)) == k-1 && val != 0)
		// {
			// if (row == ia_buf[tidl+k])
			// {
				// val_buf[tidl+k] += val;
			// }
			// else
			// {
				// atomicAdd(&y[row], val);
			// }
		// }
		// g.sync();
		val = val_buf[tidl];
		if ((tidl & (2*k-1)) == k-1)
		{
			if (row == ia_buf[tidl+k])
			{
				val_buf[tidl+k] += val;
			}
			else
			{
				atomicAdd(&y[row], val);
			}
		}
		g.sync();
	}
}
inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &ia_buf[tidb_div*32], &val_buf[tidb_div*32], y);
	// __syncthreads();
	// if (tidb_mod == 31)
	// {
		// ia_buf[tidb_mod] = ia_buf[tidb];
		// val_buf[tidb_mod] = val_buf[tidb];
	// }
	// __syncthreads();
	// if (tidb_div == 0)
		// reduce_warp(tile32, ia_buf, val_buf, y);
	// if (tidb == 0)
		// atomicAdd(&y[ia_buf[31]], val_buf[31]);
	if (tidb_mod == 31)
		atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * row_ptr, ValueType * val_ptr, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	INT_T row = *row_ptr;
	ValueType val = *val_ptr;
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_next;
		ValueType val_next;
		row_next = __shfl_sync(0xffffffff, row, tidl+k);
		val_next = __shfl_sync(0xffffffff, val, tidl+k);
		if ((tidl & (2*k-1)) == 0)
		{
			if (row == row_next)
			{
				val += val_next;
			}
			else
			{
				atomicAdd(&y[row], val);
				val = val_next;
				row = row_next;
			}
		}
		g.sync();
	}
	*row_ptr = row;
	*val_ptr = val;
	// if (tidl == 0)
		// atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &row, &val, y);
	if (tidb_mod == 0)
		atomicAdd(&y[row], val);
	// extern __shared__ char sm[];
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[32 * sizeof(ValueType)];
	// if (tidb_mod == 0)
	// {
		// ia_buf[tidb_div] = row;
		// val_buf[tidb_div] = val;
	// }
	// __syncthreads();
	// if (tidb_div == 0)
	// {
		// row = ia_buf[tidb];
		// val = val_buf[tidb];
		// reduce_warp(tile32, &row, &val, y);
	// }
	// if (tidb == 0)
		// atomicAdd(&y[row], val);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_prev;
		ValueType val_prev;
		row_prev = __shfl_sync(0xffffffff, row, tidl-k);
		val_prev = __shfl_sync(0xffffffff, val, tidl-k);
		if ((tidl & (2*k-1)) == 2*k-1)
		{
			if (row == row_prev)
			{
				val += val_prev;
			}
			else
			{
				atomicAdd(&y[row_prev], val_prev);
			}
		}
		g.sync();
	}
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
} */


// Threads may only read data from another thread which is actively participating in the __shfl_sync() command.
// If the target thread is inactive, the retrieved value is undefined.
template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	const int tidl_one_hot = 1 << tidl;
	int flag;
	INT_T row_prev;
	ValueType val_prev;
	flag = 0xaaaaaaaa;
	row_prev = __shfl_sync(flag, row, tidl-1);
	val_prev = __shfl_sync(flag, val, tidl-1);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x88888888;
	row_prev = __shfl_sync(flag, row, tidl-2);
	val_prev = __shfl_sync(flag, val, tidl-2);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80808080;
	row_prev = __shfl_sync(flag, row, tidl-4);
	val_prev = __shfl_sync(flag, val, tidl-4);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80008000;
	row_prev = __shfl_sync(flag, row, tidl-8);
	val_prev = __shfl_sync(flag, val, tidl-8);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80000000;
	row_prev = __shfl_sync(flag, row, tidl-16);
	val_prev = __shfl_sync(flag, val, tidl-16);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	g.sync();
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
}

__device__ void spmv_last_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int block_size = blockDim.x;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	const int total_j = j_e - j_s;
	const int mod = total_j % block_size;
	int j_l_s, j_l_e;
	j_l_s = j_s + tidb * (total_j / block_size);
	j_l_e = j_l_s + (total_j / block_size);
	if (tidb < mod)
	{
		j_l_s += tidb;
		j_l_e += tidb + 1;
	}
	else
	{
		j_l_s += mod;
		j_l_e += mod;
	}
	// int m = (i_e + i_s) / 2;
	// while (i_s < i_e)
	// {
		// if (j_l_s >= row_ptr[m])
		// {
			// i_s = m + 1;
		// }
		// else
		// {
			// i_e = m;
		// }
		// m = (i_e + i_s) / 2;
	// }
	// i = i_s - 1;
	i = ia[j_l_s];
	// if (tidb == block_size-1)
	// {
		// if (j_l_e != j_e)
		// {
			// printf("wrong");
		// }
	// }
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_l_s;j<j_l_e;j++)
	{
		// if (ia[j] != i)
		// {
			// atomicAdd(&y[i], sum);
			// sum = 0;
			// i = ia[j];
		// }
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			// y[i] += sum;
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
			// i = ia[j];
		}
		// sum += a[j] * x[ja[j]];
		sum = __fma_rn(a[j], x[ja[j]], sum);
	}
	// if (j_l_s < j_l_e)
		// atomicAdd(&y[i], sum);
	// val_buf[tidb] = sum;
	// ia_buf[tidb] = i;
	// __syncthreads();
	// reduce_block(ia_buf, val_buf, y);
	reduce_block(i, sum, y);
}

__device__ void spmv_full_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int block_size = blockDim.x;
	const int nnz_per_block = block_size * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = block_id * nnz_per_block;
	// j_e = (block_id + 1) * nnz_per_block;
	int j_l_s, j_l_e;
	j_l_s = j_s + tidb * NNZ_PER_THREAD;
	j_l_e = j_l_s + NNZ_PER_THREAD;
	int m = (i_e + i_s) / 2;
	while (i_s < i_e)
	{
		if (j_l_s >= row_ptr[m])
		{
			i_s = m + 1;
		}
		else
		{
			i_e = m;
		}
		m = (i_e + i_s) / 2;
	}
	i = i_s - 1;
	// i = ia[j_l_s];
	// if (tidb == block_size-1)
	// {
		// if (j_l_e != j_e)
		// {
			// printf("wrong");
		// }
	// }
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_l_s;j<j_l_e;j++)
	{
		// if (ia[j] != i)
		// {
			// atomicAdd(&y[i], sum);
			// sum = 0;
			// i = ia[j];
		// }
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			// y[i] += sum;
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
			// i = ia[j];
		}
		// sum += a[j] * x[ja[j]];
		sum = __fma_rn(a[j], x[ja[j]], sum);
	}
	reduce_block(i, sum, y);
}

__global__ void gpu_kernel_spmv_row_indices_continuous(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int grid_size = gridDim.x;
	int block_id = blockIdx.x;
	if (block_id == grid_size - 1)
		spmv_last_block(thread_block_i_s, thread_block_i_e, thread_block_j_s, thread_block_j_e, row_ptr, ia, ja, a, x, y);
	else
		spmv_full_block(thread_block_i_s, thread_block_i_e, row_ptr, ia, ja, a, x, y);
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);
	// long shared_mem_size = block_size * (sizeof(ValueType));
	// long shared_mem_size = block_size * (sizeof(ValueType) + sizeof(INT_T));
	long shared_mem_size = 0;

	if (csr->x == NULL)
	{
		printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);
		csr->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(csr->x_d, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}
	}

	hipMemset(csr->y_d, 0, csr->m * sizeof(csr->y_d));
	gpu_kernel_spmv_row_indices_continuous<<<grid_dims, block_dims, shared_mem_size>>>(csr->thread_block_i_s_d, csr->thread_block_i_e_d, csr->thread_block_j_s_d, csr->thread_block_j_e_d, csr->row_ptr_d, csr->ia_d, csr->ja_d, csr->a_d, csr->x_d, csr->y_d);
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y));
		gpuCudaErrorCheck(hipMemcpy(csr->y, csr->y_d, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

