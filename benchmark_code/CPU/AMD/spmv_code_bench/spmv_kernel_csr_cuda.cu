#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef NUM_THREADS
#define NUM_THREADS 1024
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

// #ifndef VERIFIED
// #define VERIFIED 1
// #endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	INT_T * thread_i_s = NULL;
	INT_T * thread_i_e = NULL;

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;
	INT_T * thread_i_s_d;
	INT_T * thread_i_e_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;
	INT_T * thread_i_s_h;
	INT_T * thread_i_e_h;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	ValueType * x_h = NULL;
	ValueType * y_h = NULL;

	hipStream_t stream;
	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;
	
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;
	hipEvent_t startEvent_memcpy_thread_i_s;
	hipEvent_t endEvent_memcpy_thread_i_s;
	hipEvent_t startEvent_memcpy_thread_i_e;
	hipEvent_t endEvent_memcpy_thread_i_e;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_persistent_l2_cache;
	int num_threads;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		double time_balance;
		long i;

		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_persistent_l2_cache, cudaDevAttrMaxPersistingL2CacheSize, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_persistent_l2_cache=%d\n", max_persistent_l2_cache);

		num_threads = NUM_THREADS;
		printf("NUM_THREADS=%d\n", num_threads);

		thread_i_s = (INT_T *) malloc(num_threads * sizeof(*thread_i_s));
		thread_i_e = (INT_T *) malloc(num_threads * sizeof(*thread_i_e));
		time_balance = time_it(1,
			for (i=0;i<num_threads;i++)
			{
				// loop_partitioner_balance_iterations(num_threads, i, 0, m, &thread_i_s[i], &thread_i_e[i]);
				loop_partitioner_balance_prefix_sums(num_threads, i, ia, m, nnz, &thread_i_s[i], &thread_i_e[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_i_s_d, num_threads * sizeof(*thread_i_s_d)));
		gpuCudaErrorCheck(hipMalloc(&thread_i_e_d, num_threads * sizeof(*thread_i_e_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_i_s));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_i_s));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_i_e));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_i_e));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		gpuCudaErrorCheck(hipHostMalloc(&ia_h, (m+1) * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc(&ja_h, nnz * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc(&a_h, nnz * sizeof(*a_h)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_i_s_h, num_threads * sizeof(*thread_i_s_h)));
		gpuCudaErrorCheck(hipHostMalloc(&thread_i_e_h, num_threads * sizeof(*thread_i_e_h)));
		gpuCudaErrorCheck(hipHostMalloc(&x_h, n * sizeof(*x_h)));
		gpuCudaErrorCheck(hipHostMalloc(&y_h, m * sizeof(*y_h)));

		memcpy(ia_h, ia, (m+1) * sizeof(*ia_h));
		memcpy(ja_h, ja, nnz * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));
		memcpy(thread_i_s_h, thread_i_s, num_threads * sizeof(*thread_i_s_h));
		memcpy(thread_i_e_h, thread_i_e, num_threads * sizeof(*thread_i_e_h));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_i_s));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_i_s_d, thread_i_s_h, num_threads * sizeof(*thread_i_s_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_i_s));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_i_e));
		gpuCudaErrorCheck(hipMemcpyAsync(thread_i_e_d, thread_i_e_h, num_threads * sizeof(*thread_i_e_d), hipMemcpyHostToDevice, stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_i_e));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_i_s));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_thread_i_e));

			float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_i_s, memcpyTime_cuda_thread_i_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_i_s, startEvent_memcpy_thread_i_s, endEvent_memcpy_thread_i_s));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_i_e, startEvent_memcpy_thread_i_e, endEvent_memcpy_thread_i_e));
			printf("(CUDA) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, thread_s = %.4lf ms, thread_e = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_i_s, memcpyTime_cuda_thread_i_e);
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		free(thread_i_s);
		free(thread_i_e);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(thread_i_s_d));
		gpuCudaErrorCheck(hipFree(thread_i_e_d));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));
		gpuCudaErrorCheck(hipHostFree(thread_i_s_h));
		gpuCudaErrorCheck(hipHostFree(thread_i_e_h));

		gpuCudaErrorCheck(hipStreamDestroy(stream));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_i_s));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_i_s));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_i_e));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_i_e));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_t%d", csr->num_threads);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


__global__ void gpu_kernel_csr_basic(INT_T * thread_i_s, INT_T * thread_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int tgid = cuda_get_thread_num();
	long i, i_s, i_e, j, j_e;
	ValueType sum;
	i_s = thread_i_s[tgid];
	i_e = thread_i_e[tgid];
	j = ia[i_s];
	// printf("%d: %ld %ld\n", tgid, i_s, i_e);
	for (i=i_s;i<i_e;i++)
	{
		j_e = ia[i+1];
		sum = 0;
		for (;j<j_e;j++)
		{
			sum += a[j] * x[ja[j]];
		}
		y[i] = sum;
	}
}


__global__ void gpu_kernel_csr_flat(INT_T * thread_i_s, INT_T * thread_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int tgid = cuda_get_thread_num();
	long i, i_s, i_e, j, j_e;
	ValueType sum;
	i_s = thread_i_s[tgid];
	i_e = thread_i_e[tgid];
	i = i_s;
	j = ia[i_s];
	j_e = ia[i_s+1];
	sum = 0;
	for (j=ia[i_s];i<i_e;j++)
	{
		if (j == j_e)
		{
			y[i] = sum;
			sum = 0;
			i++;
			j_e = ia[i+1];
			// if (i == i_e)
				// break;
		}
		sum += a[j] * x[ja[j]];
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	int num_threads = csr->num_threads;
	int block_size = csr->warp_size;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_threads / block_size);

	if (csr->x == NULL)
	{
		printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);
		csr->x = x;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x, csr->stream));
		memcpy(csr->x_h, x, csr->n * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x, csr->stream));
		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		#ifdef PERSISTENT_L2_PREFETCH
			int x_d_size = csr->n * sizeof(*csr->x);
			gpuCudaErrorCheck(cudaCtxResetPersistingL2Cache()); // This needs to happen every time before running kernel for 1st time for a matrix...
			if(x_d_size < csr->max_persistent_l2_cache){
				hipLaunchAttributeValue attribute;
				auto &window = attribute.accessPolicyWindow;
				window.base_ptr = csr->x_d;
				window.num_bytes = x_d_size;
				window.hitRatio = 1.0;
				window.hitProp = hipAccessPropertyPersisting;
				window.missProp = hipAccessPropertyStreaming;
				gpuCudaErrorCheck(hipStreamSetAttribute(csr->stream, hipLaunchAttributeAccessPolicyWindow, &attribute));
			}
		#endif
	}

	// if(VERIFIED){
	// 	int num_loops = 1000;
	// 	for(int k=0;k<num_loops;k++)
	// 		gpu_kernel_csr_basic<<<grid_dims, block_dims>>>(thread_i_s_d, thread_i_e_d, csr->ia_d, csr->ja_d, csr->a_d, csr->x_d, csr->y_d);
	// 	gpuCudaErrorCheck(hipPeekAtLastError());
	// 	gpuCudaErrorCheck(hipDeviceSynchronize());
	// }

	// gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution));

	// int num_loops = 128;
	// double time_execution = time_it(1,
	// 	for(int k=0;k<num_loops;k++){
	gpu_kernel_csr_basic<<<grid_dims, block_dims, 0, csr->stream>>>(csr->thread_i_s_d, csr->thread_i_e_d, csr->ia_d, csr->ja_d, csr->a_d, csr->x_d, csr->y_d);
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());
	// 	}
	// );

	// double gflops = csr->nnz / time_execution * num_loops * 2 * 1e-9;
	// printf("(DGAL timing) Execution time = %.4lf ms (%.4lf GFLOPS scalar-%d)\n", time_execution*1e3, gflops, csr->num_threads);

	// gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution));
	// float executionTime_cuda;
	// gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_execution));
	// gpuCudaErrorCheck(hipEventElapsedTime(&executionTime_cuda, csr->startEvent_execution, csr->endEvent_execution));

	// double gflops_cuda = csr->nnz / executionTime_cuda * num_loops * 2 * 1e-6;
	// printf("(CUDA) Execution time = %.4lf ms (%.4lf GFLOPS @ %d threads for %.2lf MB workload)\n", executionTime_cuda, gflops_cuda, csr->num_threads, csr->mem_footprint/(1024*1024.0));

	if (csr->y == NULL)
	{
		csr->y = y;
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * sizeof(*csr->y_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * sizeof(ValueType));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

