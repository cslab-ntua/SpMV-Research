#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


INT_T * thread_i_s = NULL;
INT_T * thread_i_e = NULL;

INT_T * thread_i_s_dev = NULL;
INT_T * thread_i_e_dev = NULL;


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;


struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
	int num_threads;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		double time_balance;
		long i;

		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		// num_threads = 1024;
		// num_threads = 3584;
		// num_threads = 4096;
		num_threads = 8192;
		// num_threads = 14336;
		// num_threads = 16384;

		thread_i_s = (INT_T *) malloc(num_threads * sizeof(*thread_i_s));
		thread_i_e = (INT_T *) malloc(num_threads * sizeof(*thread_i_e));
		time_balance = time_it(1,
			for (i=0;i<num_threads;i++)
			{
				// loop_partitioner_balance_iterations(num_threads, i, 0, m, &thread_i_s[i], &thread_i_e[i]);
				loop_partitioner_balance_prefix_sums(num_threads, i, ia, m, nnz, &thread_i_s[i], &thread_i_e[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		hipMalloc(&ia_dev, (m+1) * sizeof(*ia_dev));
		hipMalloc(&ja_dev, nnz * sizeof(*ja_dev));
		hipMalloc(&a_dev, nnz * sizeof(*a_dev));
		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));
		hipMalloc(&thread_i_s_dev, num_threads * sizeof(*thread_i_s_dev));
		hipMalloc(&thread_i_e_dev, num_threads * sizeof(*thread_i_e_dev));

		hipMemcpy(ia_dev, ia, (m+1) * sizeof(*ia_dev), hipMemcpyHostToDevice);
		hipMemcpy(ja_dev, ja, nnz * sizeof(*ja_dev), hipMemcpyHostToDevice);
		hipMemcpy(a_dev, a, nnz * sizeof(*a_dev), hipMemcpyHostToDevice);
		hipMemcpy(thread_i_s_dev, thread_i_s, num_threads * sizeof(*thread_i_s_dev), hipMemcpyHostToDevice);
		hipMemcpy(thread_i_e_dev, thread_i_e, num_threads * sizeof(*thread_i_e_dev), hipMemcpyHostToDevice);

	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		free(thread_i_s);
		free(thread_i_e);
		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);


		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Custom_CSR_CUDA";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


// __device__ int add(int a, int b)
// {
	// return a + b;
// }


__global__ void gpu_kernel_csr_basic(INT_T * thread_i_s, INT_T * thread_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int tgid = cuda_get_thread_num();
	long i, i_s, i_e, j, j_e;
	ValueType sum;
	i_s = thread_i_s[tgid];
	i_e = thread_i_e[tgid];
	j = ia[i_s];
	// printf("%d: %ld %ld\n", tgid, i_s, i_e);
	for (i=i_s;i<i_e;i++)
	{
		j_e = ia[i+1];
		sum = 0;
		for (;j<j_e;j++)
		{
			sum += a[j] * x[ja[j]];
		}
		y[i] = sum;
	}
}


__global__ void gpu_kernel_csr_flat(INT_T * thread_i_s, INT_T * thread_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int tgid = cuda_get_thread_num();
	long i, i_s, i_e, j, j_e;
	ValueType sum;
	i_s = thread_i_s[tgid];
	i_e = thread_i_e[tgid];
	i = i_s;
	j = ia[i_s];
	j_e = ia[i_s+1];
	sum = 0;
	for (j=ia[i_s];i<i_e;j++)
	{
		if (j == j_e)
		{
			y[i] = sum;
			sum = 0;
			i++;
			j_e = ia[i+1];
			// if (i == i_e)
				// break;
		}
		sum += a[j] * x[ja[j]];
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	int num_threads = csr->num_threads;
	int block_size = csr->warp_size;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_threads / block_size);
	// long shared_mem_size = block_size * sizeof(*C_dev);

	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
	}

	gpu_kernel_csr_basic<<<grid_dims, block_dims>>>(thread_i_s_dev, thread_i_e_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);
	// gpu_kernel_csr_flat<<<grid_dims, block_dims>>>(thread_i_s_dev, thread_i_e_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);

	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));

	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}

	// exit(0);
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

