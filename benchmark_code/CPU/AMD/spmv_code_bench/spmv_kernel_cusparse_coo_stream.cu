#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cublas_util.h"
	#include "cuda/cusparse_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;

#ifndef NUM_STREAMS
#define NUM_STREAMS 128
#endif

#ifndef TIME_IT
#define TIME_IT 1
#endif

#ifndef VERIFIED
#define VERIFIED 1
#endif

struct COOArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	INT_T * ia_h[NUM_STREAMS];
	INT_T * ja_h[NUM_STREAMS];
	ValueType * a_h[NUM_STREAMS];

	INT_T * ia_d[NUM_STREAMS];
	INT_T * ja_d[NUM_STREAMS];
	ValueType * a_d[NUM_STREAMS];

	hipStream_t stream[NUM_STREAMS];
	INT_T n_stream[NUM_STREAMS];
	INT_T nnz_stream[NUM_STREAMS];

	hipsparseHandle_t     handle[NUM_STREAMS];
	hipsparseSpMatDescr_t matA[NUM_STREAMS];
	void*                dBuffer[NUM_STREAMS];
	size_t               bufferSize[NUM_STREAMS];

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h[NUM_STREAMS];
	ValueType * y_h[NUM_STREAMS];
	ValueType * x_d[NUM_STREAMS];
	// ValueType * y_d[NUM_STREAMS];
	ValueType * y_d2;
	ValueType * y_d_reduction;

	hipsparseDnVecDescr_t vecX[NUM_STREAMS];
	hipsparseDnVecDescr_t vecY[NUM_STREAMS];

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution[NUM_STREAMS];
	hipEvent_t endEvent_execution[NUM_STREAMS];

	hipEvent_t startEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipEvent_t startEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_a[NUM_STREAMS];

	hipEvent_t startEvent_create_matA[NUM_STREAMS];
	hipEvent_t endEvent_create_matA[NUM_STREAMS];
	hipEvent_t startEvent_spmv_buffersize[NUM_STREAMS];
	hipEvent_t endEvent_spmv_buffersize[NUM_STREAMS];

	hipEvent_t startEvent_create_vecX[NUM_STREAMS];
	hipEvent_t endEvent_create_vecX[NUM_STREAMS];
	hipEvent_t startEvent_create_vecY[NUM_STREAMS];
	hipEvent_t endEvent_create_vecY[NUM_STREAMS];

	hipblasHandle_t handle_blas;

	int num_streams;

	COOArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		// printf("max_smem_per_block=%d\n", max_smem_per_block);
		// printf("multiproc_count=%d\n", multiproc_count);
		// printf("max_threads_per_block=%d\n", max_threads_per_block);
		// printf("warp_size=%d\n", warp_size);
		// printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		num_streams = NUM_STREAMS;

		/********************************************************************************************************/
		printf("/********************************************************************************************************/\n");
		// Convert COO representation ton CSC
		INT_T * row_indices; //for CSC format
		INT_T * row_idx;
		INT_T * col_ptr;
		ValueType * val_c;

		row_indices = (typeof(row_indices)) malloc(nnz * sizeof(*row_indices));
		row_idx = (typeof(row_idx)) malloc(nnz * sizeof(*row_idx));
		col_ptr = (typeof(col_ptr)) malloc((n+1) * sizeof(*col_ptr));
		val_c = (typeof(val_c)) malloc(nnz * sizeof(*val_c));

		double time = time_it(1,
			csr_row_indices(ia, ja, m, n, nnz, &row_indices);
			coo_to_csc(row_indices, ja, a, m, n, nnz, row_idx, col_ptr, val_c, 1);
			free(row_indices);
		);
		printf("time coo_to_csc = %g ms\n", time*1e3);

		INT_T *local_stream_j_s = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_s));
		INT_T *local_stream_j_e = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_e));
		double time_balance = time_it(1,
			for (int i=0;i<num_streams;i++)
				loop_partitioner_balance_prefix_sums(num_streams, i, col_ptr, n, nnz, &local_stream_j_s[i], &local_stream_j_e[i]);
		);

		int cnt=0, cnt2=0;
		for(int i=0; i<num_streams; i++){
			nnz_stream[i] = col_ptr[local_stream_j_e[i]] - col_ptr[local_stream_j_s[i]];
			n_stream[i] = local_stream_j_e[i] - local_stream_j_s[i];
			// printf("local_stream[%d] = %d - %d (%d cols) (%d nnz)\n", i, local_stream_j_s[i], local_stream_j_e[i], n_stream[i], nnz_stream[i]);

			cnt  += nnz_stream[i];
			cnt2 += n_stream[i];
		}
		printf("balance time (col) = %g ms\n", time_balance*1e3);

		INT_T * row_idx_stream[num_streams];
		INT_T * col_ptr_stream[num_streams];
		ValueType * val_c_stream[num_streams];
		
		double time_memcpy_stream_locals = time_it(1,
		for(int i=0; i<num_streams; i++){
			col_ptr_stream[i] = (INT_T *) malloc((n_stream[i]+1) * sizeof(INT_T));
			row_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_c_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			memcpy(col_ptr_stream[i], col_ptr + local_stream_j_s[i], (n_stream[i] + 1) * sizeof(INT_T));
			// col_ptr needs to be fixed, so that it will start from 0 again...
			for(int j=0; j<n_stream[i]+1; j++)
				col_ptr_stream[i][j] -= col_ptr[local_stream_j_s[i]];
			memcpy(row_idx_stream[i], row_idx + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(INT_T));
			memcpy(val_c_stream[i], val_c + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy_stream_locals = %lf ms\n", time_memcpy_stream_locals*1e3);
		free(local_stream_j_s);
		free(local_stream_j_e);

		INT_T * row_ptr_stream[num_streams];
		INT_T * col_idx_stream[num_streams];
		ValueType * val_stream[num_streams];

		for(int i=0; i<num_streams; i++){
			INT_T * col_indices;
			csc_col_indices(row_idx_stream[i], col_ptr_stream[i], m, n_stream[i], nnz_stream[i], &col_indices);

			row_ptr_stream[i] = (INT_T *) malloc((m+1) * sizeof(INT_T));
			col_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			coo_to_csr(row_idx_stream[i], col_indices, val_c_stream[i], m, n_stream[i], nnz_stream[i], row_ptr_stream[i], col_idx_stream[i], val_stream[i], 1, 0);

			free(col_indices);
		}

		for(int i=0; i<num_streams; i++){
			// free(row_idx_stream[i]);
			free(col_ptr_stream[i]);
			free(val_c_stream[i]);
		}
		free(row_idx);
		free(col_ptr);
		free(val_c);

		printf("/********************************************************************************************************/\n");
		/********************************************************************************************************/

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipMalloc(&ia_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ja_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&a_d[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&x_d[i], n_stream[i] * sizeof(ValueType)));
			// gpuCudaErrorCheck(hipMalloc(&y_d[i], m * sizeof(ValueType)));
		}
		gpuCudaErrorCheck(hipMalloc(&y_d2, m * num_streams * sizeof(ValueType)));
		gpuCudaErrorCheck(hipMalloc(&y_d_reduction, m * sizeof(ValueType)));
		gpuCublasErrorCheck(hipblasCreate(&handle_blas));

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipHostMalloc(&ia_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ja_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&a_h[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&x_h[i], n_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&y_h[i], m * sizeof(ValueType)));
		}

		double time_memcpy = time_it(1,
		for(int i=0; i<num_streams; i++){
			memcpy(ia_h[i], row_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(ja_h[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(a_h[i], val_stream[i], nnz_stream[i] * sizeof(ValueType));
		}
		for(int i=0; i<num_streams; i++)
			free(row_idx_stream[i]);
		);
		printf("time_memcpy (ia_h, ja_h, a_h) = %lf ms\n", time_memcpy*1e3);

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipStreamCreate(&stream[i]));
			gpuCusparseErrorCheck(hipsparseCreate(&handle[i]));
			gpuCusparseErrorCheck(hipsparseSetStream(handle[i], stream[i]));

			// cuda events for timing measurements
			gpuCudaErrorCheck(hipEventCreate(&startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_execution[i]));
		}
		gpuCublasErrorCheck(hipblasSetStream(handle_blas, stream[0]));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_create_matA[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_create_matA[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_spmv_buffersize[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_spmv_buffersize[i]));

				gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecX[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecX[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecY[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecY[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		for(int i=0; i<num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ia_d[i], ia_h[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ja_d[i], ja_h[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(a_d[i], a_h[i], nnz_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a[i], stream[i]));

			// Create sparse matrix A in COO format
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_create_matA[i], stream[i]));
			gpuCusparseErrorCheck(hipsparseCreateCoo(&matA[i], m, n_stream[i], nnz_stream[i], ia_d[i], ja_d[i], a_d[i], HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_create_matA[i], stream[i]));
		}

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(stream[i]));
				float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, create_matA_Time;//memcpyTime_cuda_thread_i_e;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia[i], endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja[i], endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a[i], endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&create_matA_Time, startEvent_create_matA[i], endEvent_create_matA[i]));
				printf("(CUDA) (stream %d): Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, matA time = %.4lf ms\n", i, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, create_matA_Time);
			}			
		}
	}

	~COOArrays()
	{
		free(a);
		free(ia);
		free(ja);

		// destroy matrix/vector descriptors
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipFree(ia_d[i]));
			gpuCudaErrorCheck(hipFree(ja_d[i]));
			gpuCudaErrorCheck(hipFree(a_d[i]));
			gpuCudaErrorCheck(hipFree(x_d[i]));
			// gpuCudaErrorCheck(hipFree(y_d[i]));

			gpuCudaErrorCheck(hipHostFree(ia_h[i]));
			gpuCudaErrorCheck(hipHostFree(ja_h[i]));
			gpuCudaErrorCheck(hipHostFree(a_h[i]));
			gpuCudaErrorCheck(hipHostFree(x_h[i]));
			gpuCudaErrorCheck(hipHostFree(y_h[i]));

			gpuCudaErrorCheck(hipStreamDestroy(stream[i]));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_execution[i]));

			gpuCusparseErrorCheck(hipsparseDestroy(handle[i]));
			gpuCusparseErrorCheck(hipsparseDestroySpMat(matA[i]));
			gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecX[i]));
			gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecY[i]));
			
			gpuCudaErrorCheck(hipFree(dBuffer[i]));
		}
		gpuCudaErrorCheck(hipFree(y_d2));
		gpuCudaErrorCheck(hipFree(y_d_reduction));
		gpuCublasErrorCheck(hipblasDestroy(handle_blas));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_create_matA[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_create_matA[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_spmv_buffersize[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_spmv_buffersize[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecX[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecX[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecY[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecY[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_coo(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);

void
COOArrays::spmv(ValueType * x, ValueType * y)
{
	compute_coo(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct COOArrays * coo = new COOArrays(row_ptr, col_ind, values, m, n, nnz);
	coo->mem_footprint = nnz * (sizeof(ValueType) + 2 * sizeof(INT_T));
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "CUSPARSE_COO_STREAM_%d", coo->num_streams);
	coo->format_name = format_name;
	return coo;
}


//==========================================================================================================================================
//= COO Custom
//==========================================================================================================================================


void
compute_coo(COOArrays * restrict coo, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (coo->x == NULL)
	{
		coo->x = x;
		int offset = 0;
		for(int i=0; i<coo->num_streams; i++){
			memcpy(coo->x_h[i], x + offset, coo->n_stream[i] * sizeof(ValueType));
			offset += coo->n_stream[i];
		}

		for(int i=0; i<coo->num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_x[i], coo->stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(coo->x_d[i], coo->x_h[i], coo->n_stream[i] * sizeof(*coo->x), hipMemcpyHostToDevice, coo->stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_x[i], coo->stream[i]));
		}

		for(int i=0; i<coo->num_streams; i++)
			gpuCudaErrorCheck(hipStreamSynchronize(coo->stream[i]));

		if(TIME_IT){
			for(int i=0; i<coo->num_streams; i++){
				float memcpyTime_cuda;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_x[i], coo->endEvent_memcpy_x[i]));
				printf("(CUDA) (stream %d) Memcpy x time = %.4lf ms\n", i, memcpyTime_cuda);
			}
		}

		for(int i=0; i<coo->num_streams; i++){
			// Create dense vector X
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_create_vecX[i], coo->stream[i]));
			gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecX[i], coo->n_stream[i], coo->x_d[i], ValueTypeCuda));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_create_vecX[i], coo->stream[i]));

			// Create dense vector y
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_create_vecY[i], coo->stream[i]));
			// gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecY[i], coo->m, coo->y_d[i], ValueTypeCuda));
			gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecY[i], coo->m, coo->y_d2 + i*coo->m, ValueTypeCuda));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_create_vecY[i], coo->stream[i]));

			// Allocate an external buffer if needed
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_spmv_buffersize[i], coo->stream[i]));
			gpuCusparseErrorCheck(hipsparseSpMV_bufferSize(coo->handle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA[i], coo->vecX[i], &beta, coo->vecY[i], ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &coo->bufferSize[i]));
			gpuCudaErrorCheck(hipMalloc(&coo->dBuffer[i], coo->bufferSize[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_spmv_buffersize[i], coo->stream[i]));
			// printf("(stream %d) SpMV_bufferSize = %ld\n", i, coo->bufferSize[i]); // size of the workspace that is needed by hipsparseSpMV()
		}

		if(TIME_IT){
			for(int i=0; i<coo->num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(coo->stream[i]));
				float create_vecX_time, create_vecY_time, spmv_buffersize_time;
				gpuCudaErrorCheck(hipEventElapsedTime(&create_vecX_time, coo->startEvent_create_vecX[i], coo->endEvent_create_vecX[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&create_vecY_time, coo->startEvent_create_vecY[i], coo->endEvent_create_vecY[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&spmv_buffersize_time, coo->startEvent_spmv_buffersize[i], coo->endEvent_spmv_buffersize[i]));
				printf("(CUDA) (stream %d) Create vecX time = %.4lf ms, vecY time = %.4lf ms, spmv_buffersize time = %.4lf (SpMV_bufferSize = %zu)\n", i, create_vecX_time, create_vecY_time, spmv_buffersize_time, coo->bufferSize[i]);
			}
		}
	}

	for(int i=0; i<coo->num_streams; i++)
		gpuCudaErrorCheck(hipEventRecord(coo->startEvent_execution[i], coo->stream[i]));

	int num_loops = 128;
	double time_execution = time_it(1,
		for(int k=0;k<num_loops;k++){
			for(int i=0; i<coo->num_streams; i++){
				gpuCusparseErrorCheck(hipsparseSpMV(coo->handle[i], HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA[i], coo->vecX[i], &beta, coo->vecY[i], ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, coo->dBuffer[i]));
			}
			// gpuCudaErrorCheck(hipPeekAtLastError());
			gpuCudaErrorCheck(hipDeviceSynchronize());
		}
	);

	double gflops = coo->nnz / time_execution * num_loops * 2 * 1e-9;
	printf("(DGAL timing) Execution time = %.4lf ms (%.4lf GFLOPS %d streams for %.2lf MB workload)\n", time_execution*1e3, gflops, coo->num_streams, coo->mem_footprint/(1024*1024.0));

	for(int i=0; i<coo->num_streams; i++)
		gpuCudaErrorCheck(hipEventRecord(coo->endEvent_execution[i], coo->stream[i]));

	for(int i=0; i<coo->num_streams; i++)
		gpuCudaErrorCheck(hipStreamSynchronize(coo->stream[i]));

	if(TIME_IT){
		for(int i=0; i<coo->num_streams; i++){
			float executionTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&executionTime_cuda, coo->startEvent_execution[i], coo->endEvent_execution[i]));

			double gflops_cuda = coo->nnz_stream[i] / executionTime_cuda * num_loops * 2 * 1e-6;
			double mem_footprint = (coo->nnz_stream[i] * (sizeof(ValueType) + sizeof(INT_T)) + (coo->m+1) * sizeof(INT_T))/(1024*1024.0);
			printf("(CUDA) (stream %d) Execution time = %.4lf ms (%.4lf GFLOPS for %.2lf MB workload)\n", i, executionTime_cuda, gflops_cuda, mem_footprint);
		}
	}

	if (coo->y == NULL)
	{
		coo->y = y;

		// for(int i=0; i<coo->num_streams; i++){
		// 	if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_y[i], coo->stream[i]));
		// 	gpuCudaErrorCheck(hipMemcpyAsync(coo->y_h[i], coo->y_d[i], coo->m * sizeof(ValueType), hipMemcpyDeviceToHost, coo->stream[i]));
		// 	if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_y[i], coo->stream[i]));
		// }

		// if(TIME_IT){
		// 	for(int i=0; i<coo->num_streams; i++){
		// 		gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_memcpy_y[i]));
		// 		float memcpyTime_cuda;
		// 		gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_y[i], coo->endEvent_memcpy_y[i]));
		// 		printf("(CUDA) (stream %d) Memcpy y time = %.4lf ms\n", i, memcpyTime_cuda);
		// 	}
		// }

		// // for(int i=0; i<coo->num_streams; i++)
		// // 	memcpy(y, coo->y_h[i], coo->m * sizeof(ValueType));
		// double time_y_reduction = time_it(1,
		// 	for(int i=0; i<coo->m; i++){
		// 		coo->y_h_final[i] = 0;
		// 		for(int k=0; k<coo->num_streams; k++)
		// 			coo->y_h_final[i] += coo->y_h[k][i];
		// 	}
		// 	memcpy(y, coo->y_h_final, coo->m * sizeof(ValueType));
		// );
		// printf("time_y_reduction = %.4lf ms\n", time_y_reduction);

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_y, coo->stream[0]));

		ValueType *ones_host, *ones_device;

		gpuCudaErrorCheck(hipHostMalloc(&ones_host, coo->num_streams * sizeof(ValueType)));
		for (int i=0; i<coo->num_streams; i++) ones_host[i] = 1.0;
		gpuCudaErrorCheck(hipMalloc(&ones_device, coo->num_streams * sizeof(ValueType)));	
		gpuCudaErrorCheck(hipMemcpyAsync(ones_device, ones_host, coo->num_streams * sizeof(ValueType), hipMemcpyHostToDevice, coo->stream[0]));
	
		ValueType  alpha = 1.0, beta = 0.0;
		gpuCublasErrorCheck(hipblasDgemv(coo->handle_blas, HIPBLAS_OP_N, coo->m, coo->num_streams, &alpha, coo->y_d2, coo->m, ones_device, 1, &beta, coo->y_d_reduction, 1));

		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipMemcpyAsync(coo->y, coo->y_d_reduction, coo->m * sizeof(coo->y), hipMemcpyDeviceToHost, coo->stream[0]));

		gpuCudaErrorCheck(hipHostFree(ones_host));
		gpuCudaErrorCheck(hipFree(ones_device));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_y, coo->stream[0]));
		gpuCudaErrorCheck(hipStreamSynchronize(coo->stream[0]));
		if(TIME_IT){
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_y, coo->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
COOArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
COOArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

