#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	// #include "string_util.h" // REMOVE THIS

	#include "cuda/cuda_util.h"
	#include "cuda/cublas_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef NUM_THREADS
#define NUM_THREADS 1024
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 1
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

INT_T * thread_i_s[NUM_STREAMS];
INT_T * thread_i_e[NUM_STREAMS];

// REMOVE THIS
/*
void replace_substring(char* str, const char* find, const char* replace)
{
	char* pos = strstr(str, find);
	if (pos != NULL) {
		size_t find_len = strlen(find);
		size_t replace_len = strlen(replace);
		size_t tail_len = strlen(pos + find_len);

		memmove(pos + replace_len, pos + find_len, tail_len + 1);
		memcpy(pos, replace, replace_len);
	}
}
// REMOVE THIS
char * fig_name_gen(const char * file_basename, const char * replace_str)
{
	long buf_n = 1000;
	char buf[buf_n];

	char * path, * filename, * filename_base;
	str_path_split_path(file_basename, strlen(file_basename) + 1, buf, buf_n, &path, &filename);

	path = strdup(path);
	filename = strdup(filename);
	char file_new[1000];
	char replace[1000];

	sprintf(file_new, "%s", file_basename);
	sprintf(replace, "_%s.mtx", replace_str);
	// replace_substring(file_new, ".mtx", replace);
	if(replace_str == NULL || replace_str[0] == '\0')
		replace_substring(file_new, ".mtx", "");
	else{
		replace_substring(file_new, ".mtx", "_|");
		replace_substring(file_new, "|", replace_str);
	}

	char * file_fig;
	str_path_split_path(file_new, strlen(file_new) + 1, buf, buf_n, &path, &filename);
	
	path = strdup(path);
	filename = strdup(filename);
	str_path_split_ext(filename, strlen(filename) + 1, buf, buf_n, &filename_base, NULL);
	filename_base = strdup(filename_base);
	snprintf(buf, buf_n, "figures/%s", filename_base);
	file_fig = strdup(buf);
	return file_fig;
}
*/

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	INT_T * ia_h[NUM_STREAMS];
	INT_T * ja_h[NUM_STREAMS];
	ValueType * a_h[NUM_STREAMS];
	INT_T * thread_i_s_h[NUM_STREAMS];
	INT_T * thread_i_e_h[NUM_STREAMS];

	INT_T * ia_d[NUM_STREAMS];
	INT_T * ja_d[NUM_STREAMS];
	ValueType * a_d[NUM_STREAMS];
	INT_T * thread_i_s_d[NUM_STREAMS];
	INT_T * thread_i_e_d[NUM_STREAMS];

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h[NUM_STREAMS];
	ValueType * y_h[NUM_STREAMS];
	ValueType * x_d[NUM_STREAMS];
	// ValueType * y_d[NUM_STREAMS];
	ValueType * y_d2;
	ValueType * y_d_reduction;

	hipStream_t stream[NUM_STREAMS];
	INT_T n_stream[NUM_STREAMS];
	INT_T nnz_stream[NUM_STREAMS];

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution[NUM_STREAMS];
	hipEvent_t endEvent_execution[NUM_STREAMS];
	
	hipEvent_t startEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_i_s[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_i_s[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_i_e[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_i_e[NUM_STREAMS];

	hipEvent_t startEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipblasHandle_t handle;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
	int num_threads;
	int num_streams;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		double time_balance;

		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		num_threads = NUM_THREADS;
		num_streams = NUM_STREAMS;

		/********************************************************************************************************/
		printf("/********************************************************************************************************/\n");
		// Convert CSR representation ton CSC
		INT_T * row_indices; //for CSC format
		INT_T * row_idx;
		INT_T * col_ptr;
		ValueType * val_c;

		row_indices = (typeof(row_indices)) malloc(nnz * sizeof(*row_indices));
		row_idx = (typeof(row_idx)) malloc(nnz * sizeof(*row_idx));
		col_ptr = (typeof(col_ptr)) malloc((n+1) * sizeof(*col_ptr));
		val_c = (typeof(val_c)) malloc(nnz * sizeof(*val_c));

		double time = time_it(1,
			csr_row_indices(ia, ja, m, n, nnz, &row_indices);
			coo_to_csc(row_indices, ja, a, m, n, nnz, row_idx, col_ptr, val_c, 1);
			free(row_indices);
		);
		printf("time coo_to_csc = %g ms\n", time*1e3);

		INT_T *local_stream_j_s = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_s));
		INT_T *local_stream_j_e = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_e));
		time_balance = time_it(1,
			for (int i=0;i<num_streams;i++)
				loop_partitioner_balance_prefix_sums(num_streams, i, col_ptr, n, nnz, &local_stream_j_s[i], &local_stream_j_e[i]);
		);

		int cnt=0, cnt2=0;
		for(int i=0; i<num_streams; i++){
			nnz_stream[i] = col_ptr[local_stream_j_e[i]] - col_ptr[local_stream_j_s[i]];
			n_stream[i] = local_stream_j_e[i] - local_stream_j_s[i];
			// printf("local_stream[%d] = %d - %d (%d cols) (%d nnz)\n", i, local_stream_j_s[i], local_stream_j_e[i], n_stream[i], nnz_stream[i]);

			cnt  += nnz_stream[i];
			cnt2 += n_stream[i];
		}

		printf("balance time (col) = %g ms\n", time_balance*1e3);

		INT_T * row_idx_stream[num_streams];
		INT_T * col_ptr_stream[num_streams];
		ValueType * val_c_stream[num_streams];
		
		double time_memcpy_stream_locals = time_it(1,
		for(int i=0; i<num_streams; i++){
			col_ptr_stream[i] = (INT_T *) malloc((n_stream[i]+1) * sizeof(INT_T));
			row_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_c_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			memcpy(col_ptr_stream[i], col_ptr + local_stream_j_s[i], (n_stream[i] + 1) * sizeof(INT_T));
			// col_ptr needs to be fixed, so that it will start from 0 again...
			for(int j=0; j<n_stream[i]+1; j++)
				col_ptr_stream[i][j] -= col_ptr[local_stream_j_s[i]];
			memcpy(row_idx_stream[i], row_idx + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(INT_T));
			memcpy(val_c_stream[i], val_c + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy_stream_locals = %lf ms\n", time_memcpy_stream_locals*1e3);
		free(local_stream_j_s);
		free(local_stream_j_e);

		INT_T * row_ptr_stream[num_streams];
		INT_T * col_idx_stream[num_streams];
		ValueType * val_stream[num_streams];

		for(int i=0; i<num_streams; i++){
			INT_T * col_indices;
			csc_col_indices(row_idx_stream[i], col_ptr_stream[i], m, n_stream[i], nnz_stream[i], &col_indices);

			row_ptr_stream[i] = (INT_T *) malloc((m+1) * sizeof(INT_T));
			col_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			coo_to_csr(row_idx_stream[i], col_indices, val_c_stream[i], m, n_stream[i], nnz_stream[i], row_ptr_stream[i], col_idx_stream[i], val_stream[i], 1, 0);
			// REMOVE THIS
			/*
			if(0){
				char * replace_str;
				replace_str = (char *)malloc(100*sizeof(char));
				sprintf(replace_str, "stream_%d", i);
				char * file_fig = fig_name_gen("matrix.mtx", replace_str);
				printf("file_fig = %s\n", file_fig);

				long num_pixels = 4096;
				long num_pixels_x = (n_stream[i] < num_pixels) ? n_stream[i] : num_pixels;
				long num_pixels_y = (m < num_pixels) ? m : num_pixels;
				if(m!=n_stream[i]) {
					double ratio = n_stream[i]*1.0 / m;
					// if((ratio>16.0) || (ratio<(1/16.0)))
					if(ratio>16.0)
						ratio=16.0;
					if(ratio < (1/16.0))
						ratio=1/16.0;
					// in order to keep both below 1024
					if(ratio>1) // n > m
						num_pixels_y = (1/ratio) * num_pixels_x;
					else // m > n
						num_pixels_x = ratio * num_pixels_y;
				}
				csr_plot(file_fig, row_ptr_stream[i], col_idx_stream[i], val_stream[i], m, n_stream[i], nnz_stream[i], 0, num_pixels_x, num_pixels_y);
			}
			*/

			free(col_indices);
		}

		for(int i=0; i<num_streams; i++){
			free(row_idx_stream[i]);
			free(col_ptr_stream[i]);
			free(val_c_stream[i]);
		}
		free(row_idx);
		free(col_ptr);
		free(val_c);

		printf("/********************************************************************************************************/\n");
		/********************************************************************************************************/

		time_balance = time_it(1,
			for(int i=0; i<num_streams; i++){
				thread_i_s[i] = (INT_T *) malloc(num_threads * sizeof(INT_T));
				thread_i_e[i] = (INT_T *) malloc(num_threads * sizeof(INT_T));
				for (int j=0; j<num_threads; j++)
				{
					// loop_partitioner_balance_iterations(num_threads, i, 0, m, &thread_i_s[i], &thread_i_e[i]);
					loop_partitioner_balance_prefix_sums(num_threads, j, row_ptr_stream[i], m, nnz_stream[i], &(thread_i_s[i][j]), &(thread_i_e[i][j]));
					// if(j==10)
					// 	printf("%d %d  =%d, %d\n", i, j, thread_i_s[i][j], thread_i_e[i][j]);
				}				
			}
		);
		printf("balance time = %g\n", time_balance);

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipMalloc(&ia_d[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ja_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&a_d[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&x_d[i], n_stream[i] * sizeof(ValueType)));
			// gpuCudaErrorCheck(hipMalloc(&y_d[i], m * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&thread_i_s_d[i], num_threads * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&thread_i_e_d[i], num_threads * sizeof(INT_T)));
		}
		gpuCudaErrorCheck(hipMalloc(&y_d2, m * num_streams * sizeof(ValueType)));
		gpuCudaErrorCheck(hipMalloc(&y_d_reduction, m * sizeof(ValueType)));
		gpuCublasErrorCheck(hipblasCreate(&handle));

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipHostMalloc(&ia_h[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ja_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&a_h[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&x_h[i], n_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&y_h[i], m * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_i_s_h[i], num_threads * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_i_e_h[i], num_threads * sizeof(INT_T)));
		}

		double time_memcpy = time_it(1,
		for(int i=0; i<num_streams; i++){
			memcpy(ia_h[i], row_ptr_stream[i], (m + 1) * sizeof(INT_T));
			memcpy(ja_h[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(a_h[i], val_stream[i], nnz_stream[i] * sizeof(ValueType));
			memcpy(thread_i_s_h[i], thread_i_s[i], num_threads * sizeof(INT_T));
			memcpy(thread_i_e_h[i], thread_i_e[i], num_threads * sizeof(INT_T));
		}
		);
		printf("time_memcpy (ia_h, ja_h, a_h, thr_i_s, thr_i_e) = %lf ms\n", time_memcpy*1e3);

		// cuda events for timing measurements
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipStreamCreate(&stream[i]));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_execution[i]));
		}
		gpuCublasErrorCheck(hipblasSetStream(handle, stream[0]));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_i_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_i_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_i_e[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_i_e[i]));

				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		for(int i=0; i<num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ia_d[i], row_ptr_stream[i], (m+1) * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ja_d[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(a_d[i], val_stream[i], nnz_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_i_s[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_i_s_d[i], thread_i_s_h[i], num_threads * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_i_s[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_i_e[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_i_e_d[i], thread_i_e_h[i], num_threads * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_i_e[i], stream[i]));
		}

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(stream[i]));
				float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_i_s, memcpyTime_cuda_thread_i_e;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia[i], endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja[i], endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a[i], endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_i_s, startEvent_memcpy_thread_i_s[i], endEvent_memcpy_thread_i_s[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_i_e, startEvent_memcpy_thread_i_e[i], endEvent_memcpy_thread_i_e[i]));
				printf("(CUDA) (stream %d) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, thread_s = %.4lf ms, thread_e = %.4lf ms\n", i, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_thread_i_s, memcpyTime_cuda_thread_i_e);
			}
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		for(int i=0; i<num_streams; i++){
			free(thread_i_s[i]);
			free(thread_i_e[i]);
		}

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipFree(ia_d[i]));
			gpuCudaErrorCheck(hipFree(ja_d[i]));
			gpuCudaErrorCheck(hipFree(a_d[i]));
			gpuCudaErrorCheck(hipFree(x_d[i]));
			// gpuCudaErrorCheck(hipFree(y_d[i]));
			gpuCudaErrorCheck(hipFree(thread_i_s_d[i]));
			gpuCudaErrorCheck(hipFree(thread_i_e_d[i]));

			gpuCudaErrorCheck(hipHostFree(ia_h[i]));
			gpuCudaErrorCheck(hipHostFree(ja_h[i]));
			gpuCudaErrorCheck(hipHostFree(a_h[i]));
			gpuCudaErrorCheck(hipHostFree(x_h[i]));
			gpuCudaErrorCheck(hipHostFree(y_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_i_s_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_i_e_h[i]));

			gpuCudaErrorCheck(hipStreamDestroy(stream[i]));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_execution[i]));
		}
		gpuCudaErrorCheck(hipFree(y_d2));
		gpuCudaErrorCheck(hipFree(y_d_reduction));
		gpuCublasErrorCheck(hipblasDestroy(handle));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_i_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_i_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_i_e[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_i_e[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a[i]));
			}
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_s%d_t%d", csr->num_streams, csr->num_threads);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


__global__ void gpu_kernel_csr_basic(INT_T * thread_i_s, INT_T * thread_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int tgid = cuda_get_thread_num();
	long i, i_s, i_e, j, j_e;
	ValueType sum;
	i_s = thread_i_s[tgid];
	i_e = thread_i_e[tgid];
	j = ia[i_s];
	// printf("%d: %ld %ld\n", tgid, i_s, i_e);
	for (i=i_s;i<i_e;i++)
	{
		j_e = ia[i+1];
		sum = 0;
		for (;j<j_e;j++)
		{
			sum += a[j] * x[ja[j]];
		}
		y[i] = sum;
	}
}


__global__ void gpu_kernel_csr_flat(INT_T * thread_i_s, INT_T * thread_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int tgid = cuda_get_thread_num();
	long i, i_s, i_e, j, j_e;
	ValueType sum;
	i_s = thread_i_s[tgid];
	i_e = thread_i_e[tgid];
	i = i_s;
	j = ia[i_s];
	j_e = ia[i_s+1];
	sum = 0;
	for (j=ia[i_s];i<i_e;j++)
	{
		if (j == j_e)
		{
			y[i] = sum;
			sum = 0;
			i++;
			j_e = ia[i+1];
			// if (i == i_e)
				// break;
		}
		sum += a[j] * x[ja[j]];
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	int num_threads = csr->num_threads;
	int block_size = csr->warp_size;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_threads / block_size);
	// printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);
	// long shared_mem_size = block_size * sizeof(*C_d);

	if (csr->x == NULL)
	{
		csr->x = x;
		int offset = 0;
		for(int i=0; i<csr->num_streams; i++){
			memcpy(csr->x_h[i], x + offset, csr->n_stream[i] * sizeof(ValueType));
			offset += csr->n_stream[i];
		}

		for(int i=0; i<csr->num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x[i], csr->stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d[i], csr->x_h[i], csr->n_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x[i], csr->stream[i]));
		}

		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

		if(TIME_IT){
			for(int i=0; i<csr->num_streams; i++){
				float memcpyTime_cuda;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x[i], csr->endEvent_memcpy_x[i]));
				printf("(CUDA) (stream %d) Memcpy x time = %.4lf ms\n", i, memcpyTime_cuda);
			}
		}
	}

	for(int i=0; i<csr->num_streams; i++){
		// gpu_kernel_csr_basic<<<grid_dims, block_dims, 0, csr->stream[i]>>>(csr->thread_i_s_d[i], csr->thread_i_e_d[i], csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->x_d[i], csr->y_d[i]);
		// gpu_kernel_csr_flat<<<grid_dims, block_dims, 0, csr->stream[i]>>>(csr->thread_i_s_d[i], csr->thread_i_e_d[i], csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->x_d[i], csr->y_d[i]);
		gpu_kernel_csr_basic<<<grid_dims, block_dims, 0, csr->stream[i]>>>(csr->thread_i_s_d[i], csr->thread_i_e_d[i], csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->x_d[i], csr->y_d2 + i*csr->m);
	}

	gpuCudaErrorCheck(hipPeekAtLastError());
	for(int i=0; i<csr->num_streams; i++)
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream[0]));

		ValueType *ones_host, *ones_device;
		gpuCudaErrorCheck(hipHostMalloc(&ones_host, csr->num_streams * sizeof(ValueType)));
		for (int i=0; i<csr->num_streams; i++) ones_host[i] = 1.0;
		gpuCudaErrorCheck(hipMalloc(&ones_device, csr->num_streams * sizeof(ValueType)));	
		gpuCudaErrorCheck(hipMemcpyAsync(ones_device, ones_host, csr->num_streams * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[0]));
	
		ValueType  alpha = 1.0, beta = 0.0;
		gpuCublasErrorCheck(hipblasDgemv(csr->handle, HIPBLAS_OP_N, csr->m, csr->num_streams, &alpha, csr->y_d2, csr->m, ones_device, 1, &beta, csr->y_d_reduction, 1));

		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y, csr->y_d_reduction, csr->m * sizeof(csr->y), hipMemcpyDeviceToHost, csr->stream[0]));

		gpuCudaErrorCheck(hipHostFree(ones_host));
		gpuCudaErrorCheck(hipFree(ones_device));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream[0]));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[0]));
		if(TIME_IT){
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

