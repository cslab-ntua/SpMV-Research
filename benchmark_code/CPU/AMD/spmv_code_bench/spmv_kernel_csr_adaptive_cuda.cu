#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

// #ifndef NUM_STREAMS
// #define NUM_STREAMS 1
// #endif

#ifndef TIME_IT
#define TIME_IT 1
#endif

#ifndef VERIFIED
#define VERIFIED 1
#endif

INT_T spmv_csr_adaptive_rowblocks(INT_T *row_ptr, INT_T m, INT_T *row_blocks){
	row_blocks[0] = 0; 
	INT_T sum = 0; 
	INT_T last_i = 0; 
	INT_T cnt = 1;
	for (INT_T i = 1; i < m; i++) {
		// Count non-zeroes in this row 
		sum += row_ptr[i] - row_ptr[i-1];
		if (sum == BLOCK_SIZE){
			// This row fills up LOCAL_SIZE 
			last_i = i;
			row_blocks[cnt++] = i;
			sum = 0;
		}
		else if (sum > BLOCK_SIZE){
			if (i - last_i > 1) {
				// This extra row will not fit 
				row_blocks[cnt++] = i - 1;
				i--;
			}
			else if (i - last_i == 1){
				// This one row is too large
				row_blocks[cnt++] = i;
			}
			last_i = i;
			sum = 0;
		}
	}
	row_blocks[cnt++] = m;
	return cnt;
}

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * row_blocks;
	INT_T row_blocks_cnt;

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	INT_T * row_blocks_d;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;
	
	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_row_blocks;
	hipEvent_t endEvent_memcpy_row_blocks;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, block_size, max_threads_per_multiproc;
	// int num_streams;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		block_size = BLOCK_SIZE;

		row_blocks = (typeof(row_blocks)) malloc(m * sizeof(*row_blocks));
		row_blocks_cnt = spmv_csr_adaptive_rowblocks(ia, m, row_blocks);
		printf("%ld nnz, %d row_blocks ( %.0lf nnz/row_block )\n", nnz, row_blocks_cnt, nnz*1.0/row_blocks_cnt);

		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&row_blocks_d, row_blocks_cnt * sizeof(*row_blocks_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_row_blocks));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_row_blocks));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpy(ia_d, ia, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_row_blocks));
		gpuCudaErrorCheck(hipMemcpy(row_blocks_d, row_blocks, row_blocks_cnt * sizeof(*row_blocks_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_row_blocks));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpy(ja_d, ja, nnz * sizeof(*ja_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpy(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_row_blocks));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));

			float memcpyTime_cuda_ia, memcpyTime_cuda_row_blocks, memcpyTime_cuda_ja, memcpyTime_cuda_a;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_row_blocks, startEvent_memcpy_row_blocks, endEvent_memcpy_row_blocks));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			printf("(CUDA) Memcpy ia time = %.4lf ms, row_blocks time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_row_blocks, memcpyTime_cuda_ja, memcpyTime_cuda_a);
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(row_blocks);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(row_blocks_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_row_blocks));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_row_blocks));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_ADAPTIVE_b%d", csr->block_size);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================

__global__ void gpu_kernel_csr_adaptive(INT_T * ia, INT_T * ja, ValueType * a, INT_T * row_blocks, ValueType * restrict x, ValueType * restrict y)
{
	INT_T startRow = row_blocks[blockIdx.x];
	INT_T nextStartRow = row_blocks[blockIdx.x + 1];
	INT_T num_rows = nextStartRow - startRow;
	INT_T i = threadIdx.x;
	__shared__ volatile ValueType LDS[BLOCK_SIZE];
	
	// If the block consists of more than one row then run CSR Stream
	if (num_rows > 1) {
		// how many nonzeros does this rowblock hold?
		// they will be less than the BLOCK_SIZE (the size of LDS)
		int nnz = ia[nextStartRow] - ia[startRow];
		int col_offset = ia[startRow];

		// Each thread writes to shared memory the result of multiplication for one nonzero
		if (i < nnz)
			LDS[i] = a[col_offset + i] * x[ja[col_offset + i]];
 		// After all positions of LDS have been filled, proceed. 
		__syncthreads();
		
		// Threads that fall within a range sum up the partial results
		// Thread0 of the block will be assigned with the first row of the thread block (startRow+0) and then the next row will be (startRow+BLOCK_SIZE) etc...
		// How many rows per thread depends on how few nonzeros this specific block can hold...
		for (int k = startRow + i; k < nextStartRow; k += BLOCK_SIZE){
			ValueType temp = 0;
			// Sum partial results that this row (k) has results in LDS
			for (INT_T j = (ia[k] - col_offset); j < (ia[k + 1] - col_offset); j++)
				temp = temp + LDS[j];
			// And finally store result in the output y vector.
			y[k] = temp;
		}
	}
	// If the block consists of only one row then run CSR Vector
	else {
		// Thread ID in warp
		INT_T ia_Start = ia[startRow];
		INT_T ia_End   = ia[nextStartRow];
		ValueType sum  = 0;

		// Use all threads in a warp to accumulate multiplied elements
		// Due to the fact that each for loop starts from "ia_Start" + some i (the index inside the thread block) 
		// LDS will be filled with all partial results from this specific row
		// It may be underutilized, considering the fact that this row will consist of less than BLOCK_SIZE elements
		for (INT_T j = ia_Start + i; j < ia_End; j += BLOCK_SIZE){
			INT_T col = ja[j];
			sum += a[j] * x[col];
		}
		// write partial sum at position i (index in thread block) in the LDS array
		LDS[i] = sum;
		__syncthreads();

		// Reduce partial sums
		// reduce results as in 
		// (BS/2 sums)  LDS[i] = LDS[i] + LDS[i + BS/2];, LDS[i+1] = LDS[i+1] + LDS[i+1 + BS/2];
		// (BS/4 sums)  LDS[i] = LDS[i] + LDS[i + BS/4]
		// ...
		// (1 sum)      LDS[i] = LDS[i] + LDS[i+1]; and then finish
		for (int stride = BLOCK_SIZE >> 1; stride > 0; stride >>= 1) {
			__syncthreads();
			if (i < stride)
				LDS[i] += LDS[i + stride]; 
		}
		// Write result
		if (i == 0){
			y[startRow] = LDS[i];
		}
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	dim3 block_dims(csr->block_size);
	dim3 grid_dims(csr->row_blocks_cnt-1);
	printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);

	if (csr->x == NULL)
	{
		csr->x = x;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(csr->x_d, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}
	}

	if(VERIFIED){
		int num_loops = 1000;
		for(int k=0;k<num_loops;k++)
			gpu_kernel_csr_adaptive<<<grid_dims, block_dims>>>(csr->ia_d, csr->ja_d, csr->a_d, csr->row_blocks_d, csr->x_d, csr->y_d);
		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipDeviceSynchronize());
	}

	gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution));

	int num_loops = 128;
	double time_execution = time_it(1,
		for(int k=0;k<num_loops;k++){
			gpu_kernel_csr_adaptive<<<grid_dims, block_dims>>>(csr->ia_d, csr->ja_d, csr->a_d, csr->row_blocks_d, csr->x_d, csr->y_d);
			gpuCudaErrorCheck(hipPeekAtLastError());
			gpuCudaErrorCheck(hipDeviceSynchronize());
		}
	);

	double gflops = csr->nnz / time_execution * num_loops * 2 * 1e-9;
	printf("(DGAL timing) Execution time = %.4lf ms (%.4lf GFLOPS adaptive-b%d)\n", time_execution*1e3, gflops, csr->block_size);

	gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution));
	float executionTime_cuda;
	gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_execution));
	gpuCudaErrorCheck(hipEventElapsedTime(&executionTime_cuda, csr->startEvent_execution, csr->endEvent_execution));

	double gflops_cuda = csr->nnz / executionTime_cuda * num_loops * 2 * 1e-6;
	printf("(CUDA) Execution time = %.4lf ms (%.4lf GFLOPS @ %d block_size for %.2lf MB workload)\n", executionTime_cuda, gflops_cuda, csr->block_size, csr->mem_footprint/(1024*1024.0));

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y));
		gpuCudaErrorCheck(hipMemcpy(csr->y, csr->y_d, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

