#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;


struct COOArrays : Matrix_Format
{
	INT_T * rowind;      // the usual rowptr (of size m+1)
	INT_T * colind;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * rowind_dev;
	INT_T * colind_dev;
	ValueType * a_dev;

	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;
	hipsparseDnVecDescr_t vecX;
	hipsparseDnVecDescr_t vecY;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;

	COOArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), colind(ja), a(a)
	{
		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		hipMalloc(&rowind_dev, nnz * sizeof(*rowind_dev));
		hipMalloc(&colind_dev, nnz * sizeof(*colind_dev));
		hipMalloc(&a_dev, nnz * sizeof(*a_dev));
		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));

		rowind = (typeof(rowind)) malloc(nnz * sizeof(*rowind));

		#pragma omp parallel
		{
			long i, j, j_s, j_e;
			#pragma omp for
			for (i=0;i<nnz;i++)
			{
				rowind[i] = 0;
			}
			#pragma omp for
			for (i=0;i<m;i++)
			{
				j_s = ia[i];
				j_e = ia[i+1];
				for (j=j_s;j<j_e;j++)
					rowind[j] = i;
			}
		}

		hipMemcpy(rowind_dev, rowind, nnz * sizeof(*rowind_dev), hipMemcpyHostToDevice);
		hipMemcpy(colind_dev, colind, nnz * sizeof(*colind_dev), hipMemcpyHostToDevice);
		hipMemcpy(a_dev, a, nnz * sizeof(*a_dev), hipMemcpyHostToDevice);

		hipsparseCreate(&handle);
		// Create sparse matrix A in CSR format
		hipsparseCreateCoo(&matA, m, n, nnz, rowind_dev, colind_dev, a_dev, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda);
	}

	~COOArrays()
	{
		free(a);
		free(rowind);
		free(colind);

		// destroy matrix/vector descriptors
		hipsparseDestroySpMat(matA);
		hipsparseDestroyDnVec(vecX);
		hipsparseDestroyDnVec(vecY);
		hipsparseDestroy(handle);

		hipFree(rowind_dev);
		hipFree(colind_dev);
		hipFree(a_dev);

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(COOArrays * restrict coo, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);


void
COOArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct COOArrays * coo = new COOArrays(row_ptr, col_ind, values, m, n, nnz);
	coo->mem_footprint = nnz * (sizeof(ValueType) + 2 * sizeof(INT_T));
	coo->format_name = (char *) "CUSPARSE_COO";
	return coo;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


void
compute_csr(COOArrays * restrict coo, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (coo->x == NULL)
	{
		coo->x = x;
		hipMemcpy(coo->x_dev, coo->x, coo->n * sizeof(*coo->x), hipMemcpyHostToDevice);
		// Create dense vector X
		hipsparseCreateDnVec(&coo->vecX, coo->n, coo->x_dev, ValueTypeCuda);
		// Create dense vector y
		hipsparseCreateDnVec(&coo->vecY, coo->m, coo->y_dev, ValueTypeCuda);
		// Allocate an external buffer if needed
		hipsparseSpMV_bufferSize(coo->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA, coo->vecX, &beta, coo->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &coo->bufferSize);
		hipMalloc(&coo->dBuffer, coo->bufferSize);
	}
	// Execute SpMV
	hipsparseSpMV(coo->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA, coo->vecX, &beta, coo->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, coo->dBuffer);
	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));
	if (coo->y == NULL)
	{
		coo->y = y;
		hipMemcpy(coo->y, coo->y_dev, coo->m * sizeof(*coo->y), hipMemcpyDeviceToHost);
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
COOArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
COOArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

