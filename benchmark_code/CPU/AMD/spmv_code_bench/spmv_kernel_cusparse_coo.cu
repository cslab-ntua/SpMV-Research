#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cusparse_util.h"
#ifdef __cplusplus
}
#endif


#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;

#ifndef TIME_IT
#define TIME_IT 0
#endif

struct COOArrays : Matrix_Format
{
	INT_T * rowind;      // the usual rowptr (of size m+1)
	INT_T * colind;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * rowind_d;
	INT_T * colind_d;
	ValueType * a_d;

	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	hipsparseDnVecDescr_t vecX;
	hipsparseDnVecDescr_t vecY;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipEvent_t startEvent_memcpy_rowind;
	hipEvent_t endEvent_memcpy_rowind;
	hipEvent_t startEvent_memcpy_colind;
	hipEvent_t endEvent_memcpy_colind;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_create_matA;
	hipEvent_t endEvent_create_matA;
	hipEvent_t startEvent_spmv_buffersize;
	hipEvent_t endEvent_spmv_buffersize;

	hipEvent_t startEvent_create_vecX;
	hipEvent_t endEvent_create_vecX;
	hipEvent_t startEvent_create_vecY;
	hipEvent_t endEvent_create_vecY;


	COOArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), colind(ja), a(a)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		// printf("max_smem_per_block=%d\n", max_smem_per_block);
		// printf("multiproc_count=%d\n", multiproc_count);
		// printf("max_threads_per_block=%d\n", max_threads_per_block);
		// printf("warp_size=%d\n", warp_size);
		// printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		gpuCudaErrorCheck(hipMalloc(&rowind_d, nnz * sizeof(*rowind_d)));
		gpuCudaErrorCheck(hipMalloc(&colind_d, nnz * sizeof(*colind_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCusparseErrorCheck(hipsparseCreate(&handle));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_spmv_buffersize));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		rowind = (typeof(rowind)) malloc(nnz * sizeof(*rowind));
		#pragma omp parallel
		{
			long i, j, j_s, j_e;
			#pragma omp for
			for (i=0;i<nnz;i++)
			{
				rowind[i] = 0;
			}
			#pragma omp for
			for (i=0;i<m;i++)
			{
				j_s = ia[i];
				j_e = ia[i+1];
				for (j=j_s;j<j_e;j++)
					rowind[j] = i;
			}
		}

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_rowind));
		gpuCudaErrorCheck(hipMemcpy(rowind_d, rowind, nnz * sizeof(*rowind_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_rowind));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_colind));
		gpuCudaErrorCheck(hipMemcpy(colind_d, colind, nnz * sizeof(*colind_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_colind));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpy(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		// Create sparse matrix A in COO format
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_create_matA));
		gpuCusparseErrorCheck(hipsparseCreateCoo(&matA, m, n, nnz, rowind_d, colind_d, a_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_create_matA));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_create_matA));

			float memcpyTime_cuda_rowind, memcpyTime_cuda_colind, memcpyTime_cuda_a, create_matA_Time;//memcpyTime_cuda_thread_i_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_rowind, startEvent_memcpy_rowind, endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_colind, startEvent_memcpy_colind, endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_matA_Time, startEvent_create_matA, endEvent_create_matA));

			printf("(CUDA) Memcpy rowind time = %.4lf ms, colind time = %.4lf ms, a time = %.4lf ms, matA time = %.4lf ms\n", memcpyTime_cuda_rowind, memcpyTime_cuda_colind, memcpyTime_cuda_a, create_matA_Time);
		}
	}

	~COOArrays()
	{
		free(a);
		free(rowind);
		free(colind);

		// destroy matrix/vector descriptors
		gpuCusparseErrorCheck(hipsparseDestroySpMat(matA));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecX));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecY));
		gpuCusparseErrorCheck(hipsparseDestroy(handle));

		gpuCudaErrorCheck(hipFree(rowind_d));
		gpuCudaErrorCheck(hipFree(colind_d));
		gpuCudaErrorCheck(hipFree(a_d));

		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(dBuffer));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_rowind));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_colind));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_spmv_buffersize));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_coo(COOArrays * restrict coo, ValueType * restrict x , ValueType * restrict y);


void
COOArrays::spmv(ValueType * x, ValueType * y)
{
	compute_coo(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct COOArrays * coo = new COOArrays(row_ptr, col_ind, values, m, n, nnz);
	coo->mem_footprint = nnz * (sizeof(ValueType) + 2 * sizeof(INT_T));
	coo->format_name = (char *) "CUSPARSE_COO";
	return coo;
}


//==========================================================================================================================================
//= COO Custom
//==========================================================================================================================================


void
compute_coo(COOArrays * restrict coo, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (coo->x == NULL)
	{
		coo->x = x;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(coo->x_d, coo->x, coo->n * sizeof(*coo->x), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_x));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_x, coo->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		// Create dense vector X
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_create_vecX));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecX, coo->n, coo->x_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_create_vecX));

		// Create dense vector y
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_create_vecY));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&coo->vecY, coo->m, coo->y_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_create_vecY));

		// Allocate an external buffer if needed
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_spmv_buffersize));
		gpuCusparseErrorCheck(hipsparseSpMV_bufferSize(coo->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA, coo->vecX, &beta, coo->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &coo->bufferSize));
		gpuCudaErrorCheck(hipMalloc(&coo->dBuffer, coo->bufferSize));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_spmv_buffersize));
		// printf("SpMV_bufferSize = %zu bytes\n", coo->bufferSize, coo->bufferSize); // size of the workspace that is needed by hipsparseSpMV()

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_spmv_buffersize));
			float create_vecX_time, create_vecY_time, spmv_buffersize_time;
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecX_time, coo->startEvent_create_vecX, coo->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecY_time, coo->startEvent_create_vecY, coo->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventElapsedTime(&spmv_buffersize_time, coo->startEvent_spmv_buffersize, coo->endEvent_spmv_buffersize));
			printf("(CUDA) Create vecX time = %.4lf ms, vecY time = %.4lf ms, spmv_buffersize time = %.4lf (SpMV_bufferSize = %zu)\n", create_vecX_time, create_vecY_time, spmv_buffersize_time, coo->bufferSize);
		}
	}

	gpuCusparseErrorCheck(hipsparseSpMV(coo->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, coo->matA, coo->vecX, &beta, coo->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, coo->dBuffer));
	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (coo->y == NULL)
	{
		coo->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->startEvent_memcpy_y));
		gpuCudaErrorCheck(hipMemcpy(coo->y, coo->y_d, coo->m * sizeof(*coo->y), hipMemcpyDeviceToHost));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(coo->endEvent_memcpy_y));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(coo->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, coo->startEvent_memcpy_y, coo->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
COOArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
COOArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

