#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cublas_util.h"
	#include "aux/csr_util.h"
	#include "aux/csr_converter.h"
	#include "aux/csc_util.h"
	#include "aux/csc_converter.h"
#ifdef __cplusplus
}
#endif


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

#ifndef NUM_THREADS
#define NUM_THREADS 1024
#endif

// int row_cluster_size = 256;
// int row_cluster_size = 192;
// int row_cluster_size = 128;
// int row_cluster_size = 64;
// int row_cluster_size = 32;
// int row_cluster_size = 16;
// int row_cluster_size = 8;
#ifndef ROW_CLUSTER_SIZE
#define ROW_CLUSTER_SIZE 4
#endif

#ifndef NUM_STREAMS
#define NUM_STREAMS 1
#endif

#ifndef TIME_IT
#define TIME_IT 0
#endif

INT_T * thread_block_i_s[NUM_STREAMS];
INT_T * thread_block_i_e[NUM_STREAMS];
unsigned char * rel_row_idx[NUM_STREAMS];

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)
	INT_T * ia_h[NUM_STREAMS];
	INT_T * ja_h[NUM_STREAMS];
	ValueType * a_h[NUM_STREAMS];
	INT_T * thread_block_i_s_h[NUM_STREAMS];
	INT_T * thread_block_i_e_h[NUM_STREAMS];
	unsigned char * rel_row_idx_h[NUM_STREAMS];

	INT_T * ia_d[NUM_STREAMS];
	INT_T * ja_d[NUM_STREAMS];
	ValueType * a_d[NUM_STREAMS];
	INT_T * thread_block_i_s_d[NUM_STREAMS];
	INT_T * thread_block_i_e_d[NUM_STREAMS];
	unsigned char * rel_row_idx_d[NUM_STREAMS];

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_h[NUM_STREAMS];
	ValueType * y_h[NUM_STREAMS];
	ValueType * x_d[NUM_STREAMS];
	// ValueType * y_d[NUM_STREAMS];
	ValueType * y_d2;
	ValueType * y_d_reduction;

	hipStream_t stream[NUM_STREAMS];
	INT_T n_stream[NUM_STREAMS];
	INT_T nnz_stream[NUM_STREAMS];

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution[NUM_STREAMS];
	hipEvent_t endEvent_execution[NUM_STREAMS];
	
	hipEvent_t startEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ia[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_ja[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_a[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_block_i_s[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_block_i_s[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_thread_block_i_e[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_thread_block_i_e[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_rel_row_idx[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_rel_row_idx[NUM_STREAMS];

	hipEvent_t startEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t endEvent_memcpy_x[NUM_STREAMS];
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipblasHandle_t handle;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
	int num_threads;
	int block_size;
	int num_blocks;

	int row_cluster_size;

	int num_streams;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		double time_balance;

		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		// block_size = warp_size / 2;
		block_size = warp_size;
		// block_size = warp_size * 2;
		// block_size = warp_size * 4;
		row_cluster_size = ROW_CLUSTER_SIZE;

		// num_threads = 128;
		// num_threads = 1ULL << 10;
		// num_threads = 3584;
		// num_threads = 1ULL << 12;
		// num_threads = 1ULL << 13;
		// num_threads = 1ULL << 14;
		// num_threads = 1ULL << 15;
		// num_threads = 1ULL << 16;
		// num_threads = 1ULL << 17;
		// num_threads = 1ULL << 21;
		// num_threads = 1ULL << 22;
		num_threads = NUM_THREADS;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		num_streams = NUM_STREAMS;

		/********************************************************************************************************/
		printf("/********************************************************************************************************/\n");
		// Convert CSR representation ton CSC
		INT_T * row_indices; //for CSC format
		INT_T * row_idx;
		INT_T * col_ptr;
		ValueType * val_c;

		row_indices = (typeof(row_indices)) malloc(nnz * sizeof(*row_indices));
		row_idx = (typeof(row_idx)) malloc(nnz * sizeof(*row_idx));
		col_ptr = (typeof(col_ptr)) malloc((n+1) * sizeof(*col_ptr));
		val_c = (typeof(val_c)) malloc(nnz * sizeof(*val_c));

		double time = time_it(1,
			csr_row_indices(ia, ja, m, n, nnz, &row_indices);
			coo_to_csc(row_indices, ja, a, m, n, nnz, row_idx, col_ptr, val_c, 1);
			free(row_indices);
		);
		printf("time coo_to_csc = %g ms\n", time*1e3);

		INT_T *local_stream_j_s = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_s));
		INT_T *local_stream_j_e = (INT_T *) malloc(num_streams * sizeof(*local_stream_j_e));
		time_balance = time_it(1,
			for (int i=0;i<num_streams;i++)
				loop_partitioner_balance_prefix_sums(num_streams, i, col_ptr, n, nnz, &local_stream_j_s[i], &local_stream_j_e[i]);
		);

		int cnt=0, cnt2=0;
		for(int i=0; i<num_streams; i++){
			nnz_stream[i] = col_ptr[local_stream_j_e[i]] - col_ptr[local_stream_j_s[i]];
			n_stream[i] = local_stream_j_e[i] - local_stream_j_s[i];
			// printf("local_stream[%d] = %d - %d (%d cols) (%d nnz)\n", i, local_stream_j_s[i], local_stream_j_e[i], n_stream[i], nnz_stream[i]);

			cnt  += nnz_stream[i];
			cnt2 += n_stream[i];
		}

		printf("balance time (col) = %g ms\n", time_balance*1e3);

		INT_T * row_idx_stream[num_streams];
		INT_T * col_ptr_stream[num_streams];
		ValueType * val_c_stream[num_streams];
		
		double time_memcpy_stream_locals = time_it(1,
		for(int i=0; i<num_streams; i++){
			col_ptr_stream[i] = (INT_T *) malloc((n_stream[i]+1) * sizeof(INT_T));
			row_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_c_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			memcpy(col_ptr_stream[i], col_ptr + local_stream_j_s[i], (n_stream[i] + 1) * sizeof(INT_T));
			// col_ptr needs to be fixed, so that it will start from 0 again...
			for(int j=0; j<n_stream[i]+1; j++)
				col_ptr_stream[i][j] -= col_ptr[local_stream_j_s[i]];
			memcpy(row_idx_stream[i], row_idx + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(INT_T));
			memcpy(val_c_stream[i], val_c + col_ptr[local_stream_j_s[i]], nnz_stream[i] * sizeof(ValueType));
		}
		);
		printf("time_memcpy_stream_locals = %lf ms\n", time_memcpy_stream_locals*1e3);
		free(local_stream_j_s);
		free(local_stream_j_e);

		INT_T * row_ptr_stream[num_streams];
		INT_T * col_idx_stream[num_streams];
		ValueType * val_stream[num_streams];

		for(int i=0; i<num_streams; i++){
			INT_T * col_indices;
			csc_col_indices(row_idx_stream[i], col_ptr_stream[i], m, n_stream[i], nnz_stream[i], &col_indices);

			row_ptr_stream[i] = (INT_T *) malloc((m+1) * sizeof(INT_T));
			col_idx_stream[i] = (INT_T *) malloc(nnz_stream[i] * sizeof(INT_T));
			val_stream[i] = (ValueType *) malloc(nnz_stream[i] * sizeof(ValueType));

			coo_to_csr(row_idx_stream[i], col_indices, val_c_stream[i], m, n_stream[i], nnz_stream[i], row_ptr_stream[i], col_idx_stream[i], val_stream[i], 1, 0);
			// REMOVE THIS
			/*
			if(0){
				char * replace_str;
				replace_str = (char *)malloc(100*sizeof(char));
				sprintf(replace_str, "stream_%d", i);
				char * file_fig = fig_name_gen("matrix.mtx", replace_str);
				printf("file_fig = %s\n", file_fig);

				long num_pixels = 4096;
				long num_pixels_x = (n_stream[i] < num_pixels) ? n_stream[i] : num_pixels;
				long num_pixels_y = (m < num_pixels) ? m : num_pixels;
				if(m!=n_stream[i]) {
					double ratio = n_stream[i]*1.0 / m;
					// if((ratio>16.0) || (ratio<(1/16.0)))
					if(ratio>16.0)
						ratio=16.0;
					if(ratio < (1/16.0))
						ratio=1/16.0;
					// in order to keep both below 1024
					if(ratio>1) // n > m
						num_pixels_y = (1/ratio) * num_pixels_x;
					else // m > n
						num_pixels_x = ratio * num_pixels_y;
				}
				csr_plot(file_fig, row_ptr_stream[i], col_idx_stream[i], val_stream[i], m, n_stream[i], nnz_stream[i], 0, num_pixels_x, num_pixels_y);
			}
			*/

			free(col_indices);
		}

		for(int i=0; i<num_streams; i++){
			free(row_idx_stream[i]);
			free(col_ptr_stream[i]);
			free(val_c_stream[i]);
		}
		free(row_idx);
		free(col_ptr);
		free(val_c);

		printf("/********************************************************************************************************/\n");
		/********************************************************************************************************/

		for(int i=0; i<num_streams; i++){
			thread_block_i_s[i] = (INT_T *) malloc(num_blocks * sizeof(INT_T));
			thread_block_i_e[i] = (INT_T *) malloc(num_blocks * sizeof(INT_T));
			for (int j=0; j<num_blocks; j++)
			{
				// loop_partitioner_balance_iterations(num_blocks, i, 0, m, &thread_block_i_s[i], &thread_block_i_e[i]);
				loop_partitioner_balance_prefix_sums(num_blocks, j, row_ptr_stream[i], m, nnz_stream[i], &(thread_block_i_s[i][j]), &(thread_block_i_e[i][j]));
			}

			rel_row_idx[i] = (unsigned char *) malloc(nnz_stream[i] * sizeof(unsigned char));
			#pragma omp parallel
			{
				long ii, i_s, i_e, i_rel, j, k;
				#pragma omp for
				for (k=0;k<num_blocks;k++)
				{
					i_s = thread_block_i_s[i][k];
					i_e = thread_block_i_e[i][k];
					for (ii=i_s;ii<i_e;ii++)
					{
						i_rel = (ii - i_s) % row_cluster_size;
						for (j=row_ptr_stream[i][ii];j<row_ptr_stream[i][ii+1];j++)
						{
							rel_row_idx[i][j] = i_rel;
						}
					}
				}
			}
		}

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipMalloc(&ia_d[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&ja_d[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&a_d[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&x_d[i], n_stream[i] * sizeof(ValueType)));
			// gpuCudaErrorCheck(hipMalloc(&y_d[i], m * sizeof(ValueType)));
			gpuCudaErrorCheck(hipMalloc(&thread_block_i_s_d[i], num_blocks * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&thread_block_i_e_d[i], num_blocks * sizeof(INT_T)));
			gpuCudaErrorCheck(hipMalloc(&rel_row_idx_d[i], nnz_stream[i] * sizeof(unsigned char)));
		}
		gpuCudaErrorCheck(hipMalloc(&y_d2, m * num_streams * sizeof(ValueType)));
		gpuCudaErrorCheck(hipMalloc(&y_d_reduction, m * sizeof(ValueType)));
		gpuCublasErrorCheck(hipblasCreate(&handle));

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipHostMalloc(&ia_h[i], (m+1) * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&ja_h[i], nnz_stream[i] * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&a_h[i], nnz_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&x_h[i], n_stream[i] * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&y_h[i], m * sizeof(ValueType)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_s_h[i], num_blocks * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&thread_block_i_e_h[i], num_blocks * sizeof(INT_T)));
			gpuCudaErrorCheck(hipHostMalloc(&rel_row_idx_h[i], nnz_stream[i] * sizeof(unsigned char)));
		}

		double time_memcpy = time_it(1,
		for(int i=0; i<num_streams; i++){
			memcpy(ia_h[i], row_ptr_stream[i], (m + 1) * sizeof(INT_T));
			memcpy(ja_h[i], col_idx_stream[i], nnz_stream[i] * sizeof(INT_T));
			memcpy(a_h[i], val_stream[i], nnz_stream[i] * sizeof(ValueType));
			memcpy(thread_block_i_s_h[i], thread_block_i_s[i], num_blocks * sizeof(INT_T));
			memcpy(thread_block_i_e_h[i], thread_block_i_e[i], num_blocks * sizeof(INT_T));
			memcpy(rel_row_idx_h[i], rel_row_idx[i], nnz_stream[i] * sizeof(unsigned char));
		}
		);
		printf("time_memcpy (ia_h, ja_h, a_h, thr_i_s, thr_i_e) = %lf ms\n", time_memcpy*1e3);

		// cuda events for timing measurements
		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipStreamCreate(&stream[i]));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_execution[i]));
		}
		gpuCublasErrorCheck(hipblasSetStream(handle, stream[0]));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_rel_row_idx[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_rel_row_idx[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_thread_block_i_e[i]));

				gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x[i]));
			}
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		for(int i=0; i<num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ia_d[i], ia_h[i], (m+1) * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(ja_d[i], ja_h[i], nnz_stream[i] * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(a_d[i], a_h[i], nnz_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_s[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_s_d[i], thread_block_i_s_h[i], num_blocks * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_s[i], stream[i]));
			
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_thread_block_i_e[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(thread_block_i_e_d[i], thread_block_i_e_h[i], num_blocks * sizeof(INT_T), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_thread_block_i_e[i], stream[i]));

			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_rel_row_idx[i], stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(rel_row_idx_d[i], rel_row_idx_h[i], nnz_stream[i] * sizeof(unsigned char), hipMemcpyHostToDevice, stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_rel_row_idx[i], stream[i]));
		}

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipStreamSynchronize(stream[i]));
				float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_rel_row_idx, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia[i], endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja[i], endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a[i], endEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_rel_row_idx, startEvent_memcpy_rel_row_idx[i], endEvent_memcpy_rel_row_idx[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_s, startEvent_memcpy_thread_block_i_s[i], endEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_thread_block_i_e, startEvent_memcpy_thread_block_i_e[i], endEvent_memcpy_thread_block_i_e[i]));
				printf("(CUDA) (stream %d) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, rel_row_idx time = %.4lf ms, thread_block_s = %.4lf ms, thread_block_e = %.4lf ms\n", i, memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, memcpyTime_cuda_rel_row_idx, memcpyTime_cuda_thread_block_i_s, memcpyTime_cuda_thread_block_i_e);
			}
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);
		for(int i=0; i<num_streams; i++){
			free(thread_block_i_s[i]);
			free(thread_block_i_e[i]);
			free(rel_row_idx[i]);
		}

		for(int i=0; i<num_streams; i++){
			gpuCudaErrorCheck(hipFree(ia_d[i]));
			gpuCudaErrorCheck(hipFree(ja_d[i]));
			gpuCudaErrorCheck(hipFree(a_d[i]));
			gpuCudaErrorCheck(hipFree(x_d[i]));
			// gpuCudaErrorCheck(hipFree(y_d[i]));
			gpuCudaErrorCheck(hipFree(thread_block_i_s_d[i]));
			gpuCudaErrorCheck(hipFree(thread_block_i_e_d[i]));
			gpuCudaErrorCheck(hipFree(rel_row_idx_d[i]));

			gpuCudaErrorCheck(hipHostFree(ia_h[i]));
			gpuCudaErrorCheck(hipHostFree(ja_h[i]));
			gpuCudaErrorCheck(hipHostFree(a_h[i]));
			gpuCudaErrorCheck(hipHostFree(x_h[i]));
			gpuCudaErrorCheck(hipHostFree(y_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_block_i_s_h[i]));
			gpuCudaErrorCheck(hipHostFree(thread_block_i_e_h[i]));
			gpuCudaErrorCheck(hipHostFree(rel_row_idx_h[i]));

			gpuCudaErrorCheck(hipStreamDestroy(stream[i]));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_execution[i]));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_execution[i]));
		}
		gpuCudaErrorCheck(hipFree(y_d2));
		gpuCudaErrorCheck(hipFree(y_d_reduction));
		gpuCublasErrorCheck(hipblasDestroy(handle));

		if(TIME_IT){
			for(int i=0; i<num_streams; i++){
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_s[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_thread_block_i_e[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_rel_row_idx[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_rel_row_idx[i]));

				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja[i]));
				gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a[i]));
				gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a[i]));
			}
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);

void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	char *format_name;
	format_name = (char *)malloc(100*sizeof(char));
	snprintf(format_name, 100, "Custom_CSR_CUDA_BUFFER_s%d_t%d_rc_%d", csr->num_streams, csr->num_threads, csr->row_cluster_size);
	csr->format_name = format_name;
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


__global__ void gpu_kernel_csr_basic(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y, unsigned char * rel_row_idx, int row_cluster_size)
{
	extern __shared__ ValueType sdata[];
	int tidg = cuda_get_thread_num();
	int tidb = threadIdx.x;
	int block_id = blockIdx.x;
	int block_size = blockDim.x;
	long i, i_s, i_e, i_rel, i_rel_e, j, j_s, j_e, k;
	ValueType sum;
	for (i=0;i<row_cluster_size;i++)
		sdata[i*block_size + tidb] = 0;
	__syncthreads();
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	// printf("%d,%d: bs=%d , bid=%d , %ld %ld\n", tidg, tidb, block_size, block_id, i_s, i_e);
	for (k=i_s;k<i_e;k+=row_cluster_size)
	{
		i_rel_e = k + row_cluster_size > i_e ? i_e - k : row_cluster_size;
		j_s = ia[k];
		j_e = ia[k+i_rel_e];
		for (j=j_s+tidb;j<j_e;j+=block_size)
		{
			i_rel = rel_row_idx[j];
			sdata[i_rel*block_size + tidb] += a[j] * x[ja[j]];
		}

		__syncthreads();

		for (i_rel=tidb;i_rel<i_rel_e;i_rel+=block_size)
		{
			sum = 0;
			for (j=0;j<block_size;j++)
			{
				sum += sdata[i_rel*block_size + j];
				sdata[i_rel*block_size + j] = 0;
			}
			y[k + i_rel] = sum;
		}

		// for (i_rel=0;i_rel<i_rel_e;i_rel++)
		// {
			// sum = 0;
			// for (j=1;j<block_size;j*=2)
			// {
				// if (tidb % (2*j) == 0)
				// {
					// sdata[i_rel*block_size + tidb] += sdata[i_rel*block_size + tidb + j];
					// sdata[i_rel*block_size + tidb + j] = 0;
				// }
				// __syncthreads();
			// }
			// if (tidb == 0)
			// {
				// y[k + i_rel] = sdata[i_rel*block_size];
				// sdata[i_rel*block_size] = 0;
			// }
		// }

		__syncthreads();
	}
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);
	// printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n", grid_dims.x, grid_dims.y, grid_dims.z, block_dims.x, block_dims.y, block_dims.z);
	// long shared_mem_size = block_size * sizeof(*C_d);

	if (csr->x == NULL)
	{
		csr->x = x;
		int offset = 0;
		for(int i=0; i<csr->num_streams; i++){
			memcpy(csr->x_h[i], x + offset, csr->n_stream[i] * sizeof(ValueType));
			offset += csr->n_stream[i];
		}

		for(int i=0; i<csr->num_streams; i++){
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x[i], csr->stream[i]));
			gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d[i], csr->x_h[i], csr->n_stream[i] * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[i]));
			if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x[i], csr->stream[i]));
		}

		for(int i=0; i<csr->num_streams; i++)
			gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

		if(TIME_IT){
			for(int i=0; i<csr->num_streams; i++){
				float memcpyTime_cuda;
				gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x[i], csr->endEvent_memcpy_x[i]));
				printf("(CUDA) (stream %d) Memcpy x time = %.4lf ms\n", i, memcpyTime_cuda);
			}
		}
	}

	for(int i=0; i<csr->num_streams; i++){
		gpu_kernel_csr_basic<<<grid_dims, block_dims, (csr->row_cluster_size*block_size*sizeof(ValueType)), csr->stream[i]>>>(csr->thread_block_i_s_d[i], csr->thread_block_i_e_d[i], csr->ia_d[i], csr->ja_d[i], csr->a_d[i], csr->x_d[i], csr->y_d2 + i*csr->m, csr->rel_row_idx_d[i], csr->row_cluster_size);
	}

	gpuCudaErrorCheck(hipPeekAtLastError());
	for(int i=0; i<csr->num_streams; i++)
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[i]));

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y, csr->stream[0]));

		ValueType *ones_host, *ones_device;
		gpuCudaErrorCheck(hipHostMalloc(&ones_host, csr->num_streams * sizeof(ValueType)));
		for (int i=0; i<csr->num_streams; i++) ones_host[i] = 1.0;
		gpuCudaErrorCheck(hipMalloc(&ones_device, csr->num_streams * sizeof(ValueType)));	
		gpuCudaErrorCheck(hipMemcpyAsync(ones_device, ones_host, csr->num_streams * sizeof(ValueType), hipMemcpyHostToDevice, csr->stream[0]));
	
		ValueType  alpha = 1.0, beta = 0.0;
		gpuCublasErrorCheck(hipblasDgemv(csr->handle, HIPBLAS_OP_N, csr->m, csr->num_streams, &alpha, csr->y_d2, csr->m, ones_device, 1, &beta, csr->y_d_reduction, 1));

		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y, csr->y_d_reduction, csr->m * sizeof(csr->y), hipMemcpyDeviceToHost, csr->stream[0]));

		gpuCudaErrorCheck(hipHostFree(ones_host));
		gpuCudaErrorCheck(hipFree(ones_device));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y, csr->stream[0]));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream[0]));
		if(TIME_IT){
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

