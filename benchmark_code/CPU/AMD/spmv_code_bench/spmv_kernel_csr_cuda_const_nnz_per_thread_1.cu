#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


using namespace cooperative_groups;


#define NNZ_PER_THREAD  4


INT_T * thread_block_i_s = NULL;
INT_T * thread_block_i_e = NULL;

INT_T * thread_block_j_s = NULL;
INT_T * thread_block_j_e = NULL;


INT_T * thread_block_i_s_dev = NULL;
INT_T * thread_block_i_e_dev = NULL;

INT_T * thread_block_j_s_dev = NULL;
INT_T * thread_block_j_e_dev = NULL;


extern int prefetch_distance;

double * thread_time_compute, * thread_time_barrier;

void
cuda_push_duplicate_base(void ** dst_ptr, void * src, long bytes)
{
	hipMalloc(dst_ptr, bytes);
	hipMemcpy(*((char **) dst_ptr), src, bytes, hipMemcpyHostToDevice);
}
#define cuda_push_duplicate(dst_ptr, src, bytes) cuda_push_duplicate_base((void **) dst_ptr, src, bytes)


struct CSRArrays : Matrix_Format
{
	INT_T * row_ptr;
	INT_T * ia;
	INT_T * ja;
	ValueType * a;

	INT_T * row_ptr_dev;
	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	ValueType * multres_dev;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc, max_block_dim_x, max_num_threads;
	int num_threads;
	int block_size;
	int num_blocks;

	CSRArrays(INT_T * row_ptr, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), row_ptr(row_ptr), ja(ja), a(a)
	{
		double time_balance;
		long i;

		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		hipDeviceGetAttribute(&max_block_dim_x, hipDeviceAttributeMaxBlockDimX, 0);
		max_num_threads = max_threads_per_multiproc * multiproc_count;
		printf("max_smem_per_block(bytes)=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);
		printf("max_block_dim_x=%d\n", max_block_dim_x);
		printf("max_num_threads=%d\n", max_num_threads);

		// block_size = 32;
		// block_size = 64;
		// block_size = 128;
		// block_size = 256;
		// block_size = 512;
		block_size = 1024;

		num_threads = (nnz + NNZ_PER_THREAD - 1) / NNZ_PER_THREAD;

		num_threads = ((num_threads + block_size - 1) / block_size) * block_size;

		num_blocks = num_threads / block_size;

		printf("num_threads=%d, block_size=%d, num_blocks=%d\n", num_threads, block_size, num_blocks);

		thread_block_i_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_s));
		thread_block_i_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_i_e));
		thread_block_j_s = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_s));
		thread_block_j_e = (INT_T *) malloc(num_blocks * sizeof(*thread_block_j_e));
		time_balance = time_it(1,
			long lower_boundary;
			// for (i=0;i<num_blocks;i++)
			// {
				// loop_partitioner_balance_iterations(num_blocks, i, 0, nnz, &thread_block_j_s[i], &thread_block_j_e[i]);
				// macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				// thread_block_i_s[i] = lower_boundary;
			// }
			long nnz_per_block = block_size * NNZ_PER_THREAD;
			for (i=0;i<num_blocks;i++)
			{
				thread_block_j_s[i] = nnz_per_block * i;
				thread_block_j_e[i] = nnz_per_block * (i+ 1);
				if (thread_block_j_s[i] > nnz)
					thread_block_j_s[i] = nnz;
				if (thread_block_j_e[i] > nnz)
					thread_block_j_e[i] = nnz;
				macros_binary_search(row_ptr, 0, m, thread_block_j_s[i], &lower_boundary, NULL);           // Index boundaries are inclusive.
				thread_block_i_s[i] = lower_boundary;
			}
			for (i=0;i<num_blocks;i++)
			{
				if (i == num_blocks - 1)   // If we calculate each thread's boundaries individually some empty rows might be unassigned.
					thread_block_i_e[i] = m;
				else
					thread_block_i_e[i] = thread_block_i_s[i+1] + 1;
				if ((thread_block_j_s[i] >= row_ptr[thread_block_i_e[i]]) || (thread_block_j_s[i] < row_ptr[thread_block_i_s[i]]))
					error("bad binary search of row start: i=%d j:[%d, %d] j=%d", thread_block_i_s[i], row_ptr[thread_block_i_s[i]], row_ptr[thread_block_i_e[i]], thread_block_j_s[i]);
			}
		);
		printf("balance time = %g\n", time_balance);

		ia = (typeof(ia)) malloc(nnz * sizeof(*ia));
		_Pragma("omp parallel")
		{
			long i, j;
			_Pragma("omp for")
			for (i=0;i<m;i++)
			{
				for (j=row_ptr[i];j<row_ptr[i+1];j++)
				{
					ia[j] = i;
				}
			}
		}

		cuda_push_duplicate(&row_ptr_dev, row_ptr, (m+1) * sizeof(*row_ptr_dev));
		cuda_push_duplicate(&ia_dev, ia, nnz * sizeof(*ia_dev));
		cuda_push_duplicate(&ja_dev, ja, nnz * sizeof(*ja_dev));
		cuda_push_duplicate(&a_dev, a, nnz * sizeof(*a_dev));
		hipMalloc(&multres_dev, nnz * sizeof(*y_dev));

		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));

		cuda_push_duplicate(&thread_block_i_s_dev, thread_block_i_s, num_blocks * sizeof(*thread_block_i_s_dev));
		cuda_push_duplicate(&thread_block_i_e_dev, thread_block_i_e, num_blocks * sizeof(*thread_block_i_e_dev));
		cuda_push_duplicate(&thread_block_j_s_dev, thread_block_j_s, num_blocks * sizeof(*thread_block_j_s_dev));
		cuda_push_duplicate(&thread_block_j_e_dev, thread_block_j_e, num_blocks * sizeof(*thread_block_j_e_dev));

	}

	~CSRArrays()
	{
		free(a);
		free(row_ptr);
		free(ia);
		free(ja);
		free(thread_block_i_s);
		free(thread_block_i_e);

		hipFree(row_ptr_dev);
		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	// for (long i=0;i<10;i++)
		// printf("%d\n", row_ptr[i]);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Custom_CSR_CUDA_reduce";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int block_size = blockDim.x;
	int row = ia_buf[tidb];
	int k;
	for (k=1;k<block_size;k*=2)
	{
		if ((tidb & (2*k-1)) == k-1)
		{
			ValueType val = val_buf[tidb];
			if (row == ia_buf[tidb+k])
			{
				val_buf[tidb+k] += val;
				// val_buf[tidb] = 0;
			}
			else
			{
				atomicAdd(&y[row], val);
				// y[row] += val;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[block_size-1]], val_buf[block_size-1]);
} */


/* inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int block_size = blockDim.x;
	int k;
	INT_T row = ia_buf[tidb];
	for (k=1;k<block_size;k*=2)
	{
		if ((tidb & (2*k-1)) == 0)
		{
			INT_T row_next = ia_buf[tidb+k];
			ValueType val_next = val_buf[tidb+k];
			if (row == row_next)
			{
				val_buf[tidb] += val_next;
			}
			else
			{
				atomicAdd(&y[row], val_buf[tidb]);
				val_buf[tidb] = val_next;
				ia_buf[tidb] = row_next;
			}
		}
		__syncthreads();
	}
	if (tidb == 0)
		atomicAdd(&y[ia_buf[0]], val_buf[0]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int row = ia_buf[tidl];
	ValueType val;
	int k;
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		// val = val_buf[tidl];
		// if ((tidl & (2*k-1)) == k-1)
		// {
			// if (tidl >= k && row == ia_buf[tidl-k])
			// {
				// val_buf[tidl-k] += val;
				// val = 0;
			// }
		// }
		// g.sync();
		// if ((tidl & (2*k-1)) == k-1 && val != 0)
		// {
			// if (row == ia_buf[tidl+k])
			// {
				// val_buf[tidl+k] += val;
			// }
			// else
			// {
				// atomicAdd(&y[row], val);
			// }
		// }
		// g.sync();
		val = val_buf[tidl];
		if ((tidl & (2*k-1)) == k-1)
		{
			if (row == ia_buf[tidl+k])
			{
				val_buf[tidl+k] += val;
			}
			else
			{
				atomicAdd(&y[row], val);
			}
		}
		g.sync();
	}
}
inline
__device__ void reduce_block(INT_T * ia_buf, ValueType * val_buf, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &ia_buf[tidb_div*32], &val_buf[tidb_div*32], y);
	// __syncthreads();
	// if (tidb_mod == 31)
	// {
		// ia_buf[tidb_mod] = ia_buf[tidb];
		// val_buf[tidb_mod] = val_buf[tidb];
	// }
	// __syncthreads();
	// if (tidb_div == 0)
		// reduce_warp(tile32, ia_buf, val_buf, y);
	// if (tidb == 0)
		// atomicAdd(&y[ia_buf[31]], val_buf[31]);
	if (tidb_mod == 31)
		atomicAdd(&y[ia_buf[tidb]], val_buf[tidb]);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T * row_ptr, ValueType * val_ptr, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	INT_T row = *row_ptr;
	ValueType val = *val_ptr;
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_next;
		ValueType val_next;
		row_next = __shfl_sync(0xffffffff, row, tidl+k);
		val_next = __shfl_sync(0xffffffff, val, tidl+k);
		if ((tidl & (2*k-1)) == 0)
		{
			if (row == row_next)
			{
				val += val_next;
			}
			else
			{
				atomicAdd(&y[row], val);
				val = val_next;
				row = row_next;
			}
		}
		g.sync();
	}
	*row_ptr = row;
	*val_ptr = val;
	// if (tidl == 0)
		// atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidb = threadIdx.x;
	const int tidb_div = tidb / 32;
	const int tidb_mod = tidb % 32;
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, &row, &val, y);
	if (tidb_mod == 0)
		atomicAdd(&y[row], val);
	// extern __shared__ char sm[];
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[32 * sizeof(ValueType)];
	// if (tidb_mod == 0)
	// {
		// ia_buf[tidb_div] = row;
		// val_buf[tidb_div] = val;
	// }
	// __syncthreads();
	// if (tidb_div == 0)
	// {
		// row = ia_buf[tidb];
		// val = val_buf[tidb];
		// reduce_warp(tile32, &row, &val, y);
	// }
	// if (tidb == 0)
		// atomicAdd(&y[row], val);
} */


/* template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	int k;
	g.sync();
	#pragma unroll
	for (k=1;k<g.size();k*=2)
	{
		INT_T row_prev;
		ValueType val_prev;
		row_prev = __shfl_sync(0xffffffff, row, tidl-k);
		val_prev = __shfl_sync(0xffffffff, val, tidl-k);
		if ((tidl & (2*k-1)) == 2*k-1)
		{
			if (row == row_prev)
			{
				val += val_prev;
			}
			else
			{
				atomicAdd(&y[row_prev], val_prev);
			}
		}
		g.sync();
	}
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
} */


// Threads may only read data from another thread which is actively participating in the __shfl_sync() command.
// If the target thread is inactive, the retrieved value is undefined.
template <typename group_t>
__device__ void reduce_warp(group_t g, INT_T row, ValueType val, ValueType * restrict y)
{
	const int tidl = g.thread_rank();   // Group lane.
	const int tidl_one_hot = 1 << tidl;
	int flag;
	INT_T row_prev;
	ValueType val_prev;
	flag = 0xaaaaaaaa;
	row_prev = __shfl_sync(flag, row, tidl-1);
	val_prev = __shfl_sync(flag, val, tidl-1);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x88888888;
	row_prev = __shfl_sync(flag, row, tidl-2);
	val_prev = __shfl_sync(flag, val, tidl-2);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80808080;
	row_prev = __shfl_sync(flag, row, tidl-4);
	val_prev = __shfl_sync(flag, val, tidl-4);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80008000;
	row_prev = __shfl_sync(flag, row, tidl-8);
	val_prev = __shfl_sync(flag, val, tidl-8);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	flag = 0x80000000;
	row_prev = __shfl_sync(flag, row, tidl-16);
	val_prev = __shfl_sync(flag, val, tidl-16);
	if (tidl_one_hot & flag)
	{
		if (row == row_prev)
		{
			val += val_prev;
		}
		else
		{
			atomicAdd(&y[row_prev], val_prev);
		}
	}
	g.sync();
	if (tidl == 31)
		atomicAdd(&y[row], val);
}
inline
__device__ void reduce_block(INT_T row, ValueType val, ValueType * restrict y)
{
	thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());
	reduce_warp(tile32, row, val, y);
}


__device__ void spmv_last_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int block_size = blockDim.x;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = thread_block_j_s[block_id];
	j_e = thread_block_j_e[block_id];
	const int total_j = j_e - j_s;
	const int mod = total_j % block_size;
	int j_l_s, j_l_e;
	j_l_s = j_s + tidb * (total_j / block_size);
	j_l_e = j_l_s + (total_j / block_size);
	if (tidb < mod)
	{
		j_l_s += tidb;
		j_l_e += tidb + 1;
	}
	else
	{
		j_l_s += mod;
		j_l_e += mod;
	}
	// int m = (i_e + i_s) / 2;
	// while (i_s < i_e)
	// {
		// if (j_l_s >= row_ptr[m])
		// {
			// i_s = m + 1;
		// }
		// else
		// {
			// i_e = m;
		// }
		// m = (i_e + i_s) / 2;
	// }
	// i = i_s - 1;
	i = ia[j_l_s];
	// if (tidb == block_size-1)
	// {
		// if (j_l_e != j_e)
		// {
			// printf("wrong");
		// }
	// }
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_l_s;j<j_l_e;j++)
	{
		// if (ia[j] != i)
		// {
			// atomicAdd(&y[i], sum);
			// sum = 0;
			// i = ia[j];
		// }
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			// y[i] += sum;
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
			// i = ia[j];
		}
		// sum += a[j] * x[ja[j]];
		sum = __fma_rn(a[j], x[ja[j]], sum);
	}
	// if (j_l_s < j_l_e)
		// atomicAdd(&y[i], sum);
	// val_buf[tidb] = sum;
	// ia_buf[tidb] = i;
	// __syncthreads();
	// reduce_block(ia_buf, val_buf, y);
	reduce_block(i, sum, y);
}


__device__ void spmv_full_block(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	// extern __shared__ char sm[];
	const int tidb = threadIdx.x;
	const int block_id = blockIdx.x;
	const int block_size = blockDim.x;
	const int nnz_per_block = block_size * NNZ_PER_THREAD;
	// ValueType * val_buf = (typeof(val_buf)) sm;
	// INT_T * ia_buf = (typeof(ia_buf)) &sm[block_size * sizeof(ValueType)];
	[[gnu::unused]] int i, i_s, i_e, j, j_s, j_e, k, l, p;
	i_s = thread_block_i_s[block_id];
	i_e = thread_block_i_e[block_id];
	j_s = block_id * nnz_per_block;
	// j_e = (block_id + 1) * nnz_per_block;
	int j_l_s, j_l_e;
	j_l_s = j_s + tidb * NNZ_PER_THREAD;
	j_l_e = j_l_s + NNZ_PER_THREAD;
	// int m = (i_e + i_s) / 2;
	// while (i_s < i_e)
	// {
		// if (j_l_s >= row_ptr[m])
		// {
			// i_s = m + 1;
		// }
		// else
		// {
			// i_e = m;
		// }
		// m = (i_e + i_s) / 2;
	// }
	// i = i_s - 1;
	i = ia[j_l_s];
	// if (tidb == block_size-1)
	// {
		// if (j_l_e != j_e)
		// {
			// printf("wrong");
		// }
	// }
	double sum = 0;
	int ptr_next = row_ptr[i+1];
	for (j=j_l_s;j<j_l_e;j++)
	{
		// if (ia[j] != i)
		// {
			// atomicAdd(&y[i], sum);
			// sum = 0;
			// i = ia[j];
		// }
		if (j >= ptr_next)
		{
			atomicAdd(&y[i], sum);
			// y[i] += sum;
			sum = 0;
			while (j >= ptr_next)
			{
				i++;
				ptr_next = row_ptr[i+1];
			}
			// i = ia[j];
		}
		// sum += a[j] * x[ja[j]];
		sum = __fma_rn(a[j], x[ja[j]], sum);
	}
	// if (j_l_s < j_l_e)
		// atomicAdd(&y[i], sum);
	// val_buf[tidb] = sum;
	// ia_buf[tidb] = i;
	// __syncthreads();
	// reduce_block(ia_buf, val_buf, y);
	reduce_block(i, sum, y);
}


__global__ void gpu_kernel_spmv_row_indices_continuous(INT_T * thread_block_i_s, INT_T * thread_block_i_e, INT_T * thread_block_j_s, INT_T * thread_block_j_e, INT_T * row_ptr, INT_T * ia, INT_T * ja, ValueType * a, ValueType * restrict x, ValueType * restrict y)
{
	int grid_size = gridDim.x;
	int block_id = blockIdx.x;
	if (block_id == grid_size - 1)
		spmv_last_block(thread_block_i_s, thread_block_i_e, thread_block_j_s, thread_block_j_e, row_ptr, ia, ja, a, x, y);
	else
		spmv_full_block(thread_block_i_s, thread_block_i_e, row_ptr, ia, ja, a, x, y);
}


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	// int num_threads = csr->num_threads;
	int block_size = csr->block_size;
	int num_blocks = csr->num_blocks;
	dim3 block_dims(block_size);
	dim3 grid_dims(num_blocks);
	// long shared_mem_size = block_size * (sizeof(ValueType));
	// long shared_mem_size = block_size * (sizeof(ValueType) + sizeof(INT_T));
	long shared_mem_size = 0;

	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
	}

	hipMemset(csr->y_dev, 0, csr->m * sizeof(csr->y_dev));

	gpu_kernel_spmv_row_indices_continuous<<<grid_dims, block_dims, shared_mem_size>>>(thread_block_i_s_dev, thread_block_i_e_dev, thread_block_j_s_dev, thread_block_j_e_dev, csr->row_ptr_dev, csr->ia_dev, csr->ja_dev, csr->a_dev, csr->x_dev, csr->y_dev);

	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("hipDeviceSynchronize: %s\n", hipGetErrorString(err));
	err = hipGetLastError();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));

	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}

	// exit(0);
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

