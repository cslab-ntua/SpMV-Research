#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif


#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;


struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_dev;
	INT_T * ja_dev;
	ValueType * a_dev;

	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_dev = NULL;
	ValueType * y_dev = NULL;
	hipsparseDnVecDescr_t vecX;
	hipsparseDnVecDescr_t vecY;

	int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
		hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0);
		hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0);
		hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0);
		hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0);
		printf("max_smem_per_block=%d\n", max_smem_per_block);
		printf("multiproc_count=%d\n", multiproc_count);
		printf("max_threads_per_block=%d\n", max_threads_per_block);
		printf("warp_size=%d\n", warp_size);
		printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);


		hipMalloc(&ia_dev, (m+1) * sizeof(*ia_dev));
		hipMalloc(&ja_dev, nnz * sizeof(*ja_dev));
		hipMalloc(&a_dev, nnz * sizeof(*a_dev));
		hipMalloc(&x_dev, n * sizeof(*x_dev));
		hipMalloc(&y_dev, m * sizeof(*y_dev));

		hipMemcpy(ia_dev, ia, (m+1) * sizeof(*ia_dev), hipMemcpyHostToDevice);
		hipMemcpy(ja_dev, ja, nnz * sizeof(*ja_dev), hipMemcpyHostToDevice);
		hipMemcpy(a_dev, a, nnz * sizeof(*a_dev), hipMemcpyHostToDevice);

		hipsparseCreate(&handle);
		// Create sparse matrix A in CSR format
		hipsparseCreateCsr(&matA, m, n, nnz, ia_dev, ja_dev, a_dev, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda);
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		// destroy matrix/vector descriptors
		hipsparseDestroySpMat(matA);
		hipsparseDestroyDnVec(vecX);
		hipsparseDestroyDnVec(vecY);
		hipsparseDestroy(handle);

		hipFree(ia_dev);
		hipFree(ja_dev);
		hipFree(a_dev);

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_kahan(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y);
void compute_csr_prefetch(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_omp_simd(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);
void compute_csr_vector_perfect_nnz_balance(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "CUSPARSE_CSR";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		hipMemcpy(csr->x_dev, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice);
		// Create dense vector X
		hipsparseCreateDnVec(&csr->vecX, csr->n, csr->x_dev, ValueTypeCuda);
		// Create dense vector y
		hipsparseCreateDnVec(&csr->vecY, csr->m, csr->y_dev, ValueTypeCuda);
		// Allocate an external buffer if needed
		hipsparseSpMV_bufferSize(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &csr->bufferSize);
		hipMalloc(&csr->dBuffer, csr->bufferSize);
	}
	// Execute SpMV
	hipsparseSpMV(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, csr->dBuffer);
	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		error("gpu kernel error: %s\n", hipGetErrorString(err));
	if (csr->y == NULL)
	{
		csr->y = y;
		hipMemcpy(csr->y, csr->y_dev, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost);
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

