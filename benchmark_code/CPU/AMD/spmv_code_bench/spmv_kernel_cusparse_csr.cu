#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "spmv_bench_common.h"
#include "spmv_kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cusparse_util.h"
#ifdef __cplusplus
}
#endif


#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

double * thread_time_compute, * thread_time_barrier;

#ifndef TIME_IT
#define TIME_IT 1
#endif

#ifndef VERIFIED
#define VERIFIED 1
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	hipsparseDnVecDescr_t vecX;
	hipsparseDnVecDescr_t vecY;

	// hipEvent_t is useful for timing, but for performance use " hipEventCreateWithFlags ( &event, hipEventDisableTiming) "
	hipEvent_t startEvent_execution;
	hipEvent_t endEvent_execution;

	hipEvent_t startEvent_memcpy_x;
	hipEvent_t endEvent_memcpy_x;
	hipEvent_t startEvent_memcpy_y;
	hipEvent_t endEvent_memcpy_y;

	hipEvent_t startEvent_memcpy_ia;
	hipEvent_t endEvent_memcpy_ia;
	hipEvent_t startEvent_memcpy_ja;
	hipEvent_t endEvent_memcpy_ja;
	hipEvent_t startEvent_memcpy_a;
	hipEvent_t endEvent_memcpy_a;

	hipEvent_t startEvent_create_matA;
	hipEvent_t endEvent_create_matA;
	hipEvent_t startEvent_spmv_buffersize;
	hipEvent_t endEvent_spmv_buffersize;

	hipEvent_t startEvent_create_vecX;
	hipEvent_t endEvent_create_vecX;
	hipEvent_t startEvent_create_vecY;
	hipEvent_t endEvent_create_vecY;


	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		int max_smem_per_block, multiproc_count, max_threads_per_block, warp_size, max_threads_per_multiproc;
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&multiproc_count, hipDeviceAttributeMultiprocessorCount, 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize , 0));
		gpuCudaErrorCheck(hipDeviceGetAttribute(&max_threads_per_multiproc, hipDeviceAttributeMaxThreadsPerMultiProcessor, 0));
		// printf("max_smem_per_block=%d\n", max_smem_per_block);
		// printf("multiproc_count=%d\n", multiproc_count);
		// printf("max_threads_per_block=%d\n", max_threads_per_block);
		// printf("warp_size=%d\n", warp_size);
		// printf("max_threads_per_multiproc=%d\n", max_threads_per_multiproc);

		gpuCudaErrorCheck(hipMalloc(&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc(&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc(&a_d, nnz * sizeof(*a_d)));
		gpuCudaErrorCheck(hipMalloc(&x_d, n * sizeof(*x_d)));
		gpuCudaErrorCheck(hipMalloc(&y_d, m * sizeof(*y_d)));

		gpuCusparseErrorCheck(hipsparseCreate(&handle));

		// cuda events for timing measurements
		gpuCudaErrorCheck(hipEventCreate(&startEvent_execution));
		gpuCudaErrorCheck(hipEventCreate(&endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_matA));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_spmv_buffersize));

			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventCreate(&startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventCreate(&endEvent_memcpy_y));
		}

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ia));
		gpuCudaErrorCheck(hipMemcpy(ia_d, ia, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ia));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_ja));
		gpuCudaErrorCheck(hipMemcpy(ja_d, ja, nnz * sizeof(*ja_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_ja));

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_memcpy_a));
		gpuCudaErrorCheck(hipMemcpy(a_d, a, nnz * sizeof(*a_d), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_memcpy_a));

		// Create sparse matrix A in CSR format
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(startEvent_create_matA));
		gpuCusparseErrorCheck(hipsparseCreateCsr(&matA, m, n, nnz, ia_d, ja_d, a_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(endEvent_create_matA));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventSynchronize(endEvent_create_matA));

			float memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, create_matA_Time;//memcpyTime_cuda_thread_i_e;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ia, startEvent_memcpy_ia, endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_ja, startEvent_memcpy_ja, endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda_a, startEvent_memcpy_a, endEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_matA_Time, startEvent_create_matA, endEvent_create_matA));

			printf("(CUDA) Memcpy ia time = %.4lf ms, ja time = %.4lf ms, a time = %.4lf ms, matA time = %.4lf ms\n", memcpyTime_cuda_ia, memcpyTime_cuda_ja, memcpyTime_cuda_a, create_matA_Time);
		}
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		// destroy matrix/vector descriptors
		gpuCusparseErrorCheck(hipsparseDestroySpMat(matA));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecX));
		gpuCusparseErrorCheck(hipsparseDestroyDnVec(vecY));
		gpuCusparseErrorCheck(hipsparseDestroy(handle));

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));

		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(dBuffer));

		gpuCudaErrorCheck(hipEventDestroy(startEvent_execution));
		gpuCudaErrorCheck(hipEventDestroy(endEvent_execution));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ia));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_ja));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_a));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_a));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_matA));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_spmv_buffersize));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_spmv_buffersize));

			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_x));
			gpuCudaErrorCheck(hipEventDestroy(startEvent_memcpy_y));
			gpuCudaErrorCheck(hipEventDestroy(endEvent_memcpy_y));
		}

		#ifdef PRINT_STATISTICS
			free(thread_time_barrier);
			free(thread_time_compute);
		#endif
	}

	void spmv(ValueType * x, ValueType * y);
	void statistics_start();
	int statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n);
};


void compute_csr(CSRArrays * restrict csr, ValueType * restrict x , ValueType * restrict y);


void
CSRArrays::spmv(ValueType * x, ValueType * y)
{
	compute_csr(this, x, y);
}


struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "CUSPARSE_CSR";
	return csr;
}


//==========================================================================================================================================
//= CSR Custom
//==========================================================================================================================================


void
compute_csr(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y)
{
	const double alpha = 1.0;
	const double beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_x));
		gpuCudaErrorCheck(hipMemcpy(csr->x_d, csr->x, csr->n * sizeof(*csr->x), hipMemcpyHostToDevice));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_x));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_x));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_x, csr->endEvent_memcpy_x));
			printf("(CUDA) Memcpy x time = %.4lf ms\n", memcpyTime_cuda);
		}

		// Create dense vector X
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_create_vecX));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&csr->vecX, csr->n, csr->x_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_create_vecX));

		// Create dense vector y
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_create_vecY));
		gpuCusparseErrorCheck(hipsparseCreateDnVec(&csr->vecY, csr->m, csr->y_d, ValueTypeCuda));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_create_vecY));

		// Allocate an external buffer if needed
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_spmv_buffersize));
		gpuCusparseErrorCheck(hipsparseSpMV_bufferSize(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, &csr->bufferSize));
		gpuCudaErrorCheck(hipMalloc(&csr->dBuffer, csr->bufferSize));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_spmv_buffersize));
		// printf("SpMV_bufferSize = %zu bytes\n", csr->bufferSize, csr->bufferSize); // size of the workspace that is needed by hipsparseSpMV()

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_spmv_buffersize));
			float create_vecX_time, create_vecY_time, spmv_buffersize_time;
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecX_time, csr->startEvent_create_vecX, csr->endEvent_create_vecX));
			gpuCudaErrorCheck(hipEventElapsedTime(&create_vecY_time, csr->startEvent_create_vecY, csr->endEvent_create_vecY));
			gpuCudaErrorCheck(hipEventElapsedTime(&spmv_buffersize_time, csr->startEvent_spmv_buffersize, csr->endEvent_spmv_buffersize));
			printf("(CUDA) Create vecX time = %.4lf ms, vecY time = %.4lf ms, spmv_buffersize time = %.4lf (SpMV_bufferSize = %zu)\n", create_vecX_time, create_vecY_time, spmv_buffersize_time, csr->bufferSize);
		}
	}

	if(VERIFIED){
		int num_loops = 1000;
		for(int k=0;k<num_loops;k++)
			gpuCusparseErrorCheck(hipsparseSpMV(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, csr->dBuffer));
		gpuCudaErrorCheck(hipPeekAtLastError());
		gpuCudaErrorCheck(hipDeviceSynchronize());
	}

	// Execute SpMV
	gpuCudaErrorCheck(hipEventRecord(csr->startEvent_execution));

	int num_loops = 128;
	double time_execution = time_it(1,
		for(int k=0;k<num_loops;k++){
			gpuCusparseErrorCheck(hipsparseSpMV(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->vecX, &beta, csr->vecY, ValueTypeCuda, HIPSPARSE_SPMV_ALG_DEFAULT, csr->dBuffer));
			gpuCudaErrorCheck(hipPeekAtLastError());
			gpuCudaErrorCheck(hipDeviceSynchronize());
		}
	);

	double gflops = csr->nnz / time_execution * num_loops * 2 * 1e-9;
	printf("(DGAL timing) Execution time = %.4lf ms (%.4lf GFLOPS for %.2lf MB workload)\n", time_execution*1e3, gflops, csr->mem_footprint/(1024*1024.0));

	gpuCudaErrorCheck(hipEventRecord(csr->endEvent_execution));
	float executionTime_cuda;
	gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_execution));
	gpuCudaErrorCheck(hipEventElapsedTime(&executionTime_cuda, csr->startEvent_execution, csr->endEvent_execution));

	double gflops_cuda = csr->nnz / executionTime_cuda * num_loops * 2 * 1e-6;
	printf("(CUDA) Execution time = %.4lf ms (%.4lf GFLOPS for %.2lf MB workload)\n", executionTime_cuda, gflops_cuda, csr->mem_footprint/(1024*1024.0));

	if (csr->y == NULL)
	{
		csr->y = y;

		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->startEvent_memcpy_y));
		gpuCudaErrorCheck(hipMemcpy(csr->y, csr->y_d, csr->m * sizeof(*csr->y), hipMemcpyDeviceToHost));
		if(TIME_IT) gpuCudaErrorCheck(hipEventRecord(csr->endEvent_memcpy_y));

		if(TIME_IT){
			gpuCudaErrorCheck(hipEventSynchronize(csr->endEvent_memcpy_y));
			float memcpyTime_cuda;
			gpuCudaErrorCheck(hipEventElapsedTime(&memcpyTime_cuda, csr->startEvent_memcpy_y, csr->endEvent_memcpy_y));
			printf("(CUDA) Memcpy y time = %.4lf ms\n", memcpyTime_cuda);
		}
	}
}


//==========================================================================================================================================
//= Print Statistics
//==========================================================================================================================================


void
CSRArrays::statistics_start()
{
}


int
statistics_print_labels(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}


int
CSRArrays::statistics_print_data(__attribute__((unused)) char * buf, __attribute__((unused)) long buf_n)
{
	return 0;
}

