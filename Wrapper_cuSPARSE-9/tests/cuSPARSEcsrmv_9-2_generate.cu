#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///

#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

#include "nvem.hpp"

int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 11,
		  "Incorrect arguments.\nUsage:  ./Executable logfilename Mpakos_9_parameters_with_spaces");
		            
	// Set/Check for device
	int device_id = 0;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *outfile = argv[1];
	int start_of_matrix_generation_args = 2, verbose = 0;
#ifdef DDEBUG
	verbose = 1;
#endif
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(argc, argv, start_of_matrix_generation_args, verbose);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf, mem_footprint = %lf MB, mem_range=%s\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf, bw_scaled = %lf, skew =%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, op.A_mem_footprint, op.mem_range,
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering, op.bw_scaled, op.skew);
		
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_AX *out = (VALUE_TYPE_AX *)malloc(op.m * sizeof(VALUE_TYPE_AX));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc((VALUE_TYPE_AX*)x);

	SpmvOperator cuSPARSE_op(op);
	cuSPARSE_op.cuSPARSE_init();
	cuSPARSE_op.format_convert(SPMV_FORMAT_CSR);
	cuSPARSE_op.mem_convert(SPMV_MEMTYPE_DEVICE);
  
#ifdef TEST
  
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
	spmv_csr<VALUE_TYPE_AX>(data->rowPtr, data->colInd, (VALUE_TYPE_AX *) data->values, (VALUE_TYPE_AX *)op.x,
		   (VALUE_TYPE_AX *) op.y, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");


	fprintf(stdout,"Testing cuSPARSE_csr...\t");

	cuSPARSE_op.cuSPARSE_csr();
	hipDeviceSynchronize();
	hipMemcpy(out, cuSPARSE_op.y, op.m * sizeof(VALUE_TYPE_AX), hipMemcpyDeviceToHost);
	check_result<VALUE_TYPE_AX>((VALUE_TYPE_AX *)out, (VALUE_TYPE_AX *)op.y, cuSPARSE_op.m);


#endif
	
	// Warmup
	for (int i = 0; i < 100; i++) cuSPARSE_op.cuSPARSE_csr();
	hipDeviceSynchronize();

	// Run cuSPARSE csr
	fprintf(stdout,"Timing cuSPARSE_csr...\n");
	short CUDA_VALUE_TYPE_AX;
	if (std::is_same<VALUE_TYPE_AX, float>::value)  CUDA_VALUE_TYPE_AX = 0;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = 1;
	char powa_filename[256];
	sprintf(powa_filename, "cuSPARSEcsrmv_9-2_generate_cudatype-%d_format-CSR.log", CUDA_VALUE_TYPE_AX);
	NvemStartMeasure(device_id, powa_filename, 0); // Set to 1 for NVEM log messages. 
	op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			cuSPARSE_op.cuSPARSE_csr();
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	op.timer = (csecond() - op.timer)/NR_ITER;
	unsigned int extra_itter = 0; 
	if (op.timer*NR_ITER < 1.0){
		extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			cuSPARSE_op.cuSPARSE_csr();
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	NvemStats_p nvem_data = NvemStopMeasure(device_id, "Energy measure cuSPARSEcsrmv_9-2_generate");
	gflops_s = op.flops*1e-9/op.timer;
	double W_avg = nvem_data->W_avg, J_estimated = nvem_data->J_estimated/(NR_ITER+extra_itter); 
	fprintf(stdout, "cuSPARSE_csr9-2: t = %lf ms (%lf Gflops/s ). Average Watts = %lf, Estimated Joules = %lf\n", op.timer*1000, gflops_s, W_avg, J_estimated);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.nz << "," << op.density << 
	"," << op.A_mem_footprint << "," << op.mem_range << "," << op.avg_nz_row << "," << op.std_nz_row <<
	"," << op.avg_bandwidth << "," << op.std_bandwidth <<
	"," << op.avg_scattering << "," << op.std_scattering << "," << op.bw_scaled << "," << op.skew <<
	"," << "cuSPARSE_csr9-2" <<  "," << op.timer << "," << gflops_s << "," << W_avg <<  "," << J_estimated << endl;

	foutp.close();

}
