///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Input functions for .mtx
///

#include <stdio.h>
#include <stdlib.h>
#include "gpu_utils.hpp"
#include "mmio.h"
#include "spmv_utils.hpp"

//From cuda 11 - cuSPARSE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

void SpmvOperator::mtx_read_uni(){
    ddebug(" -> SpmvOperator::mtx_read_uni()\n");
    int nnzA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE_AX *csrValA;
    
	// read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;
    // load matrix
    if ((f = fopen(mtx_name, "r")) == NULL)
        exit(1);

    if (mm_read_banner(f, &matcode) != 0)
    {
        cout << "Could not process Matrix Market banner." << endl;
        exit(2);
    }

    if ( mm_is_complex( matcode ) )
    {
        cout <<"Sorry, data type 'COMPLEX' is not supported. " << endl;
        exit(3);
    }

    if ( mm_is_pattern( matcode ) )  { isPattern = 1; /*cout << "type = Pattern" << endl;*/ }
    if ( mm_is_real ( matcode) )     { isReal = 1; /*cout << "type = real" << endl;*/ }
    if ( mm_is_integer ( matcode ) ) { isInteger = 1; /*cout << "type = integer" << endl;*/ }

    /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
        exit(4);

    if ( mm_is_symmetric( matcode ) || mm_is_hermitian( matcode ) )
    {
        isSymmetric = 1;
        //cout << "symmetric = true" << endl;
    }
    else
    {
        //cout << "symmetric = false" << endl;
    }

    int *csrRowPtrA_counter = (int *)malloc((m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    int *csrRowIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    int *csrColIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    VALUE_TYPE_AX *csrValA_tmp    = (VALUE_TYPE_AX *)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE_AX));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (int i = 0; i < nnzA_mtx_report; i++)
    {
        int idxi, idxj;
        VALUE_TYPE_AX fval;
        int ival;

        if (isReal)
            fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
        else if (isInteger)
        {
            fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
            fval = ival;
        }
        else if (isPattern)
        {
            fscanf(f, "%d %d\n", &idxi, &idxj);
            fval = 1.0;
        }

        // adjust from 1-based to 0-based
        idxi--;
        idxj--;

        csrRowPtrA_counter[idxi]++;
        csrRowIdxA_tmp[i] = idxi;
        csrColIdxA_tmp[i] = idxj;
        csrValA_tmp[i] = fval;
    }

    if (f != stdin)
        fclose(f);

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
        }
    }

    // exclusive scan for csrRowPtrA_counter
    int old_val, new_val;

    old_val = csrRowPtrA_counter[0];
    csrRowPtrA_counter[0] = 0;
    for (int i = 1; i <= m; i++)
    {
        new_val = csrRowPtrA_counter[i];
        csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i-1];
        old_val = new_val;
    }

	hipMallocManaged(&csrRowPtrA, (m+1) * sizeof(int));
	hipMallocManaged(&csrColIdxA, nnzA * sizeof(int));
	hipMallocManaged(&csrValA, nnzA * sizeof(VALUE_TYPE_AX));
	hipDeviceSynchronize();
	cudaCheckErrors();
  
    nnzA = csrRowPtrA_counter[m];
    memcpy(csrRowPtrA, csrRowPtrA_counter, (m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

                offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
                csrColIdxA[offset] = csrRowIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
            }
            else
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
            }
        }
    }
    else
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
            csrColIdxA[offset] = csrColIdxA_tmp[i];
            csrValA[offset] = csrValA_tmp[i];
            csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
        }
    }

    // free tmp space
    free(csrColIdxA_tmp);
    free(csrValA_tmp);
    free(csrRowIdxA_tmp);
    free(csrRowPtrA_counter);
    
    SpmvCsrData* csr_output = (SpmvCsrData *) malloc(sizeof(SpmvCsrData));
	nz = nnzA;
	mem_bytes += (nz) * sizeof(VALUE_TYPE_AX) + (2 * nz) * sizeof(int);
	gpu_mem_bytes += (nz) * sizeof(VALUE_TYPE_AX) + (2 * nz) * sizeof(int);
  	csr_output->rowPtr = csrRowPtrA;
  	csr_output->colInd = csrColIdxA;
  	csr_output->values = csrValA;
  	
  	format_data = csr_output;
  	ddebug(" <- SpmvOperator::mtx_read_uni()\n");
}

void SpmvOperator::mtx_read_device() {
  ddebug(" -> SpmvOperator::mtx_read_device()\n");
  massert(0, "SpmvOperator::mtx_read_device -> Not implemented");
  ddebug(" <- SpmvOperator::mtx_read_device()\n");
}

void SpmvOperator::mtx_generate_device(int argc, char *argv[], int start_of_matrix_generation_args, int verbose) {
  ddebug(" -> SpmvOperator::mtx_generate_device()\n");
  massert(0, "SpmvOperator::mtx_generate_device -> Not implemented");
  ddebug(" <- SpmvOperator::mtx_generate_device()\n");
}

void SpmvOperator::mtx_generate_uni(int argc, char *argv[], int start_of_matrix_generation_args, int verbose) {
  ddebug(" -> SpmvOperator::mtx_generate_uni()\n");
  massert(0, "SpmvOperator::mtx_generate_uni -> Not implemented");
  ddebug(" <- SpmvOperator::mtx_generate_uni()\n");
}
