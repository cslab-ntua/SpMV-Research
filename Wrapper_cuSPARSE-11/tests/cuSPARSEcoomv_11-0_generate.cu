#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///


#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

//From cuda 11 - cuSPARSE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "nvem.hpp"


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

/* definition to expand macro then apply to pragma message */
#define VALUE_TO_STRING(x) #x
#define VALUE(x) VALUE_TO_STRING(x)
#define VAR_NAME_VALUE(var) #var "="  VALUE(var)

/* Some example here */
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_AX))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_Y))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_COMP))

//Add here any supported combinations. CUDA data types I hate you for this. 
hipDataType CUDA_VALUE_TYPE_AX, CUDA_VALUE_TYPE_Y, CUDA_VALUE_TYPE_COMP;
void cpp_compargs_to_cuda_dtype(){
	if (std::is_same<VALUE_TYPE_AX, int8_t>::value) CUDA_VALUE_TYPE_AX = HIP_R_8I;
	else if (std::is_same<VALUE_TYPE_AX, int>::value) CUDA_VALUE_TYPE_AX = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_AX, float>::value) CUDA_VALUE_TYPE_AX = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_AX");
	
	if (std::is_same<VALUE_TYPE_Y, int>::value) CUDA_VALUE_TYPE_Y = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_Y, float>::value) CUDA_VALUE_TYPE_Y = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_Y, double>::value) CUDA_VALUE_TYPE_Y = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_Y");
	
	if (std::is_same<VALUE_TYPE_COMP, int>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32I;
	else if (std::is_same<VALUE_TYPE_COMP, float>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32F;
	else if (std::is_same<VALUE_TYPE_COMP, double>::value) CUDA_VALUE_TYPE_COMP = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_COMP");
	cout << "CUDA_VALUE_TYPE_AX: " << CUDA_VALUE_TYPE_AX << ", CUDA_VALUE_TYPE_Y: " << CUDA_VALUE_TYPE_Y << ", CUDA_VALUE_TYPE_COMP: " << CUDA_VALUE_TYPE_COMP << endl;
}

int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 9,
		  "Incorrect arguments.\nUsage:  ./Executable logfilename Mpakos_7_parameters_with_spaces");
		            
	// Set/Check for device
	int device_id = 0;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *outfile = argv[1];
	int start_of_matrix_generation_args = 2, verbose = 0;
#ifdef DDEBUG
	verbose = 1;
#endif
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(argc, argv, start_of_matrix_generation_args, verbose);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, diagonal_factor = %lf, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf, mem_footprint = %lf MB, mem_range=%s\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.diagonal_factor, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, op.A_mem_footprint, op.mem_range,
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering );
	  
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_Y *out = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc((VALUE_TYPE_AX*)x);

	op.cuSPARSE_init();
	cpp_compargs_to_cuda_dtype();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
		   
    VALUE_TYPE_COMP alpha = 1.0;
    VALUE_TYPE_COMP beta = 0.0;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    VALUE_TYPE_AX *dA_values, *dX;
    VALUE_TYPE_Y *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (op.m + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, op.nz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  op.nz * sizeof(VALUE_TYPE_AX))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         op.n * sizeof(VALUE_TYPE_AX)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         op.m * sizeof(VALUE_TYPE_Y)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, data->rowPtr,
                           (op.m + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, data->colInd, op.nz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, data->values, op.nz * sizeof(VALUE_TYPE_AX),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, op.x, op.n * sizeof(VALUE_TYPE_AX),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, op.y, op.m * sizeof(VALUE_TYPE_Y),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Convert CSR input to COO
    int   *dA_rows; 
    CHECK_CUDA( hipMalloc((void**) &dA_rows,
                           op.nz * sizeof(int)) )
    CHECK_CUSPARSE( hipsparseXcsr2coo(handle, dA_csrOffsets, op.nz, op.m, dA_rows, HIPSPARSE_INDEX_BASE_ZERO))
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    // Create sparse matrix A in Coo format
    CHECK_CUSPARSE( hipsparseCreateCoo(&matA, op.m, op.n, op.nz,
                                      dA_rows, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, CUDA_VALUE_TYPE_AX) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, op.n, dX, CUDA_VALUE_TYPE_AX) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, op.m, dY, CUDA_VALUE_TYPE_Y) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
    
#ifdef TEST
  
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	spmv_csr<VALUE_TYPE_AX, VALUE_TYPE_Y, VALUE_TYPE_COMP>(data->rowPtr, data->colInd, (VALUE_TYPE_AX *) data->values, (VALUE_TYPE_AX *)op.x,
		   (VALUE_TYPE_Y*) out, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");


	fprintf(stdout,"Testing cuSPARSE_coo...\t");
    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
	hipDeviceSynchronize();
	// device result check
    CHECK_CUDA( hipMemcpy(op.y, dY, op.m * sizeof(VALUE_TYPE_Y),
                           hipMemcpyDeviceToHost) )
	check_result<VALUE_TYPE_Y>((VALUE_TYPE_Y*)op.y, out, op.m);


#endif

	// Warmup
	for (int i = 0; i < 100; i++)     CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
	hipDeviceSynchronize();

	// Run cuSPARSE coo
	fprintf(stdout,"Timing cuSPARSE_coo...\n");
	char powa_filename[256];
	sprintf(powa_filename, "cuSPARSECoomv_11-0_generate_cudatype-%d_format-COO.log", CUDA_VALUE_TYPE_AX);
	NvemStartMeasure(device_id, powa_filename, 1); // Set to 1 for NVEM log messages. 
	op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	op.timer = (csecond() - op.timer)/NR_ITER;
	unsigned int extra_itter = 0;
	if (op.timer*NR_ITER < 1.0){
		extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	NvemStats_p nvem_data = NvemStopMeasure(device_id, "Energy measure cuSPARSEcoomv_11-0_generate");
	gflops_s = op.flops*1e-9/op.timer;
	double W_avg = nvem_data->W_avg, J_estimated = nvem_data->J_estimated/(NR_ITER+extra_itter); 
	fprintf(stdout, "cuSPARSE_coo11: t = %lf ms (%lf Gflops/s ). Average Watts = %lf, Estimated Joules = %lf\n", op.timer*1000, gflops_s, W_avg, J_estimated);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.diagonal_factor << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.nz << "," << op.density << 
	"," << op.A_mem_footprint << "," << op.mem_range << "," << op.avg_nz_row << "," << op.std_nz_row <<
	"," << op.avg_bandwidth << "," << op.std_bandwidth <<
	"," << op.avg_scattering << "," << op.std_scattering <<
	"," << "cuSPARSE_coo11" <<  "," << op.timer << "," << gflops_s << "," << W_avg <<  "," << J_estimated << endl;

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------

    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    
	foutp.close();
	return EXIT_SUCCESS;

}
