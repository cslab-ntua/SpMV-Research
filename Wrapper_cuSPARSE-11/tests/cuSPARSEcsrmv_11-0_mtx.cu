#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///


#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

//From cuda 11 - cuSPARSE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "nvmlPower.hpp"


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

/* definition to expand macro then apply to pragma message */
#define VALUE_TO_STRING(x) #x
#define VALUE(x) VALUE_TO_STRING(x)
#define VAR_NAME_VALUE(var) #var "="  VALUE(var)

/* Some example here */
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_AX))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_Y))
#pragma message(VAR_NAME_VALUE(VALUE_TYPE_COMP))

//Add here any supported combinations. CUDA data types I hate you for this. 
hipDataType CUDA_VALUE_TYPE_AX, CUDA_VALUE_TYPE_Y, CUDA_VALUE_TYPE_COMP;
hipDataType cpp_compargs_to_cuda_dtype(){
	if (std::is_same<VALUE_TYPE_AX, int8_t>::value) CUDA_VALUE_TYPE_AX = HIP_R_8I;
	if (std::is_same<VALUE_TYPE_AX, float>::value) CUDA_VALUE_TYPE_AX = HIP_R_32F;
	if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_AX");
	
	if (std::is_same<VALUE_TYPE_Y, int>::value) CUDA_VALUE_TYPE_Y = HIP_R_32I;
	if (std::is_same<VALUE_TYPE_Y, float>::value) CUDA_VALUE_TYPE_Y = HIP_R_32F;
	if (std::is_same<VALUE_TYPE_Y, double>::value) CUDA_VALUE_TYPE_Y = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_Y");
	
	if (std::is_same<VALUE_TYPE_COMP, int>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32I;
	if (std::is_same<VALUE_TYPE_COMP, float>::value) CUDA_VALUE_TYPE_COMP = HIP_R_32F;
	if (std::is_same<VALUE_TYPE_COMP, double>::value) CUDA_VALUE_TYPE_COMP = HIP_R_64F;
	else massert(0, "cpp_compargs_to_cuda_dtype: Invalid/not implemented VALUE_TYPE_COMP");
}

	
int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 3,
	  "Incorrect arguments.\nUsage:\t./Executable logfilename Matrix_name.mtx");
	  
	// Set/Check for device
	int device_id = 0;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *name = argv[2], *outfile = argv[1];
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, diagonal_factor = %lf, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.diagonal_factor, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, 
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering );
	  
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_Y *out = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc(x);

	op.cuSPARSE_init();
	
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
    VALUE_TYPE_COMP alpha = (VALUE_TYPE_COMP) 1.0;
    VALUE_TYPE_COMP beta = (VALUE_TYPE_COMP) 0.0;
    cout << "alpha: " << alpha << ", beta: " << beta << endl;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    VALUE_TYPE_AX *dA_values, *dX;
    VALUE_TYPE_Y *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (op.m + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, op.nz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  op.nz * sizeof(VALUE_TYPE_AX))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         op.n * sizeof(VALUE_TYPE_AX)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         op.m * sizeof(VALUE_TYPE_Y)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, data->rowPtr,
                           (op.m + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, data->colInd, op.nz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, data->values, op.nz * sizeof(VALUE_TYPE_AX),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, x, op.n * sizeof(VALUE_TYPE_AX),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, out, op.m * sizeof(VALUE_TYPE_Y),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, op.m, op.n, op.nz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, CUDA_VALUE_TYPE_AX) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, op.n, dX, CUDA_VALUE_TYPE_AX) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, op.m, dY, CUDA_VALUE_TYPE_Y) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
    
#ifdef TEST

	VALUE_TYPE_Y *out1 = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	spmv_csr<VALUE_TYPE_AX, VALUE_TYPE_Y, VALUE_TYPE_COMP>(data->rowPtr, data->colInd, data->values, x,
		   out1, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");

	fprintf(stdout,"Testing cuSPARSE_csr...\t");
    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
	hipDeviceSynchronize();
	// device result check
    CHECK_CUDA( hipMemcpy(out, dY, op.m * sizeof(VALUE_TYPE_Y),
                           hipMemcpyDeviceToHost) )
	hipDeviceSynchronize();
	check_result<VALUE_TYPE_Y>((VALUE_TYPE_Y*)out, out1, op.m);


#endif

	// Warmup
	for (int i = 0; i < 100; i++)     CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
	hipDeviceSynchronize();

	// Run cuSPARSE csr
	fprintf(stdout,"Timing cuSPARSE_csr...\n");
	nvmlAPIRun();
	op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	op.timer = (csecond() - op.timer)/NR_ITER;
	if (op.timer*NR_ITER < 1.0){
		unsigned int extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	nvmlAPIEnd();
	gflops_s = op.flops*1e-9/op.timer;
	fprintf(stdout, "cuSPARSE_csr: t = %lf ms (%lf Gflops/s )\n", op.timer*1000, gflops_s);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.diagonal_factor << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.density << 
	"," << op.nz << "," << op.avg_nz_row << "," << op.std_nz_row <<
	"," << op.avg_bandwidth << "," << op.std_bandwidth <<
	"," << op.avg_scattering << "," << op.std_scattering <<
	"," << "cuSPARSE_csr" <<  "," << op.timer << "," << gflops_s << "\n";

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------

    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    
	foutp.close();
	return EXIT_SUCCESS;

}
