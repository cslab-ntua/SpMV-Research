#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A benchmark script for SpMV implementations
///


#include <cstdio>
#include <gpu_utils.hpp>
#include <numeric>
#include <spmv_utils.hpp>
#include "cuSPARSE.hpp"
#include <iostream>
#include <fstream>

//From cuda 11 - cuSPARSE
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "nvmlPower.hpp"


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

//Add any wanted combinations
#if VALUE_TYPE_AX == double
#define CUDA_VALUE_TYPE_AX HIP_R_64F
#elif VALUE_TYPE_AX == float
#define CUDA_VALUE_TYPE_AX HIP_R_32F
#elif VALUE_TYPE_AX == _int8
#define CUDA_VALUE_TYPE_AX HIP_R_8I
#else 
#error
#endif

#if VALUE_TYPE_Y == double
#define CUDA_VALUE_TYPE_Y HIP_R_64F
#elif VALUE_TYPE_Y == float
#define CUDA_VALUE_TYPE_Y HIP_R_32F
#elif VALUE_TYPE_Y == _int32
#define CUDA_VALUE_TYPE_Y HIP_R_32I
#elif VALUE_TYPE_Y == int
#define CUDA_VALUE_TYPE_Y HIP_R_32I
#else 
#error
#endif

#if VALUE_TYPE_COMP == double
#define CUDA_VALUE_TYPE_COMP HIP_R_64F
#elif VALUE_TYPE_COMP == float
#define CUDA_VALUE_TYPE_COMP HIP_R_32F
#elif VALUE_TYPE_COMP == _int32
#define CUDA_VALUE_TYPE_COMP HIP_R_32I
#elif VALUE_TYPE_COMP == int
#define CUDA_VALUE_TYPE_COMP HIP_R_32I
#else 
#error
#endif

int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 9,
		  "Incorrect arguments.\nUsage:  ./Executable logfilename Mpakos_7_parameters_with_spaces");
		            
	// Set/Check for device
	int device_id = 1;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *outfile = argv[1];
	int start_of_matrix_generation_args = 2, verbose = 0;
#ifdef DDEBUG
	verbose = 1;
#endif
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(argc, argv, start_of_matrix_generation_args, verbose);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, diagonal_factor = %lf, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.diagonal_factor, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, 
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering );
	  
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_Y *out = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc((VALUE_TYPE_AX*)x);

	op.cuSPARSE_init();
	
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
		   
    VALUE_TYPE_COMP alpha = 1.0;
    VALUE_TYPE_COMP beta = 0.0;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    VALUE_TYPE_AX *dA_values, *dX;
    VALUE_TYPE_Y *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (op.m + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, op.nz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  op.nz * sizeof(VALUE_TYPE_AX))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         op.n * sizeof(VALUE_TYPE_AX)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         op.m * sizeof(VALUE_TYPE_Y)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, data->rowPtr,
                           (op.m + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, data->colInd, op.nz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, data->values, op.nz * sizeof(VALUE_TYPE_AX),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, op.x, op.n * sizeof(VALUE_TYPE_AX),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, op.y, op.m * sizeof(VALUE_TYPE_Y),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, op.m, op.n, op.nz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, CUDA_VALUE_TYPE_AX) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, op.n, dX, CUDA_VALUE_TYPE_AX) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, op.m, dY, CUDA_VALUE_TYPE_Y) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
    
#ifdef TEST
  
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	spmv_csr<VALUE_TYPE_AX, VALUE_TYPE_Y, VALUE_TYPE_COMP>(data->rowPtr, data->colInd, (VALUE_TYPE_AX *) data->values, (VALUE_TYPE_AX *)op.x,
		   (VALUE_TYPE_Y*) out, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");


	fprintf(stdout,"Testing cuSPARSE_csr...\t");
    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
	hipDeviceSynchronize();
	// device result check
    CHECK_CUDA( hipMemcpy(op.y, dY, op.m * sizeof(VALUE_TYPE_Y),
                           hipMemcpyDeviceToHost) )
	check_result<VALUE_TYPE_Y>((VALUE_TYPE_Y*)op.y, out, op.m);


#endif

	// Warmup
	for (int i = 0; i < 100; i++)     CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
	hipDeviceSynchronize();

	// Run cuSPARSE csr
	fprintf(stdout,"Timing cuSPARSE_csr...\n");
	nvmlAPIRun();
	op.timer = csecond();
	for (int i = 0; i < NR_ITER; i++) {
			CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
			hipDeviceSynchronize();
	}
	cudaCheckErrors();
	op.timer = (csecond() - op.timer)/NR_ITER;
	if (op.timer*NR_ITER < 1.0){
		unsigned int extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
			CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, CUDA_VALUE_TYPE_COMP,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
			hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
	nvmlAPIEnd();
	gflops_s = op.flops*1e-9/op.timer;
	fprintf(stdout, "cuSPARSE_csr: t = %lf ms (%lf Gflops/s )\n", op.timer*1000, gflops_s);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.diagonal_factor << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.density << 
	"," << op.nz << "," << op.avg_nz_row << "," << op.std_nz_row <<
	"," << op.avg_bandwidth << "," << op.std_bandwidth <<
	"," << op.avg_scattering << "," << op.std_scattering <<
	"," << "cuSPARSE_csr" <<  "," << op.timer << "," << gflops_s << "\n";

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------

    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    
	foutp.close();
	return EXIT_SUCCESS;

}
