#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>

#include "anonymouslib_cuda.h"

#include "gpu_utils.hpp"
#include "spmv_utils.hpp"
#include "cuSPARSE.hpp"
#include "nvem.hpp"

using namespace std;

#ifndef VALUE_TYPE_AX
#error
#endif

#ifndef VALUE_TYPE_Y
#error
#endif

#ifndef VALUE_TYPE_COMP
#error
#endif

#ifndef NR_ITER
#error
#endif
	
int main(int argc, char **argv) {
	/// Check Input
	massert(argc == 3,
	  "Incorrect arguments.\nUsage:\t./Executable logfilename Matrix_name.mtx");
	  
	// Set/Check for device
	int device_id = 0;
	hipSetDevice(device_id);
	hipGetDevice(&device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

	char *name = argv[2], *outfile = argv[1];
	double cpu_timer, gpu_timer, exc_timer = 0, trans_timer[4] = {0, 0, 0, 0}, gflops_s = -1.0;

	FILE *fp = fopen(name, "r");
	massert(fp && strstr(name, ".mtx") && !fclose(fp), "Invalid .mtx File");

	/// Mix C & C++ file inputs, because...?
	ofstream foutp;
	foutp.open(outfile, ios::out | ios::app ); 
	massert(foutp.is_open() , "Invalid output File");
	// print_devices();

	exc_timer = csecond();
	SpmvOperator op(name);
	exc_timer = csecond() - exc_timer;

	fprintf(stdout,
	  "File=%s ( distribution = %s, placement = %s, diagonal_factor = %lf, seed = %d ) -> Input time=%lf s\n\t\
	  nr_rows(m)=%d, nr_cols(n)=%d, bytes = %d, density =%lf, mem_footprint = %lf MB, mem_range=%s\n\t\
	  nr_nnzs=%d, avg_nnz_per_row=%lf, std_nnz_per_row=%lf\n\t\
	  avg_bandwidth=%lf, std_bandwidth = %lf\n\t\
	  avg_scattering=%lf, std_scattering=%lf\n",
	  op.mtx_name, op.distribution, op.placement, op.diagonal_factor, op.seed, exc_timer, 
	  op.m, op.n, op.bytes, op.density, op.A_mem_footprint, op.mem_range,
	  op.nz, op.avg_nz_row,  op.std_nz_row, 
	  op.avg_bandwidth,  op.std_bandwidth, 
	  op.avg_scattering,  op.std_scattering );
	  
	VALUE_TYPE_AX *x = (VALUE_TYPE_AX *)malloc(op.n * sizeof(VALUE_TYPE_AX));
	VALUE_TYPE_Y *out = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	vec_init_rand<VALUE_TYPE_AX>(x, op.n, 0);
	op.vec_alloc(x);
    
	op.cuSPARSE_init();
	SpmvCsrData *data = (SpmvCsrData *)op.format_data;
    VALUE_TYPE_COMP alpha = (VALUE_TYPE_COMP) 1.0;
    VALUE_TYPE_COMP beta = (VALUE_TYPE_COMP) 0.0;
    cout << "alpha: " << alpha << ", beta: " << beta << endl;

    srand(time(NULL));

    cout << " ( " << op.m << ", " << op.n << " ) nnz = " << op.nz << endl;

    double gb = getB<int, VALUE_TYPE_AX>(op.m, op.nz);
    double gflop = getFLOP<int>(op.nz);
    
     int err = 0;
    hipError_t err_cuda = hipSuccess;

    // Define pointers of matrix A, vector x and y
    int *d_csrRowPtrA;
    int *d_csrColIdxA;
    VALUE_TYPE_AX *d_csrValA;
    VALUE_TYPE_AX *dX;
    VALUE_TYPE_Y *dY;

    // Matrix A
    checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, (op.m+1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrColIdxA, op.nz  * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrValA,    op.nz  * sizeof(VALUE_TYPE_AX)));

    checkCudaErrors(hipMemcpy(d_csrRowPtrA, data->rowPtr, (op.m+1) * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIdxA, data->colInd, op.nz  * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA,    data->values,    op.nz  * sizeof(VALUE_TYPE_AX),   hipMemcpyHostToDevice));

    // Vector x
    checkCudaErrors(hipMalloc((void **)&dX, op.n * sizeof(VALUE_TYPE_AX)));
    checkCudaErrors(hipMemcpy(dX, x, op.n * sizeof(VALUE_TYPE_AX), hipMemcpyHostToDevice));

    // Vector y
    checkCudaErrors(hipMalloc((void **)&dY, op.m  * sizeof(VALUE_TYPE_Y)));
    checkCudaErrors(hipMemcpy(dY, out, op.m * sizeof(VALUE_TYPE_Y), hipMemcpyHostToDevice));

    anonymouslibHandle<int, unsigned int, VALUE_TYPE_AX> A(op.m, op.n);
    err = A.inputCSR(op.nz, d_csrRowPtrA, d_csrColIdxA, d_csrValA);
    //cout << "inputCSR err = " << err << endl;

    err = A.setX(dX); // you only need to do it once!
    //cout << "setX err = " << err << endl;

    A.setSigma(ANONYMOUSLIB_AUTO_TUNED_SIGMA);

    // warmup device
    A.warmup();

    anonymouslib_timer asCSR5_timer;
    asCSR5_timer.start();

    err = A.asCSR5();

    cout << "CSR->CSR5 time = " << asCSR5_timer.stop() << " ms." << endl;
    //cout << "asCSR5 err = " << err << endl;
    
#ifdef TEST

	VALUE_TYPE_Y *out1 = (VALUE_TYPE_Y *)calloc(op.m, sizeof(VALUE_TYPE_Y));
	fprintf(stdout,"Serial-CSR: ");
	op.timer = csecond();
	spmv_csr<VALUE_TYPE_AX, VALUE_TYPE_Y, VALUE_TYPE_COMP>(data->rowPtr, data->colInd, data->values, x,
		   out1, op.m);
	op.timer = csecond() - op.timer;
	report_results(op.timer * NR_ITER, op.flops, op.bytes);
	fprintf(stdout,"\n");

	fprintf(stdout,"\nRunning tests.. \n");

	fprintf(stdout,"Testing CSR5_9_csr...\t");
    // execute SpMV
    err = A.spmv(alpha, dY);
    
	hipDeviceSynchronize();
	// device result check
    hipMemcpy(out, dY, op.m * sizeof(VALUE_TYPE_Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	check_result<VALUE_TYPE_Y>((VALUE_TYPE_Y*)out, out1, op.m);
	free(out1);
#endif

    // warm up by running 50 times
    if (NR_ITER)
    {
        for (int i = 0; i < 50; i++)
            err = A.spmv(alpha, dY);
    }

    err_cuda = hipDeviceSynchronize();

	short CUDA_VALUE_TYPE_AX;
	if (std::is_same<VALUE_TYPE_AX, float>::value)  CUDA_VALUE_TYPE_AX = 0;
	else if (std::is_same<VALUE_TYPE_AX, double>::value) CUDA_VALUE_TYPE_AX = 1;
	char powa_filename[256];
	sprintf(powa_filename, "CSR5_CUDA_SPMV_9.2_mtx_dtype-%d.log", CUDA_VALUE_TYPE_AX);
	NvemStartMeasure(device_id, powa_filename, 0); // Set to 1 for NVEM log messages. ;
	op.timer = csecond();

    // time spmv by running NR_ITER times
    for (int i = 0; i < NR_ITER; i++){
        err = A.spmv(alpha, dY);
    	err_cuda = hipDeviceSynchronize();
    }
	op.timer = (csecond() - op.timer)/NR_ITER;
	unsigned int extra_itter = 0;
	if (op.timer*NR_ITER < 1.0){
		extra_itter = ((unsigned int) 1.0/op.timer) - NR_ITER;
		fprintf(stdout,"Performing extra %d itter for more power measurments (min benchmark time : 1s)...\n", extra_itter);
		for (int i = 0; i <  extra_itter; i++) {
        	err = A.spmv(alpha, dY);
    		err_cuda = hipDeviceSynchronize();
		}
		cudaCheckErrors();
	}
    NvemStats_p nvem_data = NvemStopMeasure(device_id, "Energy measure CSR5_9_mtx");
	gflops_s = op.flops*1e-9/op.timer;
	double W_avg = nvem_data->W_avg, J_estimated = nvem_data->J_estimated/(NR_ITER+extra_itter); 
	fprintf(stdout, "CSR5_9: t = %lf ms (%lf Gflops/s ). Average Watts = %lf, Estimated Joules = %lf\n", op.timer*1000, gflops_s, W_avg, J_estimated);
	foutp << op.mtx_name << "," << op.distribution << "," << op.placement << "," << op.diagonal_factor << "," << op.seed <<
	"," << op.m << "," << op.n << "," << op.density << "," << op.A_mem_footprint << "," << op.mem_range << 
	"," << op.nz << "," << op.avg_nz_row << "," << op.std_nz_row <<
	"," << op.avg_bandwidth << "," << op.std_bandwidth <<
	"," << op.avg_scattering << "," << op.std_scattering <<
	"," << "CSR5_9" <<  "," << op.timer << "," << gflops_s << "," << W_avg <<  "," << J_estimated << endl;

    A.destroy();

    checkCudaErrors(hipFree(d_csrRowPtrA));
    checkCudaErrors(hipFree(d_csrColIdxA));
    checkCudaErrors(hipFree(d_csrValA));
    checkCudaErrors(hipFree(dX));
    checkCudaErrors(hipFree(dY));
    foutp.close();
    // compare reference and anonymouslib results

    free(x);
    free(out);


    return 0;
}

