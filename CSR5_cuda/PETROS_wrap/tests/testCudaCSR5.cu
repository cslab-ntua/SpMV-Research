#include "hip/hip_runtime.h"
#include <iostream>

#include "spmv_utils.hpp"
#include "anonymouslib_cuda.h"

#include "mmio.h"
#include "nvmlPower.hpp"

using namespace std;

#ifndef VALUE_TYPE
#error
#endif

#ifndef NUM_RUN
#error
#endif

double call_anonymouslib(int m, int n, int nnzA,
                  int *csrRowPtrA, int *csrColIdxA, VALUE_TYPE *csrValA,
                  VALUE_TYPE *x, VALUE_TYPE *y, VALUE_TYPE alpha)
{
    int err = 0;
    hipError_t err_cuda = hipSuccess;

    // set device
    int device_id = 1;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

    double gb = getB<int, VALUE_TYPE>(m, nnzA);
    double gflop = 2*nnzA;

    // Define pointers of matrix A, vector x and y
    int *d_csrRowPtrA;
    int *d_csrColIdxA;
    VALUE_TYPE *d_csrValA;
    VALUE_TYPE *d_x;
    VALUE_TYPE *d_y;

    // Matrix A
    checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, (m+1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrColIdxA, nnzA  * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrValA,    nnzA  * sizeof(VALUE_TYPE)));

    checkCudaErrors(hipMemcpy(d_csrRowPtrA, csrRowPtrA, (m+1) * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIdxA, csrColIdxA, nnzA  * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA,    csrValA,    nnzA  * sizeof(VALUE_TYPE),   hipMemcpyHostToDevice));

    // Vector x
    checkCudaErrors(hipMalloc((void **)&d_x, n * sizeof(VALUE_TYPE)));
    checkCudaErrors(hipMemcpy(d_x, x, n * sizeof(VALUE_TYPE), hipMemcpyHostToDevice));

    // Vector y
    checkCudaErrors(hipMalloc((void **)&d_y, m  * sizeof(VALUE_TYPE)));
    checkCudaErrors(hipMemset(d_y, 0, m * sizeof(VALUE_TYPE)));

    anonymouslibHandle<int, unsigned int, VALUE_TYPE> A(m, n);
    err = A.inputCSR(nnzA, d_csrRowPtrA, d_csrColIdxA, d_csrValA);
    //cout << "inputCSR err = " << err << endl;

    err = A.setX(d_x); // you only need to do it once!
    //cout << "setX err = " << err << endl;

    A.setSigma(ANONYMOUSLIB_AUTO_TUNED_SIGMA);

    // warmup device
    A.warmup();

    anonymouslib_timer asCSR5_timer;
    asCSR5_timer.start();

    err = A.asCSR5();

    cout << "CSR->CSR5 time = " << asCSR5_timer.stop() << " ms." << endl;
    //cout << "asCSR5 err = " << err << endl;

    // check correctness by running 1 time
    err = A.spmv(alpha, d_y);
    //cout << "spmv err = " << err << endl;
    checkCudaErrors(hipMemcpy(y, d_y, m * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost));

    // warm up by running 50 times
    if (NUM_RUN)
    {
        for (int i = 0; i < 50; i++)
            err = A.spmv(alpha, d_y);
    }

    err_cuda = hipDeviceSynchronize();

    nvmlAPIRun();
    double timer = csecond();

    // time spmv by running NUM_RUN times
    for (int i = 0; i < NUM_RUN; i++)
        err = A.spmv(alpha, d_y);
    err_cuda = hipDeviceSynchronize();

    timer = (csecond() - timer)/NUM_RUN;
    nvmlAPIEnd();
    
    if (NUM_RUN)
        cout << "CSR5-based SpMV time = " << timer
             << " ms. Bandwidth = " << gb/(1.0e+9 * timer)
             << " GB/s. GFlops = " << gflop/(1.0e+9 * timer)  << " GFlops." << endl;

    A.destroy();

    checkCudaErrors(hipFree(d_csrRowPtrA));
    checkCudaErrors(hipFree(d_csrColIdxA));
    checkCudaErrors(hipFree(d_csrValA));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    return timer;
}

int main(int argc, char ** argv)
{
    int argi = 2;

    char  *filename, *logname;
    massert(argc == argi + 1, "Wrong args. Usage ./ExecMe file.mtx logfilename");
    filename = argv[1];
    logname = argv[2];
    
    cout << "Filename: " << filename << endl;
    cout << "Logfile Name: " << logname << endl;

    /// Mix C & C++ file inputs, because...?
    ofstream foutp;
    foutp.open(logname, ios::out | ios::app ); 
    massert(foutp.is_open() , "Invalid output File");

    SpmvCsrData* csr_matrix = mtx_read_csr(filename);

    int m = csr_matrix->m, n = csr_matrix->n, nnzA = csr_matrix->nz;
    int *csrRowPtrA = csr_matrix->rowPtr;
    int *csrColIdxA = csr_matrix->colInd;
    VALUE_TYPE *csrValA = (VALUE_TYPE*) csr_matrix->values;


    srand(time(NULL));

/*
    // set csrValA to 1, easy for checking floating-point results
    for (int i = 0; i < nnzA; i++)
    {
        csrValA[i] = rand() % 10;
    }
*/

    cout << " ( " << m << ", " << n << " ) nnz = " << nnzA << endl;

    VALUE_TYPE *x = (VALUE_TYPE *)malloc(n * sizeof(VALUE_TYPE));
    for (int i = 0; i < n; i++)
        x[i] = rand() % 10;

    VALUE_TYPE *y = (VALUE_TYPE *)malloc(m * sizeof(VALUE_TYPE));
    VALUE_TYPE *y_ref = (VALUE_TYPE *)malloc(m * sizeof(VALUE_TYPE));

    double gb = getB<int, VALUE_TYPE>(m, nnzA);
    double gflop = getFLOP<int>(nnzA);

    VALUE_TYPE alpha = 1.0;

    // compute reference results on a cpu core
    anonymouslib_timer ref_timer;
    ref_timer.start();

    int ref_iter = 1;
    for (int iter = 0; iter < ref_iter; iter++)
    {
        for (int i = 0; i < m; i++)
        {
            VALUE_TYPE sum = 0;
            for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
                sum += x[csrColIdxA[j]] * csrValA[j] * alpha;
            y_ref[i] = sum;
        }
    }

    double ref_time = ref_timer.stop() / (double)ref_iter;
    cout << "cpu sequential time = " << ref_time
         << " ms. Bandwidth = " << gb/(1.0e+6 * ref_time)
         << " GB/s. GFlops = " << gflop/(1.0e+6 * ref_time)  << " GFlops." << endl << endl;

    // launch compute
    double spmv_seconds = call_anonymouslib(m, n, nnzA, csrRowPtrA, csrColIdxA, csrValA, x, y, alpha);
    // compare reference and anonymouslib results
    int error_count = 0;
    for (int i = 0; i < m; i++)
        if (abs(y_ref[i] - y[i]) > 0.01 * abs(y_ref[i]))
        {
            error_count++;
//            cout << "ROW [ " << i << " ], NNZ SPAN: "
//                 << csrRowPtrA[i] << " - "
//                 << csrRowPtrA[i+1]
//                 << "\t ref = " << y_ref[i]
//                 << ", \t csr5 = " << y[i]
//                 << ", \t error = " << y_ref[i] - y[i]
//                 << endl;
//            break;

//            //if (abs(y_ref[i] - y[i]) > 0.00001)
//            //    cout << ", \t error = " << y_ref[i] - y[i] << endl;
//            //else
//            //    cout << ". \t CORRECT!" << endl;
        }

    if (error_count == 0)
        cout << "Check... PASS!" << endl;
    else
        cout << "Check... NO PASS! #Error = " << error_count << " out of " << m << " entries." << endl;

    double gflops_s = 2*nnzA/spmv_seconds*1e-9;
    foutp << filename << "," << "CSR5-cuda" << "," << m << "," << n << "," << nnzA << "," << spmv_seconds << "," << gflops_s << "\n";
    foutp.close();

    free(csrRowPtrA);
    free(csrColIdxA);
    free(csrValA);
    free(x);
    free(y);
    free(y_ref);


    return 0;
}

